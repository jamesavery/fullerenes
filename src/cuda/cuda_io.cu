#include "hip/hip_runtime.h"
#include "fullerenes/gpu/cuda_io.hh"
#include "type_traits"

namespace cuda_io{

    hipError_t output_to_queue(std::queue<std::pair<Polyhedron, size_t>>& queue, IsomerBatch& batch, const bool copy_2d_layout){
        //Batch needs to exist on the host. For performance reasons we don't want to create a new batch here and copy to that, hipMalloc is expensive.
        printLastCudaError();
        if (batch.buffer_type != HOST_BUFFER) assert(false); 

        size_t N = batch.n_atoms;
        for (size_t isomer_idx = 0; isomer_idx < batch.isomer_capacity; isomer_idx++){   
            //Only insert the isomer if it has finished (either CONVERGED or FAILED)
            if(!(batch.statuses[isomer_idx] == CONVERGED || batch.statuses[isomer_idx] == FAILED)) continue;
            
            //Graphs always have a neighbour array.
            neighbours_t out_neighbours(N); 
            std::vector<coord2d> output_2D;

            for (size_t i = 0; i < N; i++){
                //Fill in cubic neighbours
                out_neighbours[i] = {batch.neighbours[isomer_idx*N*3 + i*3], batch.neighbours[isomer_idx*N*3 + i*3 + 1], batch.neighbours[isomer_idx*N*3 + i*3 + 2]};
            }

            //If 2D layout is true, allocate memory and copy 2D layout. If this is not needed disable it for performance gains.
            if (copy_2d_layout){
                output_2D = std::vector<coord2d>(N);
                for (size_t i = 0; i < N; i++){
                    output_2D[i] = {batch.xys[isomer_idx*N + i*2], batch.xys[isomer_idx*N + i*2 + 1]};
                }
            }
            //If T is of type Polyhedron, copy 3D geometry, construct Polyhedron object and insert in queue.
            //if(std::is_same<Polyhedron,T>::value) {
            std::vector<coord3d> output_X(N);
            for (size_t i = 0; i < N; i++){
                output_X[i] = {batch.X[isomer_idx*N + i*3], batch.X[isomer_idx + i*3 + 1], batch.X[isomer_idx + i*3 + 2]};
            }
            queue.push({Polyhedron(PlanarGraph(out_neighbours, output_2D),output_X),batch.IDs[isomer_idx]});
            
            
        }
        return hipGetLastError();
    }


    hipError_t copy(   IsomerBatch& destination, //Copy data to this batch
                        const IsomerBatch& source, //Copy data from this batch
                        const LaunchCtx& ctx, //Optional: specify which launch context to perform the copy operation in.
                        const LaunchPolicy policy, //Optional: specifies whether to synchronize the stream before and after copying)
                        const std::pair<int,int>& lhs_range, //Optional: provide a range of indices to assign similar to slices in numpy eg. {0,5} = [0:5]
                        const std::pair<int,int>& rhs_range //Optional: provide a range of indices to copy from similar to slices in numpy eg. {0,5} = [0:5]
                        ){
        //Iterate over the data fields of the IsomerBatch (pseudo reflection) and copy the contents of each using the provided stream.
        if(policy == LaunchPolicy::SYNC) ctx.wait();
        for (size_t i = 0; i < source.pointers.size(); i++)
        {

            int num_isomers = (lhs_range.second > -1 && lhs_range.first > -1) ? lhs_range.second - lhs_range.first : destination.isomer_capacity;
            int num_elements = get<3>(source.pointers[i]) ?  source.n_atoms * num_isomers : num_isomers;
            int lhs_offset = lhs_range.first > 0 ? lhs_range.first * max((size_t)1, source.n_atoms*get<3>(source.pointers[i])) * get<2>(source.pointers[i]) : 0;
            int rhs_offset = rhs_range.first > 0 ? rhs_range.first * max((size_t)1, source.n_atoms*get<3>(source.pointers[i])) * get<2>(source.pointers[i]) : 0;
            char* lhs_ptr = (char*)(*(get<1>(destination.pointers[i])));
            char* rhs_ptr = (char*)(*(get<1>(source.pointers[i])));
            hipMemcpyAsync(lhs_ptr + lhs_offset, rhs_ptr + rhs_offset, get<2>(source.pointers[i])*num_elements, hipMemcpyKind(2*source.buffer_type +  destination.buffer_type), ctx.stream);
        }
        destination.n_isomers = source.n_isomers;
        printLastCudaError("Failed to copy struct");
        if(policy == LaunchPolicy::SYNC) ctx.wait();
        return hipGetLastError();
    }

    hipError_t free(IsomerBatch& batch){
        for (int i = 0; i < batch.pointers.size(); i++){
            if(batch.buffer_type == DEVICE_BUFFER) {hipFree(*get<1>(batch.pointers[i]));} else {hipHostFree(*get<1>(batch.pointers[i]));}
        }
        return hipGetLastError();
    }
    
    hipError_t resize(IsomerBatch& batch, const size_t new_capacity, const LaunchCtx& ctx, const LaunchPolicy policy, int front, int back){
        //Construct a tempory batch: allocates the needed amount of memory.
        IsomerBatch temp_batch = IsomerBatch(batch.n_atoms, new_capacity, batch.buffer_type);
        //Copy contents of old batch into newly allocated memory.
        IsomerBatch::copy(temp_batch, batch, ctx.stream);
        for (int i = 0; i < batch.pointers.size(); i++)
        {
            void* temp_ptr = *get<1>(batch.pointers[i]);
            printLastCudaError("Free failed");
            //Reassign pointers of the input batch, to the new memory
            *get<1>(batch.pointers[i]) = *get<1>(temp_batch.pointers[i]);
            //Assign old pointers to temporary object, let destructor take care of cleanup.
            *get<1>(temp_batch.pointers[i]) = temp_ptr;
        }
        batch.isomer_capacity = temp_batch.isomer_capacity;
        return hipGetLastError();
    }
}
 
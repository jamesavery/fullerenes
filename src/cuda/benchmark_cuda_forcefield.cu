#include "kernel_clean.cu"
#include "coord3d.cu"
#include "C256ih.cu"


typedef uint16_t node_t;
constexpr
int minpow2(int v)
{
  v--;
  v |= v >> 1;
  v |= v >> 2;
  v |= v >> 4;
  v |= v >> 8;
  v |= v >> 16;
  v++;
  return v;
}

int main(){

    /**
     * Args: 
     * 
     * real_t X*                    -Pointer to starting geometry 
     * node_t cubic_neighbours      -Pointer to neighbour array
     * node_t next_on_face          -Pointer to array with cyclical neighbour information
     * node_t prev_on_face          -Pointer to array with cyclical (counter clockwise) neighbour information
     * uint8_t face_right           -Pointer to array with face information to the right of the vertex A->B
     * 
     * These arrays are assumed to be contigous containers with information for all fullerenes in the batch. I.e. 
     * real_t X[N*3*M] = {{x0_0,y0_0,z0_0, x0_1,y0_1,z0_1 , ..., x0_N,y0_N,z0_N},{x1_0,y1_0,z1_0, x1_1,y1_1,z1_1 , ..., x1_N,y1_N,z1_N}, ..., {x2_0,y2_0,z2_0, x2_1,y2_1,z2_1 , ..., x2_N,y2_N,z2_N}}
     * Dimensions: N x 3 = Fullerene size * 3 coordinates, M = Batch size. 
     * 
     * 
     * Batch size depends on the fullerene size N in the following way:
     * batchSize = ( maxThreadsPerBlock // N ) * numOfMultiProcessors
     * However the API call hipOccupancyMaxActiveBlocksPerMultiprocessor() should be used.
     * 
    **/
    const size_t N = 256;
    const int Block_Size_Pow2 = minpow2((int)N);

    size_t batch_size = IsomerspaceForcefield::computeBatchSize<Block_Size_Pow2>(N);
    
    printf("Solving %d fullerenes of size: %d \n", (int)batch_size, (int)N);

    /** Generates a synthetic load from a single set of fullerene pointers **/
    real_t* synth_X = reinterpret_cast<real_t*>(IsomerspaceForcefield::synthetic_array<real_t>(N, batch_size, &X[0]));
    node_t* synth_cubic_neighbours = reinterpret_cast<node_t*>(IsomerspaceForcefield::synthetic_array<node_t>(N, batch_size, &cubic_neighbours[0]));
    node_t* synth_next_on_face = reinterpret_cast<node_t*>(IsomerspaceForcefield::synthetic_array<node_t>(N, batch_size, &next_on_face[0]));
    node_t* synth_prev_on_face = reinterpret_cast<node_t*>(IsomerspaceForcefield::synthetic_array<node_t>(N, batch_size, &prev_on_face[0]));
    uint8_t* synth_face_right = reinterpret_cast<uint8_t*>(IsomerspaceForcefield::synthetic_array<uint8_t>(N, batch_size, &face_right[0]));
    
    real_t* bonds = new real_t[batch_size*N*3];
    real_t* angles = new real_t[batch_size*N*3];
    real_t* dihedrals = new real_t[batch_size*N*3];
    real_t* bond_0 = new real_t[batch_size*N*3];
    real_t* angle_0 = new real_t[batch_size*N*3];
    real_t* dihedral_0 = new real_t[batch_size*N*3];
    real_t* gradients =  new real_t[batch_size*N*3];
    
    real_t* d_X; real_t* d_X_temp; real_t* d_X2; node_t* d_neighbours; node_t* d_prev_on_face; node_t* d_next_on_face; uint8_t* d_face_right; real_t* d_gdata; real_t* d_bonds; real_t* d_angles; real_t* d_dihedrals; real_t* d_angle_0; real_t* d_bond_0; real_t* d_dihedral_0; real_t* d_gradients;
    IsomerspaceForcefield::DevicePointers d_pointers = IsomerspaceForcefield::DevicePointers(d_X,d_X_temp,d_X2,d_neighbours,d_prev_on_face, d_next_on_face, d_face_right, d_gdata,d_bonds,d_angles,d_dihedrals,d_bond_0,d_angle_0,d_dihedral_0,d_gradients);
    IsomerspaceForcefield::HostPointers h_pointers = IsomerspaceForcefield::HostPointers(synth_X, synth_cubic_neighbours, synth_next_on_face, synth_prev_on_face, synth_face_right);
    IsomerspaceForcefield::AllocateDevicePointers(d_pointers, N, batch_size);
    IsomerspaceForcefield::OptimizeBatch<Block_Size_Pow2>(d_pointers,h_pointers,N,batch_size,N*5);
    IsomerspaceForcefield::CheckBatch<Block_Size_Pow2>(d_pointers, h_pointers, N, batch_size);
    IsomerspaceForcefield::InternalCoordinates<Block_Size_Pow2>(d_pointers,h_pointers,N,batch_size,bonds,angles,dihedrals);
    IsomerspaceForcefield::HarmonicConstants<Block_Size_Pow2>(d_pointers,h_pointers,N,batch_size,bond_0,angle_0,dihedral_0);
    IsomerspaceForcefield::Gradients<Block_Size_Pow2>(d_pointers,h_pointers,N,batch_size,gradients);
    IsomerspaceForcefield::FreePointers(d_pointers);

    //IsomerspaceForcefield::print_array(reinterpret_cast<IsomerspaceForcefield::coord3d*>(bonds),N,0);


}

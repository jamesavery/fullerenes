#include "isomerspace_forcefield.cu"
#include "coord3d.cu"
#include "C512ih.cu"
#include "fullerenes/gpu/isomerspace_forcefield.hh"

#include <unistd.h>
constexpr
int minpow2(int v)
{
  v--;
  v |= v >> 1;
  v |= v >> 2;
  v |= v >> 4;
  v |= v >> 8;
  v |= v >> 16;
  v++;
  return v;
}

int main(){

    /**
     * Args: 
     * 
     * device_real_t X*                    -Pointer to starting geometry 
     * device_node_t cubic_neighbours      -Pointer to neighbour array
     * device_node_t next_on_face          -Pointer to array with cyclical neighbour information
     * device_node_t prev_on_face          -Pointer to array with cyclical (counter clockwise) neighbour information
     * uint8_t face_right           -Pointer to array with face information to the right of the vertex A->B
     * 
     * These arrays are assumed to be contigous containers with information for all fullerenes in the batch. I.e. 
     * device_real_t X[N*3*M] = {{x0_0,y0_0,z0_0, x0_1,y0_1,z0_1 , ..., x0_N,y0_N,z0_N},{x1_0,y1_0,z1_0, x1_1,y1_1,z1_1 , ..., x1_N,y1_N,z1_N}, ..., {x2_0,y2_0,z2_0, x2_1,y2_1,z2_1 , ..., x2_N,y2_N,z2_N}}
     * Dimensions: N x 3 = Fullerene size * 3 coordinates, M = Batch size. 
     * 
     * 
     * Batch size depends on the fullerene size N in the following way:
     * batchSize = ( maxThreadsPerBlock // N ) * numOfMultiProcessors
     * However the API call hipOccupancyMaxActiveBlocksPerMultiprocessor() should be used.
     * 
    **/
    const size_t N = 512;


    //size_t batch_size = IsomerspaceForcefield::computeBatchSize(N);
    size_t batch_size = IsomerspaceForcefield::get_batch_capacity(N);
    printf("Solving %d fullerenes of size: %d \n", (int)batch_size, (int)N);

    /** Generates a synthetic load from a single set of fullerene pointers **/

    device_real_t* synth_X                = reinterpret_cast<device_real_t*>(synthetic_array<device_real_t>(N, batch_size, &X[0]));
    device_node_t* synth_cubic_neighbours = reinterpret_cast<device_node_t*>(synthetic_array<device_node_t>(N, batch_size, &cubic_neighbours[0]));
    device_node_t* synth_next_on_face     = reinterpret_cast<device_node_t*>(synthetic_array<device_node_t>(N, batch_size, &next_on_face[0]));
    device_node_t* synth_prev_on_face     = reinterpret_cast<device_node_t*>(synthetic_array<device_node_t>(N, batch_size, &prev_on_face[0]));
    uint8_t* synth_face_right             = reinterpret_cast<uint8_t*>(synthetic_array<uint8_t>(N, batch_size, &face_right[0]));

    IsomerspaceForcefield::IsomerspaceGraph graph = IsomerspaceForcefield::IsomerspaceGraph(synth_X,synth_cubic_neighbours, synth_next_on_face, synth_prev_on_face, synth_face_right);
    graph.N = N; graph.batch_size = batch_size;
    IsomerspaceForcefield kernel = IsomerspaceForcefield(N);

    kernel.insert_isomer_batch(graph);
    kernel.optimize_batch(N*5);
    kernel.check_batch();
    kernel.to_file(0);
    kernel.batch_statistics_to_file();
    //IsomerspaceForcefield::print_array(reinterpret_cast<IsomerspaceForcefield::coord3d*>(kernel.h_graph.X),N,0);


}

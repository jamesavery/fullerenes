#include "hip/hip_runtime.h"
#include "fullerenes/gpu/isomerspace_forcefield.hh"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include ""
#include <stdio.h>
#define getLastCudaError(x) 
#include <iostream>
#include <fstream>
#include <chrono>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>

namespace IsomerspaceForcefield {

typedef device_real_t real_t;
typedef device_node_t node_t;

#include "coord3d.cu"
#include "helper_functions.cu"


using namespace std::literals;
namespace cg = cooperative_groups;

struct DevicePointers; struct HostPointers;


struct ArcData{
    //124 FLOPs;
    __device__ ArcData(const node_t a, const uint8_t j, const coord3d* __restrict__ X, const BookkeepingData& bdat){   
        this->j = j;   

        real_t r_rmp;
        coord3d ap, am, ab, ac, ad, mp;
        coord3d X_a = X[a]; coord3d X_b = X[bdat.neighbours[j]];
        //printf("Index: %d \n", a*3 + j);

        //Compute the arcs ab, ac, ad, bp, bm, ap, am, mp, bc and cd
        ab = (X_b - X_a);  r_rab = bond_length(ab); ab_hat = r_rab * ab; rab = non_resciprocal_bond_length(ab);
        ac = (X[bdat.neighbours[(j+1)%3]] - X_a); r_rac = bond_length(ac); ac_hat = r_rac * ac;
        ad = (X[bdat.neighbours[(j+2)%3]] - X_a); r_rad = bond_length(ad); ad_hat = r_rad * ad;
        
        coord3d bp = (X[bdat.next_on_face[j]] - X_b); bp_hat = unit_vector(bp);
        coord3d bm = (X[bdat.prev_on_face[j]] - X_b); bm_hat = unit_vector(bm);

        ap = bp + ab; r_rap = bond_length(ap); ap_hat = r_rap * ap;
        am = bm + ab; r_ram = bond_length(am); am_hat = r_ram * am;
        mp = bp - bm; r_rmp = bond_length(mp); mp_hat = r_rmp * mp;

        bc_hat = unit_vector(ac - ab);
        cd_hat = unit_vector(ad - ac);

        //Compute inverses of some arcs, these are subject to be omitted if the equations are adapted appropriately with inversion of signs.
        ba_hat = -ab_hat;
        mb_hat = -bm_hat;
        pa_hat = -ap_hat;
        pb_hat = -bp_hat;
    }

    //3 FLOPs
    INLINE real_t harmonic_energy(const real_t p0, const real_t p) const{
        return (real_t)0.5*(p-p0)*(p-p0);
    }
    //4 FLOPs
    INLINE coord3d  harmonic_energy_gradient(const real_t p0, const real_t p, const coord3d gradp) const{
        return (p-p0)*gradp;     
    }

    //1 FLOP
    INLINE real_t bond() const {return (real_t)rab;}

    //5 FLOPs
    INLINE real_t angle() const {return dot(ab_hat,ac_hat);}

    //Returns the inner dihedral angle for the current arc. Used here only for energy calculation, 
    //otherwise embedded in dihedral computation because the planes and angles that make up the dihedral angle computation are required for derivative computation.
    //50 FLOPs
    INLINE real_t dihedral() const 
    { 
        coord3d nabc, nbcd; real_t cos_b, cos_c, r_sin_b, r_sin_c;
        cos_b = dot(ba_hat,bc_hat);  r_sin_b = rsqrt((real_t)1.0 - cos_b*cos_b); nabc = cross(ba_hat, bc_hat) * r_sin_b;
        cos_c = dot(-bc_hat,cd_hat); r_sin_c = rsqrt((real_t)1.0 - cos_c*cos_c); nbcd = cross(-bc_hat,cd_hat) * r_sin_c;
        return dot(nabc, nbcd);
    }
    
    // Chain rule terms for angle calculation
    //Computes gradient related to bending term. ~24 FLOPs
    INLINE coord3d inner_angle_gradient(const Constants<coord3d>& c) const
    {   
        real_t cos_angle = angle(); //Inner angle of arcs ab,ac.
        coord3d grad = cos_angle * (ab_hat * r_rab + ac_hat * r_rac) - ab_hat * r_rac - ac_hat* r_rab; //Derivative of inner angle: Eq. 21. 
        return get(c.f_inner_angle,j) * harmonic_energy_gradient(get(c.angle0,j), cos_angle, grad); //Harmonic Energy Gradient: Eq. 21. multiplied by harmonic term.
    }
    //Computes gradient related to bending of outer angles. ~20 FLOPs
    INLINE coord3d outer_angle_gradient_m(const Constants<coord3d>& c) const
    {
        real_t cos_angle = -dot(ab_hat, bm_hat); //Compute outer angle. ab,bm
        coord3d grad = (bm_hat + ab_hat * cos_angle) * r_rab; //Derivative of outer angles Eq. 30. Buster Thesis
        return get(c.f_outer_angle_m,j) * harmonic_energy_gradient(get(c.outer_angle_m0,j),cos_angle,grad); //Harmonic Energy Gradient: Eq. 30 multiplied by harmonic term.
    }
    INLINE coord3d outer_angle_gradient_p(const Constants<coord3d>& c) const
    {
        real_t cos_angle = -dot(ab_hat, bp_hat); //Compute outer angle. ab,bp
        coord3d grad = (bp_hat + ab_hat * cos_angle) * r_rab; //Derivative of outer angles Eq. 28. Buster Thesis
        return get(c.f_outer_angle_p,j) * harmonic_energy_gradient(get(c.outer_angle_p0,j),cos_angle,grad); //Harmonic Energy Gradient: Eq. 28 multiplied by harmonic term.
    }
    // Chain rule terms for dihedral calculation
    //Computes gradient related to dihedral/out-of-plane term. ~75 FLOPs
    INLINE coord3d inner_dihedral_gradient(const Constants<coord3d>& c) const
    {
        coord3d nabc, nbcd; real_t cos_b, cos_c, r_sin_b, r_sin_c;
        cos_b = dot(ba_hat,bc_hat); r_sin_b = rsqrtf((real_t)1.0 - cos_b*cos_b); nabc = cross(ba_hat, bc_hat) * r_sin_b;
        cos_c = dot(-bc_hat,cd_hat); r_sin_c = rsqrtf((real_t)1.0 - cos_c*cos_c); nbcd = cross(-bc_hat,cd_hat) * r_sin_c;

        real_t cos_beta = dot(nabc, nbcd); //Inner dihedral angle from planes abc,bcd.
        real_t cot_b = cos_b * r_sin_b * r_sin_b; //cos(b)/sin(b)^2

        //Derivative w.r.t. inner dihedral angle F and G in Eq. 26
        coord3d grad = cross(bc_hat, nbcd) * r_sin_b * r_rab - ba_hat * cos_beta * r_rab + (cot_b * cos_beta * r_rab) * (bc_hat - ba_hat * cos_b);

        return get(c.f_inner_dihedral,j) * harmonic_energy_gradient(get(c.inner_dih0,j), cos_beta, grad); //Eq. 26.
    }

    //Computes gradient from dihedral angles constituted by the planes bam, amp ~162 FLOPs
    INLINE coord3d outer_a_dihedral_gradient(const Constants<coord3d>& c) const
    {
        coord3d nbam_hat, namp_hat; real_t cos_a, cos_m, r_sin_a, r_sin_m;

        cos_a = dot(ab_hat,am_hat); r_sin_a = rsqrtf((real_t)1.0 - cos_a*cos_a); nbam_hat = cross(ab_hat,am_hat) * r_sin_a;
        cos_m = dot(-am_hat,mp_hat); r_sin_m = rsqrtf((real_t)1.0 - cos_m*cos_m); namp_hat = cross(-am_hat,mp_hat) * r_sin_m;
        
        real_t cos_beta = dot(nbam_hat, namp_hat); //Outer Dihedral angle bam, amp
        real_t cot_a = cos_a * r_sin_a * r_sin_a;
        real_t cot_m = cos_m * r_sin_m * r_sin_m;

        //Derivative w.r.t. outer dihedral angle, factorized version of Eq. 31.
        coord3d grad = cross(mp_hat,nbam_hat)*r_ram*r_sin_m - (cross(namp_hat,ab_hat)*r_ram + cross(am_hat,namp_hat)*r_rab)*r_sin_a +
                        cos_beta*(ab_hat*r_rab + r_ram * ((real_t)2.0*am_hat + cot_m*(mp_hat+cos_m*am_hat)) - cot_a*(r_ram*(ab_hat - am_hat*cos_a) + r_rab*(am_hat-ab_hat*cos_a)));
        
        //Eq. 31 multiplied by harmonic term.
        return get(c.f_outer_dihedral,j) * harmonic_energy_gradient(get(c.outer_dih0_a,j), cos_beta, grad);
    }

    //Computes gradient from dihedral angles constituted by the planes nbmp, nmpa ~92 FLOPs
    INLINE coord3d outer_m_dihedral_gradient(const Constants<coord3d>& c) const
    {
        coord3d nbmp_hat, nmpa_hat; real_t cos_m, cos_p, r_sin_m, r_sin_p;
        cos_m = dot(mb_hat,mp_hat);  r_sin_m = rsqrtf((real_t)1.0 - cos_m*cos_m); nbmp_hat = cross(mb_hat,mp_hat) * r_sin_m;
        cos_p = dot(-mp_hat,pa_hat); r_sin_p = rsqrtf((real_t)1.0 - cos_p*cos_p); nmpa_hat = cross(-mp_hat,pa_hat) * r_sin_p;
        
        //Cosine to the outer dihedral angle constituted by the planes bmp and mpa
        real_t cos_beta = dot(nbmp_hat, nmpa_hat); //Outer dihedral angle bmp,mpa.
        real_t cot_p = cos_p * r_sin_p * r_sin_p;
        
        //Derivative w.r.t. outer dihedral angle, factorized version of Eq. 32.
        coord3d grad = r_rap * (cot_p*cos_beta * (-mp_hat - pa_hat*cos_p) - cross(nbmp_hat, mp_hat)*r_sin_p - pa_hat*cos_beta );

        //Eq. 32 multiplied by harmonic term.
        return get(c.f_outer_dihedral,j) * harmonic_energy_gradient(get(c.outer_dih0_m,j), cos_beta, grad);
    }

    //Computes gradient from dihedral angles constituted by the planes bpa, pam ~162 FLOPs
    INLINE coord3d outer_p_dihedral_gradient(const Constants<coord3d>& c) const
    {
        coord3d nbpa_hat, npam_hat; real_t cos_p, cos_a, r_sin_p, r_sin_a;
        cos_a = dot(ap_hat,am_hat);  r_sin_a = rsqrtf((real_t)1.0 - cos_a*cos_a); npam_hat = cross(ap_hat,am_hat)  * r_sin_a;
        cos_p = dot(pb_hat,-ap_hat); r_sin_p = rsqrtf((real_t)1.0 - cos_p*cos_p); nbpa_hat = cross(pb_hat,-ap_hat) * r_sin_p;

        real_t cos_beta = dot(nbpa_hat, npam_hat); //Outer dihedral angle bpa, pam.
        real_t cot_p = cos_p * r_sin_p * r_sin_p;
        real_t cot_a = cos_a * r_sin_a * r_sin_a;

        //Derivative w.r.t. outer dihedral angle, factorized version of Eq. 33.
        coord3d grad = cross(npam_hat,pb_hat)*r_rap*r_sin_p - (cross(am_hat,nbpa_hat)*r_rap + cross(nbpa_hat,ap_hat)*r_ram)*r_sin_a +
                        cos_beta*(am_hat*r_ram + r_rap * ((real_t)2.0*ap_hat + cot_p*(pb_hat+cos_p*ap_hat)) - cot_a*(r_rap*(am_hat - ap_hat*cos_a) + r_ram*(ap_hat-am_hat*cos_a)));
        
        //Eq. 33 multiplied by harmonic term.
        return get(c.f_outer_dihedral,j) * harmonic_energy_gradient(get(c.outer_dih0_p,j), cos_beta, grad);
    }
    // Internal coordinate gradients
    INLINE coord3d bond_length_gradient(const Constants<coord3d>& c) const { return - get(c.f_bond,j) * harmonic_energy_gradient(get(c.r0,j),bond(),ab_hat);}
    //Sum of angular gradient components.
    INLINE coord3d angle_gradient(const Constants<coord3d>& c) const { return inner_angle_gradient(c) + outer_angle_gradient_p(c) + outer_angle_gradient_m(c);}
    //Sum of inner and outer dihedral gradient components.
    INLINE coord3d dihedral_gradient(const Constants<coord3d>& c) const { return inner_dihedral_gradient(c) + outer_a_dihedral_gradient(c) + outer_m_dihedral_gradient(c) + outer_p_dihedral_gradient(c);}
    //coord3d flatness()             const { return ;  }   
    
    INLINE real_t bond_energy(const Constants<coord3d>& c) const {return (real_t)0.5 *get(c.f_bond,j) *harmonic_energy(bond(),get(c.r0,j));}
    INLINE real_t bend_energy(const Constants<coord3d>& c) const {return get(c.f_inner_angle,j)* harmonic_energy(angle(),get(c.angle0,j));}
    INLINE real_t dihedral_energy(const Constants<coord3d>& c) const {return get(c.f_inner_dihedral,j)* harmonic_energy(dihedral(),get(c.inner_dih0,j));}
    //Harmonic energy contribution from bond stretching, angular bending and dihedral angle bending.
    //71 FLOPs
    INLINE real_t energy(const Constants<coord3d>& c) const {return bond_energy(c) + bend_energy(c) + dihedral_energy(c); }
    //Sum of bond, angular and dihedral gradient components.
    INLINE coord3d gradient(const Constants<coord3d>& c) const{return bond_length_gradient(c) + angle_gradient(c) + dihedral_gradient(c);}

    
    uint8_t j;

    //Residual lengths of arcs ab, ac, am, ap.
    real_t
        rab,
        r_rab,
        r_rac,
        r_rad,
        r_ram,
        r_rap;

    //Base Arcs,
    coord3d
        ab,
        ac,
        ad;

    /*
    All normalized arcs required to perform energy & gradient calculations.
    Note that all these arcs are cyclical the arc ab becomes: ab->ac->ad,  the arc ac becomes: ac->ad->ab , the arc bc becomes: bc->cd->db (For iterations 0, 1, 2)
    As such the naming convention here is related to the arcs as they are used in the 0th iteration. */
    coord3d 
        ab_hat,
        ac_hat,
        ad_hat,
        bp_hat,
        bm_hat,
        am_hat,
        ap_hat,
        ba_hat,
        bc_hat,
        cd_hat,
        mp_hat,
        mb_hat,
        pa_hat,
        pb_hat;
};

__device__ coord3d gradient(const coord3d* __restrict__ X, const node_t node_id, const BookkeepingData &dat, const Constants<coord3d> &constants) {
    coord3d grad = {0.0, 0.0, 0.0};

    for (uint8_t j = 0; j < 3; j++ ){
        ArcData arc = ArcData(node_id, j, X, dat);
        grad += arc.gradient(constants);
    }
    return grad;
}

__device__ real_t energy(const coord3d* __restrict__ X, const node_t node_id, const BookkeepingData &dat, const Constants<coord3d> &constants, real_t* __restrict__ reduction_array, real_t* __restrict__ gdata, const node_t N, bool single_block_fullerenes) {
    real_t arc_energy = (real_t)0.0;

    //(71 + 124) * 3 * N  = 585*N FLOPs
    for (uint8_t j = 0; j < 3; j++ ){
        ArcData arc = ArcData(node_id, j, X, dat);
        arc_energy += arc.energy(constants);
    }
    cg::sync(cg::this_thread_block());
    reduction_array[threadIdx.x] = arc_energy;
    // (/N // 32) * log2(32) = N//32  * 5 FLOPs 
    reduction(reduction_array,gdata, N, single_block_fullerenes); 
     return reduction_array[0];
}

__device__ void golden_section_search(coord3d* __restrict__ X, coord3d& direction, coord3d& new_direction,coord3d* __restrict__ X1, coord3d* __restrict__ X2, real_t* __restrict__ reduction_array, real_t* __restrict__ gdata, const node_t node_id, const node_t N, const BookkeepingData& dat, const Constants<coord3d>& constants, cg::thread_group sync_group, bool single_block_fullerenes){
    real_t tau = (sqrtf(5) - 1) / 2;
    //Actual coordinates resulting from each traversal 
    //Line search x - values;
    real_t a = 0.0; real_t b = 1.0;
    real_t x1,  x2;/* , dfc; */	/* TODO: dfc is not used: should it be? */


    x1 = (a + (1 - tau) * (b - a));
    x2 = (a + tau * (b - a));

    X1[node_id] = X[node_id] + x1 * direction;
    X2[node_id] = X[node_id] + x2 * direction;


    cg::sync(sync_group);

    real_t f1 = energy(X1, node_id, dat, constants, reduction_array, gdata, N, single_block_fullerenes);
    real_t f2 = energy(X2, node_id, dat, constants, reduction_array, gdata, N, single_block_fullerenes);

    for (uint8_t i = 0; i < 20; i++){
        if (f1 > f2){
            a = x1;
            x1 = x2;
            f1 = f2;
            x2 = a + tau * (b - a);
            cg::sync(sync_group);
            X2[node_id] = X[node_id] + x2 * direction;
            cg::sync(sync_group);
            f2 = energy(X2, node_id, dat, constants, reduction_array, gdata, N, single_block_fullerenes);
        }else
        {
            b = x2;
            x2 = x1;
            f2 = f1;
            x1 = a + (1 - tau) * (b - a);
            cg::sync(sync_group);
            X1[node_id] = X[node_id] + x1 * direction;
            cg::sync(sync_group);
            f1 = energy(X1, node_id, dat, constants, reduction_array, gdata, N, single_block_fullerenes);
        }
    }
    //Line search coefficient
    real_t alfa = (a+b)/2;
    cg::sync(sync_group);
    X1[node_id] = X[node_id] + alfa*direction;
    cg::sync(sync_group);
    new_direction = -gradient(X1,node_id,dat, constants);
    
}

__global__ void conjugate_gradient(DevicePointers p, const size_t N, const bool single_block_fullerenes, const size_t MaxIter){
    extern __shared__ real_t smem[];
    

    cg::grid_group grid = cg::this_grid();
    cg::thread_block block = cg::this_thread_block();

    coord3d delta_x0, delta_x1, direction;
    node_t node_id;
    size_t offset;
    size_t gradient_evals = 0;
    size_t energy_evals = 0;
    
    real_t beta, dnorm, r0_norm, direction_norm;
    beta = dnorm = r0_norm = direction_norm = 0.0;

    //If fullerenes are localized to individual blocks then use block threadIdx else use grid ID and use 1 grid per fullerene with concurrent launches.
    //If fullerenes are localized to individual blocks then use block size to determine pointer in array, else assume concurrent kernel launches with pointers to individual fullerenes.
    if (single_block_fullerenes){
        node_id = threadIdx.x;
        offset = blockIdx.x * blockDim.x;
    } else
    {
        node_id = blockDim.x * blockIdx.x + threadIdx.x;
        offset = 0;
    }
    
    coord3d* X = &reinterpret_cast<coord3d*>(p.X)[offset];
    coord3d* X1 = &reinterpret_cast<coord3d*>(p.X1)[offset];
    coord3d* X2 = &reinterpret_cast<coord3d*>(p.X2)[offset];
    
    if (single_block_fullerenes)
    {
        coord3d* sX =&reinterpret_cast<coord3d*>(smem)[(int)ceilf(N/3) + 1];
        coord3d* sX1 =&reinterpret_cast<coord3d*>(smem)[(int)ceilf(N/3) + 2 + N];
        coord3d* sX2 =&reinterpret_cast<coord3d*>(smem)[(int)ceilf(N/3)+ 3 +2*N];  
        sX[node_id] = X[node_id];
        sX1[node_id] = sX[node_id];

        X = &sX[0]; X1 = &sX1[0]; X2 = &sX2[0];
    } else {
        X1[node_id] = X[node_id];
    }

    
    //Pre-compute force constants and store in registers.
    BookkeepingData bookit = BookkeepingData(&p.neighbours[3*offset],&p.face_right[3*offset],&p.next_on_face[3*offset],&p.prev_on_face[3*offset]);
    Constants<coord3d> constants = compute_constants<coord3d>(bookit, node_id);

    //Load constant bookkeeping data into registers.
    const node_t neighbours[3] = {p.neighbours[3*(offset+node_id)],p.neighbours[3*(offset+node_id) + 1],p.neighbours[3*(offset+node_id) + 2]};
    const uint8_t face_right[3] = {p.face_right[3*(offset+node_id)],p.face_right[3*(offset+node_id) + 1],p.face_right[3*(offset+node_id) + 2]};;
    const node_t next_on_face[3] = {p.next_on_face[3*(offset+node_id)],p.next_on_face[3*(offset+node_id) + 1],p.next_on_face[3*(offset+node_id) + 2]};
    const node_t prev_on_face[3] = {p.prev_on_face[3*(offset+node_id)],p.prev_on_face[3*(offset+node_id) + 1],p.prev_on_face[3*(offset+node_id) + 2]};
    BookkeepingData bookkeeping = BookkeepingData(&neighbours[0],&face_right[0],&next_on_face[0],&prev_on_face[0]);   

    cg::sync(grid);
    direction = gradient(X, node_id ,bookkeeping, constants);
    gradient_evals ++;
    
    smem[threadIdx.x] = dot(direction,direction);

    reduction(smem,p.gdata,N,single_block_fullerenes);
    dnorm = sqrtf(smem[0]);
    direction = -direction/dnorm;
    delta_x0 = direction;
    cg::sync(grid);
    for (size_t i = 0; i < MaxIter; i++)
    {   
        beta = 0.0; direction_norm = 0.0; dnorm=0.0; r0_norm = 0.0;
        cg::sync(block);
        if (single_block_fullerenes){golden_section_search(X, direction, delta_x1, X1, X2, smem, p.gdata, node_id, N, bookkeeping, constants, block, single_block_fullerenes);} 
        else { golden_section_search(X, direction, delta_x1, X1, X2, smem, p.gdata, node_id, N, bookkeeping, constants, grid, single_block_fullerenes);}
        
        cg::sync(block);

        gradient_evals++;
        energy_evals += 22;
        //Polak Ribiere method
        
        smem[threadIdx.x] = dot(delta_x0, delta_x0); reduction(smem,p.gdata,N,single_block_fullerenes); r0_norm = smem[0];
        cg::sync(block);
        smem[threadIdx.x] = dot(delta_x1, (delta_x1 - delta_x0)); reduction(smem,p.gdata,N,single_block_fullerenes); beta = smem[0] / r0_norm;
        cg::sync(block);
        real_t E1 = energy(X1, node_id, bookkeeping, constants, smem, p.gdata, N, single_block_fullerenes);
        cg::sync(block);
        real_t E2 = energy(X, node_id, bookkeeping, constants, smem, p.gdata, N, single_block_fullerenes);
        cg::sync(block);

        if (E1> E2)
        {   
            X1[node_id] =  X[node_id];
            delta_x1 =  delta_x0;
            beta = 0.0;
        }
        else
        {   
            X[node_id] = X1[node_id];
            delta_x0 = delta_x1;
        }
        direction = delta_x1 + beta*direction;
        //Calculate gradient and residual gradient norms..
        cg::sync(block);
        smem[threadIdx.x] = dot(direction,direction); reduction(smem,p.gdata,N,single_block_fullerenes); direction_norm = sqrtf(smem[0]);
        cg::sync(block);
        smem[threadIdx.x] = dot(delta_x1,delta_x1); reduction(smem,p.gdata,N,single_block_fullerenes);dnorm = sqrtf(smem[0]);
        cg::sync(block);
        //Normalize gradient.
        direction /= direction_norm;
    }
    real_t Etest = energy(X, node_id, bookkeeping, constants, smem, p.gdata, N, single_block_fullerenes);
    if (threadIdx.x + blockDim.x == 0)
    {
        print(Etest);
    }
    print(Etest);
    
    reinterpret_cast<coord3d*>(p.X)[offset + threadIdx.x] = X[threadIdx.x];
}

__global__ void FullereneProperties(DevicePointers p){
    extern __shared__ coord3d sdata[];
    size_t offset = blockIdx.x * blockDim.x;
    coord3d* X = &reinterpret_cast<coord3d*>(p.X)[offset];


    BookkeepingData bookit = BookkeepingData(&p.neighbours[3*(offset)],&p.face_right[3*(offset)],&p.next_on_face[3*(offset)],&p.prev_on_face[3*(offset)]);
    Constants<coord3d> constants = compute_constants<coord3d>(bookit,threadIdx.x);
    BookkeepingData bdat = BookkeepingData(&p.neighbours[3*(offset + threadIdx.x)],&p.face_right[3*(offset + threadIdx.x)],&p.next_on_face[3*(offset + threadIdx.x)],&p.prev_on_face[3*(offset + threadIdx.x)]);

    //Use X1 buffer as storage array.
    coord3d* NodeEnergyCoord = &reinterpret_cast<coord3d*>(p.X1)[offset];
    real_t NodeEnergy = (real_t)0.0; real_t NodeBond_Error = (real_t)0.0; real_t NodeAngle_Error = (real_t)0.0; real_t NodeDihedral_Error = (real_t)0.0;
    real_t ArcEnergy = (real_t)0.0; real_t ArcBond_Error = (real_t)0.0; real_t ArcAngle_Error = (real_t)0.0; real_t ArcDihedral_Error = (real_t)0.0;
    
    
    for (uint8_t j = 0; j < 3; j++)
    {
        ArcData arc = ArcData(threadIdx.x, j, X, bdat);
        ArcEnergy = arc.dihedral_energy(constants);
        ArcBond_Error = abs(abs(arc.rab - get(constants.r0,j))/get(constants.r0,j));
        ArcAngle_Error =  abs(abs(arc.angle() - get(constants.angle0,j))/get(constants.angle0,j));
        ArcDihedral_Error = abs(abs(arc.dihedral() - get(constants.inner_dih0,j))/get(constants.inner_dih0,j));

        NodeEnergy += ArcEnergy; NodeBond_Error += ArcBond_Error; NodeAngle_Error += ArcAngle_Error; NodeDihedral_Error += ArcDihedral_Error;
        reinterpret_cast<real_t*>(NodeEnergyCoord)[threadIdx.x*3 + j] =  ArcEnergy;
        sdata[threadIdx.x*3 + j] = arc.gradient(constants);
    }
    real_t Energy = energy(X,threadIdx.x,bdat,constants,reinterpret_cast<real_t*>(sdata),p.gdata,blockDim.x,true);
    reinterpret_cast<real_t*>(sdata)[threadIdx.x] = NodeDihedral_Error/3.0; reduction(reinterpret_cast<real_t*>(sdata)); real_t AvgDihedralErr = reinterpret_cast<real_t*>(sdata)[0]/blockDim.x; cg::sync(cg::this_thread_block());
    reinterpret_cast<real_t*>(sdata)[threadIdx.x] = NodeAngle_Error/3.0; reduction(reinterpret_cast<real_t*>(sdata)); real_t AvgAngleErr = reinterpret_cast<real_t*>(sdata)[0]/blockDim.x; cg::sync(cg::this_thread_block());
    reinterpret_cast<real_t*>(sdata)[threadIdx.x] = NodeBond_Error/3.0 ; reduction(reinterpret_cast<real_t*>(sdata)); real_t AvgBondErr = reinterpret_cast<real_t*>(sdata)[0]/blockDim.x; cg::sync(cg::this_thread_block());
    /*
    if ((threadIdx.x + blockIdx.x * blockDim.x)== 0)
    {
        printf("Average Node Energy: "); print(Energy/blockDim.x); 
        printf("Average Bond Error: "); print(AvgBondErr);
        printf("Average Angle Error: "); print(AvgAngleErr);
        printf("Average Dihedral Error: "); print(AvgDihedralErr);
    }*/

    //Check convergence via gradient:
    reinterpret_cast<real_t*>(sdata)[threadIdx.x] = dot(gradient(X,threadIdx.x, bdat, constants),gradient(X,threadIdx.x, bdat, constants)); reduction(reinterpret_cast<real_t*>(sdata)); real_t GradSum = sqrt(reinterpret_cast<real_t*>(sdata)[0])/blockDim.x; cg::sync(cg::this_thread_block());

    if (threadIdx.x == 0){
    if (AvgBondErr < 5e-2)
    {
        p.gdata[blockIdx.x] = (real_t)1.0;
    } else
    {
       p.gdata[blockIdx.x] = (real_t)0.0;
    }}
    cg::sync(cg::this_grid());
    real_t Success = 0;
    if ((threadIdx.x + blockIdx.x * blockDim.x) == 0)
    {
        for (size_t i = 0; i < gridDim.x; i++)
        {
            Success += p.gdata[i];
        }
    }
    
    if( (threadIdx.x + blockIdx.x * blockDim.x)== 0){
        printf("%d", (size_t)Success); printf("/ %d Fullerenes Converged in Batch \n", gridDim.x);
    }   
}
__global__ void kernel_InternalCoordinates(DevicePointers p){
    size_t offset = blockIdx.x * blockDim.x;
    coord3d* X = &reinterpret_cast<coord3d*>(p.X)[offset];
    BookkeepingData bookit = BookkeepingData(&p.neighbours[3*(offset)],&p.face_right[3*(offset)],&p.next_on_face[3*(offset)],&p.prev_on_face[3*(offset)]);
    Constants<coord3d> constants = compute_constants<coord3d>(bookit,threadIdx.x);
    BookkeepingData bdat = BookkeepingData(&p.neighbours[3*(offset + threadIdx.x)],&p.face_right[3*(offset + threadIdx.x)],&p.next_on_face[3*(offset + threadIdx.x)],&p.prev_on_face[3*(offset + threadIdx.x)]);

    size_t tid = threadIdx.x + blockDim.x*blockIdx.x;
    for (uint8_t j = 0; j < 3; j++)
    {   
        ArcData arc = ArcData(threadIdx.x, j, X, bdat);
        p.bonds[tid*3 + j] = arc.bond();
        p.angles[tid*3 + j] = arc.angle();
        p.dihedrals[tid*3 + j] = arc.dihedral();
    }
}

__global__ void kernel_HarmonicConstants(DevicePointers p){
    size_t offset = blockIdx.x * blockDim.x;
    coord3d* X = &reinterpret_cast<coord3d*>(p.X)[offset];
    BookkeepingData bookit = BookkeepingData(&p.neighbours[3*(offset)],&p.face_right[3*(offset)],&p.next_on_face[3*(offset)],&p.prev_on_face[3*(offset)]);
    Constants<coord3d> constants = compute_constants<coord3d>(bookit,threadIdx.x);
    BookkeepingData bdat = BookkeepingData(&p.neighbours[3*(offset + threadIdx.x)],&p.face_right[3*(offset + threadIdx.x)],&p.next_on_face[3*(offset + threadIdx.x)],&p.prev_on_face[3*(offset + threadIdx.x)]);

    size_t tid = threadIdx.x + blockDim.x*blockIdx.x;
    for (uint8_t j = 0; j < 3; j++)
    {   
        p.bond_0[tid*3 + j] = get(constants.r0,j);
        p.angle_0[tid*3 + j] = get(constants.angle0,j);
        p.dihedral_0[tid*3 + j] = get(constants.inner_dih0,j);
    }
}

__global__ void kernel_Gradients(DevicePointers p){
    size_t offset = blockIdx.x * blockDim.x;
    coord3d* X = &reinterpret_cast<coord3d*>(p.X)[offset];
    BookkeepingData bookit = BookkeepingData(&p.neighbours[3*(offset)],&p.face_right[3*(offset)],&p.next_on_face[3*(offset)],&p.prev_on_face[3*(offset)]);
    Constants<coord3d> constants = compute_constants<coord3d>(bookit,threadIdx.x);
    BookkeepingData bdat = BookkeepingData(&p.neighbours[3*(offset + threadIdx.x)],&p.face_right[3*(offset + threadIdx.x)],&p.next_on_face[3*(offset + threadIdx.x)],&p.prev_on_face[3*(offset + threadIdx.x)]);

    size_t tid = threadIdx.x + blockDim.x*blockIdx.x;
    for (uint8_t j = 0; j < 3; j++)
    {   
        ArcData arc = ArcData(threadIdx.x, j, X, bdat);
        reinterpret_cast<coord3d*>(p.gradients)[tid] += gradient(X,threadIdx.x, bdat, constants);
    }
}
    
size_t computeBatchSize(size_t N){
    hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties,0);

    /** Compiling with --maxrregcount=64   is necessary to easily (singular blocks / fullerene) parallelize fullerenes of size 20-1024 !**/
    int fullerenes_per_block;
    
    /** Needs 3 storage arrays for coordinates and 1 for reductions **/
    int sharedMemoryPerBlock = sizeof(coord3d)* 3 * (N + 1) + sizeof(real_t)*N;

    /** Calculates maximum number of resident fullerenes on a single Streaming Multiprocessor, multiply with multi processor count to get total batch size**/
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&fullerenes_per_block, conjugate_gradient, N, sharedMemoryPerBlock);

    return (size_t)(properties.multiProcessorCount*fullerenes_per_block);
}

void AllocateDevicePointers(DevicePointers& p, size_t N, size_t batch_size){
    hipMalloc(&p.X, sizeof(coord3d)*N*batch_size);
    hipMalloc(&p.X1, sizeof(coord3d)*N*batch_size);
    hipMalloc(&p.X2, sizeof(coord3d)*N*batch_size);
    hipMalloc(&p.neighbours, sizeof(node_t)*3*N*batch_size);
    hipMalloc(&p.next_on_face, sizeof(node_t)*3*N*batch_size);
    hipMalloc(&p.prev_on_face, sizeof(node_t)*3*N*batch_size);
    hipMalloc(&p.face_right, sizeof(uint8_t)*3*N*batch_size);
    hipMalloc(&p.gdata, sizeof(real_t)*batch_size);
    hipMalloc(&p.bonds, sizeof(real_t)*3*N*batch_size);
    hipMalloc(&p.angles, sizeof(real_t)*3*N*batch_size);
    hipMalloc(&p.dihedrals, sizeof(real_t)*3*N*batch_size);
    hipMalloc(&p.bond_0, sizeof(real_t)*3*N*batch_size);
    hipMalloc(&p.angle_0, sizeof(real_t)*3*N*batch_size);
    hipMalloc(&p.dihedral_0, sizeof(real_t)*3*N*batch_size);
    hipMalloc(&p.gradients, sizeof(real_t)*3*N*batch_size);
}

void FreePointers(DevicePointers& p){
    hipFree(p.X);
    hipFree(p.X1);
    hipFree(p.X2);
    hipFree(p.neighbours);
    hipFree(p.next_on_face);
    hipFree(p.prev_on_face);
    hipFree(p.face_right);
    hipFree(p.gdata);
    hipFree(p.bonds);
    hipFree(p.angles);
    hipFree(p.dihedrals);
    hipFree(p.bond_0);
    hipFree(p.angle_0);
    hipFree(p.dihedral_0);
    hipFree(p.gradients);
    
    hipDeviceReset();
}

void CopyToDevice(DevicePointers& p, const HostPointers& h, const size_t N, const size_t batch_size){
    hipMemcpy(p.X, h.h_X, sizeof(coord3d)*N*batch_size , hipMemcpyHostToDevice);
    hipMemcpy(p.neighbours, h.h_cubic_neighbours, sizeof(node_t)*3*N*batch_size, hipMemcpyHostToDevice);
    hipMemcpy(p.next_on_face, h.h_next_on_face, sizeof(node_t)*3*N*batch_size, hipMemcpyHostToDevice);
    hipMemcpy(p.prev_on_face, h.h_prev_on_face, sizeof(node_t)*3*N*batch_size, hipMemcpyHostToDevice);
    hipMemcpy(p.face_right, h.h_face_right, sizeof(uint8_t)*3*N*batch_size, hipMemcpyHostToDevice);
}

void CheckBatch(DevicePointers& p, const HostPointers& h, const size_t N, const size_t batch_size){
    CopyToDevice(p,h,N,batch_size);
    void* kernelArgs[] = {(void*)&p};
    hipLaunchCooperativeKernel((void*)FullereneProperties, dim3(batch_size,1,1), dim3(N,1,1), kernelArgs, sizeof(coord3d)*3*N, NULL);
}

void InternalCoordinates(DevicePointers& p, const HostPointers& h, const size_t N, const size_t batch_size, real_t* bonds, real_t* angles, real_t* dihedrals){
    CopyToDevice(p,h,N,batch_size);
    void* kernelArgs[] = {(void*)&p};
    hipLaunchCooperativeKernel((void*)kernel_InternalCoordinates, dim3(batch_size,1,1), dim3(N,1,1), kernelArgs, sizeof(coord3d)*3*N, NULL);
    hipDeviceSynchronize();
    hipMemcpy(bonds, p.bonds, sizeof(coord3d)*N*batch_size , hipMemcpyDeviceToHost);
    hipMemcpy(angles, p.angles, sizeof(coord3d)*N*batch_size , hipMemcpyDeviceToHost);
    hipMemcpy(dihedrals, p.dihedrals, sizeof(coord3d)*N*batch_size , hipMemcpyDeviceToHost);
}

void HarmonicConstants(DevicePointers& p, const HostPointers& h, const size_t N, const size_t batch_size, real_t* bond_0, real_t* angle_0, real_t* dihedral_0){
    CopyToDevice(p,h,N,batch_size);
    void* kernelArgs[] = {(void*)&p};
    hipLaunchCooperativeKernel((void*)kernel_HarmonicConstants, dim3(batch_size,1,1), dim3(N,1,1), kernelArgs, sizeof(coord3d)*3*N, NULL);
    hipDeviceSynchronize();
    hipMemcpy(bond_0, p.bond_0, sizeof(coord3d)*N*batch_size , hipMemcpyDeviceToHost);
    hipMemcpy(angle_0, p.angle_0, sizeof(coord3d)*N*batch_size , hipMemcpyDeviceToHost);
    hipMemcpy(dihedral_0, p.dihedral_0, sizeof(coord3d)*N*batch_size , hipMemcpyDeviceToHost);
}

void Gradients(DevicePointers& p, const HostPointers& h, const size_t N, const size_t batch_size, real_t* gradients){
    CopyToDevice(p,h,N,batch_size);
    void* kernelArgs[] = {(void*)&p};
    hipLaunchCooperativeKernel((void*)kernel_Gradients, dim3(batch_size,1,1), dim3(N,1,1), kernelArgs, sizeof(coord3d)*3*N, NULL);
    hipDeviceSynchronize();
    hipMemcpy(gradients, p.gradients, sizeof(coord3d)*N*batch_size , hipMemcpyDeviceToHost);
}


void OptimizeBatch(DevicePointers& p, HostPointers& h, const size_t N, const size_t batch_size, const size_t MaxIter){
    bool single_block_fullerenes = true;

    getLastCudaError("One or more Mallocs Failed! \n");
    hipMemcpy(p.X, h.h_X, sizeof(coord3d)*N*batch_size , hipMemcpyHostToDevice);
    hipMemcpy(p.neighbours, h.h_cubic_neighbours, sizeof(node_t)*3*N*batch_size, hipMemcpyHostToDevice);
    hipMemcpy(p.next_on_face, h.h_next_on_face, sizeof(node_t)*3*N*batch_size, hipMemcpyHostToDevice);
    hipMemcpy(p.prev_on_face, h.h_prev_on_face, sizeof(node_t)*3*N*batch_size, hipMemcpyHostToDevice);
    hipMemcpy(p.face_right, h.h_face_right, sizeof(uint8_t)*3*N*batch_size, hipMemcpyHostToDevice);
    getLastCudaError("Memcpy Failed! \n");
    auto start = std::chrono::system_clock::now();
    void* kernelArgs[] = {
    (void*)&p,
    (void*)&N,
    (void*)&single_block_fullerenes,
    (void*)&MaxIter
    };

    hipLaunchCooperativeKernel((void*)conjugate_gradient, dim3(batch_size, 1, 1), dim3(N, 1, 1), kernelArgs, sizeof(coord3d)*3*(N+1) + sizeof(real_t)*N, NULL);
    hipDeviceSynchronize();
    auto end = std::chrono::system_clock::now();
    getLastCudaError("Failed to launch kernel: ");
    
    hipMemcpy(h.h_X, p.X, sizeof(coord3d)*N*batch_size , hipMemcpyDeviceToHost);
    getLastCudaError("Failed to copy back: ");
    
    std::cout << "Elapsed time: " << (end-start)/ 1ms << "ms\n" ;
    std::cout << "Estimated Performance " << ((real_t)(batch_size)/(std::chrono::duration_cast<std::chrono::microseconds>(end-start)).count()) * 1.0e6 << "Fullerenes/s \n";
}

};

#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <cooperative_groups/scan.h>
#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
//#define FLOAT_TYPE 2
#include "fullerenes/gpu/cuda_definitions.h"
#include "fullerenes/gpu/cu_array.hh"
#include "fullerenes/gpu/isomer_batch.hh"
#include "fullerenes/gpu/kernels.hh"

namespace gpu_kernels{
    namespace isomerspace_properties{
        #include "device_includes.cu"
        symMat3 __device__ inertia_matrix(const device_coord3d* X){
            DEVICE_TYPEDEFS;
            extern __shared__ real_t smem[];
            clear_cache(smem, blockDim.x);
            int tid = threadIdx.x;
            symMat3 I;
            real_t diag = reduction(smem, dot(X[tid], X[tid]));
            I.a = diag;
            I.d = diag;
            I.f = diag;
            I.a -= reduction(smem, X[tid][0]*X[tid][0]);
            I.b -= reduction(smem, X[tid][0]*X[tid][1]);
            I.c -= reduction(smem, X[tid][0]*X[tid][2]);
            I.d -= reduction(smem, X[tid][1]*X[tid][1]);
            I.e -= reduction(smem, X[tid][1]*X[tid][2]);
            I.f -= reduction(smem, X[tid][2]*X[tid][2]);
            return I;
        }

     
      std::array<device_coord3d,3> __device__ principal_axes(const device_coord3d* X){
            DEVICE_TYPEDEFS;
            auto I = inertia_matrix(X);
	    auto [V,lambdas] = I.eigensystem();
	    return V;
        }

        //Returns the best ellipsoid for the given coordinates, lambda0 = a, lambda1 = b, lambda2 = c.
        device_coord3d __device__ best_ellipsoid (const device_coord3d* X){
            DEVICE_TYPEDEFS;
            auto I = inertia_matrix(X);
            return rsqrt3(d_sort(d_abs(I.eigenvalues()))); 
        }

        void __global__ transform_coordinates_(IsomerBatch B){
            DEVICE_TYPEDEFS;
            extern __shared__ real_t shared_memory[];
            const int tid = threadIdx.x;
            auto limit = ((B.isomer_capacity + gridDim.x - 1) / gridDim.x ) * gridDim.x;  //Fast ceiling integer division.
            for (int isomer_idx = blockIdx.x; isomer_idx < limit; isomer_idx += gridDim.x){
                if (isomer_idx < B.isomer_capacity) if(B.statuses[isomer_idx] != IsomerStatus::EMPTY)
                {
                clear_cache(shared_memory, blockDim.x);
                BLOCK_SYNC
                size_t offset = isomer_idx * blockDim.x;
                coord3d* X              = reinterpret_cast<coord3d*>(shared_memory) + B.n_atoms;
                assign(X[threadIdx.x],reinterpret_cast<std::array<float,3>*>(B.X+offset*3)[threadIdx.x]);
                BLOCK_SYNC
                coord3d centroid = {reduction(shared_memory, X[tid][0]), reduction(shared_memory, X[tid][1]), reduction(shared_memory, X[tid][2])};
                X[tid] -= centroid/real_t(B.n_atoms);
                BLOCK_SYNC
		            mat3 P{principal_axes(X)};
                if (ISNAN(P[0][0]) || ISNAN(P[0][1]) || ISNAN(P[0][2]) || ISNAN(P[1][0]) || ISNAN(P[1][1]) || ISNAN(P[1][2]) || ISNAN(P[2][0]) || ISNAN(P[2][1]) || ISNAN(P[2][2])) {
                    //assert(false);
                    return;
                } 
                BLOCK_SYNC
                X[tid] = dot(P, X[tid]);
                BLOCK_SYNC
                assign(reinterpret_cast<std::array<float,3>*>(B.X)[offset + threadIdx.x], X[threadIdx.x]);
                //sequential_print(P[0][0],0);
                }
            }
            
        }

        void __global__ moments_of_inertia_(IsomerBatch B, CuArray<device_real_t> lambdas){
            DEVICE_TYPEDEFS;
            extern __shared__ real_t shared_memory[];
            const int tid = threadIdx.x;
            auto limit = ((B.isomer_capacity + gridDim.x - 1) / gridDim.x ) * gridDim.x;  //Fast ceiling integer division.
            for (int isomer_idx = blockIdx.x; isomer_idx < limit; isomer_idx += gridDim.x){
	      //TODO: simplify
                if (isomer_idx < B.isomer_capacity) if(B.statuses[isomer_idx] == IsomerStatus::CONVERGED)
                {
		BLOCK_SYNC;
                size_t offset = isomer_idx * blockDim.x;
                coord3d* X           = reinterpret_cast<coord3d*>(shared_memory) + B.n_atoms;
                assign(X[threadIdx.x],reinterpret_cast<std::array<float,3>*>(B.X+offset*3)[threadIdx.x]);
                BLOCK_SYNC;
		auto I = inertia_matrix(X);
		device_coord3d lams = I.eigenvalues(); 
                BLOCK_SYNC;
                if (tid == 0){ lambdas.data[3*isomer_idx] = lams[0];lambdas.data[3*isomer_idx+1] = lams[0];lambdas.data[3*isomer_idx+2] = lams[2]; }
                }
            }
        }
        
      
      
        void __global__ eccentricities_(IsomerBatch B, CuArray<device_real_t> ecce){
            DEVICE_TYPEDEFS;
            extern __shared__ real_t shared_memory[];
            const int tid = threadIdx.x;
            auto limit = ((B.isomer_capacity + gridDim.x - 1) / gridDim.x ) * gridDim.x;  //Fast ceiling integer division.
            for (int isomer_idx = blockIdx.x; isomer_idx < limit; isomer_idx += gridDim.x){
                if (isomer_idx < B.isomer_capacity) if(B.statuses[isomer_idx] == IsomerStatus::CONVERGED)
                {
                BLOCK_SYNC
                size_t offset = isomer_idx * blockDim.x;
                coord3d* X            = reinterpret_cast<coord3d*>(shared_memory) + B.n_atoms;
                assign(X[threadIdx.x],reinterpret_cast<std::array<float,3>*>(B.X+offset*3)[threadIdx.x]);
                BLOCK_SYNC
                auto result = best_ellipsoid(X);
                BLOCK_SYNC
                if (tid == 0) ecce.data[isomer_idx] = result[0] / result[2];
                }
            }
        }
        
        void __global__ volume_divergences_(IsomerBatch B, CuArray<device_real_t> vd){
            DEVICE_TYPEDEFS;
            typedef device_node3 tri_t;
            extern __shared__ real_t smems[];
            const int tid = threadIdx.x;
            auto limit = ((B.isomer_capacity + gridDim.x - 1) / gridDim.x ) * gridDim.x;  //Fast ceiling integer division.
            for (int isomer_idx = blockIdx.x; isomer_idx < limit; isomer_idx += gridDim.x){
                if (isomer_idx < B.isomer_capacity) if(B.statuses[isomer_idx] == IsomerStatus::CONVERGED)
                {
                BLOCK_SYNC
                size_t offset = isomer_idx * blockDim.x;
                NodeNeighbours node_graph    = NodeNeighbours(B, isomer_idx, smems);
                coord3d* X              = reinterpret_cast<coord3d*>(smems) + B.n_atoms;
                assign(X[threadIdx.x],reinterpret_cast<std::array<float,3>*>(B.X+offset*3)[threadIdx.x]);
                BLOCK_SYNC
                real_t V = 0.;
                if (tid < B.n_faces) {
                    coord3d face_center = (coord3d){0.,0.,0.};
                    for (int i = 0; i < node_graph.face_size; i++) face_center += X[node_graph.face_nodes[i]];
                    face_center /= node_graph.face_size; //The center of the threadIdx.x-th face.

                    for (int i = 0; i < node_graph.face_size; i++){
                        coord3d a = X[node_graph.face_nodes[i]];
                        coord3d b = X[node_graph.face_nodes[(i+1)%node_graph.face_size]];
                        coord3d c = face_center;
                        coord3d u = b - a;
                        coord3d v = c - a;
                        coord3d n = cross(u,v);
                        V += dot(a,n) / real_t(2.0);
                    }
                }
                clear_cache(smems, blockDim.x);
                auto result = reduction(smems, V)/real_t(3.0);
                if (tid == 0) vd.data[isomer_idx] = result;
                }
            }
        }

        void __global__ surface_areas_(IsomerBatch B, CuArray<device_real_t> sa){
            DEVICE_TYPEDEFS;
            typedef device_node3 tri_t;
            extern __shared__ real_t smems[];
            const int tid = threadIdx.x;
            auto limit = ((B.isomer_capacity + gridDim.x - 1) / gridDim.x ) * gridDim.x;  //Fast ceiling integer division.
            for (int isomer_idx = blockIdx.x; isomer_idx < limit; isomer_idx += gridDim.x){
                if (isomer_idx < B.isomer_capacity) if(B.statuses[isomer_idx] != IsomerStatus::EMPTY)
                {
                BLOCK_SYNC
                size_t offset = isomer_idx * blockDim.x;
                NodeNeighbours node_graph    = NodeNeighbours(B, isomer_idx, smems);
                coord3d* X              = reinterpret_cast<coord3d*>(smems) + B.n_atoms;
                assign(X[threadIdx.x],reinterpret_cast<std::array<float,3>*>(B.X+offset*3)[threadIdx.x]);
                BLOCK_SYNC
                real_t A = real_t(0.);
                if (tid < B.n_faces) {
                    coord3d face_center = (coord3d){0.,0.,0.};
                    for (int i = 0; i < node_graph.face_size; i++) face_center += X[node_graph.face_nodes[i]];
                    face_center /= node_graph.face_size; //The center of the threadIdx.x-th face.

                    for (int i = 0; i < node_graph.face_size; i++){
                        coord3d a = X[node_graph.face_nodes[i]];
                        coord3d b = X[node_graph.face_nodes[(i+1)%node_graph.face_size]];
                        coord3d c = face_center;
                        coord3d u = b - a;
                        coord3d v = c - a;
                        coord3d n = cross(u,v);
                        A += norm(n);
                    }
                }
                clear_cache(smems, blockDim.x);
                auto result = reduction(smems, A)/real_t(2.0);
                if (tid == 0) sa.data[isomer_idx] = result;
                }
            }
        }

        void __global__ debug_function_(IsomerBatch B, CuArray<device_real_t> eigenvalues, CuArray<device_real_t> eigenvectors, CuArray<device_real_t> inertia_matrices, CuArray<device_real_t> orthogonality){
            DEVICE_TYPEDEFS;
            extern __shared__ real_t shared_memory[];
            const int tid = threadIdx.x;
            auto limit = ((B.isomer_capacity + gridDim.x - 1) / gridDim.x ) * gridDim.x;  //Fast ceiling integer division.
            for (int isomer_idx = blockIdx.x; isomer_idx < limit; isomer_idx += gridDim.x){
                if (isomer_idx < B.isomer_capacity)
                if (B.statuses[isomer_idx] == IsomerStatus::CONVERGED)
                {
                BLOCK_SYNC
                size_t offset = isomer_idx * blockDim.x;
                Constants constants          = Constants(B, isomer_idx);
                NodeNeighbours node_graph    = NodeNeighbours(B, isomer_idx, shared_memory);
                coord3d* X              = reinterpret_cast<coord3d*>(shared_memory) + B.n_atoms;
                assign(X[threadIdx.x],reinterpret_cast<std::array<float,3>*>(B.X+offset*3)[threadIdx.x]);
                BLOCK_SYNC
                auto I = inertia_matrix(X);
                BLOCK_SYNC
                if (tid == 0){
		  auto [P,eigs] = I.eigensystem();		  

		  reinterpret_cast<coord3d*>(eigenvalues.data) [isomer_idx] = eigs;
		  reinterpret_cast<coord3d*>(eigenvectors.data)[isomer_idx*3+0] = P[0];
		  reinterpret_cast<coord3d*>(eigenvectors.data)[isomer_idx*3+1] = P[1];
		  reinterpret_cast<coord3d*>(eigenvectors.data)[isomer_idx*3+2] = P[2];
		
		  real_t orthog = real_t(1.0);

		  if(ISNAN(eigs[0]) || ISNAN(eigs[1]) || ISNAN(eigs[2]) || ISNAN(I.a) || ISNAN(I.b) || ISNAN(I.c) || ISNAN(I.d) || ISNAN(I.e) || ISNAN(I.f)){
                    orthog = real_t(2.0);
		  } else if ((ABS(eigs[0]- eigs[1])/ABS(eigs[0]) < 1e-5) && (ABS(eigs[1]- eigs[2])/ABS(eigs[0]) < 1e-5)){
                    orthog = real_t(3.0);
		  } else if (ABS(eigs[0] - eigs[1])/ABS(eigs[0]) < 1e-5) {
                    orthog = ABS(dot(P[0], P[2]));
		  } else {
                    orthog = ABS(dot(P[0], P[1]));
		  }

		  orthogonality.data[isomer_idx] = orthog;
		  reinterpret_cast<coord3d*>(inertia_matrices.data)[isomer_idx*3] = {I.a, I.b, I.c};
		  reinterpret_cast<coord3d*>(inertia_matrices.data)[isomer_idx*3+1] = {I.b, I.d, I.e};
		  reinterpret_cast<coord3d*>(inertia_matrices.data)[isomer_idx*3+2] = {I.c, I.e, I.f};
                }
                }
            }
        }



        hipError_t transform_coordinates(IsomerBatch& B, const LaunchCtx& ctx, const LaunchPolicy policy){
            hipSetDevice(B.get_device_id());
                
            //If launch ploicy is synchronous then wait.
            if(policy == LaunchPolicy::SYNC) ctx.wait();

            size_t smem = sizeof(device_coord3d)* (3*B.n_atoms + 4) + sizeof(device_real_t)*Block_Size_Pow_2;
            static LaunchDims dims((void*)transform_coordinates_, B.n_atoms, smem, B.isomer_capacity);
            dims.update_dims((void*)transform_coordinates_, B.n_atoms, smem, B.isomer_capacity);
            void* kargs[]{(void*)&B};
            auto error = safeCudaKernelCall((void*)transform_coordinates_, dims.get_grid(), dims.get_block(), kargs, smem, ctx.stream);
    
            if(policy == LaunchPolicy::SYNC) ctx.wait();
            printLastCudaError("Transformation of Coordinates Failed: ");
            return error;
        }


        hipError_t eccentricities(const IsomerBatch& B, CuArray<device_real_t>& eccentricities, const LaunchCtx& ctx, const LaunchPolicy policy){
            hipSetDevice(B.get_device_id());

            //If launch ploicy is synchronous then wait.
            if(policy == LaunchPolicy::SYNC) ctx.wait();

            size_t smem = sizeof(device_coord3d)* (3*B.n_atoms + 4) + sizeof(device_real_t)*Block_Size_Pow_2;
            static LaunchDims dims((void*)eccentricities_, B.n_atoms, smem, B.isomer_capacity);
            dims.update_dims((void*)eccentricities_, B.n_atoms, smem, B.isomer_capacity);
            void* kargs[]{(void*)&B, (void*)&eccentricities};
            auto error = safeCudaKernelCall((void*)eccentricities_, dims.get_grid(), dims.get_block(), kargs, smem, ctx.stream);

            if(policy == LaunchPolicy::SYNC) ctx.wait();
            printLastCudaError("Calculation of Eccentricities Failed: ");
            return error;
        }

      hipError_t moments_of_inertia(const IsomerBatch& B, CuArray<device_real_t>& lambdas, const LaunchCtx& ctx, const LaunchPolicy policy){
            hipSetDevice(B.get_device_id());

            //If launch ploicy is synchronous then wait.
            if(policy == LaunchPolicy::SYNC) ctx.wait();

            size_t smem = sizeof(device_coord3d)* (3*B.n_atoms + 4) + sizeof(device_real_t)*Block_Size_Pow_2;
            static LaunchDims dims((void*)moments_of_inertia_, B.n_atoms, smem, B.isomer_capacity);
            dims.update_dims((void*)moments_of_inertia_, B.n_atoms, smem, B.isomer_capacity);
            void* kargs[]{(void*)&B, (void*)&lambdas};
            auto error = safeCudaKernelCall((void*)moments_of_inertia_, dims.get_grid(), dims.get_block(), kargs, smem, ctx.stream);

            if(policy == LaunchPolicy::SYNC) ctx.wait();
            printLastCudaError("Calculation of moments of inertia Failed: ");
            return error;
        }

        hipError_t surface_areas(const IsomerBatch& B, CuArray<device_real_t>& surface_areas, const LaunchCtx& ctx, const LaunchPolicy policy){
            hipSetDevice(B.get_device_id());

            //If launch ploicy is synchronous then wait.
            if(policy == LaunchPolicy::SYNC) ctx.wait();

            size_t smem = sizeof(device_coord3d)* (3*B.n_atoms + 4) + sizeof(device_real_t)*Block_Size_Pow_2;
            static LaunchDims dims((void*)surface_areas_, B.n_atoms, smem, B.isomer_capacity);
            dims.update_dims((void*)surface_areas_, B.n_atoms, smem, B.isomer_capacity);
            void* kargs[]{(void*)&B, (void*)&surface_areas};
            auto error = safeCudaKernelCall((void*)surface_areas_, dims.get_grid(), dims.get_block(), kargs, smem, ctx.stream);

            if(policy == LaunchPolicy::SYNC) ctx.wait();
            printLastCudaError("Calculation of Volume Divergences Failed: ");
            return error;
        }

        hipError_t volume_divergences(const IsomerBatch& B, CuArray<device_real_t>& volumes, const LaunchCtx& ctx, const LaunchPolicy policy){
            hipSetDevice(B.get_device_id());

            //If launch ploicy is synchronous then wait.
            if(policy == LaunchPolicy::SYNC) ctx.wait();

            size_t smem = sizeof(device_coord3d)* (3*B.n_atoms + 4) + sizeof(device_real_t)*Block_Size_Pow_2;
            static LaunchDims dims((void*)volume_divergences_, B.n_atoms, smem, B.isomer_capacity);
            dims.update_dims((void*)volume_divergences_, B.n_atoms, smem, B.isomer_capacity);
            void* kargs[]{(void*)&B, (void*)&volumes};
            auto error = safeCudaKernelCall((void*)volume_divergences_, dims.get_grid(), dims.get_block(), kargs, smem, ctx.stream);

            if(policy == LaunchPolicy::SYNC) ctx.wait();
            printLastCudaError("Calculation of Volume Divergences Failed: ");
            return error;
        }

        hipError_t debug_function(const IsomerBatch& B, CuArray<device_real_t>& eigenvalues, CuArray<device_real_t>& eigenvectors, CuArray<device_real_t>& inertia_matrices, CuArray<device_real_t>& orthogonality, const LaunchCtx& ctx, const LaunchPolicy policy){
            hipSetDevice(B.get_device_id());

            //If launch ploicy is synchronous then wait.
            if(policy == LaunchPolicy::SYNC) ctx.wait();

            size_t smem = sizeof(device_coord3d)* (3*B.n_atoms + 4) + sizeof(device_real_t)*Block_Size_Pow_2;
            static LaunchDims dims((void*)debug_function_, B.n_atoms, smem, B.isomer_capacity);
            dims.update_dims((void*)debug_function_, B.n_atoms, smem, B.isomer_capacity);
            void* kargs[]{(void*)&B, (void*)&eigenvalues, (void*)&eigenvectors, (void*)&inertia_matrices, (void*)&orthogonality};
            auto error = safeCudaKernelCall((void*)debug_function_, dims.get_grid(), dims.get_block(), kargs, smem, ctx.stream);

            if(policy == LaunchPolicy::SYNC) ctx.wait();
            printLastCudaError("Debug Function Failed: ");
            return error;
        }





    }
}

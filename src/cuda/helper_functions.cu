#include "hip/hip_runtime.h"
#include "coord3d.cu"
#include "coord3d_aligned.cu"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include "hip/hip_runtime.h"
#include <assert.h>


namespace cg = cooperative_groups;

template <typename T>
void copy_and_append(T* memory, const T* fullerene, size_t N){
    for (size_t i = 0; i < N; i++)
    {
        memory[i] = fullerene[i];
    }
}

template <typename T>
T* synthetic_array(size_t N, const size_t num_molecules, const T* fullerene){
    size_t array_size = N;
    if (sizeof(T) != sizeof(coord3d))
    {
        array_size *= 3;
    }
    T* storage_array = new T[array_size*num_molecules];
    for (size_t i = 0; i < num_molecules; i++)
    {
        copy_and_append(&storage_array[array_size*i],fullerene,array_size);
    }
    return storage_array;
}


__device__ void align16(coord3d* input, coord3d_a* output, size_t N){
    cg::sync(cg::this_grid());
    output[threadIdx.x] = {input[threadIdx.x].x, input[threadIdx.x].y, input[threadIdx.x].z, 0};
    cg::sync(cg::this_grid());
}

template <typename T>
__device__ void pointerswap(T **r, T **s)
{
    T *pSwap = *r;
    *r = *s;
    *s = pSwap;
    return;
}



//Pentagons = 0
//Hexagons = 1
//PPP = 0, {HPP, PHP, PPH} = 1, {PHH, HPH, HHP} = 2, {HHH} = 3
__constant__ real_t optimal_corner_cos_angles[2] = {-0.3090169944, -0.5}; 
__constant__ real_t optimal_bond_lengths[3] = {1.479, 1.458, 1.401}; 
__constant__ real_t optimal_dih_cos_angles[8] = {0.79465455715, 0.87290360705, 0.87290360705, 0.9410338473, 0.816287936, 0.913965949, 0.913965949, 1}; 

__constant__ real_t angle_forces[2] = {207.924,216.787}; 
__constant__ real_t bond_forces[3] = {260.0, 353.377, 518.992}; 
__constant__ real_t dih_forces[4] = {35.0,65.0,3.772,270.0}; 

__device__ __host__ struct BookkeepingData{
    const node_t* neighbours;
    const uint8_t* face_right;
    const node_t* next_on_face;
    const node_t* prev_on_face;
    __device__ __host__ BookkeepingData (const node_t* neighbours, const uint8_t* face_right, const node_t* next_on_face, const node_t* prev_on_face) : 
        neighbours(neighbours), face_right(face_right), next_on_face(next_on_face), prev_on_face(prev_on_face) {}
};




template <typename T>
__device__ struct Constants{
    const T f_bond;
    const T f_inner_angle;
    const T f_inner_dihedral;
    const T f_outer_angle_m;
    const T f_outer_angle_p;
    const T f_outer_dihedral;

    const T r0;
    const T angle0;
    const T outer_angle_m0;
    const T outer_angle_p0;
    const T inner_dih0;
    const T outer_dih0_a;
    const T outer_dih0_m;
    const T outer_dih0_p;
    
    __device__ Constants(const T f_bond, const T f_inner_angle, const T f_inner_dihedral, const T f_outer_angle_m, const T f_outer_angle_p, const T f_outer_dihedral,
                            const T r0, const T angle0, const T outer_angle_m0, const T outer_angle_p0, const T inner_dih0, const T outer_dih0_a, const T outer_dih0_m, const T outer_dih0_p): f_bond(f_bond), f_inner_angle(f_inner_angle),
                            f_inner_dihedral(f_inner_dihedral), f_outer_angle_m(f_outer_angle_m), f_outer_angle_p(f_outer_angle_p), f_outer_dihedral(f_outer_dihedral), r0(r0), angle0(angle0), outer_angle_m0(outer_angle_m0), outer_angle_p0(outer_angle_p0),
                            inner_dih0(inner_dih0), outer_dih0_a(outer_dih0_a), outer_dih0_m(outer_dih0_m), outer_dih0_p(outer_dih0_p) {}

};

__device__ struct EnergyConstants{
    const coord3d f_bond;
    const coord3d f_inner_angle;
    const coord3d f_inner_dihedral;

    const coord3d r0;
    const coord3d angle0;
    const coord3d inner_dih0;
    __device__ EnergyConstants(const coord3d f_bond, const coord3d f_inner_angle, const coord3d f_inner_dihedral, const coord3d r0, const coord3d angle0, const coord3d inner_dih0): f_bond(f_bond), f_inner_angle(f_inner_angle),
                            f_inner_dihedral(f_inner_dihedral),r0(r0), angle0(angle0), inner_dih0(inner_dih0) {}
};

__device__ struct ArcConstants{
    const real_t f_bond;
    const real_t f_inner_angle;
    const real_t f_inner_dihedral;
    const real_t f_outer_angle_m;
    const real_t f_outer_angle_p;
    const real_t f_outer_dihedral;

    const real_t r0;
    const real_t angle0;
    const real_t outer_angle_m0;
    const real_t outer_angle_p0;
    const real_t inner_dih0;
    const real_t outer_dih0;
    
    __device__ ArcConstants(const real_t f_bond, const real_t f_inner_angle, const real_t f_inner_dihedral, const real_t f_outer_angle_m, const real_t f_outer_angle_p, const real_t f_outer_dihedral,
                            const real_t r0, const real_t angle0, const real_t outer_angle_m0, const real_t outer_angle_p0, const real_t inner_dih0, const real_t outer_dih0): f_bond(f_bond), f_inner_angle(f_inner_angle),
                            f_inner_dihedral(f_inner_dihedral), f_outer_angle_m(f_outer_angle_m), f_outer_angle_p(f_outer_angle_p), f_outer_dihedral(f_outer_dihedral), r0(r0), angle0(angle0), outer_angle_m0(outer_angle_m0), outer_angle_p0(outer_angle_p0),
                            inner_dih0(inner_dih0), outer_dih0(outer_dih0) {}

};

__device__ __host__ uint8_t face_index(uint8_t f1, uint8_t f2, uint8_t f3){
    return f1*4 + f2*2 + f3;
}
template <typename T>
__device__ Constants<T> compute_constants(BookkeepingData& dat, node_t node_id){
    T r0, angle0, inner_dih0, outer_angle_m0, outer_angle_p0, outer_dih0_a, outer_dih0_m, outer_dih0_p;
    T f_bond, f_inner_angle, f_inner_dihedral, f_outer_angle_m, f_outer_angle_p, f_outer_dihedral ;
    
    for (uint8_t j = 0; j < 3; j++) {
        uint8_t f_r = dat.face_right[node_id * 3 + j] - 5;
        uint8_t f_l = dat.face_right[node_id * 3 + (2 + j)%3] - 5;

        uint8_t face_sum = dat.face_right[node_id * 3] - 5 + dat.face_right[node_id * 3 + 1] - 5 + dat.face_right[node_id * 3 + 2] - 5;
        uint8_t dihedral_face_sum = dat.face_right[dat.neighbours[node_id * 3 + j] * 3]-5 + dat.face_right[dat.neighbours[node_id * 3 + j] * 3 + 1]-5 +  dat.face_right[dat.neighbours[node_id * 3 + j] * 3 + 2]-5;

        //Load equillibirium distance, angles and dihedral angles from face information.
        set(r0,j,optimal_bond_lengths[ f_l + f_r ]);
        set(angle0,j,optimal_corner_cos_angles[ f_r ]);
        set(inner_dih0,j,optimal_dih_cos_angles[ face_index(dat.face_right[node_id * 3 + j] - 5, dat.face_right[node_id * 3 + (1+j)%3] - 5 , dat.face_right[node_id * 3 + (2+j)%3] - 5) ]);
        set(outer_angle_m0,j,optimal_corner_cos_angles[ f_l ]);
        set(outer_angle_p0,j,optimal_corner_cos_angles[ f_r ]);


        uint8_t dihedral_index_a = face_index(f_l,dat.face_right[node_id * 3 + (1 + j)%3] - 5,f_r);
        uint8_t dihedral_index_m =  face_index(dat.face_right[node_id * 3 + (1 + j)%3] - 5, f_r, f_l);
        uint8_t dihedral_index_p = face_index(f_r,f_l, dat.face_right[node_id * 3 + (1 + j)%3] - 5);

        set(outer_dih0_a,j,optimal_dih_cos_angles[dihedral_index_a]  );
        set(outer_dih0_m,j,optimal_dih_cos_angles[dihedral_index_m]  );
        set(outer_dih0_p,j,optimal_dih_cos_angles[dihedral_index_p]  );

        //Load force constants from neighbouring face information.
        set(f_bond,j,bond_forces[ f_l + f_r ]);
        set(f_inner_angle,j,angle_forces[ f_r ]);
        set(f_inner_dihedral,j,dih_forces[ face_sum]);
        set(f_outer_angle_m,j,angle_forces[ f_l ]);
        set(f_outer_angle_p,j,angle_forces[ f_r ]);
        set(f_outer_dihedral,j,dih_forces[ dihedral_face_sum]);
    }
    return Constants<T>(f_bond,f_inner_angle,f_inner_dihedral, f_outer_angle_m, f_outer_angle_p, f_outer_dihedral, r0, angle0, outer_angle_m0, outer_angle_p0, inner_dih0, outer_dih0_a, outer_dih0_m, outer_dih0_p);
}

__device__ ArcConstants compute_arc_constants(BookkeepingData &dat, node_t node_id, uint8_t j){
    real_t r0 ; real_t angle0 ; real_t inner_dih0 ; real_t outer_angle_m0 ; real_t outer_angle_p0 ; real_t outer_dih0 ;
    real_t f_bond ; real_t f_inner_angle ; real_t f_inner_dihedral ; real_t f_outer_angle_m ; real_t f_outer_angle_p ; real_t f_outer_dihedral ;
    
    

    uint8_t f_r = dat.face_right[node_id * 3 + j] - 5;
    uint8_t f_l = dat.face_right[node_id * 3 + (2 + j)%3] - 5;

    uint8_t face_sum = dat.face_right[node_id * 3] - 5 + dat.face_right[node_id * 3 + 1] - 5 + dat.face_right[node_id * 3 + 2] - 5;
    uint8_t dihedral_face_sum = dat.face_right[dat.neighbours[node_id * 3 + j] * 3]-5 + dat.face_right[dat.neighbours[node_id * 3 + j] * 3 + 1]-5 +  dat.face_right[dat.neighbours[node_id * 3 + j] * 3 + 2]-5;

    //Load equillibirium distance, angles and dihedral angles from face information.
    r0 = optimal_bond_lengths[ f_l + f_r ];
    angle0 = optimal_corner_cos_angles[ f_r ];
    inner_dih0 = optimal_dih_cos_angles[ face_sum ];
    outer_angle_m0 = optimal_corner_cos_angles[ f_l ];
    outer_angle_p0 = optimal_corner_cos_angles[ f_r ];
    outer_dih0 = optimal_dih_cos_angles[ dihedral_face_sum ];

    //Load force constants from neighbouring face information.
    f_bond = bond_forces[ f_l + f_r ];
    f_inner_angle = angle_forces[ f_l ];
    f_inner_dihedral = dih_forces[ face_sum];
    f_outer_angle_m = angle_forces[ f_r ];
    f_outer_angle_p = angle_forces[ f_l ];
    f_outer_dihedral = dih_forces[ dihedral_face_sum];
        

    return ArcConstants(f_bond,f_inner_angle,f_inner_dihedral, f_outer_angle_m, f_outer_angle_p, f_outer_dihedral, r0, angle0, outer_angle_m0, outer_angle_p0, inner_dih0, outer_dih0);
}


//Reduction method for single block fullerenes.
__device__ void reduction(real_t *sdata){

    cg::thread_block block = cg::this_thread_block();
    cg::sync(block);
    cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(block);
    sdata[threadIdx.x] = cg::reduce(tile32, sdata[threadIdx.x], cg::plus<real_t>());
    cg::sync(block);

    real_t beta = 0.0;
    if (block.thread_rank() == 0) {
        beta  = 0;
        for (uint16_t i = 0; i < block.size(); i += tile32.size()) {
            beta  += sdata[i];
        }
        sdata[0] = beta;
    }
    cg::sync(block);
}


__device__ void reduction(half *sdata){

    cg::thread_block block = cg::this_thread_block();
    cg::sync(block);
    cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(block);
    sdata[threadIdx.x] = cg::reduce(tile32, sdata[threadIdx.x], cg::plus<half>());
    cg::sync(block);

    half beta = 0.0;
    if (block.thread_rank() == 0) {
        beta  = 0;
        for (uint16_t i = 0; i < block.size(); i += tile32.size()) {
            beta  += sdata[i];
        }
        sdata[0] = beta;
    }
    cg::sync(block);
}

//Multi purpose reduction algorithm (Small or Large fullerenes).
__device__ void reduction(real_t *sdata, real_t *gdata, const node_t N, const bool single_block_fullerenes){
    cg::thread_block block = cg::this_thread_block();

    cg::sync(block);
    if (((threadIdx.x + blockIdx.x * blockDim.x) >= N) && !single_block_fullerenes)
    {
        sdata[threadIdx.x] = 0;
    }
    cg::sync(block);
    cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(block);
    sdata[threadIdx.x] = cg::reduce(tile32, sdata[threadIdx.x], cg::plus<real_t>());
    cg::sync(block);
    
    real_t beta = 0.0;
    if (single_block_fullerenes)
    {
        if (block.thread_rank() == 0) {
            for (uint16_t i = 0; i < block.size(); i += tile32.size()) {
                beta  += sdata[i];
            }
            sdata[0] = beta;
        }
        cg::sync(block);
    }
    else 
    {   
        auto grid = cg::this_grid();
        if (block.thread_rank() == 0) 
        {
            for (uint16_t i = 0; i < block.size(); i += tile32.size()) 
            {
                beta  += sdata[i];
            }
            gdata[blockIdx.x] = beta;
        }
        cg::sync(grid);
        beta = 0.0;
        if (grid.thread_rank() == 0)
        {
            for (uint16_t i = 0; i < gridDim.x; i++) 
            {
                beta  += gdata[i];
            }
            gdata[0] = beta;
        }
        cg::sync(grid);
        if (block.thread_rank() == 0) {sdata[0] = gdata[0];}
        cg::sync(grid);
    }
}

template < class T >
size_t optimize_block_size(size_t N, hipDeviceProp_t prop, T kernel){
    int maxActiveBlocks;
    size_t best_size = prop.warpSize;
    size_t min_waste = prop.maxThreadsPerMultiProcessor;
    for (size_t blocksize = prop.warpSize*2; blocksize < prop.maxThreadsPerBlock; blocksize +=prop.warpSize)
    {
        hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxActiveBlocks, kernel, (int)blocksize, (size_t)(sizeof(real_t)*blocksize));
        size_t wasted_threads = N % blocksize  +  blocksize*maxActiveBlocks % prop.maxThreadsPerMultiProcessor;

        if (wasted_threads < min_waste)
        {
            min_waste = wasted_threads;
            best_size = blocksize;
        }
    }
    return best_size;
}


__device__ real_t AverageBondLength(real_t* smem,const coord3d* X, const node_t* neighbours,const node_t node_id, const size_t N){
    real_t node_average_bond_length = 0.0;
    for (size_t i = 0; i < 3; i++)
    {
        node_average_bond_length += non_resciprocal_bond_length(X[node_id] - X[neighbours[i]]);
    }
    node_average_bond_length /= (real_t)3.0;
    cg::sync(cg::this_thread_block());
    smem[threadIdx.x] = node_average_bond_length;
    reduction(smem);
    return smem[0]/(real_t)N;
}

__device__ void print(const coord3d& ab){
    printf("[%.16e, %.16e, %.16e]\n",ab.x,ab.y,ab.z);
}
__device__ void print(const half4& ab){
    print_coord(ab);
}

__device__ void print(const half2& ab){
    printf("[%.16e, %.16e] \n", __half2float(ab.x), __half2float(ab.y));
}

__device__ void print(real_t a){
    printf("[%.16e]\n", a);
}

__device__ void print(const ushort3& a){
    printf("[%d, %d, %d]\n",a.x,a.y,a.z);
}

__device__ void print(const uchar3& a){
    printf("[%d, %d, %d]\n",a.x,a.y,a.z);
}

__device__ void print(const uint3& a){
    printf("[%d, %d, %d]\n",a.x,a.y,a.z);
}


template <typename T>
__device__ void sequential_print(T* Data){
    for (size_t i = 0; i < blockDim.x; i++)
    {
        if (threadIdx.x == i)
        {
            print(Data[i]);
        }
        cg::sync(cg::this_thread_block());
    }
}

template <typename T>
__device__ void sequential_print(T Data){
    for (size_t i = 0; i < blockDim.x; i++)
    {
        if (threadIdx.x == i)
        {
            print(Data);
        }
        cg::sync(cg::this_thread_block());
    }
}

template <typename T>
__device__ void sequential_print(T* Data, size_t N){
    for (size_t i = 0; i < N; i++)
    {
        if (threadIdx.x == i)
        {
            print(Data[i]);
        }
        cg::sync(cg::this_thread_block());
    }
}
#pragma once
#include <hip/hip_runtime.h>

typedef float3 coord3d;


__device__ __forceinline__ float3 operator-(const float3& a)                  { return make_float3(-a.x, -a.y, -a.z);  }
__device__ __forceinline__ float3 operator-(const float3& a, const float3& b){ return make_float3(a.x-b.x, a.y-b.y, a.z-b.z);  }
__device__ __forceinline__ float3 operator+(const float3& a, const float3& b){ return make_float3(a.x+b.x, a.y+b.y, a.z+b.z);  }
__device__ __forceinline__ float3 operator*(const float3& a, const float s)  { return make_float3(a.x*s, a.y*s, a.z*s);  }
__device__ __forceinline__ float3 operator*(const float s, const float3& a)  { return a*s; }
__device__ __forceinline__ float3 operator*(const float3& a, const float3& b) { return make_float3(a.x*b.x, a.y*b.y, a.z*b.z);}
__device__ __forceinline__ float3 operator/(const float s, const float3& a)  { return a*(1/s); }
__device__ __forceinline__ float3 operator/(const float3& a, const float s)  { return a*(1/s); }
__device__ __forceinline__ void operator+=(float3& a, const float3 b) {a = a + b;}
__device__ __forceinline__ void operator/=(float3& a, const float b) {a = a / b;}

__device__ __forceinline__ void set(float3& a, const uint8_t j, float b){
  switch (j)
  {
  case 0:
    a.x = b;
    break;
  case 1:
    a.y = b;
    break;
  case 2:
    a.z = b;
    break;
  default:
    break;
  }
}

__device__ __forceinline__ float get(const float3& a, const uint8_t j){
  switch (j)
  {
  case 0:
    return a.x;
  case 1:
    return a.y;
  case 2:
    return a.z;
  default:
    break;
  }
}
//5 FLOPs
__device__ __forceinline__  float  dot(const float3& a,  const float3& b) { return a.x*b.x + a.y*b.y + a.z*b.z; }

//6 FLOPs
__device__ __forceinline__  float norm(const float3& a)                    { return sqrt(dot(a,a)); }

//7 FLOPs
__device__ __forceinline__  float3 unit_vector(const float3& a){
  float r = rsqrt(dot(a,a));
  return (a*r);
}
//10 FLOPs
__device__ __forceinline__  float3 cross(const float3& a, const float3& b){ return make_float3(a.y*b.z-a.z*b.y,
							   -a.x*b.z+a.z*b.x,
							   a.x*b.y-a.y*b.x); }
// $(a \otimes b) \cdot c$
__device__ __forceinline__  float3 outer_dot(const float3& a, const float3& b, const float3& c){
  return make_float3(a.x*(b.x*c.x + b.y*c.y + b.z*c.z), 
  a.y*(b.x*c.x + b.y*c.y + b.z*c.z), 
  a.z*(b.x*c.x + b.y*c.y + b.z*c.z));
}

//6 FLOPs
__device__ __forceinline__  float bond_length(const float3& ab){
    return rsqrtf(dot(ab,ab));
}

__host__ __device__ void print_coord(const float3& ab){

    printf("[%.16e, %.16e, %.16e]\n",ab.x,ab.y,ab.z);
    //cout<< "[" << ab.x << ", " << ab.y << ", " << ab.z << "]\n" ;
}
/*
template <int N>
void write_to_file(const array<double3,N>& a){
    FILE* pFile;
    pFile = fopen("test.bin","wb");
    fwrite(&a, sizeof(float), N*3, pFile);
    fclose(pFile);
}*/


__device__ __forceinline__ double3 operator-(const double3& a)                  { return make_double3(-a.x, -a.y, -a.z);  }
__device__ __forceinline__ double3 operator-(const double3& a, const double3& b){ return make_double3(a.x-b.x, a.y-b.y, a.z-b.z);  }
__device__ __forceinline__ double3 operator+(const double3& a, const double3& b){ return make_double3(a.x+b.x, a.y+b.y, a.z+b.z);  }
__device__ __forceinline__ double3 operator*(const double3& a, const double s)  { return make_double3(a.x*s, a.y*s, a.z*s);  }
__device__ __forceinline__ double3 operator*(const double s, const double3& a)  { return a*s; }
__device__ __forceinline__ double3 operator*(const double3& a, const double3& b) { return make_double3(a.x*b.x, a.y*b.y, a.z*b.z);}
__device__ __forceinline__ double3 operator/(const double s, const double3& a)  { return a*(1/s); }
__device__ __forceinline__ double3 operator/(const double3& a, const double s)  { return a*(1/s); }
__device__ __forceinline__ void operator+=(double3& a, const double3 b) {a = a + b;}
__device__ __forceinline__ void operator/=(double3& a, const double b) {a = a / b;}

__device__ __forceinline__ void set(double3& a, const uint8_t j, double b){
  switch (j)
  {
  case 0:
    a.x = b;
    break;
  case 1:
    a.y = b;
    break;
  case 2:
    a.z = b;
    break;
  default:
    break;
  }
}

__device__ __forceinline__ double get(const double3& a, const uint8_t j){
  switch (j)
  {
  case 0:
    return a.x;
  case 1:
    return a.y;
  case 2:
    return a.z;
  default:
    break;
  }
}

//5 FLOPs
__device__ __forceinline__  double  dot(const double3& a,  const double3& b) { return a.x*b.x + a.y*b.y + a.z*b.z; }

//6 FLOPs
__device__ __forceinline__  double norm(const double3& a)                    { return sqrt(dot(a,a)); }

//7 FLOPs
__device__ __forceinline__  double3 unit_vector(const double3& a){
  double r = rsqrt(dot(a,a));
  return (a*r);
}
//10 FLOPs
__device__ __forceinline__  double3 cross(const double3& a, const double3& b){ return make_double3(a.y*b.z-a.z*b.y,
							   -a.x*b.z+a.z*b.x,
							   a.x*b.y-a.y*b.x); }
// $(a \otimes b) \cdot c$
__device__ __forceinline__  double3 outer_dot(const double3& a, const double3& b, const double3& c){
  return make_double3(a.x*(b.x*c.x + b.y*c.y + b.z*c.z), 
  a.y*(b.x*c.x + b.y*c.y + b.z*c.z), 
  a.z*(b.x*c.x + b.y*c.y + b.z*c.z));
}

//6 FLOPs
__device__ __forceinline__  double bond_length(const double3& ab){
    return rsqrtf(dot(ab,ab));
}

__host__ __device__ void print_coord(const double3& ab){

    printf("[%.16e, %.16e, %.16e]\n",ab.x,ab.y,ab.z);
    //cout<< "[" << ab.x << ", " << ab.y << ", " << ab.z << "]\n" ;
}

/*
template <int N>
void write_to_file(const array<double3,N>& a){
    FILE* pFile;
    pFile = fopen("test.bin","wb");
    fwrite(&a, sizeof(real_t), N*3, pFile);
    fclose(pFile);
}*/
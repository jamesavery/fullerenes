#include "hip/hip_runtime.h"
#pragma once
#include "coord3d.cu"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include "hip/hip_runtime.h"
#include <assert.h>
#include<iostream>
#include <fstream>

#ifndef AUXILIARY_CUDA_FUN
#define AUXILIARY_CUDA_FUN
#define __HD__ __device__ __host__ 
namespace cg = cooperative_groups;

template <typename T>
void copy_and_append(T* memory, const T* fullerene, size_t N){
    for (size_t i = 0; i < N; i++)
    {
        memory[i] = fullerene[i];
    }
}

template <typename T>
T* synthetic_array(size_t N, const size_t num_molecules, const T* fullerene){
    size_t array_size = N;
    if (sizeof(T) != sizeof(device_coord3d))
    {
        array_size *= 3;
    }
    T* storage_array = new T[array_size*num_molecules];
    for (size_t i = 0; i < num_molecules; i++)
    {
        copy_and_append(&storage_array[array_size*i],fullerene,array_size);
    }
    return storage_array;
}



template <typename T>
__device__ void pointerswap(T **r, T **s)
{
    T *pSwap = *r;
    *r = *s;
    *s = pSwap;
    return;
}


__host__ hipError_t safeCudaKernelCall(const void* func, dim3 gridDim, dim3 blockDim, void** args, size_t sharedMem){
    if (gridDim.x > 0 && gridDim.y > 0 && gridDim.z > 0 && blockDim.x > 0 && blockDim.y > 0 && blockDim.z > 0)
    {
        return hipLaunchCooperativeKernel(reinterpret_cast<const void*>(func),gridDim,blockDim,args,sharedMem);
    }
    else
    {
        std::cout << "WARNING: Attempted to launch kernel with 1 or more dimensions <= 0 \n";
        return hipErrorInvalidValue;
    }
    
}


#if REDUCTION_METHOD==0
    __device__ device_real_t reduction(device_real_t* sdata, const device_real_t data){
        sdata[threadIdx.x] = data;
        BLOCK_SYNC
        if((Block_Size_Pow_2 > 512)){if (threadIdx.x < 512){sdata[threadIdx.x] += sdata[threadIdx.x + 512];} BLOCK_SYNC}
        if((Block_Size_Pow_2 > 256)){if (threadIdx.x < 256){sdata[threadIdx.x] += sdata[threadIdx.x + 256];} BLOCK_SYNC}
        if((Block_Size_Pow_2 > 128)){if (threadIdx.x < 128){sdata[threadIdx.x] += sdata[threadIdx.x + 128];} BLOCK_SYNC}
        if((Block_Size_Pow_2 > 64)){if (threadIdx.x < 64){sdata[threadIdx.x] += sdata[threadIdx.x + 64];} BLOCK_SYNC}
        if(threadIdx.x < 32){
        if((Block_Size_Pow_2 > 32)){if (threadIdx.x < 32){sdata[threadIdx.x] += sdata[threadIdx.x + 32];} __syncwarp();}
        cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cg::this_thread_block());
        sdata[threadIdx.x] = cg::reduce(tile32, sdata[threadIdx.x], cg::plus<device_real_t>());
        }
        BLOCK_SYNC
        device_real_t sum = sdata[0];
        BLOCK_SYNC
        return sum;
    }
#elif REDUCTION_METHOD==1
    __device__ device_real_t reduction(device_real_t* sdata, const device_real_t data){
        sdata[threadIdx.x] = data;
        BLOCK_SYNC
        
        if (threadIdx.x < 512){sdata[threadIdx.x] += sdata[threadIdx.x + 512];} BLOCK_SYNC
        if (threadIdx.x < 256){sdata[threadIdx.x] += sdata[threadIdx.x + 256];} BLOCK_SYNC
        if (threadIdx.x < 128){sdata[threadIdx.x] += sdata[threadIdx.x + 128];} BLOCK_SYNC
        if (threadIdx.x < 64){sdata[threadIdx.x] += sdata[threadIdx.x + 64];} BLOCK_SYNC
        if(threadIdx.x < 32){
        if (threadIdx.x < 32){sdata[threadIdx.x] += sdata[threadIdx.x + 32];} __syncwarp();
        cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cg::this_thread_block());
        sdata[threadIdx.x] = cg::reduce(tile32, sdata[threadIdx.x], cg::plus<device_real_t>());
        }
        BLOCK_SYNC
        device_real_t sum = sdata[0];
        BLOCK_SYNC
        return sum;
    }
#elif REDUCTION_METHOD==2
    __device__ device_real_t reduction(device_real_t *sdata, const device_real_t data){
        sdata[threadIdx.x] = data;
        cg::thread_block block = cg::this_thread_block();
        BLOCK_SYNC
        cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(block);
        sdata[threadIdx.x] = cg::reduce(tile32, sdata[threadIdx.x], cg::plus<device_real_t>());
        BLOCK_SYNC

        device_real_t beta = 0.0;
        if (block.thread_rank() == 0) {
            beta  = 0;
            for (uint16_t i = 0; i < block.size(); i += tile32.size()) {
                beta  += sdata[i];
            }
            sdata[0] = beta;
        }
        BLOCK_SYNC
        device_real_t sum = sdata[0];
        BLOCK_SYNC
        return sum;
    }
#endif

__device__ device_node_t max(const device_node_t a, const device_node_t b){
    if (a > b){
        return a;
    }else {
        return b;
    }
}

__device__ device_real_t reduction_max(device_real_t* sdata, const device_real_t data){
    sdata[threadIdx.x] = data;
    cg::thread_block block = cg::this_thread_block();
    cg::sync(block);
    
    if((Block_Size_Pow_2 > 512)){if (threadIdx.x < 512){sdata[threadIdx.x] = max(sdata[threadIdx.x + 512],sdata[threadIdx.x]);} cg::sync(block);}
    if((Block_Size_Pow_2 > 256)){if (threadIdx.x < 256){sdata[threadIdx.x] = max(sdata[threadIdx.x + 256],sdata[threadIdx.x]);} cg::sync(block);}
    if((Block_Size_Pow_2 > 128)){if (threadIdx.x < 128){sdata[threadIdx.x] = max(sdata[threadIdx.x + 128],sdata[threadIdx.x]);} cg::sync(block);}
    if((Block_Size_Pow_2 > 64)){if (threadIdx.x < 64){sdata[threadIdx.x] = max(sdata[threadIdx.x + 64],sdata[threadIdx.x]);} cg::sync(block);}
    if(threadIdx.x < 32){
    if((Block_Size_Pow_2 > 32)){if (threadIdx.x < 32){sdata[threadIdx.x] = max(sdata[threadIdx.x + 32],sdata[threadIdx.x]);} __syncwarp();}
    cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(block);
    sdata[threadIdx.x] = cg::reduce(tile32, sdata[threadIdx.x], cg::greater<device_real_t>()); 
    }
    cg::sync(block);
    device_real_t max = sdata[0];
    cg::sync(block);
    return max;
}

__device__ device_node_t reduction_max(device_node_t* sdata, const device_node_t data){
    sdata[threadIdx.x] = data;
    cg::thread_block block = cg::this_thread_block();
    cg::sync(block);
    
    if((Block_Size_Pow_2 > 512)){if (threadIdx.x < 512){sdata[threadIdx.x] = max(sdata[threadIdx.x + 512],sdata[threadIdx.x]);} cg::sync(block);}
    if((Block_Size_Pow_2 > 256)){if (threadIdx.x < 256){sdata[threadIdx.x] = max(sdata[threadIdx.x + 256],sdata[threadIdx.x]);} cg::sync(block);}
    if((Block_Size_Pow_2 > 128)){if (threadIdx.x < 128){sdata[threadIdx.x] = max(sdata[threadIdx.x + 128],sdata[threadIdx.x]);} cg::sync(block);}
    if((Block_Size_Pow_2 > 64)){if (threadIdx.x < 64){sdata[threadIdx.x] = max(sdata[threadIdx.x + 64],sdata[threadIdx.x]);} cg::sync(block);}
    if(threadIdx.x < 32){
    if((Block_Size_Pow_2 > 32)){if (threadIdx.x < 32){sdata[threadIdx.x] = max(sdata[threadIdx.x + 32],sdata[threadIdx.x]);} __syncwarp();}
    cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(block);
    sdata[threadIdx.x] = cg::reduce(tile32, sdata[threadIdx.x], cg::greater<device_node_t>()); 
    }
    cg::sync(block);
    device_node_t max = sdata[0];
    cg::sync(block);
    return max;
}

__device__ half reduction(half *sdata){

    cg::thread_block block = cg::this_thread_block();
    cg::sync(block);
    cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(block);
    sdata[threadIdx.x] = cg::reduce(tile32, sdata[threadIdx.x], cg::plus<half>());
    cg::sync(block);

    half beta = 0.0;
    if (block.thread_rank() == 0) {
        beta  = 0;
        for (uint16_t i = 0; i < block.size(); i += tile32.size()) {
            beta  += sdata[i];
        }
        sdata[0] = beta;
    }
    cg::sync(block);
    return sdata[0];
}




__HD__ void print(const device_coord3d& ab){
    printf("[%.6f,%.6f,%.6f]",ab.x,ab.y,ab.z);
}
__device__ void print(const half4& ab){
    print_coord(ab);
}

__device__ void print(const half2& ab){
    printf("[%.6f,%.6f] \n", __half2float(ab.x), __half2float(ab.y));
}

__HD__ void print(device_real_t a){
    printf("%.6f", a);
}

__HD__ void print(bool b){
    printf("%d",int(b));
}

__HD__ void print(int a){
    printf("%d",a);
}

__device__ void print(const char* a){
    printf(a);
}

__device__ void print(const device_node3& a){
    printf("[%d,%d,%d]",a.x,a.y,a.z);
}

__device__ void print(const device_coord2d& a){
    printf("[%.6f,%.6f]",a.x,a.y);
}
template <typename T>
__device__ void print_single(T data){
    if (threadIdx.x + blockIdx.x == 0) {
        print(data);
    }
}

template <typename T>
__device__ void sequential_print(T* data){
    for (size_t i = 0; i < blockDim.x; i++)
    {
        if (threadIdx.x == i)
        {
            print(data[i]); printf(",");
        }
        cg::sync(cg::this_thread_block());
    }
}

template <typename T>
__device__ void sequential_print(T data, size_t fullerene_id){
    if (blockIdx.x == fullerene_id)
    {
    if (threadIdx.x == 0) printf("[");
    cg::sync(cg::this_thread_block());
    for (size_t i = 0; i < blockDim.x; i++)
    {
        if (threadIdx.x == i)
        {   
            if (i != blockDim.x-1)
            {
                print(data); printf(",");
            } else{
                print(data);
            }
        }
        cg::sync(cg::this_thread_block());
    }
    if (threadIdx.x == 0) printf("]\n");
    cg::sync(cg::this_thread_block());
    }
}

template <typename T>
__host__ void print_array(T* data, size_t N, size_t fullerene_id){
    for (size_t i = 0; i < N; i++)
    {
        print(data[fullerene_id + i]);
    }
}

template <typename T>
__host__ void to_binary(std::string filename,T* data, size_t bytes){
    T* pointer =  data;
    std::fstream myFile (filename, std::fstream::out | std::fstream::in | std::fstream::trunc | std::fstream::binary );

    myFile.write(reinterpret_cast<const char*>(pointer), bytes);
    if(!myFile)
      std::cout<<"error";
    myFile.close();
}

template <typename T>
__device__ void sequential_print(T* data, size_t fullerene_id){
    if (blockIdx.x == fullerene_id)
    {
    if (threadIdx.x == 0) printf("[");
    cg::sync(cg::this_thread_block());
    for (size_t i = 0; i < blockDim.x; i++)
    {   

            if (threadIdx.x == i)
            {
            if (i != blockDim.x-1)
            {
                print(data[i]); printf(",");
            } else{
                print(data[i]);
            }

        }
        cg::sync(cg::this_thread_block());
    }
    if(threadIdx.x == 0) printf("]\n");
    cg::sync(cg::this_thread_block());
    }
}

template <typename T>
__device__ void grid_print(T data){

    if (threadIdx.x + blockIdx.x == 0) printf("[");
    cg::sync(cg::this_grid());
    for (size_t i = 0; i < gridDim.x; i++)
    {   
            if(threadIdx.x == 0){
            if (blockIdx.x == i)
            {
            if (i != gridDim.x-1)
            {
                print(data); printf(",");
            } else{
                print(data);
            }}

        }
        cg::sync(cg::this_grid());
    }
    if(threadIdx.x + blockIdx.x == 0) printf("]\n");
    cg::sync(cg::this_grid());
}

template <typename T>
__HD__ void swap_reals(T& a, T& b){
    T temp = a;
    a = b;
    b = temp;
}

void printLastCudaError(std::string message = ""){
    hipError_t error = hipGetLastError();
    if(error != hipSuccess){
        std::cout << "\n" << message << " :\t";
        std::cout << hipGetErrorString(error);
        printf("\n");
    }
}

__device__ void clear_cache(device_real_t* sdata, size_t N){
    BLOCK_SYNC
    for (size_t index = threadIdx.x; index < N; index+=blockDim.x)
    {
        sdata[index] = (device_real_t)0.0;
    }
    BLOCK_SYNC
}
__device__ device_real_t global_reduction(device_real_t *sdata, device_real_t *gdata, device_real_t data, bool mask = true){
    GRID_SYNC
    if(!mask){data = (device_real_t)0.0;}
    device_real_t block_sum    = reduction(sdata,data);
    if(threadIdx.x == 0){gdata[blockIdx.x] = block_sum;}
    GRID_SYNC

        if (gridDim.x > 1024 && threadIdx.x == 0 && ((blockIdx.x + 1024) < gridDim.x))   {if (blockIdx.x < 1024) {gdata[blockIdx.x]  += gdata[blockIdx.x + 1024];}} GRID_SYNC
        if (gridDim.x > 512 && threadIdx.x == 0 && ((blockIdx.x + 512) < gridDim.x))    {if (blockIdx.x < 512)  {gdata[blockIdx.x]  += gdata[blockIdx.x + 512];}} GRID_SYNC
        if (gridDim.x > 256 && threadIdx.x == 0 && ((blockIdx.x + 256) < gridDim.x))    {if (blockIdx.x < 256)  {gdata[blockIdx.x]  += gdata[blockIdx.x + 256];}} GRID_SYNC
        if (gridDim.x > 128 && threadIdx.x == 0 && ((blockIdx.x + 128) < gridDim.x))    {if (blockIdx.x < 128)  {gdata[blockIdx.x]  += gdata[blockIdx.x + 128];}} GRID_SYNC
        if (gridDim.x > 64 && threadIdx.x == 0 && ((blockIdx.x + 64) < gridDim.x))     {if (blockIdx.x < 64)   {gdata[blockIdx.x]  += gdata[blockIdx.x + 64];}} GRID_SYNC
        if (gridDim.x > 32 && threadIdx.x == 0 && ((blockIdx.x + 32) < gridDim.x))     {if (blockIdx.x < 32)   {gdata[blockIdx.x]  += gdata[blockIdx.x + 32];}} GRID_SYNC
        if (threadIdx.x < 32 && blockIdx.x == 0)
        {
            cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cg::this_thread_block());
            gdata[threadIdx.x] = cg::reduce(tile32, gdata[threadIdx.x], cg::plus<device_real_t>()); 
        }
    GRID_SYNC
    device_real_t sum = (device_real_t)0.0;
    sum = gdata[0];
    GRID_SYNC
    return sum;
}

#endif
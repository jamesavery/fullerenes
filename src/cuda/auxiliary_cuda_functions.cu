#include "hip/hip_runtime.h"
#include "coord3d.cu"
#include "coord3d_aligned.cu"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include "hip/hip_runtime.h"
#include <assert.h>
#include<iostream>
#include <fstream>

#define __HD__ __device__ __host__ 
namespace cg = cooperative_groups;

template <typename T>
void copy_and_append(T* memory, const T* fullerene, size_t N){
    for (size_t i = 0; i < N; i++)
    {
        memory[i] = fullerene[i];
    }
}

template <typename T>
T* synthetic_array(size_t N, const size_t num_molecules, const T* fullerene){
    size_t array_size = N;
    if (sizeof(T) != sizeof(device_coord3d))
    {
        array_size *= 3;
    }
    T* storage_array = new T[array_size*num_molecules];
    for (size_t i = 0; i < num_molecules; i++)
    {
        copy_and_append(&storage_array[array_size*i],fullerene,array_size);
    }
    return storage_array;
}


__device__ void align16(device_coord3d* input, coord3d_a* output, size_t N){
    cg::sync(cg::this_grid());
    output[threadIdx.x] = {input[threadIdx.x].x, input[threadIdx.x].y, input[threadIdx.x].z, 0};
    cg::sync(cg::this_grid());
}

template <typename T>
__device__ void pointerswap(T **r, T **s)
{
    T *pSwap = *r;
    *r = *s;
    *s = pSwap;
    return;
}

__device__ device_real_t reduction(device_real_t* sdata, const device_real_t data){
    sdata[threadIdx.x] = data;
    cg::thread_block block = cg::this_thread_block();
    cg::sync(block);
    
    if((Block_Size_Pow_2 > 512)){if (threadIdx.x < 512){sdata[threadIdx.x] += sdata[threadIdx.x + 512];} cg::sync(block);}
    if((Block_Size_Pow_2 > 256)){if (threadIdx.x < 256){sdata[threadIdx.x] += sdata[threadIdx.x + 256];} cg::sync(block);}
    if((Block_Size_Pow_2 > 128)){if (threadIdx.x < 128){sdata[threadIdx.x] += sdata[threadIdx.x + 128];} cg::sync(block);}
    if((Block_Size_Pow_2 > 64)){if (threadIdx.x < 64){sdata[threadIdx.x] += sdata[threadIdx.x + 64];} cg::sync(block);}
    if(threadIdx.x < 32){
    if((Block_Size_Pow_2 > 32)){if (threadIdx.x < 32){sdata[threadIdx.x] += sdata[threadIdx.x + 32];} __syncwarp();}
    cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(block);
    sdata[threadIdx.x] = cg::reduce(tile32, sdata[threadIdx.x], cg::plus<device_real_t>());
    }
    cg::sync(block);
    device_real_t sum = sdata[0];
    cg::sync(block);
    return sum;
}


__device__ device_real_t reduction_max(device_real_t* sdata, const device_real_t data){
    sdata[threadIdx.x] = data;
    cg::thread_block block = cg::this_thread_block();
    cg::sync(block);
    
    if((Block_Size_Pow_2 > 512)){if (threadIdx.x < 512){sdata[threadIdx.x] = max(sdata[threadIdx.x + 512],sdata[threadIdx.x]);} cg::sync(block);}
    if((Block_Size_Pow_2 > 256)){if (threadIdx.x < 256){sdata[threadIdx.x] = max(sdata[threadIdx.x + 256],sdata[threadIdx.x]);} cg::sync(block);}
    if((Block_Size_Pow_2 > 128)){if (threadIdx.x < 128){sdata[threadIdx.x] = max(sdata[threadIdx.x + 128],sdata[threadIdx.x]);} cg::sync(block);}
    if((Block_Size_Pow_2 > 64)){if (threadIdx.x < 64){sdata[threadIdx.x] = max(sdata[threadIdx.x + 64],sdata[threadIdx.x]);} cg::sync(block);}
    if(threadIdx.x < 32){
    if((Block_Size_Pow_2 > 32)){if (threadIdx.x < 32){sdata[threadIdx.x] = max(sdata[threadIdx.x + 32],sdata[threadIdx.x]);} __syncwarp();}
    cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(block);
    sdata[threadIdx.x] = cg::reduce(tile32, sdata[threadIdx.x], cg::greater<device_real_t>()); 
    }
    cg::sync(block);
    device_real_t max = sdata[0];
    cg::sync(block);
    return max;
}

__device__ half reduction(half *sdata){

    cg::thread_block block = cg::this_thread_block();
    cg::sync(block);
    cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(block);
    sdata[threadIdx.x] = cg::reduce(tile32, sdata[threadIdx.x], cg::plus<half>());
    cg::sync(block);

    half beta = 0.0;
    if (block.thread_rank() == 0) {
        beta  = 0;
        for (uint16_t i = 0; i < block.size(); i += tile32.size()) {
            beta  += sdata[i];
        }
        sdata[0] = beta;
    }
    cg::sync(block);
    return sdata[0];
}

//Multi purpose reduction algorithm (Small or Large fullerenes).
__device__ void reduction(device_real_t *sdata, device_real_t *gdata, const device_node_t N, const bool single_block_fullerenes){
    cg::thread_block block = cg::this_thread_block();

    cg::sync(block);
    if (((threadIdx.x + blockIdx.x * blockDim.x) >= N) && !single_block_fullerenes)
    {
        sdata[threadIdx.x] = 0;
    }
    cg::sync(block);
    cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(block);
    sdata[threadIdx.x] = cg::reduce(tile32, sdata[threadIdx.x], cg::plus<device_real_t>());
    cg::sync(block);
    
    device_real_t beta = 0.0;
    if (single_block_fullerenes)
    {
        if (block.thread_rank() == 0) {
            for (uint16_t i = 0; i < block.size(); i += tile32.size()) {
                beta  += sdata[i];
            }
            sdata[0] = beta;
        }
        cg::sync(block);
    }
    else 
    {   
        auto grid = cg::this_grid();
        if (block.thread_rank() == 0) 
        {
            for (uint16_t i = 0; i < block.size(); i += tile32.size()) 
            {
                beta  += sdata[i];
            }
            gdata[blockIdx.x] = beta;
        }
        cg::sync(grid);
        beta = 0.0;
        if (grid.thread_rank() == 0)
        {
            for (uint16_t i = 0; i < gridDim.x; i++) 
            {
                beta  += gdata[i];
            }
            gdata[0] = beta;
        }
        cg::sync(grid);
        if (block.thread_rank() == 0) {sdata[0] = gdata[0];}
        cg::sync(grid);
    }
}

__HD__ void print(const device_coord3d& ab){
    printf("[%.8e, %.8e, %.8e]\n",ab.x,ab.y,ab.z);
}
__device__ void print(const half4& ab){
    print_coord(ab);
}

__device__ void print(const half2& ab){
    printf("[%.16e, %.16e] \n", __half2float(ab.x), __half2float(ab.y));
}

__HD__ void print(device_real_t a){
    printf("[%.16e]\n", a);
}

__HD__ void print(bool b){
    printf("[%d]\n",int(b));
}

__HD__ void print(int a){
    printf("[%d]\n",a);
}

__device__ void print(const ushort3& a){
    printf("[%d, %d, %d]\n",a.x,a.y,a.z);
}

__device__ void print(const uchar3& a){
    printf("[%d, %d, %d]\n",a.x,a.y,a.z);
}

__device__ void print(const uint3& a){
    printf("[%d, %d, %d]\n",a.x,a.y,a.z);
}

template <typename T>
__device__ void print_single(T data){
    if (threadIdx.x + blockIdx.x == 0) {
        print(data);
    }
}

template <typename T>
__device__ void sequential_print(T* data){
    for (size_t i = 0; i < blockDim.x; i++)
    {
        if (threadIdx.x == i)
        {
            print(data[i]);
        }
        cg::sync(cg::this_thread_block());
    }
}

template <typename T>
__device__ void sequential_print(T data, size_t fullerene_id){
    if (blockIdx.x == fullerene_id)
    {
    for (size_t i = 0; i < blockDim.x; i++)
    {
        if (threadIdx.x == i)
        {
            print(data);
        }
        cg::sync(cg::this_thread_block());
    }
    }
}

template <typename T>
__host__ void print_array(T* data, size_t N, size_t fullerene_id){
    for (size_t i = 0; i < N; i++)
    {
        print(data[fullerene_id + i]);
    }
}

template <typename T>
__host__ void toBinary(std::string filename,T* data, size_t N, size_t fullerene_id){
    T* pointer =  data + N * fullerene_id;
    std::fstream myFile (filename, std::fstream::out | std::fstream::in | std::fstream::trunc | std::fstream::binary );

    myFile.write(reinterpret_cast<const char*>(pointer), sizeof(T)*N);
    if(!myFile)
      std::cout<<"error";
    myFile.close();
}

template <typename T>
__device__ void sequential_print(T* data, size_t fullerene_id){
    if (blockIdx.x == fullerene_id)
    {
    for (size_t i = 0; i < blockDim.x; i++)
    {
        if (threadIdx.x == i)
        {
            print(data[i]);
        }
        cg::sync(cg::this_thread_block());
    }
    }
}

template <typename T>
__HD__ void swap_reals(T& a, T& b){
    T temp = a;
    a = b;
    b = temp;
}

void printLastCudaError(std::string message = ""){
    hipError_t error = hipGetLastError();
    if(error != hipSuccess){
        std::cout << "\n" << message << " :\t";
        std::cout << hipGetErrorString(error);
        printf("\n");
    }
}

__device__ void clear_cache(device_real_t* sdata, size_t N){
    cg::sync(cg::this_thread_block());
    for (size_t index = threadIdx.x; index < N; index+=blockDim.x)
    {
        sdata[index] = 0;
    }
    cg::sync(cg::this_thread_block());
}

#include "hip/hip_runtime.h"
struct symMat3
{
  device_real_t a, b, c, d, e, f;
  
  //[[a , b,  c]
  // [b,  d,  e]
  // [c,  e,  f]]
  INLINE symMat3(){}
  INLINE symMat3(device_real_t a, device_real_t b, device_real_t c, device_real_t d, device_real_t e, device_real_t f) : a(a), b(b), c(c), d(d), e(e), f(f){}
  
  //Approx 107 FLOPS
  INLINE device_coord3d eigenvalues() const{
    DEVICE_TYPEDEFS
     // Coefficients of characteristic polynomial, calculated with Mathematica
    real_t 
      A = -1.f,
      B = a+d+f,
      C = b*b + c*c - a*d + e*e - a*f - d*f,
      D = -c*c*d + (real_t)2.f*b*c*e - a*e*e - b*b*f + a*d*f;

    if(abs(D) < 1e-12){
      auto temp = B*B - real_t(4.f)*A*C;
      real_t Disc = temp > (real_t)0. ? sqrt(B*B - real_t(4.f)*A*C) : 0;

      return {0.f, (-B-Disc)/( real_t(2.f)*A),(-B+Disc)/( real_t(2.f)*A)};
    }

    // Depress characteristic polynomial - see http://en.wikipedia.org/wiki/Cubic_equation#Reduction_to_a_depressed_cubic
    real_t
      p  = ( (real_t)3.f*A*C - B*B)/( (real_t)3.f*A*A),
      q  = ( (real_t)2.f*B*B*B - (real_t)9.f*A*B*C + (real_t)27.f*A*A*D)/( (real_t)27.f*A*A*A),
      xc = B/( (real_t)3.f*A);

    // François Viète's solution to cubic polynomials with three real roots. 
    device_coord3d t;
    if(abs(p) < 1e-12) {
      t = {(real_t)0., (real_t)0., (real_t)0.};
      return t - xc;}

    //For numerical stability we must ensure that acos doesn't receive an arugment which is outside [-1,1]
    auto frac = ( (real_t)3.f*q)/( (real_t)2.f*p)*sqrt((real_t)-3.f/p);
    frac = d_max((real_t)-1.,d_min((real_t)1., frac));

    real_t K = (real_t)2.f*sqrt(-p/ (real_t)3.f), 
                  theta0 = ((real_t)1.f/ (real_t)3.f)*acos(frac);
    for(int k=0;k<3;k++) d_set(t,k,K*cos(theta0-k* (real_t)2.f* (real_t)M_PI/ (real_t)3.f) );
    // lambda = t - B/(3A)
    return t - xc;
    
  }
  //Best case 25 FLOPS
  INLINE device_coord3d eigenvector(const device_real_t lambda) const{
    // using the first two eqs
    // [ a_12 * a_23 - a_13 * (a_22 - r) ]
    // [ a_12 * a_13 - a_23 * (a_11 - r) ]
    // [ (a_11 - r) * (a_22 - r) - a_12^2 ]
    device_real_t normx;
    device_coord3d x = {b*e - c*(d-lambda),
                 b*c - e*(a-lambda),
                 (a-lambda)*(d-lambda) - b*b };
    normx = norm(x);
    if (normx / (a + d + f) > 1.e-12){ // not zero-ish
      return x/normx;
    }
  
    // using the first+last eqs
    // [ a_12 * (a_33 - r) - a_13 * a_23 ]
    // [ a_13^2 - (a_11 - r) * (a_33 - r) ]
    // [ a_23 * (a_11 - r) - a_12 * a_13 ]
    x = { b*(f-lambda) - c*e,
                 c*c - (a-lambda)*(f-lambda),
                 e*(a-lambda) - b*c };
    normx = norm(x);
    if (normx / (a + d + f) > 1.e-12){ // not zero-ish
      return x/normx;
    }

    // using the last two eqs
    // [ a_23^2 - (a_22 - r) * (a_33 - r) ]
    // [ a_12 * (a_33 - r) - a_13 * a_23 ]
    // [ a_13 * (a_22 - r) - a_12 * a_23 ]
    x ={ e*e - (d-lambda)*(f-lambda),
                 b*(f-lambda) - c*e,
                 c*(d-lambda) - b*e };
    normx = norm(x);
    if (normx / (a + d + f) > 1.e-12){ // not zero-ish
      return x/normx;
    } 
    //assert(false); // Something went wrong possibly two degenerate evals.
    return device_coord3d();
  }
};
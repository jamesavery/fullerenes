#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "fullerenes/gpu/kernels.hh"


namespace gpu_kernels{
namespace isomerspace_forcefield{

#include "hip/hip_runtime.h"
#include "hip/hip_cooperative_groups.h"

#include <hip/hip_runtime_api.h>
#include "fullerenes/gpu/cuda_definitions.h"
#include "misc_cuda.cu"
#include "reductions.cu"
#include "coord3d.cuh"
#include "forcefield_structs.cu"

// This struct was made to reduce signature cluttering of device functions, it is simply a container for default arguments which are shared between functions
struct ForceField{
    DEVICE_TYPEDEFS

    coord3d r1,                         //d_i+1 In conjugated gradient algorithm  Buster p. 36
            r0;                         //d_0 In conjugated gradient algorithm Buster p. 36

    const NodeGraph node_graph;         //Contains face-information and neighbour-information. Both of which are constant in the lifespan of this struct. 
    const Constants constants;          //Contains force-constants and equillibrium-parameters. Constant in the lifespan of this struct.

    size_t node_id = threadIdx.x;
    real_t* sdata;                      //Pointer to start of L1 cache array, used exclusively for reduction.

    __device__ ForceField(  const NodeGraph &G,
                            const Constants &c, 
                            real_t* sdata): node_graph(G), constants(c), sdata(sdata) {}


//Container for all energy and gradient evaluations with respect to an arc, eg. AB, AC or AD.
struct ArcData{
    //124 FLOPs;
    uint8_t j;
    __device__ ArcData(const uint8_t j, const coord3d* __restrict__ X, const NodeGraph& G){   
        this->j = j;   
        node_t a = threadIdx.x;
        real_t r_rmp;
        coord3d ap, am, ab, ac, ad, mp;
        coord3d X_a = X[a]; coord3d X_b = X[d_get(G.neighbours,j)];

        //Compute the arcs ab, ac, ad, bp, bm, ap, am, mp, bc and cd
        ab = (X_b - X_a);  r_rab = bond_length(ab); ab_hat = r_rab * ab;
        ac = (X[d_get(G.neighbours,(j+1)%3)] - X_a); r_rac = bond_length(ac); ac_hat = r_rac * ac; rab = non_resciprocal_bond_length(ab);
        ad = (X[d_get(G.neighbours,(j+2)%3)] - X_a); r_rad = bond_length(ad); ad_hat = r_rad * ad;
        
        coord3d bp = (X[d_get(G.next_on_face,j)] - X_b); bp_hat = unit_vector(bp);
        coord3d bm = (X[d_get(G.prev_on_face,j)] - X_b); bm_hat = unit_vector(bm);

        ap = bp + ab; r_rap = bond_length(ap); ap_hat = r_rap * ap;
        am = bm + ab; r_ram = bond_length(am); am_hat = r_ram * am;
        mp = bp - bm; r_rmp = bond_length(mp); mp_hat = r_rmp * mp;

        bc_hat = unit_vector(ac - ab);
        cd_hat = unit_vector(ad - ac);

        //Compute inverses of some arcs, these are subject to be omitted if the equations are adapted appropriately with inversion of signs.
        ba_hat = -ab_hat;
        mb_hat = -bm_hat;
        pa_hat = -ap_hat;
        pb_hat = -bp_hat;
    }

    //3 FLOPs
    INLINE real_t harmonic_energy(const real_t p0, const real_t p) const{
        return (real_t)0.5*(p-p0)*(p-p0);
    }
    //4 FLOPs
    INLINE coord3d  harmonic_energy_gradient(const real_t p0, const real_t p, const coord3d gradp) const{
        return (p-p0)*gradp;     
    }
    //1 FLOP
    INLINE real_t bond() const {return rab;}
    //5 FLOPs
    INLINE real_t angle() const {return dot(ab_hat,ac_hat);}

    //Returns outer angle m, used only diagnostically.
    INLINE real_t outer_angle_m() const {return -dot(ab_hat, bm_hat);} //Compute outer angle. ab,bm

    //Returns outer angle p, used only diagnostically.
    INLINE real_t outer_angle_p() const{return -dot(ab_hat, bp_hat);} //Compute outer angle. ab,bp

    //Returns the inner dihedral angle for the current arc. Used here only for energy calculation, 
    //otherwise embedded in dihedral computation because the planes and angles that make up the dihedral angle computation are required for derivative computation.
    //50 FLOPs
    INLINE real_t dihedral() const 
    { 
        coord3d nabc, nbcd; real_t cos_b, cos_c, r_sin_b, r_sin_c;
        cos_b = dot(ba_hat,bc_hat);  r_sin_b = (real_t)1.0/sqrt((real_t)1.0 - cos_b*cos_b); nabc = cross(ba_hat, bc_hat) * r_sin_b;
        cos_c = dot(-bc_hat,cd_hat); r_sin_c = (real_t)1.0/sqrt((real_t)1.0 - cos_c*cos_c); nbcd = cross(-bc_hat,cd_hat) * r_sin_c;
        return dot(nabc, nbcd);
    }
    //Returns the Outer-dihedral-a wrt. current arc, only accessed diagnostically (internal coordinate).
    INLINE real_t outer_dihedral_a() const
    {
        coord3d nbam_hat, namp_hat; real_t cos_a, cos_m, r_sin_a, r_sin_m;
        cos_a = dot(ab_hat,am_hat); r_sin_a = (real_t)1.0/sqrt((real_t)1.0 - cos_a*cos_a); nbam_hat = cross(ab_hat,am_hat) * r_sin_a;
        cos_m = dot(-am_hat,mp_hat); r_sin_m = (real_t)1.0/sqrt((real_t)1.0 - cos_m*cos_m); namp_hat = cross(-am_hat,mp_hat) * r_sin_m;
        real_t cos_beta = dot(nbam_hat, namp_hat); //Outer Dihedral angle bam, amp
        return cos_beta;
    }
    //Returns the Outer-dihedral-m wrt. current arc, only accessed diagnostically (internal coordinate).
    INLINE real_t outer_dihedral_m() const
    {
        coord3d nbmp_hat, nmpa_hat; real_t cos_m, cos_p, r_sin_m, r_sin_p;
        cos_m = dot(mb_hat,mp_hat);  r_sin_m = (real_t)1.0/sqrt((real_t)1.0 - cos_m*cos_m); nbmp_hat = cross(mb_hat,mp_hat) * r_sin_m;
        cos_p = dot(-mp_hat,pa_hat); r_sin_p = (real_t)1.0/sqrt((real_t)1.0 - cos_p*cos_p); nmpa_hat = cross(-mp_hat,pa_hat) * r_sin_p;
        //Cosine to the outer dihedral angle constituted by the planes bmp and mpa
        real_t cos_beta = dot(nbmp_hat, nmpa_hat); //Outer dihedral angle bmp,mpa.
        return cos_beta;    
    }
    //Returns the Outer-dihedral-p wrt. current arc, only accessed diagnostically (internal coordinate).
    INLINE real_t outer_dihedral_p() const
    {
        coord3d nbpa_hat, npam_hat; real_t cos_p, cos_a, r_sin_p, r_sin_a;
        cos_a = dot(ap_hat,am_hat);  r_sin_a = (real_t)1.0/sqrt((real_t)1.0 - cos_a*cos_a); npam_hat = cross(ap_hat,am_hat)  * r_sin_a;
        cos_p = dot(pb_hat,-ap_hat); r_sin_p = (real_t)1.0/sqrt((real_t)1.0 - cos_p*cos_p); nbpa_hat = cross(pb_hat,-ap_hat) * r_sin_p;
        real_t cos_beta = dot(nbpa_hat, npam_hat); //Outer dihedral angle bpa, pam.
        //Eq. 33 multiplied by harmonic term.
        return cos_beta;
    }
    
    // Chain rule terms for angle calculation
    //Computes gradient related to bending term. ~24 FLOPs
    INLINE coord3d inner_angle_gradient(const Constants& c) const
    {   
        real_t cos_angle = angle(); //Inner angle of arcs ab,ac.
        coord3d grad = cos_angle * (ab_hat * r_rab + ac_hat * r_rac) - ab_hat * r_rac - ac_hat* r_rab; //Derivative of inner angle: Eq. 21. 
        return d_get(c.f_inner_angle,j) * harmonic_energy_gradient(d_get(c.angle0,j), cos_angle, grad); //Harmonic Energy Gradient: Eq. 21. multiplied by harmonic term.
    }
    //Computes gradient related to bending of outer angles. ~20 FLOPs
    INLINE coord3d outer_angle_gradient_m(const Constants& c) const
    {
        real_t cos_angle = -dot(ab_hat, bm_hat); //Compute outer angle. ab,bm
        coord3d grad = (bm_hat + ab_hat * cos_angle) * r_rab; //Derivative of outer angles Eq. 30. Buster Thesis
        return d_get(c.f_outer_angle_m,j) * harmonic_energy_gradient(d_get(c.outer_angle_m0,j),cos_angle,grad); //Harmonic Energy Gradient: Eq. 30 multiplied by harmonic term.
    }
    INLINE coord3d outer_angle_gradient_p(const Constants& c) const
    {
        real_t cos_angle = -dot(ab_hat, bp_hat); //Compute outer angle. ab,bp
        coord3d grad = (bp_hat + ab_hat * cos_angle) * r_rab; //Derivative of outer angles Eq. 28. Buster Thesis
        return d_get(c.f_outer_angle_p,j) * harmonic_energy_gradient(d_get(c.outer_angle_p0,j),cos_angle,grad); //Harmonic Energy Gradient: Eq. 28 multiplied by harmonic term.
    }
    // Chain rule terms for dihedral calculation
    //Computes gradient related to dihedral/out-of-plane term. ~75 FLOPs
    INLINE coord3d inner_dihedral_gradient(const Constants& c) const
    {
        coord3d nabc, nbcd; real_t cos_b, cos_c, r_sin_b, r_sin_c;
        cos_b = dot(ba_hat,bc_hat); r_sin_b = (real_t)1.0/sqrt((real_t)1.0 - cos_b*cos_b); nabc = cross(ba_hat, bc_hat) * r_sin_b;
        cos_c = dot(-bc_hat,cd_hat); r_sin_c = (real_t)1.0/sqrt((real_t)1.0 - cos_c*cos_c); nbcd = cross(-bc_hat,cd_hat) * r_sin_c;

        real_t cos_beta = dot(nabc, nbcd); //Inner dihedral angle from planes abc,bcd.
        real_t cot_b = cos_b * r_sin_b * r_sin_b; //cos(b)/sin(b)^2

        //Derivative w.r.t. inner dihedral angle F and G in Eq. 26
        coord3d grad = cross(bc_hat, nbcd) * r_sin_b * r_rab - ba_hat * cos_beta * r_rab + (cot_b * cos_beta * r_rab) * (bc_hat - ba_hat * cos_b);

        return d_get(c.f_inner_dihedral,j) * harmonic_energy_gradient(d_get(c.inner_dih0,j), cos_beta, grad); //Eq. 26.
    }

    //Computes gradient from dihedral angles constituted by the planes bam, amp ~162 FLOPs
    INLINE coord3d outer_dihedral_gradient_a(const Constants& c) const
    {
        coord3d nbam_hat, namp_hat; real_t cos_a, cos_m, r_sin_a, r_sin_m;

        cos_a = dot(ab_hat,am_hat); r_sin_a = (real_t)1.0/sqrt((real_t)1.0 - cos_a*cos_a); nbam_hat = cross(ab_hat,am_hat) * r_sin_a;
        cos_m = dot(-am_hat,mp_hat); r_sin_m = (real_t)1.0/sqrt((real_t)1.0 - cos_m*cos_m); namp_hat = cross(-am_hat,mp_hat) * r_sin_m;
        
        real_t cos_beta = dot(nbam_hat, namp_hat); //Outer Dihedral angle bam, amp
        real_t cot_a = cos_a * r_sin_a * r_sin_a;
        real_t cot_m = cos_m * r_sin_m * r_sin_m;

        //Derivative w.r.t. outer dihedral angle, factorized version of Eq. 31.
        coord3d grad = cross(mp_hat,nbam_hat)*r_ram*r_sin_m - (cross(namp_hat,ab_hat)*r_ram + cross(am_hat,namp_hat)*r_rab)*r_sin_a +
                        cos_beta*(ab_hat*r_rab + r_ram * ((real_t)2.0*am_hat + cot_m*(mp_hat+cos_m*am_hat)) - cot_a*(r_ram*(ab_hat - am_hat*cos_a) + r_rab*(am_hat-ab_hat*cos_a)));
        
        //Eq. 31 multiplied by harmonic term.

        return d_get(c.f_outer_dihedral,j) * harmonic_energy_gradient(d_get(c.outer_dih0_a,j), cos_beta, grad);
    }

    //Computes gradient from dihedral angles constituted by the planes nbmp, nmpa ~92 FLOPs
    INLINE coord3d outer_dihedral_gradient_m(const Constants& c) const
    {
        coord3d nbmp_hat, nmpa_hat; real_t cos_m, cos_p, r_sin_m, r_sin_p;
        cos_m = dot(mb_hat,mp_hat);  r_sin_m = (real_t)1.0/sqrt((real_t)1.0 - cos_m*cos_m); nbmp_hat = cross(mb_hat,mp_hat) * r_sin_m;
        cos_p = dot(-mp_hat,pa_hat); r_sin_p = (real_t)1.0/sqrt((real_t)1.0 - cos_p*cos_p); nmpa_hat = cross(-mp_hat,pa_hat) * r_sin_p;
        
        //Cosine to the outer dihedral angle constituted by the planes bmp and mpa
        real_t cos_beta = dot(nbmp_hat, nmpa_hat); //Outer dihedral angle bmp,mpa.
        real_t cot_p = cos_p * r_sin_p * r_sin_p;
        
        //Derivative w.r.t. outer dihedral angle, factorized version of Eq. 32.
        coord3d grad = r_rap * (cot_p*cos_beta * (-mp_hat - pa_hat*cos_p) - cross(nbmp_hat, mp_hat)*r_sin_p - pa_hat*cos_beta );

        //Eq. 32 multiplied by harmonic term.
        return d_get(c.f_outer_dihedral,j) * harmonic_energy_gradient(d_get(c.outer_dih0_m,j), cos_beta, grad);
    }

    //Computes gradient from dihedral angles constituted by the planes bpa, pam ~162 FLOPs
    INLINE coord3d outer_dihedral_gradient_p(const Constants& c) const
    {
        coord3d nbpa_hat, npam_hat; real_t cos_p, cos_a, r_sin_p, r_sin_a;
        cos_a = dot(ap_hat,am_hat);  r_sin_a = (real_t)1.0/sqrt((real_t)1.0 - cos_a*cos_a); npam_hat = cross(ap_hat,am_hat)  * r_sin_a;
        cos_p = dot(pb_hat,-ap_hat); r_sin_p = (real_t)1.0/sqrt((real_t)1.0 - cos_p*cos_p); nbpa_hat = cross(pb_hat,-ap_hat) * r_sin_p;

        real_t cos_beta = dot(nbpa_hat, npam_hat); //Outer dihedral angle bpa, pam.
        real_t cot_p = cos_p * r_sin_p * r_sin_p;
        real_t cot_a = cos_a * r_sin_a * r_sin_a;

        //Derivative w.r.t. outer dihedral angle, factorized version of Eq. 33.
        coord3d grad = cross(npam_hat,pb_hat)*r_rap*r_sin_p - (cross(am_hat,nbpa_hat)*r_rap + cross(nbpa_hat,ap_hat)*r_ram)*r_sin_a +
                        cos_beta*(am_hat*r_ram + r_rap * ((real_t)2.0*ap_hat + cot_p*(pb_hat+cos_p*ap_hat)) - cot_a*(r_rap*(am_hat - ap_hat*cos_a) + r_ram*(ap_hat-am_hat*cos_a)));
        
        //Eq. 33 multiplied by harmonic term.
        return d_get(c.f_outer_dihedral,j) * harmonic_energy_gradient(d_get(c.outer_dih0_p,j), cos_beta, grad);
    }
    // Internal coordinate gradients
    INLINE coord3d bond_length_gradient(const Constants& c) const { return d_get(c.f_bond,j) * harmonic_energy_gradient(bond(),d_get(c.r0,j),ab_hat);}
    //Sum of angular gradient components.
    INLINE coord3d angle_gradient(const Constants& c) const { return inner_angle_gradient(c) + outer_angle_gradient_p(c) + outer_angle_gradient_m(c);}
    //Sum of inner and outer dihedral gradient components.
    INLINE coord3d dihedral_gradient(const Constants& c) const { return inner_dihedral_gradient(c) + outer_dihedral_gradient_a(c) + outer_dihedral_gradient_m(c) + outer_dihedral_gradient_p(c);}
    //coord3d flatness()             const { return ;  }   
    
    INLINE real_t bond_energy(const Constants& c) const {return (real_t)0.5 *d_get(c.f_bond,j) *harmonic_energy(bond(),d_get(c.r0,j));}
    INLINE real_t bend_energy(const Constants& c) const {return d_get(c.f_inner_angle,j)* harmonic_energy(angle(),d_get(c.angle0,j));}
    INLINE real_t dihedral_energy(const Constants& c) const {return d_get(c.f_inner_dihedral,j)* harmonic_energy(dihedral(),d_get(c.inner_dih0,j));}
    //Harmonic energy contribution from bond stretching, angular bending and dihedral angle bending.
    //71 FLOPs
    INLINE real_t energy(const Constants& c) const {return bond_energy(c) + bend_energy(c) + dihedral_energy(c); }
    //Sum of bond, angular and dihedral gradient components.
    INLINE coord3d gradient(const Constants& c) const{return bond_length_gradient(c) + angle_gradient(c) + dihedral_gradient(c);}


    //Reciprocal lengths of arcs ab, ac, am, ap.
    real_t
        rab,
        r_rab,
        r_rac,
        r_rad,
        r_ram,
        r_rap;

    //Base Arcs,
    coord3d
        ab,
        ac,
        ad;

    //All normalized arcs required to perform energy & gradient calculations.
    //Note that all these arcs are cyclical the arc ab becomes: ab->ac->ad,  the arc ac becomes: ac->ad->ab , the arc bc becomes: bc->cd->db (For iterations 0, 1, 2)
    //As such the naming convention here is related to the arcs as they are used in the 0th iteration.
    coord3d 
        ab_hat,
        ac_hat,
        ad_hat,
        bp_hat,
        bm_hat,
        am_hat,
        ap_hat,
        ba_hat,
        bc_hat,
        cd_hat,
        mp_hat,
        mb_hat,
        pa_hat,
        pb_hat;
};


INLINE coord3d gradient(coord3d* X) const {
    BLOCK_SYNC
    coord3d grad = {0.0, 0.0, 0.0};
    for (uint8_t j = 0; j < 3; j++ ){
        ArcData arc = ArcData(j, X, node_graph);
        grad += arc.gradient(constants);
    }
    return grad;
}

INLINE real_t energy(coord3d* X) const {
    BLOCK_SYNC
    real_t arc_energy = (real_t)0.0;

    //(71 + 124) * 3 * N  = 585*N FLOPs
    for (uint8_t j = 0; j < 3; j++ ){
        ArcData arc = ArcData(j, X, node_graph);
        arc_energy += arc.energy(constants);
    }
    return reduction(sdata, arc_energy);;

}

INLINE real_t gradnorm(coord3d* X, coord3d& d)const {
    return reduction(sdata, dot(-gradient(X),d));
}

//Bracketing method designed to find upper bound for linesearch method that matches 
//reference python implementation by Buster.
INLINE real_t FindLineSearchBound(coord3d* X, coord3d& r0, coord3d* X1) const{
    real_t bound        = 1e-5;
    bool negative_grad  = true;
    size_t iter         = 0;
    while (negative_grad && iter < 1000)
    {   
        bound *= (real_t)1.5;
        X1[node_id] = X[node_id] + bound * r0;
        real_t gradsum = reduction(sdata, dot(gradient(X1),r0));
        negative_grad = (gradsum < 0);
    }
    return bound;
}

//For compatibility with reference implementation by Buster. Warning: Extremely slow, requires a lot of gradient evaluations.
INLINE real_t Bisection(coord3d* X, coord3d& r0, coord3d* X1, coord3d* X2){
    real_t dfc = 1; size_t count = 0;
    real_t c; real_t a = 0.0; real_t b = FindLineSearchBound(X,r0,X1);
    coord3d d;
    while (abs(dfc) > 1e-10 && count < 1000){
        count++;
        c =  (a+b)/2;
        X1[node_id] = X[node_id] + c*r0;
        d  =  gradient(X1);
        dfc = reduction(sdata,dot(d,r0)); 

        if (dfc < (real_t)0.0){
            a = c;
        }
        else{
            b = c;
        }
    }
    return c;
}

//Brents Method for line-search using fixed number of iterations.
INLINE real_t BrentsMethod(coord3d* X, coord3d& r0, coord3d* X1, coord3d* X2)const{
    real_t a,b,s,d;
    a = (real_t)0.0; //b = (real_t)1.0; 

    //To match python reference implementation by Buster.
    b = (real_t)1.0;//FindLineSearchBound(X,r0,X1);

    X1[node_id] = X[node_id] + a * r0;
    X2[node_id] = X[node_id] + b * r0;

    real_t f_a = gradnorm(X1,r0);
    real_t f_b = gradnorm(X2,r0);

    if (f_a*f_b > 0)
    {
        return b;
    }
    if (abs(f_a) < abs(f_b)){swap_reals(a,b); swap_reals(f_a,f_b);}

    real_t c = a; real_t f_c = f_a;
    bool flag = true;

    for (uint8_t i = 0; i < 30; i++)
    {   
        // Inverse quadratic interpolation
        if ( (f_a != f_c) && (f_b != f_c) )
        {
            s = a*f_a*f_c / ((f_a - f_b)*(f_a - f_c)) + b * f_a * f_c / ((f_b-f_a)*(f_b-f_c)) + c*f_a*f_b/((f_c-f_a)*(f_c-f_b));
        }else // Secant Method
        {
            s = b - f_b*(b-a)/(f_b-f_a);
        }
        
        bool condition_1 = !(s > (((real_t)3.0*a + b)/(real_t)4.0) && s < b);
        bool condition_2 = flag && (abs(s-b) >= abs(b-c)/(real_t)2.0);
        bool condition_3 = !flag && (abs(s-b) >= abs(c-d)/(real_t)2.0);
        bool condition_4 = flag && (abs(b-c) < (real_t)5e-8);
        bool condition_5 = !flag && (abs(c-d) < (real_t)5e-8);

        if (condition_1 || condition_2 || condition_3 || condition_4 || condition_5)
        {
            s = (a+b) / (real_t)2.0; // Bisection Method
            flag = true;
        }else
        {
            flag = false;
        }
        X1[node_id] = X[node_id] + s * r0;
        real_t f_s = gradnorm(X1,r0);
        d = c;
        c = b; f_c = f_b;
        if (f_a*f_s < 0)
        {
            b = s; f_b = f_s;
        }else
        {
            a = s; f_a = f_s;
        }
        if (abs(f_a) < abs(f_b))
        {
            swap_reals(a,b); swap_reals(f_a,f_b);
        }
    }
    return b;
}

//Golden Section Search, using fixed iterations.
INLINE real_t GSS(coord3d* X, coord3d& r0, coord3d* X1, coord3d* X2) const{
    constexpr real_t tau = (real_t)0.6180339887;
    //Line search x - values;
    real_t a = 0.0; real_t b = (real_t)1.0;
    
    real_t x1,  x2;
    x1 = (a + (1 - tau) * (b - a));
    x2 = (a + tau * (b - a));
    //Actual coordinates resulting from each traversal 
    X1[node_id] = X[node_id] + x1 * r0;
    X2[node_id] = X[node_id] + x2 * r0;
    real_t f1 = energy(X1);
    real_t f2 = energy(X2);

    for (uint8_t i = 0; i < 20; i++){
        if (f1 > f2){
            a = x1;
            x1 = x2;
            f1 = f2;
            x2 = a + tau * (b - a);
            X2[node_id] = X[node_id] + x2 * r0;
            f2 = energy(X2);
        }else
        {
            b = x2;
            x2 = x1;
            f2 = f1;
            x1 = a + ((real_t)1.0 - tau) * (b - a);
            X1[node_id] = X[node_id] + x1 * r0;
            f1 = energy(X1);
        }
    }
    if (f1 > energy(X)) {return (real_t)0.0;}
    //Line search coefficient
    real_t alpha = (a+b)/(real_t)2.0;
    return alpha;
}

INLINE  void CG(coord3d* X, coord3d* X1, coord3d* X2, const size_t MaxIter)
{
    real_t alpha, beta, g0_norm2, s_norm;
    coord3d g0,g1,s;
    g0 = gradient(X);
    s = -g0;
    //Normalize To match reference python implementation by Buster.
    #if USE_MAX_NORM==1
        s_norm = reduction_max(sdata, max(max(s.x,s.y),s.z));
    #else
        s_norm = sqrt(reduction(sdata, dot(s,s)));
    #endif
    s /= s_norm;

    for (size_t i = 0; i < MaxIter; i++)
    {   
        alpha = LINESEARCH_METHOD(X,s,X1,X2);
        if (alpha > (real_t)0.0){X1[node_id] = X[node_id] + alpha * s;}
        g1 = gradient(X1);
        //Polak Ribiere method
        g0_norm2 = reduction(sdata, dot(g0, g0));
        beta = max(reduction(sdata, dot(g1, (g1 - g0))) / g0_norm2,(real_t)0.0);

        if (alpha > (real_t)0.0){X[node_id] = X1[node_id];}else{ g1 = g0; beta = (real_t) 0.0;}
        s = -g1 + beta*s;
        g0 = g1;
        //Normalize Search Direction using MaxNorm or 2Norm
        #if USE_MAX_NORM==1
            s_norm = reduction_max(sdata, max(max(s.x,s.y),s.z));
        #else
            s_norm = sqrt(reduction(sdata, dot(s,s)));
        #endif
        s /= s_norm;
    }   
} 
};

__device__ void check_batch(IsomerBatch &B, device_real_t* global_reduction_array, const size_t max_iterations){
    DEVICE_TYPEDEFS
    extern __shared__ real_t smem[];
    clear_cache(smem,Block_Size_Pow_2);
    if (B.statuses[blockIdx.x] == NOT_CONVERGED){
    size_t offset = blockIdx.x * blockDim.x;
    Constants constants     = Constants(B);
    NodeGraph node_graph    = NodeGraph(B);
    ForceField FF           = ForceField(node_graph, constants, smem);
    coord3d* X              = reinterpret_cast<coord3d*>(B.X+offset*3);

    coord3d rel_bond_err, rel_angle_err, rel_dihedral_err;
    BLOCK_SYNC
    for (uint8_t j = 0; j < 3; j++){
        auto arc            = ForceField::ArcData(j, X, node_graph);
        d_set(rel_bond_err,      j, abs(abs(arc.bond()       - d_get(constants.r0,j))        /d_get(constants.r0,j)));
        d_set(rel_angle_err,     j, abs(abs(arc.angle()      - d_get(constants.angle0,j))    /d_get(constants.angle0,j)));
        d_set(rel_dihedral_err,  j, abs(abs(arc.dihedral()   - d_get(constants.inner_dih0,j))/d_get(constants.inner_dih0,j)));
    }

    real_t bond_max         = reduction_max(smem, max(rel_bond_err));
    real_t angle_max        = reduction_max(smem, max(rel_angle_err));
    real_t dihedral_max     = reduction_max(smem, max(rel_dihedral_err));
    real_t bond_rms         = sqrt(reduction(smem,dot(rel_bond_err,rel_bond_err))/blockDim.x);
    real_t angle_rms        = sqrt(reduction(smem,dot(rel_angle_err,rel_angle_err))/blockDim.x);
    real_t dihedral_rms     = sqrt(reduction(smem,dot(rel_dihedral_err,rel_dihedral_err))/blockDim.x);
    real_t bond_mean        = reduction(smem,sum(rel_bond_err))/blockDim.x;
    real_t angle_mean       = reduction(smem,sum(rel_angle_err))/blockDim.x;
    real_t dihedral_mean    = reduction(smem,sum(rel_dihedral_err))/blockDim.x;
    real_t grad_norm        = sqrt(reduction(smem,dot(FF.gradient(X), FF.gradient(X))))/blockDim.x;
    real_t energy           = FF.energy(X); 
    
    bool converged = (grad_norm < 1e-2) && !isnan(grad_norm);
    //real_t num_converged    = global_reduction(smem,global_reduction_array,converged,(threadIdx.x==0) && (B.statuses[blockIdx.x] == IsomerspaceForcefield::NOT_CONVERGED));
    //if(threadIdx.x + blockIdx.x == 0){printf("%d", (int)num_converged); printf("/ %d Fullerenes Converged in Batch \n", (int)gridDim.x);}

    if(threadIdx.x == 0 && B.statuses[blockIdx.x] != EMPTY){
        if (converged)
        {
            B.statuses[blockIdx.x] = CONVERGED;
        } else if (B.iterations[blockIdx.x] >= max_iterations || isnan(grad_norm)) {
            B.statuses[blockIdx.x] = FAILED;
        }
    }
    }
    
}

__global__ void __optimize_batch(IsomerBatch B, const size_t iterations, const size_t max_iterations){
    DEVICE_TYPEDEFS
    extern __shared__ real_t smem[];
    clear_cache(smem,Block_Size_Pow_2);
    if (B.statuses[blockIdx.x] == NOT_CONVERGED)
    {
        real_t* base_pointer        = smem + Block_Size_Pow_2;
        size_t offset               = blockIdx.x * blockDim.x;
        size_t node_id              = threadIdx.x;
        size_t N                    = blockDim.x;
        
        //Set VRAM pointer to start of each fullerene, as opposed to at the start of the isomerbatch.
        coord3d* X = reinterpret_cast<coord3d*>(B.X+3*offset);

        //Assign a section of L1 cache to each set of cartesian coordinates X, X1 and X2.
        coord3d* sX =reinterpret_cast<coord3d*>(base_pointer);
        coord3d* X1 =reinterpret_cast<coord3d*>(base_pointer+3*N);
        coord3d* X2 =reinterpret_cast<coord3d*>(base_pointer+6*N);  

                                    
        sX[node_id] = X[node_id];   //Copy cartesian coordinates from DRAM to L1 Cache.
        X           = sX;       //Switch coordinate pointer from DRAM to L1 Cache.


        //Pre-compute force constants and store in registers.

        Constants constants = Constants(B);
        NodeGraph nodeG     = NodeGraph(B);

        //Create forcefield struct and use optimization algorithm to optimize the fullerene 
        ForceField FF = ForceField(nodeG, constants, smem);
        FF.CG(X,X1,X2,iterations);
        BLOCK_SYNC
        
        //Copy data back from L1 cache to DRAM 
        reinterpret_cast<coord3d*>(B.X)[offset + threadIdx.x]= X[threadIdx.x];

        if (threadIdx.x == 0) {B.iterations[blockIdx.x] += iterations;}
    }
    //Check the convergence of isomers and assign status accordingly.
    //check_batch(B, g_array, max_iterations)
}


hipError_t optimize_batch(IsomerBatch& B, const size_t iterations, const size_t max_iterations, const hipStream_t stream){
    size_t smem = sizeof(device_coord3d)*3*B.n_atoms + sizeof(device_real_t)*Block_Size_Pow_2;
    static LaunchDims dims((void*)__optimize_batch, B.n_atoms, smem);
    dims.update_dims((void*)__optimize_batch, B.n_atoms, smem);
    void* kargs[]{(void*)&B, (void*)&iterations, (void*)&max_iterations};
    return safeCudaKernelCall((void*)__optimize_batch, dims.get_grid(), dims.get_block(), kargs, smem, stream);
}

}}

#pragma once
#include "fullerenes/gpu/isomerspace_forcefield.hh"
#include "fullerenes/gpu/isomerspace_tutte.hh"
#include "auxiliary_cuda_functions.cu"
#include "fullerenes/gpu/gpudatastruct.hh"


void GPUDataStruct::allocate(GPUDataStruct& G, size_t N, const size_t batch_size, const BufferType buffer_type){
    if((!G.allocated)){
        G.buffer_type = buffer_type;
        G.batch_size  = batch_size; 
        G.N           = N; 
        size_t num_elements = N*batch_size;
        if (buffer_type == DEVICE_BUFFER){
            for (size_t i = 0; i < G.pointers.size(); i++) {
                hipMalloc(get<1>(G.pointers[i]), num_elements* get<2>(G.pointers[i])); 
            }
            printLastCudaError("Failed to allocate device struct");
        }else{
            for (size_t i = 0; i < G.pointers.size(); i++) {
                *get<1>(G.pointers[i])= malloc(num_elements* get<2>(G.pointers[i])); 
            }
        }        
        G.allocated = true;
    }
}

void GPUDataStruct::free(GPUDataStruct& G){
    if(G.allocated){
        if (G.buffer_type == DEVICE_BUFFER){    
            for (size_t i = 0; i < G.pointers.size(); i++) {
                hipFree(*get<1>(G.pointers[i]));
            }
            printLastCudaError("Failed to free device struct"); 
        } else{
            for (size_t i = 0; i < G.pointers.size(); i++) {
                std::free(*get<1>(G.pointers[i])); 
            }
        }
        G.allocated = false;
    }
}

void GPUDataStruct::copy(GPUDataStruct& destination, const GPUDataStruct& source){
    if(source.batch_size > 0){
    for (size_t i = 0; i < destination.pointers.size(); i++)
    {
        hipMemcpy(*(get<1>(destination.pointers[i])) , *(get<1>(source.pointers[i])), get<2>(source.pointers[i])*source.N*source.batch_size, hipMemcpyKind(2*source.buffer_type +  destination.buffer_type));
    }
    }
    else{
        std::cout << "WARNING: Call to copy made for 0 isomers \n";
    }
    printLastCudaError("Failed to copy struct");
}

void operator <<= (GPUDataStruct& destination, const GPUDataStruct& source){
    GPUDataStruct::copy(destination, source);
}


void operator <<= (IsomerspaceForcefield::IsomerBatch& a, const IsomerspaceForcefield::IsomerBatch& b){
    GPUDataStruct::copy(a,b);
    GPUDataStruct::copy(a.stats,b.stats);
}

void operator <<= (IsomerspaceTutte::IsomerBatch& a, const IsomerspaceTutte::IsomerBatch& b){
    GPUDataStruct::copy(a,b);
    GPUDataStruct::copy(a.stats,b.stats);
}

void IsomerspaceTutte::eject_isomer(size_t i, size_t idx){
    size_t n_offset  = idx*3*N;      //neighbour offset
    size_t c_offset  = idx*2*N;      //coords offset
    size_t f_offset  = idx*N;    //outer_face offset
    IsomerBatchStats stats = h_batch[i].stats;
    neighbours_t neighbours(N); std::vector<coord2d> xys(N);
    for (size_t j = 0; j < N; j++) {
        neighbours[j]       = std::vector<node_t>(h_batch[i].neighbours + n_offset + j*3, h_batch[i].neighbours + n_offset + j*3 + 3);
        xys[j]   = {reinterpret_cast<GPU_REAL2*>(h_batch[i].xys + c_offset)[j].x,reinterpret_cast<GPU_REAL2*>(h_batch[i].xys + c_offset)[j].y};
                            
    }
    FullereneGraph P = FullereneGraph(Graph(neighbours,true));
    P.layout2d       = xys;
    output_queue.push({stats.isomer_IDs[idx],P});
    stats.isomer_statuses[idx]==CONVERGED ? converged_count++ : failed_count++;
}

void IsomerspaceForcefield::eject_isomer(size_t i, size_t idx){
    IsomerBatchStats stats = h_batch[i].stats;
    size_t offset   = idx*3*N;
    neighbours_t output(N); std::vector<coord3d> output_X(N);
    for (size_t j = 0; j < N; j++) {
        output[j] = std::vector<node_t>(h_batch[i].neighbours + offset + j*3, h_batch[i].neighbours + offset + j*3 + 3);
        output_X[j] = {h_batch[i].X[offset + j*3], h_batch[i].X[offset + j*3 + 1], h_batch[i].X[offset + j*3 + 2]};
    }
    Polyhedron P = Polyhedron(FullereneGraph(Graph(output,true)), output_X);
    output_queue.push({stats.isomer_IDs[idx],P});
    stats.isomer_statuses[idx]==CONVERGED ? converged_count++ : failed_count++;
}

void IsomerspaceTutte::update_batch(){
    while (batch_size < batch_capacity && !insert_queue.empty()){
        for (size_t i = 0; i < this->device_count; i++)
        if (batch_sizes[i] < device_capacities[i]){
            IsomerBatchStats stats = h_batch[i].stats;
            size_t idx       = index_queue[i].front();
            size_t n_offset  = idx*3*N;      //neighbour offset
            size_t f_offset  = idx*N;    //outer_face offset
            if ((stats.isomer_statuses[idx] == CONVERGED) || (stats.isomer_statuses[idx]==FAILED))
            {
                eject_isomer(i,idx);
            }

            size_t ID        = insert_queue.front().first;
            FullereneGraph P = insert_queue.front().second;
            P.outer_face     = P.get_face_oriented({0,P.neighbours[0][0]}, INT32_MAX);

            for (device_node_t u = 0; u < N; u++){
                for (int j = 0; j < 3; j++) h_batch[i].neighbours[u*3 + j + n_offset]     = P.neighbours[u][j];
            }
            for (device_node_t u = 0; u < P.outer_face.size(); u++) {
                h_batch[i].outer_face[u + f_offset] = P.outer_face[u]; 
            }
            //TODO: Construct polyhedrons from neighbour list and coordinates and store that along with convergence status in an output_queue instead.
            //This is currently what happens to all isomers that are finished, we simply store their ID and energy.

            stats.iteration_counts[idx]   = 0;
            stats.isomer_statuses[idx]    = NOT_CONVERGED;
            stats.isomer_IDs[idx]         = ID;
            stats.Nface[idx]              = P.outer_face.size();
            
            batch_size++;
            batch_sizes[i]++;
            insert_queue.pop();
            index_queue[i].pop();
            break;
    }

    }
    if (insert_queue.empty()){
        for (size_t i = 0; i < device_count; i++)
        for (size_t j = 0; j < device_capacities[i]; j++){   
            if ((h_batch[i].stats.isomer_statuses[j] == CONVERGED) || (h_batch[i].stats.isomer_statuses[j]==FAILED)){
                eject_isomer(i,j);
                h_batch[i].stats.isomer_statuses[j] = EMPTY;
            }
        }
        
    }
}

void IsomerspaceForcefield::update_batch(){
    while (batch_size < batch_capacity && !insert_queue.empty()){
        for (size_t i = 0; i < this->device_count; i++)
        if (batch_sizes[i] < device_capacities[i]){
            IsomerBatchStats stats = h_batch[i].stats;
            size_t idx      = index_queue[i].front();
            size_t offset   = idx*3*N;
            if ((stats.isomer_statuses[idx] == CONVERGED) || (stats.isomer_statuses[idx]==FAILED))
            {
                eject_isomer(i,idx);
            }
            
            size_t ID       = insert_queue.front().first;
            Polyhedron P    = insert_queue.front().second;

            for (device_node_t u = 0; u < N; u++){
                for (int j = 0; j < 3; j++){
                    device_node_t v = P.neighbours[u][j];
                    size_t arc_index = u*3 + j + offset;
                    h_batch[i].neighbours  [arc_index] = v;
                    h_batch[i].next_on_face[arc_index] = P.next_on_face(u,v);
                    h_batch[i].prev_on_face[arc_index] = P.prev_on_face(u,v);
                    h_batch[i].face_right  [arc_index] = P.face_size(u,v);
                    h_batch[i].X           [arc_index] = P.points[u][j];
                }   
            }
            


            stats.iteration_counts[idx]   = 0;
            stats.isomer_statuses[idx]    = NOT_CONVERGED;
            stats.isomer_IDs[idx]         = ID;
            
            batch_size++;
            batch_sizes[i]++;
            insert_queue.pop();
            index_queue[i].pop();
            break;
        }
    }
    if (insert_queue.empty()){
        for (size_t i = 0; i < device_count; i++){
        IsomerBatchStats stats = h_batch[i].stats;
        for (size_t j = 0; j < device_capacities[i]; j++){   
            if ((stats.isomer_statuses[j] == CONVERGED) || (stats.isomer_statuses[j]==FAILED)){
                eject_isomer(i,j);
                stats.isomer_statuses[j] = EMPTY;
            }
        }
        }
    }
}


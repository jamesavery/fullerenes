#pragma once
#include "fullerenes/gpu/isomerspace_forcefield.hh"
#include "fullerenes/gpu/isomerspace_tutte.hh"
#include "auxiliary_cuda_functions.cu"
#include "fullerenes/gpu/gpudatastruct.hh"
#include "fullerenes/gpu/isomerspace_kernel.hh"

void GPUDataStruct::allocate(GPUDataStruct& G, size_t N, const size_t batch_size, const BufferType buffer_type){
    if((!G.allocated)){
        G.buffer_type = buffer_type;
        G.batch_size  = batch_size; 
        G.N           = N; 
        if (buffer_type == DEVICE_BUFFER){
            for (size_t i = 0; i < G.pointers.size(); i++) {
                size_t num_elements = get<3>(G.pointers[i]) ?  N*batch_size : batch_size;
                hipMalloc(get<1>(G.pointers[i]), num_elements* get<2>(G.pointers[i])); 
            }
            printLastCudaError("Failed to allocate device struct");
        }else{
            for (size_t i = 0; i < G.pointers.size(); i++) {
                size_t num_elements = get<3>(G.pointers[i]) ?  N*batch_size : batch_size;
                *get<1>(G.pointers[i])= malloc(num_elements* get<2>(G.pointers[i])); 
            }
        }        
        G.allocated = true;
    }
}

void GPUDataStruct::free(GPUDataStruct& G){
    if(G.allocated){
        if (G.buffer_type == DEVICE_BUFFER){    
            for (size_t i = 0; i < G.pointers.size(); i++) {
                hipFree(*get<1>(G.pointers[i]));
            }
            printLastCudaError("Failed to free device struct"); 
        } else{
            for (size_t i = 0; i < G.pointers.size(); i++) {
                std::free(*get<1>(G.pointers[i])); 
            }
        }
        G.allocated = false;
    }
}

void GPUDataStruct::copy(GPUDataStruct& destination, const GPUDataStruct& source){
    if(source.batch_size > 0){
    for (size_t i = 0; i < destination.pointers.size(); i++)
    {
        size_t num_elements = get<3>(destination.pointers[i]) ?  destination.N*destination.batch_size : destination.batch_size;
        hipMemcpy(*(get<1>(destination.pointers[i])) , *(get<1>(source.pointers[i])), get<2>(source.pointers[i])*num_elements, hipMemcpyKind(2*source.buffer_type +  destination.buffer_type));
    }
    }
    else{
        std::cout << "WARNING: Call to copy made for 0 isomers \n";
    }
    printLastCudaError("Failed to copy struct");
}

void operator <<= (GPUDataStruct& destination, const GPUDataStruct& source){
    GPUDataStruct::copy(destination, source);
}

template<typename T>
void IsomerspaceKernel<T>::eject_isomer(size_t i, size_t idx){
    IsomerBatch B    = h_batch[i];
    size_t offset    = idx*3*N;
    size_t c_offset  = idx*2*N;
    neighbours_t output(N); std::vector<coord3d> output_X(N); std::vector<coord2d> xys(N);
    for (size_t j = 0; j < N; j++) {
        output[j] = std::vector<node_t>(B.neighbours + offset + j*3, B.neighbours + offset + j*3 + 3);
        xys[j]      = {reinterpret_cast<device_coord2d*>(B.xys + c_offset)[j].x,reinterpret_cast<device_coord2d*>(B.xys + c_offset)[j].y};
        output_X[j] = {B.X[offset + j*3], B.X[offset + j*3 + 1], B.X[offset + j*3 + 2]};
    }
    Polyhedron P = Polyhedron(FullereneGraph(Graph(output,true)), output_X);
    P.layout2d = xys;
    output_queue.push({B.IDs[idx],P});
    B.statuses[idx]==CONVERGED ? converged_count++ : failed_count++;
}

template<typename T>
void IsomerspaceKernel<T>::update_batch(){
    while (batch_size < batch_capacity && !insert_queue.empty()){
        for (size_t i = 0; i < this->device_count; i++)
        if (batch_sizes[i] < device_capacities[i]){
            IsomerBatch B = h_batch[i];
            size_t idx       = index_queue[i].front();
            size_t offset  = idx*3*N;      //neighbour offset
            if ((B.statuses[idx] == CONVERGED) || (B.statuses[idx]==FAILED))
            {
                eject_isomer(i,idx);
            }

            size_t ID        = insert_queue.front().first;
            Polyhedron P     = insert_queue.front().second;

            for (device_node_t u = 0; u < N; u++){
                for (int j = 0; j < 3; j++){
                    device_node_t v = P.neighbours[u][j];
                    size_t arc_index = u*3 + j + offset;
                    B.neighbours  [arc_index] = v;
                    B.X           [arc_index] = !P.points.empty() ? P.points[u][j] : 0.0;
                }   
            }

            B.iterations[idx]   = 0;
            B.statuses[idx]    = NOT_CONVERGED;
            B.IDs[idx]         = ID;
            
            batch_size++;
            batch_sizes[i]++;
            insert_queue.pop();
            index_queue[i].pop();
            break;
    }

    }
    if (insert_queue.empty()){
        for (size_t i = 0; i < device_count; i++)
        for (size_t j = 0; j < device_capacities[i]; j++){   
            if ((h_batch[i].statuses[j] == CONVERGED) || (h_batch[i].statuses[j]==FAILED)){
                eject_isomer(i,j);
                h_batch[i].statuses[j] = EMPTY;
            }
        }
    }
}


#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <cooperative_groups/scan.h>
#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include "fullerenes/gpu/cuda_definitions.h"
#include "fullerenes/gpu/cu_array.hh"
#include "fullerenes/gpu/isomer_batch.hh"
#include "fullerenes/gpu/kernels.hh"
#include <hipsolver.h>
#include <hipsparse.h>

#define N_STREAMS 16

namespace gpu_kernels{
    namespace isomerspace_eigen{
        #include "device_includes.cu"

        void eigensolve_cusolver(const IsomerBatch& B, const CuArray<device_real_t>& hessians, const CuArray<device_node_t>& cols, CuArray<device_real_t>& eigenvalues, const LaunchCtx& ctx, const LaunchPolicy policy){
            static std::vector<hipsolverHandle_t> cusolverHs(N_STREAMS, NULL);
            static int m = B.n_atoms*3;
            static int lda = m;
            static bool initialized = false;
            static std::vector<CuArray<device_real_t>> As(N_STREAMS);
            static std::vector<LaunchCtx> ctxs(N_STREAMS);
            static int nisomers = B.isomer_capacity;
            static std::vector<CuArray<device_real_t>> d_works(N_STREAMS);
            static std::vector<std::vector<device_real_t>> h_works(N_STREAMS);
            static std::vector<int*> d_infos(N_STREAMS, nullptr);
            static std::vector<int> infos(N_STREAMS, 1);
            static std::vector<int> lworks(N_STREAMS, 0);
            static std::vector<size_t> workspaceInBytesOnDevice(N_STREAMS, 0);
            static std::vector<size_t> workspaceInBytesOnHost(N_STREAMS, 0);
            int ncols = 10*3;
            int nn = m*ncols;
            hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_NOVECTOR; // compute eigenvalues and eigenvectors.
            hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
            auto T0 = std::chrono::steady_clock::now();
            if (!initialized){
                for (size_t I = 0; I < N_STREAMS ; I++){
                    As[I] = CuArray<device_real_t>(m*m, 0.);
                    ctxs[I] = LaunchCtx(0);
                    hipsolverDnCreate(&cusolverHs[I]);
                    hipsolverSetStream(cusolverHs[I], ctxs[I].stream);   
                    hipMalloc(reinterpret_cast<void **>(&(d_infos[I])), sizeof(int));
                  
                    #if FLOAT_TYPE == 3
                    //    hipsolverDnDsyevd_bufferSize(cusolverHs[I], jobz, uplo, m, As[I].data, lda, &(eigenvalues.data[I*m]), &(lworks[I]));
                        cusolverDnXsyevd_bufferSize(cusolverHs[I], NULL, jobz, uplo, m, HIP_R_64F, (const void*)As[I].data, lda, HIP_R_64F, (const void*)&(eigenvalues.data[I*m]), HIP_R_64F, &workspaceInBytesOnDevice[I], &workspaceInBytesOnHost[I]);  
                    #elif FLOAT_TYPE == 2
                        cusolverDnXsyevd_bufferSize(cusolverHs[I], NULL, jobz, uplo, m, HIP_R_32F, (const void*)As[I].data, lda, HIP_R_32F, (const void*)&(eigenvalues.data[I*m]), HIP_R_32F, &workspaceInBytesOnDevice[I], &workspaceInBytesOnHost[I]);  
                    //  hipsolverDnSsyevd_bufferSize(cusolverHs[I], jobz, uplo, m, As[I].data, lda, &(eigenvalues.data[I*m]), &(lworks[I]));
                    #endif
                    d_works[I] = CuArray<device_real_t>(workspaceInBytesOnDevice[I]/sizeof(device_real_t));
                    h_works[I].resize(workspaceInBytesOnHost[I]/sizeof(device_real_t));
                    //hipMalloc(reinterpret_cast<void **>(&(d_works[I])), sizeof(device_real_t) * workspaceInBytesOnDevice[I]);
                    //hipHostMalloc(reinterpret_cast<void **>(&(h_works[I])), sizeof(device_real_t) * workspaceInBytesOnHost[I]);
                    printLastCudaError("eigensolve");
                }    
                initialized = true;
            }
            auto T1 = std::chrono::steady_clock::now();
            std::cout << "Initializing the eigensolver took " << std::chrono::duration_cast<std::chrono::microseconds>(T1 - T0).count() / (float)nisomers << " us / isomer" << std::endl;
            int counter = 0;
            //Loading the sparse hessians into dense matrices; Might be a bottleneck.
            auto start = std::chrono::steady_clock::now();
            auto fill_As = [&] (){
                int I = 0;
                auto start_counter = counter;
                for (int II = start_counter; II < std::min(start_counter + N_STREAMS, nisomers)  ; II++){
                counter++;
                for (size_t i = 0; i < m; i++){ //Number of rows in the hessian
                    for (size_t j = 0; j < ncols; j++){ //Number of columns in the hessian, it is 10*3 because we have a term for the node itself, it's 3 neighbours and 6 outer neighbours, each with 3 dx, dy, dz terms
                        int col = cols.data[int(II*nn + i*ncols + j)];
                        As[I][int(i*m + col)] = hessians.data[int(II*nn + i*ncols + j)];
                    }
                }
                ++I;
            }
            };
            
            auto end = std::chrono::steady_clock::now();
            std::cout << "Loading the sparse hessians into dense matrices took " << std::chrono::duration_cast<std::chrono::microseconds>(end - start).count() / (float)nisomers << " us / isomer" << std::endl;   

            if (policy == LaunchPolicy::SYNC){
                ctx.wait();
            }

            start = std::chrono::steady_clock::now();
            for (size_t I = 0; I < nisomers ; I++){
                int idx = I % N_STREAMS;
                if (idx == 0) fill_As();
                //std::cout  << "Inum: " << counter << std::endl;
                #if FLOAT_TYPE == 3
                    cusolverDnXsyevd( cusolverHs[idx], NULL, jobz, uplo, m, HIP_R_64F, As[idx].data, lda, HIP_R_64F, &eigenvalues.data[I*m], HIP_R_64F, (void *) d_works[idx].data, workspaceInBytesOnDevice[idx], (void *) h_works[idx].data(), workspaceInBytesOnHost[idx], d_infos[idx]);
                //    hipsolverDnDsyevd(cusolverHs[idx], jobz, uplo, m, As[idx].data,    lda, &eigenvalues.data[I*m], d_works[idx],   lworks[idx], d_infos[idx]);
                #elif FLOAT_TYPE == 2
                    cusolverDnXsyevd( cusolverHs[idx], NULL, jobz, uplo, m, HIP_R_32F, As[idx].data, lda, HIP_R_32F, &eigenvalues.data[I*m], HIP_R_32F, (void *) d_works[idx].data, workspaceInBytesOnDevice[idx], (void *) h_works[idx].data(), workspaceInBytesOnHost[idx], d_infos[idx]);
                //    hipsolverDnSsyevd(cusolverHs[idx], jobz, uplo, m, As[idx].data,    lda, &eigenvalues.data[I*m], d_works[idx],   lworks[idx], d_infos[idx]);
                #endif
                //hipMemcpyAsync(&infos[idx], d_infos[idx], sizeof(int), hipMemcpyDeviceToHost, ctx.stream);
            }

            if (policy == LaunchPolicy::SYNC){
                for (size_t I = 0; I < N_STREAMS ; I++){
                    ctxs[I].wait();
                }
            }
            end = std::chrono::steady_clock::now();
            std::cout << "eigensolve time: " << std::chrono::duration_cast<std::chrono::microseconds>(end - start).count() / (float)nisomers <<  " us / isomer" <<std::endl;
        
        }
        void __global__ eigensolve_(const IsomerBatch B, CuArray<device_real_t> Qglobal, const CuArray<device_real_t> H, const CuArray<device_node_t> cols, CuArray<device_real_t> eigenvalues){
            DEVICE_TYPEDEFS
            extern __shared__ real_t smem[];
            int N = B.n_atoms * 3; //Number of rows in the hessian
            constexpr int M = 10*3;          //Number of columns in the hessian
            real_t* betas = smem + N;
            real_t* alphas = smem + N*2;
            real_t A[M]; //Hessian matrix, threadIdx.x'th row
            node_t C[M]; //Column indices of the threadIdx.x'th row 3-fold degenerate
            real_t* Q;
            clear_cache(smem, N);
            auto mat_vect = [&](const real_t x){
                real_t result = real_t(0);
                smem[threadIdx.x] = x;
                BLOCK_SYNC
                #pragma unroll
                for (int j = 0; j < M; j++){
                    int col = C[j];
                    result += A[j] * smem[col];
                }
                BLOCK_SYNC
                return result;
            };
            //Modified Gram-Schmidt
            auto MGS = [&](int index){
                BLOCK_SYNC
                real_t result = Q[index*N];
                smem[threadIdx.x] = 0;
                #pragma unroll
                for (int j = 0; j < index; j++){
                    auto proj = reduction(smem, result * Q[j*N]) * Q[j*N];
                    result -= proj; //Remove the component along Q[j*N] from result
                }
                result /= sqrt(reduction(smem, result * result));
                return result;
            };
           
            hiprandState state;            
            hiprand_init(42 + threadIdx.x, 0, 0, &state);

            for (int I = blockIdx.x; I < B.isomer_capacity; I += gridDim.x){
                Q = Qglobal.data + I * N * N + threadIdx.x;
                //Load the hessian and cols into local memory
                memcpy(A, &H.data[I*N*M + threadIdx.x*M], M*sizeof(real_t));
                for (int j = 0; j < M; j++){ 
                    A[j] = H.data[I*N*M + threadIdx.x*M + j];
                    C[j] = cols.data[I*N*M + threadIdx.x*M + j];
                }

                //Lanczos algorithm 
                if(threadIdx.x == 0) betas[0] = real_t(0);
                real_t beta = real_t(0);
                real_t alpha = real_t(0);
                Q[0*N] = hiprand_uniform(&state);
                Q[0*N] /= SQRT(reduction(smem, Q[0*N] * Q[0*N]));
                for (int i = 0; i < N; i++){
                    if (i % 2 == 0 && i > 1){
                        Q[(i-1)*N] = MGS(i-1);
                        Q[i*N] = MGS(i);
                        //if(threadIdx.x + blockIdx.x == 0) printf("i = %d, N = %d, Q[i*N] = %f\n", i, N, Q[i*N]);
                    }
                    real_t v = mat_vect(Q[i*N]);
                    alpha = reduction(smem, v * Q[i*N]);
                    if (threadIdx.x == i) alphas[i] = alpha;
                    if (i == 0){
                        v -= alpha * Q[i*N];
                    } else {
                        v -= betas[i-1] * Q[(i-1)*N] + alpha * Q[i*N];
                    }
                    beta = SQRT(reduction(smem, v * v));
                    if (threadIdx.x == i) betas[i] = beta;
                    if (i < N-1) Q[(i+1)*N] = v / beta;
                    //if (i < N-1) Q[(i+1)*N] = beta;
                }
                eigenvalues.data[I*N*2 + threadIdx.x] = ISNAN(alphas[threadIdx.x])  ? real_t(0) : alphas[threadIdx.x];
                eigenvalues.data[I*N*2 + N + threadIdx.x] = ISNAN(betas[threadIdx.x]) ? real_t(0) : betas[threadIdx.x];
            }   
        }

        void eigensolve(const IsomerBatch& B, CuArray<device_real_t>& Q, const CuArray<device_real_t>& hessians, const CuArray<device_node_t>& cols, CuArray<device_real_t>& eigenvalues, const LaunchCtx& ctx, const LaunchPolicy policy){
            if (policy == LaunchPolicy::SYNC) ctx.wait();
            hipSetDevice(B.get_device_id());
            auto dev = B.get_device_id();

            if(policy == LaunchPolicy::SYNC) {ctx.wait();}

            size_t smem = sizeof(device_coord3d)*B.n_atoms*3 + sizeof(device_real_t)*Block_Size_Pow_2;
            static LaunchDims dims((void*)eigensolve_, B.n_atoms*3, smem, B.isomer_capacity);
            dims.update_dims((void*)eigensolve_, B.n_atoms*3, smem, B.isomer_capacity);
            void* kargs[]{(void*)&B, (void*)&Q, (void*)&hessians, (void*)&cols, (void*)&eigenvalues};
            safeCudaKernelCall((void*)eigensolve_, dims.get_grid(), dims.get_block(), kargs, smem, ctx.stream);  
            
            
            printLastCudaError("Cubic Layout: ");

            if (policy == LaunchPolicy::SYNC) ctx.wait();
        }


    }
}
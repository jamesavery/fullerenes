#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <cooperative_groups/scan.h>
#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include "fullerenes/config.h"
#include "fullerenes/gpu/cu_array.hh"
#include "fullerenes/isomer_batch.hh"
#include "fullerenes/gpu/kernels.hh"

#if(CUSOLVER)
# include <hipsolver.h>
# include <hipsparse.h>
#endif

#define N_STREAMS 16


// TODO: T_QTQ based on Givens rotations (should be possible to do with fewer operations)
//size_t QTQ_calls = 0;
void T_QTQ(const int n, device_real_t *D, device_real_t *L, device_real_t *Vout, device_real_t shift=0)
{
  //  QTQ_calls ++;
  // Unrolled
  //  device_real_t numerical_zero = T.max_norm()*10*std::numeric_limits<device_real_t>::epsilon();
  // specialized max_norm = max(sum(abs(A),axis=1)) for tridiagonal matrix. 
  device_real_t max_norm = 0, numerical_zero = 10*max_norm*std::numeric_limits<device_real_t>::epsilon();
  for(int i=0;i<n;i++) max_norm = std::max(max_norm, std::abs(D[i]) + 2*std::abs(L[i]));
  
  device_real_t a[2], v[2], U[2*(n+1)];//, D[n+1], L[n+1];
  device_real_t d_n, l_n, l_nm1;
    d_n = D[n]; l_n = L[n]; l_nm1 = L[n-1];
  for(int i=0;i<n+1;i++){
    D[i] -= shift;		// Diagonal
    //L[i] = 0;			// Zero padding to avoid branching in inner loop
    //U[i] = 0;                   // Zero padding to avoid branching in inner loop
    U[(n+1)+i] = 0;		// Second upper diagonal for fill-in. U[n+k] = T(k,k+2) is the element two rows above (k+2)st diagonal element.
    if(i<n-1){
      L[ i ] = L[i];	// First lower subdiagonal. L[k] = T(k+1,k) is element below kth diagonal element.
      U[ i ] = L[i];	// First upper subdiagonal. U[k] = T(k,k+1) is element above (k+1)st diagonal element.
      Vout[2*i] = 0; Vout[2*i+1] = 0;	// i'th reflection vector. (0,0) yields "no reflection". Must be initialized due to skipped steps.          
    } else {
        L[ i ] = 0;		// Zero padding to avoid branching in inner loop
        U[ i ] = 0;		// Zero padding to avoid branching in inner loop
    }
  }
   
  for(int k=0;k<n-1;k++)
    if(fabs(L[k]) > numerical_zero)  // Only process if subdiagonal element is not already zero.
    {
      a[0] = D[k]; a[1] = L[k];       // a = T[k:k+2,k] is the vector of nonzeros in kth subdiagonal column.
      
      device_real_t anorm = sqrt(a[0]*a[0] + a[1]*a[1]); 

      // // Udrullet
      // //    reflection_vector(a,anorm,v);
      v[0] = D[k]; v[1] = L[k];
      device_real_t alpha = -copysign(anorm,a[0]); // Koster ingenting
      v[0] -= alpha;

      device_real_t vnorm = sqrt(v[0]*v[0]+v[1]*v[1]);
      device_real_t norm_inv = 1/vnorm;               /* Normalize */
      v[0] *= norm_inv;  v[1] *= norm_inv;

      Vout[2*k] = v[0]; Vout[2*k+1] = v[1];
      
      // // Udrullet 
      // //    apply_reflection(T({k,k+2},{k,k+3}),v);
      // //      if(k+1<n){			// k=n-1 case handled by padding with zeros
      device_real_t   vTA[3] = {D[ k ]*v[0] + L[ k ]*v[1],  // T(k,k  )*v[0] + T(k+1,k  )*v[1]
      			 U[ k ]*v[0] + D[k+1]*v[1],  // T(k,k+1)*v[0] + T(k+1,k+1)*v[1]
      			 U[(n+1)+k]*v[0] + U[k+1]*v[1]}; // T(k,k+2)*v[0] + T(k+1,k+2)*v[1]

      D[ k ]     -= 2*v[0]*vTA[0];
      L[ k ]     -= 2*v[1]*vTA[0];
      U[ k ]     -= 2*v[0]*vTA[1];
      D[k+1]     -= 2*v[1]*vTA[1];
      U[(n+1)+k] -= 2*v[0]*vTA[2];
      U[k+1]     -= 2*v[1]*vTA[2];
        
    }

  // Transform from the right = transform columns of the transpose.
  {
    int k = 0;
    const device_real_t *v = &Vout[0];
    device_real_t   vTA[2] = {D[ k ]*v[0] + U[  k  ]*v[1],  // T(k,k  )*v[0] + T(k,  k+1)*v[1]
  		          0        + D[ k+1 ]*v[1]}; // T(k+1,k)*v[0] + T(k+1,k+1)*v[1]. Lower subdiagonal is zero at this stage.
    
    D[k]       -= 2*v[0]*vTA[0]; // T(k,k)     -= 2*v[0]*vTA[0]
    U[k]       -= 2*v[1]*vTA[0]; // T(k,k+1)   -= 2*v[1]*vTA[0]
    L[k]       -= 2*v[0]*vTA[1]; // T(k+1,k)   -= 2*v[0]*vTA[1]
    D[k+1]     -= 2*v[1]*vTA[1]; // T(k+1,k+1) -= 2*v[1]*vTA[1]        
  }    

  for(int k=1;k<n-1;k++){
    const device_real_t *v = &Vout[2*k];

    device_real_t   vTA[3] = {U[k-1]*v[0] + U[(n+1)+k-1]*v[1], // T(k-1,k)*v[0] + T(k-1,k+1)*v[1]  
  		       D[ k ]*v[0] + U[  k  ]*v[1],     // T(k,k  )*v[0] + T(k,  k+1)*v[1]
  		       L[ k ]*v[0] + D[ k+1 ]*v[1]};    // T(k+1,k)*v[0] + T(k+1,k+1)*v[1]. Lower subdiagonal is zero at this stage

    U[k-1]     -= 2*v[0]*vTA[0];     // T(k-1,k)   -= 2*v[0]*vTA[0]
    U[(n+1)+(k-1)] -= 2*v[1]*vTA[0]; // T(k-1,k+1) -= 2*v[1]*vTA[0]
    D[k]       -= 2*v[0]*vTA[1];     // T(k,  k)     -= 2*v[0]*vTA[1]
    U[k]       -= 2*v[1]*vTA[1];     // T(k,  k+1)   -= 2*v[1]*vTA[1]
    L[k]       -= 2*v[0]*vTA[2];     // T(k+1,k)   -= 2*v[0]*vTA[2]
    D[k+1]     -= 2*v[1]*vTA[2];     // T(k+1,k+1) -= 2*v[1]*vTA[2]        
  } 

  // Copy working diagonals to output
  for(int i=0;i<n;i++){
    D[i] = D[i] + shift;	  // Diagonal
    if(i<n-1){
      L[i] = U[i];  // First lower subdiagonal. L[k] = T(k+1,k) is element below kth diagonal element.
    }
  }
  D[n] = d_n;
  L[n-1] = l_nm1;
  L[n] = l_n;
}

void apply_all_reflections(const device_real_t *V, const int n, const int m, vector<device_real_t>& Q)
{
    for(int k=0;k<n;k++){
        const device_real_t &v0 = V[2*k], &v1 = V[2*k+1];      
        // Udrullet:
        //       apply_reflection(Q({k,k+2},{0,m}), v);
        for(int l=0;l<m;l++){
            device_real_t &q0 = Q[k*m+l], &q1 = Q[(k+1)*m+l];
            device_real_t vTA = q0*v0 + q1*v1;
            q0 -= 2*v0*vTA;
            q1 -= 2*v1*vTA;
        }      
    }  
}

array<device_real_t,2> eigvalsh2x2(const array<device_real_t,4> &A){
  auto [a,b,c,d] = A;
  device_real_t D = sqrt(4*b*c+(a-d)*(a-d));
  return {(a+d-D)/2, (a+d+D)/2};
}


int nth_time = 0;

// TODO: Til tridiagonale matricer er Givens-rotation nemmere/hurtigere (kun een sqrt)
// TODO: Assumes all different eigenvalues. Does this break with multiples?
// TODO: Stop after max_steps for fixed k. Return max Gershgorin radius as convergence -- or max Rayleigh quotient residual?
// TODO: Implement implicit QR iteration using Francis' Q theorem/bulge chasing
std::pair<device_real_t,size_t> eigensystem_hermitian(const int n, const 
                            vector<device_real_t>& D_, 
                            const vector<device_real_t>& L_, 
                            vector<device_real_t>& Q, 
					        vector<device_real_t>& lambdas,
					        const device_real_t tolerance=1e4*std::numeric_limits<device_real_t>::epsilon(),
					        const int max_iterations=5)
{
  device_real_t max_error = 0;
  int n_iterations = 0;

  //@Jonas: Herfra arbejder vi med en tridiagonal reel matrix. 
  device_real_t D[n + 1], L[n + 1], V[2*(n-1)];
  for(int i=0;i<n;i++){
    D[i] = D_[i];
    L[i] = (i+1<n)? L_[i] : 0;
  }

    for (int i = 0; i < n; i++) {
        Q[i*n+i] = device_real_t(1);
    }

  // 2. After tridiagonal decomposition, we can do an eigenvalue
  //    QR-iteration step in O(n), and an eigenvector QR-iteration
  //    step in O(n^2).
  for(int k=n-1;k>=0;k--){
    // We start by targeting the (n,n)-eigenvalue, and gradually
    // deflate, working on smaller and smaller submatrices.
    device_real_t d = D[k];		// d = T(k,k)
    device_real_t shift = d;

    // The Gershgorin disk radius is defined by just the row-sums of
    // absolute off-diagonal elements, since T is symmetric. As T is
    // tridiagonal, only T(k,k-1),T(k,k), and T(k,k+1) are nonzero.
    // Thus, the k'th Gershgorin radius is just |T(k,k-1)| +
    // |T(k,k+1)| = |T(k,k-1)| + |T(k+1,k)| = |L[k-1]|+|L[k]|.
    int i=0;
    device_real_t GR = (k>0?fabs(L[k-1]):0)+fabs(L[k]);
    int not_done = 1;    
    while(not_done > 0){	// GPU NB: Kan erstattes med fornuftig konstant antal iterationer, f.eks. 4-5 stykker.
      i++;   
      T_QTQ(k+1, D,L, V, shift);  // 
      apply_all_reflections(V,k,n,Q);
      
      GR = (k>0?fabs(L[k-1]):0)+(k+1<n?fabs(L[k]):0);      

      // Best guess to eigenvalue in position n-1,n-1.
      if(k>0){
	auto [l0,l1]  = eigvalsh2x2({D[k-1],L[k-1],   /* Diagonalize T[(k-1):k, (k-1):k] 2x2 submatrix */
				     L[k-1],D[k]  });

	shift    = fabs(l0-d) < fabs(l1-d)? l0 : l1; // Pick closest eigenvalue
      } else
	shift    = D[k];
      
      if(GR <= tolerance) not_done--; // Do one (or optionally more) steps after reaching tolerance, to get all off-diagonals below.
                                      // GPU NB: Se GPU NB ovenfor.
      if(i>max_iterations){
	//printf("%dth run: Cannot converge eigenvalue %d to tolerance using machine precision %g (d=%g, shift=%g, G=%g)\n D[k] = %g, L[k-1] = %g, L[k] = %g\n", nth_time,k,tolerance, std::numeric_limits<device_real_t>::epsilon(),d,shift,GR, D[k], (k>0)?L[k-1]:0, (k+1<n)?L[k]:0);
	
	max_error = std::max(max_error,GR);
	break;
      }
      n_iterations++;
    }
  }
  for(int k=0;k<n;k++) lambdas[k] = D[k]; // Extract eigenvalues into result.
  
  return {max_error,n_iterations};
}





namespace gpu_kernels{
    namespace isomerspace_eigen{
        template void spectrum_ends<GPU, float, uint16_t>(const IsomerBatch<GPU>& B, const CuArray<float>& hessians, const CuArray<uint16_t>& cols, CuArray<float>& lambda_mins, CuArray<float>& lambda_maxs, int m_lanczos, const LaunchCtx& ctx, const LaunchPolicy policy);
        template void spectrum_ends<GPU, double, uint16_t>(const IsomerBatch<GPU>& B, const CuArray<double>& hessians, const CuArray<uint16_t>& cols, CuArray<double>& lambda_mins, CuArray<double>& lambda_maxs, int m_lanczos, const LaunchCtx& ctx, const LaunchPolicy policy);
        template void spectrum_ends<GPU, float, uint16_t>(const IsomerBatch<GPU>& B, const CuArray<float>& hessians, const CuArray<uint16_t>& cols, CuArray<float>& lambda_mins, CuArray<float>& lambda_maxs, CuArray<float>& eigvect_mins, CuArray<float>& eigvect_maxs, int m_lanczos, const LaunchCtx& ctx, const LaunchPolicy policy);
        template void spectrum_ends<GPU, double, uint16_t>(const IsomerBatch<GPU>& B, const CuArray<double>& hessians, const CuArray<uint16_t>& cols, CuArray<double>& lambda_mins, CuArray<double>& lambda_maxs, CuArray<double>& eigvect_mins, CuArray<double>& eigvect_maxs, int m_lanczos, const LaunchCtx& ctx, const LaunchPolicy policy);
        template void eigensolve<GPU, float, uint16_t>(const IsomerBatch<GPU>& B, CuArray<float>& Q, const CuArray<float>& hessians, const CuArray<uint16_t>& cols, CuArray<float>& eigenvalues, const LaunchCtx& ctx, const LaunchPolicy policy);
        template void eigensolve<GPU, double, uint16_t>(const IsomerBatch<GPU>& B, CuArray<double>& Q, const CuArray<double>& hessians, const CuArray<uint16_t>& cols, CuArray<double>& eigenvalues, const LaunchCtx& ctx, const LaunchPolicy policy);
        template void eigensolve_special<GPU, float, uint16_t>(const IsomerBatch<GPU>& B, CuArray<float>& Q, const CuArray<float>& hessians, const CuArray<uint16_t>& cols, CuArray<float>& eigenvalues, const LaunchCtx& ctx, const LaunchPolicy policy);

        #include "device_includes.cu"
        enum class EigensolveMode {NO_VECTORS, VECTORS, ENDS, FULL_SPECTRUM, FULL_SPECTRUM_MOLECULE}; 
#if(CUSOLVER)      
        void eigensolve_cusolver(const IsomerBatch& B, const CuArray<T>& hessians, const CuArray<K>& cols, CuArray<T>& eigenvalues, const LaunchCtx& ctx, const LaunchPolicy policy){
            static std::vector<hipsolverHandle_t> cusolverHs(N_STREAMS, NULL);
            static int m = B.n_atoms*3;
            static int lda = m;
            static bool initialized = false;
            static std::vector<CuArray<device_real_t>> As(N_STREAMS);
            static std::vector<LaunchCtx> ctxs(N_STREAMS);
            static int nisomers = B.isomer_capacity;
            static std::vector<CuArray<device_real_t>> d_works(N_STREAMS);
            static std::vector<std::vector<device_real_t>> h_works(N_STREAMS);
            static std::vector<int*> d_infos(N_STREAMS, nullptr);
            static std::vector<int> infos(N_STREAMS, 1);
            static std::vector<int> lworks(N_STREAMS, 0);
            static std::vector<size_t> workspaceInBytesOnDevice(N_STREAMS, 0);
            static std::vector<size_t> workspaceInBytesOnHost(N_STREAMS, 0);
            int ncols = 10*3;
            int nn = m*ncols;
            hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_NOVECTOR; // compute eigenvalues and eigenvectors.
            hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
            auto T0 = std::chrono::steady_clock::now();
            if (!initialized){
                for (size_t I = 0; I < N_STREAMS ; I++){
                    As[I] = CuArray<device_real_t>(m*m, 0.);
                    ctxs[I] = LaunchCtx(0);
                    hipsolverDnCreate(&cusolverHs[I]);
                    hipsolverSetStream(cusolverHs[I], ctxs[I].stream);   
                    hipMalloc(reinterpret_cast<void **>(&(d_infos[I])), sizeof(int));
                  
                    #if FLOAT_TYPE == 3
                    //    hipsolverDnDsyevd_bufferSize(cusolverHs[I], jobz, uplo, m, As[I].data, lda, &(eigenvalues.data[I*m]), &(lworks[I]));
                        cusolverDnXsyevd_bufferSize(cusolverHs[I], NULL, jobz, uplo, m, HIP_R_64F, (const void*)As[I].data, lda, HIP_R_64F, (const void*)&(eigenvalues.data[I*m]), HIP_R_64F, &workspaceInBytesOnDevice[I], &workspaceInBytesOnHost[I]);  
                    #elif FLOAT_TYPE == 2
                        cusolverDnXsyevd_bufferSize(cusolverHs[I], NULL, jobz, uplo, m, HIP_R_32F, (const void*)As[I].data, lda, HIP_R_32F, (const void*)&(eigenvalues.data[I*m]), HIP_R_32F, &workspaceInBytesOnDevice[I], &workspaceInBytesOnHost[I]);  
                    //  hipsolverDnSsyevd_bufferSize(cusolverHs[I], jobz, uplo, m, As[I].data, lda, &(eigenvalues.data[I*m]), &(lworks[I]));
                    #endif
                    d_works[I] = CuArray<device_real_t>(workspaceInBytesOnDevice[I]/sizeof(device_real_t));
                    h_works[I].resize(workspaceInBytesOnHost[I]/sizeof(device_real_t));
                    //hipMalloc(reinterpret_cast<void **>(&(d_works[I])), sizeof(device_real_t) * workspaceInBytesOnDevice[I]);
                    //hipHostMalloc(reinterpret_cast<void **>(&(h_works[I])), sizeof(device_real_t) * workspaceInBytesOnHost[I]);
                    printLastCudaError("eigensolve");
                }    
                initialized = true;
            }
            auto T1 = std::chrono::steady_clock::now();
            std::cout << "Initializing the eigensolver took " << std::chrono::duration_cast<std::chrono::microseconds>(T1 - T0).count() / (device_real_t)nisomers << " us / isomer" << std::endl;
            int counter = 0;
            //Loading the sparse hessians into dense matrices; Might be a bottleneck.
            auto start = std::chrono::steady_clock::now();
            auto fill_As = [&] (){
                int I = 0;
                auto start_counter = counter;
                for (int II = start_counter; II < std::min(start_counter + N_STREAMS, nisomers)  ; II++){
                counter++;
                for (size_t i = 0; i < m; i++){ //Number of rows in the hessian
                    for (size_t j = 0; j < ncols; j++){ //Number of columns in the hessian, it is 10*3 because we have a term for the node itself, it's 3 neighbours and 6 outer neighbours, each with 3 dx, dy, dz terms
                        int col = cols.data[int(II*nn + i*ncols + j)];
                        As[I][int(i*m + col)] = hessians.data[int(II*nn + i*ncols + j)];
                    }
                }
                ++I;
            }
            };
            
            auto end = std::chrono::steady_clock::now();
            std::cout << "Loading the sparse hessians into dense matrices took " << std::chrono::duration_cast<std::chrono::microseconds>(end - start).count() / (device_real_t)nisomers << " us / isomer" << std::endl;   

            if (policy == LaunchPolicy::SYNC){
                ctx.wait();
            }

            start = std::chrono::steady_clock::now();
            for (size_t I = 0; I < nisomers ; I++){
                int idx = I % N_STREAMS;
                if (idx == 0) fill_As();
                //std::cout  << "Inum: " << counter << std::endl;
                #if FLOAT_TYPE == 3
                    cusolverDnXsyevd( cusolverHs[idx], NULL, jobz, uplo, m, HIP_R_64F, As[idx].data, lda, HIP_R_64F, &eigenvalues.data[I*m], HIP_R_64F, (void *) d_works[idx].data, workspaceInBytesOnDevice[idx], (void *) h_works[idx].data(), workspaceInBytesOnHost[idx], d_infos[idx]);
                //    hipsolverDnDsyevd(cusolverHs[idx], jobz, uplo, m, As[idx].data,    lda, &eigenvalues.data[I*m], d_works[idx],   lworks[idx], d_infos[idx]);
                #elif FLOAT_TYPE == 2
                    cusolverDnXsyevd( cusolverHs[idx], NULL, jobz, uplo, m, HIP_R_32F, As[idx].data, lda, HIP_R_32F, &eigenvalues.data[I*m], HIP_R_32F, (void *) d_works[idx].data, workspaceInBytesOnDevice[idx], (void *) h_works[idx].data(), workspaceInBytesOnHost[idx], d_infos[idx]);
                //    hipsolverDnSsyevd(cusolverHs[idx], jobz, uplo, m, As[idx].data,    lda, &eigenvalues.data[I*m], d_works[idx],   lworks[idx], d_infos[idx]);
                #endif
                //hipMemcpyAsync(&infos[idx], d_infos[idx], sizeof(int), hipMemcpyDeviceToHost, ctx.stream);
            }

            if (policy == LaunchPolicy::SYNC){
                for (size_t I = 0; I < N_STREAMS ; I++){
                    ctxs[I].wait();
                }
            }
            end = std::chrono::steady_clock::now();
            std::cout << "eigensolve time: " << std::chrono::duration_cast<std::chrono::microseconds>(end - start).count() / (device_real_t)nisomers <<  " us / isomer" <<std::endl;
        
        }
#endif
      
        /* template <typename T>
        struct array_wrapper{
            T* data;
            int stride;
            __device__ array_wrapper(T* data, int stride) : data(data), stride(stride) {}
            T& __device__ operator[](int i) const{
                return data[i*stride];
            }
        };

        typedef array_wrapper<device_real_t> real_wrap; */
        template <typename T>
        void __device__ apply_all_reflections(const T *V, const int n, const int m, T* Q)
        {   
            static_assert(std::is_floating_point<T>::value, "T must be floating point");

            for(int k=0;k<n;k++){
                const T &v0 = V[2*k], &v1 = V[2*k+1];      
                // Udrullet:
                //       apply_reflection(Q({k,k+2},{0,m}), v);
                for(int l=threadIdx.x;l<m; l+=blockDim.x){
                    T &q0 = Q[k*m+l], &q1 = Q[(k+1)*m+l];
                    T vTA = q0*v0 + q1*v1;
                    q0 -= 2*v0*vTA;
                    q1 -= 2*v1*vTA;
                }      
            }  
        }
        //Customized diagonalization routine for symmetric tridiagonal matrices
        template <typename T>
        void __device__ T_QTQ(const int n, T* D, T* L, T* U, T* Vout, T shift=0)
        {
        int tix = threadIdx.x;
        FLOAT_TYPEDEFS(T);
        SMEM(T);
        //  QTQ_calls ++;
        // Unrolled
        //  real_t numerical_zero = T.max_norm()*10*std::numeric_limits<real_t>::epsilon();
        // specialized max_norm = max(sum(abs(A),axis=1)) for tridiagonal matrix. 
        real_t local_max = real_t(0.);
        for (int i = tix; i < n; i += blockDim.x){
            local_max = std::max(local_max, ABS(D[i]) + 2*ABS(L[i]));
        }
        real_t max_norm = reduction_max(smem, local_max);
        real_t numerical_zero = 10*std::numeric_limits<real_t>::epsilon();
        device_real_t d_n, l_n, l_nm1;
        d_n = D[n]; l_n = L[n]; l_nm1 = L[n-1];
        BLOCK_SYNC
        //real_t a[2], v[2], D[n+1], L[n+1], U[2*(n+1)];
        real_t a[2], v[2];//, D[n+1], L[n+1], U[2*(n+1)];
        for(int k = tix; k < n + 1; k += blockDim.x){
            D[k] -= shift;
            U[n+1 + k] = real_t(0.);
            if(k < n-1){
                U[k] = L[k];
                Vout[2*k] = real_t(0.); Vout[2*k+1] = real_t(0.);
            } else {
                L[k] = real_t(0.);
                U[k] = real_t(0.);
            }
        }
        
        BLOCK_SYNC
        if(tix == 0)
            for(int k=0;k<n-1;k++){
                if (ABS(L[k]) > numerical_zero){
                a[0] = D[k]; a[1] = L[k];       // a = T[k:k+2,k] is the vector of nonzeros in kth subdiagonal column.
                
                real_t anorm = SQRT(a[0]*a[0] + a[1]*a[1]); 

                // // Udrullet
                // //    reflection_vector(a,anorm,v);
                v[0] = D[k]; v[1] = L[k];
                real_t alpha = -copysign(anorm,a[0]); // Koster ingenting
                v[0] -= alpha;

                real_t vnorm = SQRT(v[0]*v[0]+v[1]*v[1]);
                real_t norm_inv = real_t(1.)/vnorm;               //Normalize
                v[0] *= norm_inv;  v[1] *= norm_inv;

                Vout[2*k] = v[0]; Vout[2*k+1] = v[1];
                
                // // Udrullet 
                // //    apply_reflection(T({k,k+2},{k,k+3}),v);
                // //      if(k+1<n){			// k=n-1 case handled by padding with zeros
                coord3d vTA = { D[ k ]*v[0] + L[ k ]*v[1],  // T(k,k  )*v[0] + T(k+1,k  )*v[1]
                                U[ k ]*v[0] + D[k+1]*v[1],  // T(k,k+1)*v[0] + T(k+1,k+1)*v[1]
                                U[(n+1)+k]*v[0] + U[k+1]*v[1]}; // T(k,k+2)*v[0] + T(k+1,k+2)*v[1]

            
                D[k]     -= real_t(2.)*v[0]*vTA[0];
                L[k]     -= real_t(2.)*v[1]*vTA[0];
                U[k]     -= real_t(2.)*v[0]*vTA[1];
                D[k+1]     -= real_t(2.)*v[1]*vTA[1];
                U[(n+1)+k] -= real_t(2.)*v[0]*vTA[2];
                U[k+1]     -= real_t(2.)*v[1]*vTA[2];
                }
            }
        
        if(tix == 0)
        { // Transform from the right = transform columns of the transpose.
            int k = 0;
            const real_t *v = &Vout[0];
            real_t   vTA[2] = {D[ k ]*v[0] + U[  k  ]*v[1],  // T(k,k  )*v[0] + T(k,  k+1)*v[1]
                        0        + D[ k+1 ]*v[1]}; // T(k+1,k)*v[0] + T(k+1,k+1)*v[1]. Lower subdiagonal is zero at this stage.
            
            D[k]       -= real_t(2.)*v[0]*vTA[0]; // T(k,k)     -= 2*v[0]*vTA[0]
            U[k]       -= real_t(2.)*v[1]*vTA[0]; // T(k,k+1)   -= 2*v[1]*vTA[0]
            L[k]       -= real_t(2.)*v[0]*vTA[1]; // T(k+1,k)   -= 2*v[0]*vTA[1]
            D[k+1]     -= real_t(2.)*v[1]*vTA[1]; // T(k+1,k+1) -= 2*v[1]*vTA[1]        
        }
        BLOCK_SYNC

        
        if(tix == 0){
            for(int k=1;k<n-1;k++){
                const real_t *v = &Vout[2*k];
                coord3d vTA = {U[k-1]*v[0] + U[(n+1)+k-1]*v[1], // T(k-1,k)*v[0] + T(k-1,k+1)*v[1]  
                                D[ k ]*v[0] + U[  k  ]*v[1],     // T(k,k  )*v[0] + T(k,  k+1)*v[1]
                                L[ k ]*v[0] + D[ k+1 ]*v[1]};    // T(k+1,k)*v[0] + T(k+1,k+1)*v[1]. Lower subdiagonal is zero at this stage

                U[k-1]     -= real_t(2.)*v[0]*vTA[0];     // T(k-1,k)   -= 2*v[0]*vTA[0]
                U[(n+1)+(k-1)] -= real_t(2.)*v[1]*vTA[0]; // T(k-1,k+1) -= 2*v[1]*vTA[0]
                U[k]       -= real_t(2.)*v[1]*vTA[1];     // T(k,  k+1)   -= 2*v[1]*vTA[1]
                D[k]       -= real_t(2.)*v[0]*vTA[1];     // T(k,  k)     -= 2*v[0]*vTA[1]
                L[k]       -= real_t(2.)*v[0]*vTA[2];     // T(k+1,k)   -= 2*v[0]*vTA[2]
                D[k+1]     -= real_t(2.)*v[1]*vTA[2];     // T(k+1,k+1) -= 2*v[1]*vTA[2]        
            }
        }
       

        BLOCK_SYNC
        for (int k = tix; k<n; k+=blockDim.x){  // Copy working diagonals to output
            D[k] += shift;
            if(k < n-1){
                L[k] = U[k];
            }
        }
        BLOCK_SYNC
        if (tix==0){
         D[n] = d_n;
         L[n-1] = l_nm1;
         L[n] = l_n;
        }
        BLOCK_SYNC
        
        }

        template <typename T>
        array<T,2> INLINE eigvalsh2x2(const array<T,4> &A){
            auto [a,b,c,d] = A;
            T D = SQRT(4*b*c+(a-d)*(a-d));
            return {(a+d-D)/2, (a+d+D)/2};
        }

        //Takes a set of tridiagonal matrices and solves them

        template<EigensolveMode mode, Device DEV, typename T, typename K>
        void __global__ eigensolve_(const IsomerBatch<DEV> B, CuArray<T> D_, CuArray<T> L_, CuArray<T> U_, CuArray<T> Q_, int n){
            FLOAT_TYPEDEFS(T);
            SMEM(T);
            T *D = smem + blockDim.x*2, *L = D + (n+1), *U = L + (n+1), *V = U + (n+1)*2;
            //Expected layout is that each thread reads the (threadIdx.x + blockIdx.x*blockDim.x)^th column of D and L, in that way reads should be coalesced.
            for (int I = blockIdx.x; I < B.isomer_capacity; I += gridDim.x) if(B.statuses[I] == IsomerStatus::CONVERGED){
                for(int i = threadIdx.x; i < n; i += blockDim.x){
                    D[i] = D_.data[n*I + i];
                    L[i] = L_.data[n*I + i];
                    U[i] = L_.data[n*I + i];
                }
                for (int i = threadIdx.x; i < n; i += blockDim.x){
                    Q_.data[n*n*I + i*(n+1)] = real_t(1.); //Set Q to the identity matrix
                }
            
            BLOCK_SYNC
                

              // 2. After tridiagonal decomposition, we can do an eigenvalue
            //    QR-iteration step in O(n), and an eigenvector QR-iteration
            //    step in O(n^2).
            for(int k=n-1;k>=0;k--){
                // We start by targeting the (n,n)-eigenvalue, and gradually
                // deflate, working on smaller and smaller submatrices.
                real_t d = D[k];		// d = T(k,k)
                real_t shift = d;

                // The Gershgorin disk radius is defined by just the row-sums of
                // absolute off-diagonal elements, since T is symmetric. As T is
                // tridiagonal, only T(k,k-1),T(k,k), and T(k,k+1) are nonzero.
                // Thus, the k'th Gershgorin radius is just |T(k,k-1)| +
                // |T(k,k+1)| = |T(k,k-1)| + |T(k+1,k)| = |L[k-1]|+|L[k]|.
                int i=0;
                real_t GR = (k>0?ABS(L[k-1]):0)+ABS(L[k]);
                int not_done = 1;    
                while(not_done > 0){	// GPU NB: Kan erstattes med fornuftig konstant antal iterationer, f.eks. 4-5 stykker.
                i++;   
                T_QTQ(k+1, D,L, U, V, shift);  // 
                if(mode == EigensolveMode::VECTORS || mode == EigensolveMode::FULL_SPECTRUM_MOLECULE){
                    apply_all_reflections(V,k,n,Q_.data + n*n*I);
                }
                
                GR = (k>0?ABS(L[k-1]):0)+(k+1<n?ABS(L[k]):0);      

                // Best guess to eigenvalue in position n-1,n-1.
                if(k>0){
                    std::array<T,4> args = {D[k-1],L[k-1],L[k-1],D[k]};
                    auto [l0,l1]  = eigvalsh2x2(args);

                shift    = ABS(l0-d) < ABS(l1-d)? l0 : l1; // Pick closest eigenvalue
                } else
                shift    = D[k];
                
                if(GR <= std::numeric_limits<real_t>::epsilon()*real_t(10.)) not_done--; // Do one (or optionally more) steps after reaching tolerance, to get all off-diagonals below.
                                                // GPU NB: Se GPU NB ovenfor.
                if(i>5){
                //printf("%dth run: Cannot converge eigenvalue %d to tolerance " G " using machine precision %g (d=%g, shift=%g, G=%g)\n" "D[k] = %g, L[k-1] = %g, L[k] = %g\n", nth_time,k,tolerance, std::numeric_limits<real_t>::epsilon(),d,shift,GR, D[k], (k>0)?L[k-1]:0, (k+1<n)?L[k]:0);
                
                auto max_error = std::max(std::numeric_limits<real_t>::epsilon()*real_t(10.),GR);
                break;
                }

                }
            }
            BLOCK_SYNC
            //Copy back to global memory
            for (int i = threadIdx.x; i < n; i += blockDim.x){
                if( mode == EigensolveMode::FULL_SPECTRUM_MOLECULE){
                    if(i < 6) {D_.data[(n+6)*I + i] = 0;}
                    D_.data[(n+6)*I + 6 + i] = D[i];
                } else {
                    D_.data[n*I + i] = D[i];
                }
            }
        }
        }
        
        template<EigensolveMode mode, Device DEV, typename T, typename K>
        void __global__ eigensolve_min_max_(const IsomerBatch<DEV> B, CuArray<T> D_, CuArray<T> L_, CuArray<T> U_, CuArray<T> Q_, CuArray<T> EigMin_, CuArray<T> EigMax_, CuArray<int> MinIdx_, CuArray<int> MaxIdx_, int n){
            FLOAT_TYPEDEFS(T);
            SMEM(T);
            T *D = smem + blockDim.x*2, *L = D + (n+1), *U = L + (n+1), *V = U + (n+1)*2;
            //Expected layout is that each thread reads the (threadIdx.x + blockIdx.x*blockDim.x)^th column of D and L, in that way reads should be coalesced.
            for (int I = blockIdx.x; I < B.isomer_capacity; I += gridDim.x) if(B.statuses[I] == IsomerStatus::CONVERGED){
                for(int i = threadIdx.x; i < n; i += blockDim.x){
                    D[i] = D_.data[n*I + i];
                    L[i] = L_.data[n*I + i];
                    U[i] = L_.data[n*I + i];
                }
                if (mode == EigensolveMode::VECTORS)
                    for (int i =  threadIdx.x; i < n; i += blockDim.x){
                        Q_.data[n*n*I + i*(n+1)] = real_t(1.); //Set Q to the identity matrix
                    }
            
            BLOCK_SYNC
                

              // 2. After tridiagonal decomposition, we can do an eigenvalue
            //    QR-iteration step in O(n), and an eigenvector QR-iteration
            //    step in O(n^2).
            for(int k=n-1;k>=0;k--){
                // We start by targeting the (n,n)-eigenvalue, and gradually
                // deflate, working on smaller and smaller submatrices.
                real_t d = D[k];		// d = T(k,k)
                real_t shift = d;

                // The Gershgorin disk radius is defined by just the row-sums of
                // absolute off-diagonal elements, since T is symmetric. As T is
                // tridiagonal, only T(k,k-1),T(k,k), and T(k,k+1) are nonzero.
                // Thus, the k'th Gershgorin radius is just |T(k,k-1)| +
                // |T(k,k+1)| = |T(k,k-1)| + |T(k+1,k)| = |L[k-1]|+|L[k]|.
                int i=0;
                real_t GR = (k>0?ABS(L[k-1]):0)+ABS(L[k]);
                int not_done = 1;    
                while(not_done > 0){	// GPU NB: Kan erstattes med fornuftig konstant antal iterationer, f.eks. 4-5 stykker.
                i++;   
                T_QTQ(k+1, D,L, U, V, shift);  // 
                if(mode == EigensolveMode::VECTORS) apply_all_reflections(V,k,n,Q_.data + n*n*I);
                
                GR = (k>0?ABS(L[k-1]):0)+(k+1<n?ABS(L[k]):0);      

                // Best guess to eigenvalue in position n-1,n-1.
                if(k>0){
                std::array<T,4> args = {D[k-1],L[k-1],
                            L[k-1],D[k]};
                auto [l0,l1]  = eigvalsh2x2(args   /* Diagonalize T[(k-1):k, (k-1):k] 2x2 submatrix */);

                shift    = ABS(l0-d) < ABS(l1-d)? l0 : l1; // Pick closest eigenvalue
                } else
                shift    = D[k];
                
                if(GR <= std::numeric_limits<real_t>::epsilon()*real_t(10.)) not_done--; // Do one (or optionally more) steps after reaching tolerance, to get all off-diagonals below.
                                                // GPU NB: Se GPU NB ovenfor.
                if(i>5){
                //printf("%dth run: Cannot converge eigenvalue %d to tolerance " G " using machine precision %g (d=%g, shift=%g, G=%g)\n" "D[k] = %g, L[k-1] = %g, L[k] = %g\n", nth_time,k,tolerance, std::numeric_limits<real_t>::epsilon(),d,shift,GR, D[k], (k>0)?L[k-1]:0, (k+1<n)?L[k]:0);
                
                auto max_error = std::max(std::numeric_limits<real_t>::epsilon()*real_t(10.),GR);
                break;
                }

                }
            }
            BLOCK_SYNC
            //Copy back to global memory
            for (int i = threadIdx.x; i < n; i += blockDim.x){
                D_.data[n*I + i] = D[i];}
            smem[threadIdx.x] = real_t(0.);
            real_t local_max = real_t(0.);
            real_t local_min = numeric_limits<real_t>::max();
            int local_min_idx = 0;
            int local_max_idx = 0;
            for (int i = threadIdx.x; i < n; i += blockDim.x){
                local_max = ISNAN(D[i]) ? NAN : std::max(local_max, ABS(D[i]));
                local_min = ISNAN(D[i]) ? NAN : std::min(local_min, ABS(D[i]));
                local_min_idx = ISNAN(D[i]) ? NAN : (local_min == ABS(D[i]) ? i : local_min_idx);
                local_max_idx = ISNAN(D[i]) ? NAN : (local_max == ABS(D[i]) ? i : local_max_idx);
            }
            real_t max_eig = reduction_max(smem, local_max);
            smem[threadIdx.x] = numeric_limits<real_t>::max();
            real_t min_eig = reduction_min(smem, local_min > 1e-1 ? local_min : numeric_limits<real_t>::max());
            if(threadIdx.x == 0){
                EigMax_.data[I] = max_eig;
                EigMin_.data[I] = min_eig;
            }
            BLOCK_SYNC
            //Argmax and argmin
            if (min_eig == D[local_min_idx]){
                //If by some miracle multiple eigenvalues are exactly equal, we just pick one of them at random using atomicExch_block
                atomicExch_block(MinIdx_.data + I, local_min_idx);
            }
            if (max_eig == D[local_max_idx]){
                //If by some miracle multiple eigenvalues are exactly equal, we just pick one of them at random using atomicExch_block
                atomicExch_block(MaxIdx_.data + I, local_max_idx);
            }
        }
        }

        template <EigensolveMode mode, Device DEV, typename T, typename K>
        void __global__ lanczos_(const IsomerBatch<DEV> B, CuArray<T> V_, CuArray<T> U, CuArray<T> D, const CuArray<T> H, const CuArray<K> cols, int m){
            TEMPLATE_TYPEDEFS(T,K);
            SMEM(T);
            int N = B.n_atoms * 3; //Number of rows in the hessian
            int atom_idx = threadIdx.x/3; //Atom index
            constexpr int M = 10*3;          //Number of columns in the hessian
            real_t* betas = smem + N;
            real_t* alphas = betas + m;
            real_t A[M]; //Hessian matrix, threadIdx.x'th row
            node_t C[M]; //Column indices of the threadIdx.x'th row 3-fold degenerate
            real_t* V;
            float* X_ptr = B.X + N*blockIdx.x; //WARNING float 
            real_t Z[6]; //Eigenvectors spanning the kernel of the hessian (Rotations, Translations)
            if (mode == EigensolveMode::ENDS || mode == EigensolveMode::FULL_SPECTRUM_MOLECULE){
                Z[0] = real_t(threadIdx.x%3 == 0)/SQRT(B.n_atoms); Z[1] = real_t(threadIdx.x%3 == 1)/SQRT(B.n_atoms); Z[2] = real_t(threadIdx.x%3 == 2)/SQRT(B.n_atoms); // Translation eigenvectors
                
            }
            BLOCK_SYNC
      
            clear_cache(smem, N);


            auto mat_vect = [&](const real_t x){
                real_t result = real_t(0);
                smem[threadIdx.x] = x;
                BLOCK_SYNC
                #pragma unroll
                for (int j = 0; j < M; j++){
                    int col = C[j];
                    result += A[j] * smem[col];
                }
                BLOCK_SYNC
                return result;
            };
            
            //Modified Gram-Schmidt, Also orthogonalizes against the deflation space
            auto MGS = [&](int index){
                BLOCK_SYNC
                real_t result = V[index*N];
                smem[threadIdx.x] = 0;
                if (mode == EigensolveMode::ENDS || mode == EigensolveMode::FULL_SPECTRUM_MOLECULE){
                    #pragma unroll
                    for (int j = 0; j < 6; j++){
                        auto proj = reduction(smem, result * Z[j]) * Z[j];
                        result -= proj; //Remove the component along Z[j] from result
                    }
                }

                #pragma unroll
                for (int j = 0; j < index; j++){
                    auto proj = reduction(smem, result * V[j*N]) * V[j*N];
                    result -= proj; //Remove the component along V[j*N] from result
                }
                result /= sqrt(reduction(smem, result * result));
                return result;
            };
           
            hiprandState state;            
            hiprand_init(42 + threadIdx.x, 0, 0, &state);

            for (int I = blockIdx.x; I < B.isomer_capacity; I += gridDim.x) if(B.statuses[I] == IsomerStatus::CONVERGED){
                V = V_.data + I * m * N + threadIdx.x;
                if(mode == EigensolveMode::ENDS || mode == EigensolveMode::FULL_SPECTRUM_MOLECULE){
                    X_ptr = B.X + N*I;
                    if (threadIdx.x%3 == 0) {
                        Z[3] = real_t(0.);
                        Z[4] = -X_ptr[atom_idx*3 + 2];
                        Z[5] = -X_ptr[atom_idx*3 + 1];
                    } else if (threadIdx.x%3 == 1) {
                        Z[3] = -X_ptr[atom_idx*3 + 2];
                        Z[4] = real_t(0.);
                        Z[5] = X_ptr[atom_idx*3 + 0];
                    } else {
                        Z[3] = X_ptr[atom_idx*3 + 1];
                        Z[4] = X_ptr[atom_idx*3 + 0];
                        Z[5] = real_t(0.);
                    }
                        clear_cache(smem, N);
                        Z[3] /= SQRT(reduction(smem, Z[3]*Z[3]));
                        clear_cache(smem, N);
                        Z[4] /= SQRT(reduction(smem, Z[4]*Z[4]));
                        clear_cache(smem, N);
                        Z[5] /= SQRT(reduction(smem, Z[5]*Z[5]));

                }
                //Load the hessian and cols into local memory
                memcpy(A, &H.data[I*N*M + threadIdx.x*M], M*sizeof(real_t));
                for (int j = 0; j < M; j++){ 
                    A[j] = H.data[I*N*M + threadIdx.x*M + j];
                    C[j] = cols.data[I*N*M + threadIdx.x*M + j];
                }
                BLOCK_SYNC
                /* for(int j = 0; j< 6; j++){
                clear_cache(smem, N);
                real_t test_ = mat_vect(Z[j]);
                clear_cache (smem, N);
                real_t rayleigh_ = reduction(smem, test_ * Z[j]);
                real_t resid_ = test_ - rayleigh_ * Z[j];

                print_single("\nRayleigh: \n");
                print_single(rayleigh_);
                print_single("\nResidual: \n");
                print_single(resid_);
                print_single("\n");
                }
 */
                //Lanczos algorithm 
                if(threadIdx.x == 0) betas[0] = real_t(0);
                real_t beta = real_t(0);
                real_t alpha = real_t(0);
                V[0*N] = hiprand_uniform(&state);
                smem[threadIdx.x] = real_t(0); //Clear the shared memory
                V[0*N] /= SQRT(reduction(smem, V[0*N] * V[0*N]));
                V[0*N] = MGS(0);
                for (int i = 0; i < m; i++){
                    if (i % 2 == 0 && i > 1){
                        V[(i-1)*N] = MGS(i-1);
                        V[i*N] = MGS(i);
                    }
                    real_t v = mat_vect(V[i*N]);
                    smem[threadIdx.x] = real_t(0); //Clear the shared memory
                    alpha = reduction(smem, v * V[i*N]);
                    if (threadIdx.x == i) alphas[i] = alpha;
                    if (i == 0){
                        v -= alpha * V[i*N];
                    } else {
                        v -= betas[i-1] * V[(i-1)*N] + alpha * V[i*N];
                    }
                    smem[threadIdx.x] = real_t(0); //Clear the shared memory
                    beta = SQRT(reduction(smem, v * v));
                    if (threadIdx.x == i) betas[i] = beta;
                    if (i < m-1) V[(i+1)*N] = v / beta;
                    //if (i < N-1) V[(i+1)*N] = beta;
                }
                if (threadIdx.x < m){
                    D.data[I*m + threadIdx.x] = alphas[threadIdx.x];
                    U.data[I*m + threadIdx.x] = betas[threadIdx.x];
                }
            }   
        }
        //Assumes that N = B.n_atoms * 3
        template <EigensolveMode mode, Device DEV,typename T, typename K>
        void __global__ compute_eigenvectors_(const IsomerBatch<DEV> B, CuArray<T> Q, CuArray<T> V, CuArray<T> E, int m){
            int atom_idx = threadIdx.x/3; //Atom index (Integer division so the result is rounded down)
            TEMPLATE_TYPEDEFS(T,K);
            SMEM(T);
            int n = B.n_atoms * 3;
            int offset = 0;
            if (mode == EigensolveMode::ENDS || mode == EigensolveMode::FULL_SPECTRUM_MOLECULE){
                offset = 6;
            }
            for (int I = blockIdx.x; I < B.isomer_capacity; I += gridDim.x){
                real_t* v = V.data + I * m * n;
                real_t* e = E.data + I * n * n;
                real_t* q = Q.data + I * m * m;
                if (mode == EigensolveMode::FULL_SPECTRUM_MOLECULE) {
                    for(int i = 0; i < 3; i++){
                        e[i*n + threadIdx.x] = real_t(threadIdx.x%3 == i)/SQRT(B.n_atoms); 
                    }
                    float* X_ptr = B.X + n*I; //WARNING float hardcoded here to match the type of B.X (IsomerBatch)
                    if (threadIdx.x%3 == 0) {
                        e[3*n + threadIdx.x] = real_t(0.);
                        e[4*n + threadIdx.x] = -X_ptr[atom_idx*3 + 2];
                        e[5*n + threadIdx.x] = -X_ptr[atom_idx*3 + 1];
                    } else if (threadIdx.x%3 == 1) {
                        e[3*n + threadIdx.x] = -X_ptr[atom_idx*3 + 2];
                        e[4*n + threadIdx.x] = real_t(0.);
                        e[5*n + threadIdx.x] = X_ptr[atom_idx*3 + 0];
                    } else {
                        e[3*n + threadIdx.x] = X_ptr[atom_idx*3 + 1];
                        e[4*n + threadIdx.x] = X_ptr[atom_idx*3 + 0];
                        e[5*n + threadIdx.x] = real_t(0.);
                    }
                        clear_cache(smem,  n + warpSize);
                        e[3*n + threadIdx.x] /= SQRT(reduction(smem, e[3*n + threadIdx.x]*e[3*n + threadIdx.x]));
                        clear_cache(smem,  n + warpSize);
                        e[4*n + threadIdx.x] /= SQRT(reduction(smem, e[4*n + threadIdx.x]*e[4*n + threadIdx.x]));
                        clear_cache(smem,  n + warpSize);
                        e[5*n + threadIdx.x] /= SQRT(reduction(smem, e[5*n + threadIdx.x]*e[5*n + threadIdx.x]));
                }
                
                if(threadIdx.x < n)
                for (int k = offset; k < n; k++){
                    e = E.data + I * n * n + k * n;
                    e[threadIdx.x] = real_t(0.);
                    q = Q.data + I * m * m + (k-offset) * m;
                    for (int i = 0; i < m; i++){
                        e[threadIdx.x] += v[i*n + threadIdx.x] * q[i];
                    }
                }
            }
        }
        
        template <Device DEV,typename T, typename K>
        void __global__ compute_eigenvectors_ends_(const IsomerBatch<DEV> B, CuArray<T> Q, CuArray<T> V, CuArray<T> Emin, CuArray<T> Emax, CuArray<int> MinIdx, CuArray<int> MaxIdx, int m){
            TEMPLATE_TYPEDEFS(T,K);
            int n = B.n_atoms * 3;
            for (int I = blockIdx.x; I < B.isomer_capacity; I += gridDim.x){
                int minidx = MinIdx.data[I];
                int maxidx = MaxIdx.data[I];
                real_t* emin = Emin.data + I * n;
                real_t* emax = Emax.data + I * n;
                real_t* v = V.data + I * m * n;
                real_t* qmin = Q.data + I * m * m + minidx * m;
                real_t* qmax = Q.data + I * m * m + maxidx * m;
                emin[threadIdx.x] = real_t(0.);
                emax[threadIdx.x] = real_t(0.);
                for (int i = 0; i < m; i++){
                    emin[threadIdx.x] += v[i*n + threadIdx.x] * qmin[i];
                    emax[threadIdx.x] += v[i*n + threadIdx.x] * qmax[i];
                }
            }
        }

        template <Device DEV,typename T, typename K>
        void eigensolve(const IsomerBatch<DEV>& B, CuArray<T>& Q, const CuArray<T>& hessians, const CuArray<K>& cols, CuArray<T>& eigenvalues, const LaunchCtx& ctx, const LaunchPolicy policy){
            if (policy == LaunchPolicy::SYNC) ctx.wait();
            FLOAT_TYPEDEFS(T);
            hipSetDevice(B.get_device_id());
            auto dev = B.get_device_id();
            static int Nd = LaunchCtx::get_device_count();
            static std::vector<bool> init(Nd, false);
            if(policy == LaunchPolicy::SYNC) {ctx.wait();}
            static std::vector<CuArray<T>> Us(Nd); //Upper diagonals
            static std::vector<CuArray<T>> Ls(Nd); //Lower diagonals
            static std::vector<CuArray<T>> Vs(Nd); //Store Lanczos vectors
            static std::vector<CuArray<T>> Qs(Nd); //Store transformation matrices Q DEV,T,K Q^DEV,T,K = H
            static int m_natoms = B.n_atoms;
            static int m_isomer_capacity = B.isomer_capacity;
            static int n_deflation = 0;
            if (!init[dev] || m_natoms != B.n_atoms || m_isomer_capacity != B.isomer_capacity){
                init[dev] = true;
                Us[dev].resize(B.n_atoms*B.isomer_capacity*3); Us[dev].fill(0.); Us[dev].to_device(dev);
                Ls[dev].resize(B.n_atoms*B.isomer_capacity*3); Ls[dev].fill(0.); Ls[dev].to_device(dev);
                Vs[dev].resize(B.n_atoms*B.isomer_capacity*3*3*B.n_atoms); Vs[dev].fill(0.); Vs[dev].to_device(dev);
                Qs[dev].resize(B.n_atoms*B.isomer_capacity*3*3*B.n_atoms); Qs[dev].fill(0.); Qs[dev].to_device(dev);
                m_natoms = B.n_atoms;
                m_isomer_capacity = B.isomer_capacity;
            }

            size_t smem = sizeof(coord3d)*B.n_atoms*3 + sizeof(T)*Block_Size_Pow_2;
            size_t smem_qr = sizeof(T)*(B.n_atoms*3+1)*6 + sizeof(T)*(B.n_atoms*3)*2;
            size_t smem_eig = 0;

            static LaunchDims dims((void*)lanczos_<EigensolveMode::FULL_SPECTRUM,DEV,T,K>, B.n_atoms*3, smem, B.isomer_capacity);
            static LaunchDims qr_dims((void*)eigensolve_<EigensolveMode::VECTORS,DEV,T,K>, 64, smem_qr, B.isomer_capacity);
            static LaunchDims eig_dims((void*)compute_eigenvectors_<EigensolveMode::FULL_SPECTRUM,DEV,T,K>, B.n_atoms*3, smem_eig, B.isomer_capacity);
            dims.update_dims((void*)lanczos_<EigensolveMode::FULL_SPECTRUM,DEV,T,K>, B.n_atoms*3, smem, B.isomer_capacity);
            qr_dims.update_dims((void*)eigensolve_<EigensolveMode::VECTORS,DEV,T,K>, 64, smem_qr, B.isomer_capacity);
            eig_dims.update_dims((void*)compute_eigenvectors_<EigensolveMode::FULL_SPECTRUM,DEV,T,K>, B.n_atoms*3, smem_eig, B.isomer_capacity);
            
            //The hessian has 6 degrees of freedom, so in order to find the smallest eigenvalue we must find the 6 eigenvectors corresponding to these lambda=0
            //for (int i = 0; i <  6; i++){ 
            int Nlanczos = B.n_atoms*3;
            void* kargs[]{(void*)&B, (void*)&Vs[dev], (void*)&Ls[dev], (void*)&eigenvalues, (void*)&hessians, (void*)&cols, (void*)&Nlanczos};
            void* kargs_qr[]{(void*)&B, (void*)&eigenvalues, (void*)&Ls[dev], (void*)&Us[dev], (void*)&Qs[dev], (void*)&Nlanczos};
            void* kargs_vector[]{(void*)&B, (void*)&Qs[dev], (void*)&Vs[dev], (void*)&Q, (void*)&Nlanczos};
            safeCudaKernelCall((void*)lanczos_<EigensolveMode::FULL_SPECTRUM,DEV,T,K>, dims.get_grid(), dims.get_block(), kargs, smem, ctx.stream);
            //ctx.wait();
            //vector<T> Qhost(B.n_atoms*3*3*B.n_atoms);
            //vector<T> Diags = vector<T>(eigenvalues.data, eigenvalues.data + B.n_atoms*3);
            //vector<T> OffDiags = vector<T>(Ls[dev].data, Ls[dev].data + B.n_atoms*3);
            //vector<T> LambdaHost(B.n_atoms*3);
            //eigensystem_hermitian(Nlanczos, Diags, OffDiags, Qhost, LambdaHost);
            //ofstream out("Qhost.float32", ios::binary); out.write((char*)Qhost.data(), Qhost.size()*sizeof(T)); out.close();
            //ofstream out2("D.float32", ios::binary); out2.write((char*)eigenvalues.data, eigenvalues.size()*sizeof(T)); out2.close();
            //ofstream out3("L.float32", ios::binary); out3.write((char*)Ls[dev].data, Ls[dev].size()*sizeof(T)); out3.close();

            safeCudaKernelCall((void*)eigensolve_<EigensolveMode::VECTORS,DEV,T,K>, qr_dims.get_grid(), qr_dims.get_block(), kargs_qr, smem_qr, ctx.stream);
            safeCudaKernelCall((void*)compute_eigenvectors_<EigensolveMode::FULL_SPECTRUM,DEV,T,K>, eig_dims.get_grid(), eig_dims.get_block(), kargs_vector, smem_eig, ctx.stream);

            if (policy == LaunchPolicy::SYNC) ctx.wait();
            printLastCudaError("Full Spectrum Eigensolver Failed : ");
        }

        template <Device DEV,typename T, typename K>
        void eigensolve(const IsomerBatch<DEV>& B, const CuArray<T>& hessians, const CuArray<K>& cols, CuArray<T>& eigenvalues, const LaunchCtx& ctx, const LaunchPolicy policy){
            if (policy == LaunchPolicy::SYNC) ctx.wait();
            hipSetDevice(B.get_device_id());
            FLOAT_TYPEDEFS(T);
            auto dev = B.get_device_id();
            static int Nd = LaunchCtx::get_device_count();
            static std::vector<bool> init(Nd, false);
            if(policy == LaunchPolicy::SYNC) {ctx.wait();}
            static std::vector<CuArray<T>> Us(Nd); //Upper diagonals
            static std::vector<CuArray<T>> Ls(Nd); //Lower diagonals
            static std::vector<CuArray<T>> Vs(Nd); //Store Lanczos vectors
            static std::vector<CuArray<T>> Qs(Nd); //Store transformation matrices Q DEV,T,K Q^DEV,T,K = H
            static int m_natoms = B.n_atoms;
            static int m_isomer_capacity = B.isomer_capacity;
            static int n_deflation = 0;
            if (!init[dev] || m_natoms != B.n_atoms || m_isomer_capacity != B.isomer_capacity){
                init[dev] = true;
                Us[dev].resize(B.n_atoms*B.isomer_capacity*3); Us[dev].fill(0.); Us[dev].to_device(dev);
                Ls[dev].resize(B.n_atoms*B.isomer_capacity*3); Ls[dev].fill(0.); Ls[dev].to_device(dev);
                Vs[dev].resize(B.n_atoms*B.isomer_capacity*3*3*B.n_atoms); Vs[dev].fill(0.); Vs[dev].to_device(dev);
                Qs[dev].resize(B.n_atoms*B.isomer_capacity*3*3*B.n_atoms); Qs[dev].fill(0.); Qs[dev].to_device(dev);
                m_natoms = B.n_atoms;
                m_isomer_capacity = B.isomer_capacity;
            }

            size_t smem = sizeof(coord3d)*B.n_atoms*3 + sizeof(T)*Block_Size_Pow_2;
            size_t smem_qr = sizeof(T)*(B.n_atoms*3+1)*6 + sizeof(T)*(B.n_atoms*3)*2;
            size_t smem_eig = 0;
            static LaunchDims dims((void*)lanczos_<EigensolveMode::FULL_SPECTRUM, DEV,T,K>, B.n_atoms*3, smem, B.isomer_capacity);
            static LaunchDims qr_dims((void*)eigensolve_<EigensolveMode::NO_VECTORS, DEV,T,K>, 64, smem_qr, B.isomer_capacity);

            dims.update_dims((void*)lanczos_<EigensolveMode::FULL_SPECTRUM, DEV,T,K>, B.n_atoms*3, smem, B.isomer_capacity);
            qr_dims.update_dims((void*)eigensolve_<EigensolveMode::NO_VECTORS, DEV,T,K>, 64, smem_qr, B.isomer_capacity);

            
            //The hessian has 6 degrees of freedom, so in order to find the smallest eigenvalue we must find the 6 eigenvectors corresponding to these lambda=0
            //for (int i = 0; i <  6; i++){ 
            int Nlanczos = B.n_atoms*3;
            void* kargs[]{(void*)&B, (void*)&Vs[dev], (void*)&Ls[dev], (void*)&eigenvalues, (void*)&hessians, (void*)&cols, (void*)&Nlanczos};
            void* kargs_qr[]{(void*)&B, (void*)&eigenvalues, (void*)&Ls[dev], (void*)&Us[dev], (void*)&Qs[dev], (void*)&Nlanczos};
            safeCudaKernelCall((void*)lanczos_<EigensolveMode::FULL_SPECTRUM, DEV,T,K>, dims.get_grid(), dims.get_block(), kargs, smem, ctx.stream);
            //ctx.wait();
            //vector<T> Qhost(B.n_atoms*3*3*B.n_atoms);
            //vector<T> Diags = vector<T>(eigenvalues.data, eigenvalues.data + B.n_atoms*3);
            //vector<T> OffDiags = vector<T>(Ls[dev].data, Ls[dev].data + B.n_atoms*3);
            //vector<T> LambdaHost(B.n_atoms*3);
            //eigensystem_hermitian(Nlanczos, Diags, OffDiags, Qhost, LambdaHost);
            //ofstream out("Qhost.float32", ios::binary); out.write((char*)Qhost.data(), Qhost.size()*sizeof(T)); out.close();
            //ofstream out2("D.float32", ios::binary); out2.write((char*)eigenvalues.data, eigenvalues.size()*sizeof(T)); out2.close();
            //ofstream out3("L.float32", ios::binary); out3.write((char*)Ls[dev].data, Ls[dev].size()*sizeof(T)); out3.close();
            safeCudaKernelCall((void*)eigensolve_<EigensolveMode::NO_VECTORS, DEV,T,K>, qr_dims.get_grid(), qr_dims.get_block(), kargs_qr, smem_qr, ctx.stream);
            if (policy == LaunchPolicy::SYNC) ctx.wait();
            printLastCudaError("Full Spectrum Eigensolver Failed : ");
        }

        template <Device DEV,typename T, typename K>
        void spectrum_ends(const IsomerBatch<DEV>& B, const CuArray<T>& hessians, const CuArray<K>& cols, CuArray<T>& lambda_mins, CuArray<T>& lambda_maxs, int m_lanczos, const LaunchCtx& ctx, const LaunchPolicy policy){
            if (policy == LaunchPolicy::SYNC) ctx.wait();
            FLOAT_TYPEDEFS(T);
            hipSetDevice(B.get_device_id());
            auto dev = B.get_device_id();
            static int Nd = LaunchCtx::get_device_count();
            static std::vector<bool> init(Nd, false);
            if(policy == LaunchPolicy::SYNC) {ctx.wait();}
            static std::vector<CuArray<T>> Us(Nd); //Upper diagonals
            static std::vector<CuArray<T>> Ls(Nd); //Lower diagonals
            static std::vector<CuArray<T>> Ds(Nd); //Diagonals
            static std::vector<CuArray<T>> Vs(Nd); //Lanczos vectors
            static std::vector<CuArray<T>> Qs(Nd); //Q matrix for QR decomposition
            static std::vector<CuArray<int>> EigMinIdxs(Nd); //Indices of the smallest eigenvalues
            static std::vector<CuArray<int>> EigMaxIdxs(Nd); //Indices of the largest eigenvalues
            static int m_natoms = B.n_atoms;
            static int m_isomer_capacity = B.isomer_capacity;
            static int m = m_lanczos;

            if (!init[dev] || m != m_lanczos || m_natoms != B.n_atoms || m_isomer_capacity != B.isomer_capacity){
                init[dev] = true;
                m_natoms = B.n_atoms;
                m_isomer_capacity = B.isomer_capacity;
                m = m_lanczos;
                Us[dev].resize(B.isomer_capacity*m); Us[dev].fill(0.); Us[dev].to_device(dev);
                Ls[dev].resize(B.isomer_capacity*m); Ls[dev].fill(0.); Ls[dev].to_device(dev);
                Ds[dev].resize(B.isomer_capacity*m); Ds[dev].fill(0.); Ds[dev].to_device(dev);
                Vs[dev].resize(B.isomer_capacity*B.n_atoms*3*m); Vs[dev].fill(0.); Vs[dev].to_device(dev);
                Qs[dev].resize(B.isomer_capacity*m*m); Qs[dev].fill(0.); Qs[dev].to_device(dev);
                EigMinIdxs[dev].resize(B.isomer_capacity); EigMinIdxs[dev].fill(0); EigMinIdxs[dev].to_device(dev);
                EigMaxIdxs[dev].resize(B.isomer_capacity); EigMaxIdxs[dev].fill(0); EigMaxIdxs[dev].to_device(dev);
            }

            
            size_t smem = sizeof(T)*B.n_atoms*3*2 + m*2*sizeof(T);
            size_t smem_qr = sizeof(T)*(m+1)*6 + sizeof(T)*(64)*2;
            //size_t smem_transform = sizeof(T)*B.n_atoms*3*2;

            static LaunchDims dims((void*)lanczos_<EigensolveMode::ENDS,DEV,T,K>, B.n_atoms*3, smem, B.isomer_capacity);
            static LaunchDims qr_dims((void*)eigensolve_min_max_<EigensolveMode::NO_VECTORS,DEV,T,K>, 64, smem_qr, B.isomer_capacity);
            //static LaunchDims transform_dims((void*)compute_eigenvectors_, B.n_atoms*3, smem_transform, B.isomer_capacity);
            dims.update_dims((void*)lanczos_<EigensolveMode::ENDS,DEV,T,K>, B.n_atoms*3, smem, B.isomer_capacity);
            qr_dims.update_dims((void*)eigensolve_min_max_<EigensolveMode::NO_VECTORS,DEV,T,K>, 64, smem_qr, B.isomer_capacity);

            //transform_dims.update_dims((void*)compute_eigenvectors_, B.n_atoms*3, smem_transform, B.isomer_capacity);
            
            //The hessian has 6 degrees of freedom, so in order to find the smallest eigenvalue we must find the 6 eigenvectors corresponding to these lambda=0
            int n_deflate = 0;
            void* kargs[]{(void*)&B, (void*)&Vs[dev], (void*)&Ls[dev], (void*)&Ds[dev], (void*)&hessians, (void*)&cols, (void*)&m};
            void* kargs_qr[]{(void*)&B, (void*)&Ds[dev], (void*)&Ls[dev], (void*)&Us[dev], (void*)&Qs[dev], (void*)&lambda_mins, (void*)&lambda_maxs, (void*)&EigMinIdxs[dev], (void*)&EigMaxIdxs[dev], (void*)&m};
            //void* kargs_transform[]{(void*)&B, (void*)&Ds[dev], (void*)&Qs[dev], (void*)&Vs[dev], (void*)&Es[dev], (void*)&m, (void*)&n_deflate};
            safeCudaKernelCall((void*)lanczos_<EigensolveMode::ENDS,DEV,T,K>, dims.get_grid(), dims.get_block(), kargs, smem, ctx.stream);
            safeCudaKernelCall((void*)eigensolve_min_max_<EigensolveMode::NO_VECTORS,DEV,T,K>, qr_dims.get_grid(), qr_dims.get_block(), kargs_qr, smem_qr, ctx.stream);

            if (policy == LaunchPolicy::SYNC) ctx.wait();
            printLastCudaError("Spectrum Ends Failed: ");
        }

        template <Device DEV,typename T, typename K>
        void spectrum_ends(const IsomerBatch<DEV>& B, const CuArray<T>& hessians, const CuArray<K>& cols, CuArray<T>& lambda_mins, CuArray<T>& lambda_maxs, CuArray<T>& eigvect_mins, CuArray<T>& eigvect_maxs, int m_lanczos, const LaunchCtx& ctx, const LaunchPolicy policy){
            if (policy == LaunchPolicy::SYNC) ctx.wait();
            hipSetDevice(B.get_device_id());
            auto dev = B.get_device_id();
            static int Nd = LaunchCtx::get_device_count();
            static std::vector<bool> init(Nd, false);
            if(policy == LaunchPolicy::SYNC) {ctx.wait();}
            static std::vector<CuArray<T>> Us(Nd); //Upper diagonals
            static std::vector<CuArray<T>> Ls(Nd); //Lower diagonals
            static std::vector<CuArray<T>> Ds(Nd); //Diagonals
            static std::vector<CuArray<T>> Vs(Nd); //Lanczos vectors
            static std::vector<CuArray<T>> Qs(Nd); //Q matrix for QR decomposition
            static std::vector<CuArray<int>> EigMinIdxs(Nd); //Indices of the smallest eigenvalues
            static std::vector<CuArray<int>> EigMaxIdxs(Nd); //Indices of the largest eigenvalues
            static int m_natoms = B.n_atoms;
            static int m_isomer_capacity = B.isomer_capacity;
            static int m = m_lanczos;

            if (!init[dev] || m != m_lanczos || m_natoms != B.n_atoms || m_isomer_capacity != B.isomer_capacity){
                init[dev] = true;
                m_natoms = B.n_atoms;
                m_isomer_capacity = B.isomer_capacity;
                m = m_lanczos;
                Us[dev].resize(B.isomer_capacity*m); Us[dev].fill(0.); Us[dev].to_device(dev);
                Ls[dev].resize(B.isomer_capacity*m); Ls[dev].fill(0.); Ls[dev].to_device(dev);
                Ds[dev].resize(B.isomer_capacity*m); Ds[dev].fill(0.); Ds[dev].to_device(dev);
                Vs[dev].resize(B.isomer_capacity*B.n_atoms*3*m); Vs[dev].fill(0.); Vs[dev].to_device(dev);
                Qs[dev].resize(B.isomer_capacity*m*m); Qs[dev].fill(0.); Qs[dev].to_device(dev);
                EigMinIdxs[dev].resize(B.isomer_capacity); EigMinIdxs[dev].fill(0); EigMinIdxs[dev].to_device(dev);
                EigMaxIdxs[dev].resize(B.isomer_capacity); EigMaxIdxs[dev].fill(0); EigMaxIdxs[dev].to_device(dev);
            }

            
            size_t smem = sizeof(T)*B.n_atoms*3*2 + m*2*sizeof(T);
            size_t smem_qr = sizeof(T)*(m+1)*6 + sizeof(T)*(64)*2;
            size_t smem_eigs = 0;


            static LaunchDims dims((void*)lanczos_<EigensolveMode::ENDS,DEV,T,K>, B.n_atoms*3, smem, B.isomer_capacity);
            static LaunchDims qr_dims((void*)eigensolve_min_max_<EigensolveMode::VECTORS,DEV,T,K>, 64, smem_qr, B.isomer_capacity);
            static LaunchDims eigs_dims((void*)compute_eigenvectors_ends_<DEV,T,K>, B.n_atoms*3, smem_eigs, B.isomer_capacity);

            dims.update_dims((void*)lanczos_<EigensolveMode::ENDS,DEV,T,K>, B.n_atoms*3, smem, B.isomer_capacity);
            qr_dims.update_dims((void*)eigensolve_min_max_<EigensolveMode::VECTORS,DEV,T,K>, 64, smem_qr, B.isomer_capacity);
            eigs_dims.update_dims((void*)compute_eigenvectors_ends_<DEV,T,K>, B.n_atoms*3, smem_eigs, B.isomer_capacity);
            
            //The hessian has 6 degrees of freedom, so in order to find the smallest eigenvalue we must find the 6 eigenvectors corresponding to these lambda=0
            int n_deflate = 0;
            void* kargs[]{(void*)&B, (void*)&Vs[dev], (void*)&Ls[dev], (void*)&Ds[dev], (void*)&hessians, (void*)&cols, (void*)&m};
            void* kargs_qr[]{(void*)&B, (void*)&Ds[dev], (void*)&Ls[dev], (void*)&Us[dev], (void*)&Qs[dev], (void*)&lambda_mins, (void*)&lambda_maxs, (void*)&EigMinIdxs[dev], (void*)&EigMaxIdxs[dev], (void*)&m};
            void* kargs_eigs[]{(void*)&B, (void*)&Qs[dev], (void*)&Vs[dev], (void*)&eigvect_mins, (void*)&eigvect_maxs, (void*)&EigMinIdxs[dev], (void*)&EigMaxIdxs[dev], (void*)&m};
            //void* kargs_transform[]{(void*)&B, (void*)&Ds[dev], (void*)&Qs[dev], (void*)&Vs[dev], (void*)&Es[dev], (void*)&m, (void*)&n_deflate};

            safeCudaKernelCall((void*)lanczos_<EigensolveMode::ENDS,DEV,T,K>, dims.get_grid(), dims.get_block(), kargs, smem, ctx.stream);
            safeCudaKernelCall((void*)eigensolve_min_max_<EigensolveMode::VECTORS,DEV,T,K>, qr_dims.get_grid(), qr_dims.get_block(), kargs_qr, smem_qr, ctx.stream);
            safeCudaKernelCall((void*)compute_eigenvectors_ends_<DEV,T,K>, eigs_dims.get_grid(), eigs_dims.get_block(), kargs_eigs, smem_eigs, ctx.stream);

            if (policy == LaunchPolicy::SYNC) ctx.wait();
            printLastCudaError("Spectrum Ends Failed: ");
        }

        template <Device U, typename T, typename K>        //Compute the full spectrum of the hessian matrix in the special case where the hessian has 6 zero eigenvalues pertaining to the 3 translational and 3 rotational degrees of freedom.
        void eigensolve_special(const IsomerBatch<U>& B, CuArray<T>& Q, const CuArray<T>& hessians, const CuArray<K>& cols, CuArray<T>& eigenvalues, const LaunchCtx& ctx, const LaunchPolicy policy){
            if (policy == LaunchPolicy::SYNC) ctx.wait();
            FLOAT_TYPEDEFS(T);
            hipSetDevice(B.get_device_id());
            auto dev = B.get_device_id();
            static int Nd = LaunchCtx::get_device_count();
            static std::vector<bool> init(Nd, false);
            if(policy == LaunchPolicy::SYNC) {ctx.wait();}
            static std::vector<CuArray<T>> Us(Nd); //Upper diagonals
            static std::vector<CuArray<T>> Ls(Nd); //Lower diagonals
            static std::vector<CuArray<T>> Vs(Nd); //Store Lanczos vectors
            static std::vector<CuArray<T>> Qs(Nd); //Store transformation matrices Q DEV,T,K Q^DEV,T,K = H
            int m_natoms = B.n_atoms;
            int m_isomer_capacity = B.isomer_capacity;
            int n_deflation = 6;
            int n_eigs = B.n_atoms *3;
            if (!init[dev] || m_natoms != B.n_atoms || m_isomer_capacity != B.isomer_capacity){
                init[dev] = true;
                Us[dev].resize((n_eigs)*B.isomer_capacity); Us[dev].fill(0.); Us[dev].to_device(dev);
                Ls[dev].resize((n_eigs)*B.isomer_capacity); Ls[dev].fill(0.); Ls[dev].to_device(dev);
                Vs[dev].resize(B.isomer_capacity*(n_eigs) * n_eigs); Vs[dev].fill(0.); Vs[dev].to_device(dev);
                Qs[dev].resize(B.isomer_capacity*(n_eigs) * n_eigs); Qs[dev].fill(0.); Qs[dev].to_device(dev);
            }

            size_t smem = sizeof(coord3d)*n_eigs + sizeof(T)*Block_Size_Pow_2;
            size_t smem_qr = sizeof(T)*(n_eigs+1)*6 + sizeof(T)*(n_eigs)*2;
            size_t smem_eig = sizeof(T)*n_eigs*2; //We just need enough memory to compute reductions of eigenvectors such that we can normalize them.

            static LaunchDims dims((void*)lanczos_<EigensolveMode::FULL_SPECTRUM_MOLECULE,U,T,K>, n_eigs, smem, B.isomer_capacity);
            static LaunchDims qr_dims((void*)eigensolve_<EigensolveMode::FULL_SPECTRUM_MOLECULE,U,T,K>, 64, smem_qr, B.isomer_capacity);
            static LaunchDims eig_dims((void*)compute_eigenvectors_<EigensolveMode::FULL_SPECTRUM_MOLECULE,U,T,K>, n_eigs, smem_eig, B.isomer_capacity);
            dims.update_dims((void*)lanczos_<EigensolveMode::FULL_SPECTRUM_MOLECULE,U,T,K>, n_eigs, smem, B.isomer_capacity);
            qr_dims.update_dims((void*)eigensolve_<EigensolveMode::FULL_SPECTRUM_MOLECULE,U,T,K>, 64, smem_qr, B.isomer_capacity);
            eig_dims.update_dims((void*)compute_eigenvectors_<EigensolveMode::FULL_SPECTRUM_MOLECULE,U,T,K>, n_eigs, smem_eig, B.isomer_capacity);
            
            //The hessian has 6 degrees of freedom, so in order to find the smallest eigenvalue we must find the 6 eigenvectors corresponding to these lambda=0
            //for (int i = 0; i <  6; i++){ 
            int Nlanczos = n_eigs - n_deflation;
            void* kargs[]{(void*)&B, (void*)&Vs[dev], (void*)&Ls[dev], (void*)&eigenvalues, (void*)&hessians, (void*)&cols, (void*)&Nlanczos};
            void* kargs_qr[]{(void*)&B, (void*)&eigenvalues, (void*)&Ls[dev], (void*)&Us[dev], (void*)&Qs[dev], (void*)&Nlanczos};
            void* kargs_vector[]{(void*)&B, (void*)&Qs[dev], (void*)&Vs[dev], (void*)&Q, (void*)&Nlanczos};
            safeCudaKernelCall((void*)lanczos_<EigensolveMode::FULL_SPECTRUM_MOLECULE,U,T,K>, dims.get_grid(), dims.get_block(), kargs, smem, ctx.stream);
            //ctx.wait();
            //vector<T> Qhost(B.n_atoms*3*3*B.n_atoms);
            //vector<T> Diags = vector<T>(eigenvalues.data, eigenvalues.data + B.n_atoms*3);
            //vector<T> OffDiags = vector<T>(Ls[dev].data, Ls[dev].data + B.n_atoms*3);
            //vector<T> LambdaHost(B.n_atoms*3);
            //eigensystem_hermitian(Nlanczos, Diags, OffDiags, Qhost, LambdaHost);
            //ofstream out("Qhost.float32", ios::binary); out.write((char*)Qhost.data(), Qhost.size()*sizeof(T)); out.close();
            //ofstream out2("D.float32", ios::binary); out2.write((char*)eigenvalues.data, eigenvalues.size()*sizeof(T)); out2.close();
            //ofstream out3("L.float32", ios::binary); out3.write((char*)Ls[dev].data, Ls[dev].size()*sizeof(T)); out3.close();

            safeCudaKernelCall((void*)eigensolve_<EigensolveMode::FULL_SPECTRUM_MOLECULE,U,T,K>, qr_dims.get_grid(), qr_dims.get_block(), kargs_qr, smem_qr, ctx.stream);
            safeCudaKernelCall((void*)compute_eigenvectors_<EigensolveMode::FULL_SPECTRUM_MOLECULE,U,T,K>, eig_dims.get_grid(), eig_dims.get_block(), kargs_vector, smem_eig, ctx.stream);

            if (policy == LaunchPolicy::SYNC) ctx.wait();
            printLastCudaError("Full Spectrum Eigensolver Failed : ");

        }


    }
}

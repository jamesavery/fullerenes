#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <cooperative_groups/scan.h>
#include "fullerenes/gpu/kernels.hh"
#include "fullerenes/gpu/launch_ctx.hh"
namespace gpu_kernels{

namespace isomerspace_X0{
#include "device_includes.cu"

template hipError_t zero_order_geometry<GPU>(IsomerBatch<GPU>& B, const float scalerad, const LaunchCtx& ctx, const LaunchPolicy policy);

template <Device U> __device__
device_node_t multiple_source_shortest_paths(const IsomerBatch<U>& B, device_node_t* distances, const size_t isomer_idx){
    DEVICE_TYPEDEFS;
    
    DeviceCubicGraph FG = DeviceCubicGraph(&B.cubic_neighbours[isomer_idx*blockDim.x*3]);
    node_t outer_face[6]; memset(outer_face, 0, sizeof(node_t)*6); //Do not rely on uninitialized memory it will only be zero on first touch.
    uint8_t Nface = FG.get_face_oriented(0, FG.cubic_neighbours[0],outer_face);
    /* uint8_t local_fsize = FG.face_size(threadIdx.x, FG.cubic_neighbours[threadIdx.x*3]);
    node_t is_pentagon = node_t(local_fsize == uint8_t(5) ? 1 : 0);
    node_t* sharedmem = reinterpret_cast<node_t*>(distances);
    ex_scan(sharedmem, is_pentagon, blockDim.x);
    if(threadIdx.x > 0 && (sharedmem[threadIdx.x] == node_t(1)) && (sharedmem[threadIdx.x - 1] == node_t(0))) sharedmem[0] = node_t(threadIdx.x - 1);
    BLOCK_SYNC;
    node_t lowest_index_pentagon = sharedmem[0];
    uint8_t Nface = FG.get_face_oriented(lowest_index_pentagon, FG.cubic_neighbours[lowest_index_pentagon*3],outer_face);
    BLOCK_SYNC; */ //This code finds the pentagon with the lowest index and uses that as the starting point for the BFS. 
    distances[threadIdx.x] = node_t(NODE_MAX);    
    BLOCK_SYNC
    if (threadIdx.x < Nface)  distances[outer_face[threadIdx.x]] = 0;
    BLOCK_SYNC
    if (threadIdx.x == 0){
        CuDeque<node_t> queue = CuDeque<node_t>(distances + blockDim.x, blockDim.x);
        for (size_t i = 0; i < Nface; i++) queue.push_back(outer_face[i]);
        while (!queue.empty())
        {   
            node_t v = queue.pop_front();
            for (size_t i = 0; i < 3; i++)
            {   
                node_t w = FG.cubic_neighbours[v*3 + i];
                if(distances[w] == NODE_MAX) {
                distances[w] = distances[v]+1;
                queue.push_back(w);
                }
            }
        }
    }
    BLOCK_SYNC
    device_node_t distance = distances[threadIdx.x];
    BLOCK_SYNC
    return distance;
}


template <Device U> __device__
coord2dh spherical_projection(const IsomerBatch<U>& B, device_node_t* sdata, const size_t isomer_idx){
    DEVICE_TYPEDEFS;

    node_t distance =  multiple_source_shortest_paths(B,reinterpret_cast<node_t*>(sdata), isomer_idx);
    BLOCK_SYNC
    clear_cache(reinterpret_cast<real_t*>(sdata), Block_Size_Pow_2); 
    node_t d_max = reduction_max(sdata, distance);

    clear_cache(reinterpret_cast<real_t*>(sdata), Block_Size_Pow_2); 
    ordered_atomic_add(&reinterpret_cast<real_t*>(sdata)[distance],real_t(1.0)); 
    BLOCK_SYNC
    node_t num_of_same_dist = node_t(reinterpret_cast<real_t*>(sdata)[distance]); 
    BLOCK_SYNC
    clear_cache(reinterpret_cast<real_t*>(sdata), Block_Size_Pow_2);
    BLOCK_SYNC;
    coord2dh xys = reinterpret_cast<coord2dh*>(B.xys)[isomer_idx*blockDim.x + threadIdx.x]; BLOCK_SYNC;
    ordered_atomic_add(&reinterpret_cast<real_t*>(sdata)[distance*2], xys[0]); 
    ordered_atomic_add(&reinterpret_cast<real_t*>(sdata)[distance*2+1], xys[1]); BLOCK_SYNC
    coord2dh centroid = reinterpret_cast<coord2dh*>(sdata)[distance] / (real_t)num_of_same_dist; BLOCK_SYNC    
    coord2dh xy = xys - centroid;
    real_t dtheta = real_t(M_PI)/real_t(d_max+1); 
    real_t phi = dtheta*(distance + 0.5); 
    real_t theta = atan2(xy[0],xy[1]); 
    coord2dh spherical_layout = {theta, phi};
    

    return spherical_layout;
}

template <Device U> __global__
void zero_order_geometry_(IsomerBatch<U> B, float scalerad, int offset){
    DEVICE_TYPEDEFS;
    
    extern __shared__  device_real_t sdata[];
    clear_cache(sdata, Block_Size_Pow_2);
    size_t isomer_idx = blockIdx.x + offset;
    if (isomer_idx < B.isomer_capacity && B.statuses[isomer_idx] != IsomerStatus::EMPTY){
    NodeNeighbours node_graph = NodeNeighbours(B, isomer_idx); 
    coord2dh angles = spherical_projection(B,reinterpret_cast<device_node_t*>(sdata), isomer_idx);
    real_t theta = angles[0]; real_t phi = angles[1];
    real_t x = cos(theta)*sin(phi), y = sin(theta)*sin(phi), z = cos(phi);
    coord3d coordinate = {x, y ,z};

    clear_cache(sdata, Block_Size_Pow_2);
    x = reduction(sdata, coordinate[0]); y = reduction(sdata, coordinate[1]); z = reduction(sdata,coordinate[2]);
    coord3d cm = {x, y, z};
    cm /= (real_t)blockDim.x;
    coordinate -= cm;
    real_t Ravg = real_t(0.0);
    clear_cache(sdata, Block_Size_Pow_2);
    real_t* base_pointer = sdata + Block_Size_Pow_2; 
    coord3d* X = reinterpret_cast<coord3d*>(base_pointer);
    X[threadIdx.x] = coordinate;
    BLOCK_SYNC
    real_t local_Ravg = real_t(0.0);
    for (uint8_t i = 0; i < 3; i++) {local_Ravg += norm(X[threadIdx.x] - X[d_get(node_graph.cubic_neighbours,i)]);}
    Ravg = reduction(sdata, local_Ravg);
    Ravg /= real_t(3*blockDim.x);
    coordinate *= (real_t)scalerad*(real_t)1.5/Ravg;
    assign(reinterpret_cast<std::array<float,3>*>(B.X)[blockDim.x*isomer_idx + threadIdx.x] , coordinate);
    }
}

float kernel_time = 0.0;
std::chrono::microseconds time_spent(){
    return std::chrono::microseconds((int) (kernel_time * 1000.f));
}

void reset_time(){
    kernel_time = 0.0;
}

template <Device U>
hipError_t zero_order_geometry(IsomerBatch<U>& B, const float scalerad, const LaunchCtx& ctx, const LaunchPolicy policy){
    hipSetDevice(B.get_device_id());
    //Need a way of telling whether the kernel has been called previously.
    static std::vector<bool> first_call(16, true);
    static hipEvent_t start[16], stop[16];
    float single_kernel_time = 0.0;
    //Construct events only once
    auto dev = B.get_device_id();
    if(first_call[dev]) {hipEventCreate(&start[dev]); hipEventCreate(&stop[dev]);}

    //If launch ploicy is synchronous then wait.
    if(policy == LaunchPolicy::SYNC) {ctx.wait();}
    else if(policy == LaunchPolicy::ASYNC && !first_call[dev]){
        //Records time from previous kernel call
        hipEventElapsedTime(&single_kernel_time, start[dev], stop[dev]);
        kernel_time += single_kernel_time;
    }
    size_t smem =  sizeof(device_coord3d)*B.n_atoms + sizeof(device_real_t)*Block_Size_Pow_2;
    
    //Compute best grid dimensions once.
    static LaunchDims dims((void*)zero_order_geometry_<U>, B.n_atoms, smem, B.isomer_capacity);
    dims.update_dims((void*)zero_order_geometry_<U>, B.n_atoms, smem, B.isomer_capacity);
    hipError_t error;

    //Note: some memory bug exists when using grid-stride for loops inside the kernel launches
    hipEventRecord(start[dev], ctx.stream);
    for (int i = 0; i < B.isomer_capacity + (dims.get_grid().x - B.isomer_capacity % dims.get_grid().x ); i += dims.get_grid().x)
    {
        void* kargs[]{(void*)&B, (void*)&scalerad, (void*)&i};
        error = safeCudaKernelCall((void*)zero_order_geometry_<U>, dims.get_grid(), dims.get_block(), kargs, smem, ctx.stream);
    }
    hipEventRecord(stop[dev], ctx.stream);
    
    if(policy == LaunchPolicy::SYNC) {
        ctx.wait();
        hipEventElapsedTime(&single_kernel_time, start[dev], stop[dev]);
        kernel_time += single_kernel_time;
    }
    printLastCudaError("Zero order geometry:");
    first_call[dev] = false;
    return error;
}

}}

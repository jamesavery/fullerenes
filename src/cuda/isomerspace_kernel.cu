#pragma once
#include "fullerenes/gpu/isomerspace_kernel.hh"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"

template <typename T>
size_t IsomerspaceKernel<T>::get_batch_capacity(){
    hipDeviceProp_t properties;
    int device_count = 0;
    int total_capacity = 0;
    int fullerenes_per_SM;
    hipGetDeviceCount(&device_count);
    for (size_t i = 0; i < device_count; i++)
    {
        hipGetDeviceProperties(&properties,i);
        /** Compiling with --maxrregcount=64   is necessary to easily (singular blocks / fullerene) parallelize fullerenes of size 20-1024 !**/
        /** Needs 3 storage arrays for coordinates and 1 for reductions **/
        /** Calculates maximum number of resident fullerenes on a single Streaming Multiprocessor, multiply with multi processor count to d_get total batch size**/
        hipOccupancyMaxActiveBlocksPerMultiprocessor(&fullerenes_per_SM, kernel_pointer, N, shared_memory_bytes); // How many threads per block
        this->device_capacities[i] = properties.multiProcessorCount*fullerenes_per_SM;
        total_capacity += properties.multiProcessorCount*fullerenes_per_SM;
    }
    return (size_t)total_capacity;
}

template <typename T>
IsomerspaceKernel<T>::IsomerspaceKernel(const size_t N, void* kernel){
    hipGetDeviceCount(&this->device_count);
    kernel_pointer          = kernel;
    global_reduction_arrays = std::vector<device_real_t*>(device_count);
    batch_sizes             = std::vector<int>(device_count);
    device_capacities       = std::vector<int>(device_count);
    this->N                 = N;
    batch_capacity          = get_batch_capacity();    
    index_queue             = std::vector<std::queue<int>>(device_count);

    for (size_t i = 0; i < device_count; i++)
    {
        hipMalloc(&global_reduction_arrays[i], sizeof(device_real_t)*N*device_capacities[i]);
        batch_sizes[i] = 0;
        for (size_t j = 0; j < device_capacities[i]; j++)   index_queue[i].push(j);
        
    }
}

template <typename T>
IsomerspaceKernel<T>::~IsomerspaceKernel(){
    for (size_t i = 0; i < device_count; i++)
    {
        hipSetDevice(i);
        hipFree(global_reduction_arrays[i]);
    }
}
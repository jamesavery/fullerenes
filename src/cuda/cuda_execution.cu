#include "iostream"
#include "fullerenes/gpu/cuda_execution.hh"
#include "hip/hip_runtime_api.h"

int LaunchCtx::get_device_id() const {
    return m_device_id;
}

bool LaunchCtx::is_finished() const{
    return hipStreamQuery(stream) == hipSuccess;
}

bool LaunchCtx::is_default_stream() const{
    return m_unique_stream_idx == -1;
}

void LaunchCtx::wait() const {
    hipStreamSynchronize(stream);
}

void LaunchCtx::wait_all(){
    for (auto& it: m_all_streams) hipStreamSynchronize(**it.second);
}   

int LaunchCtx::get_device_count(){
    int count;
    hipGetDeviceCount(&count);
    return count;
}

int LaunchCtx::get_stream_count(){
    return m_all_streams.size();
}

LaunchCtx::LaunchCtx(){
    hipGetDeviceCount(&m_device_count);
    if (m_device_count < 1) {
        std::cout << "Error: no CUDA enabled devices found" << std::endl; 
        return;
    }
    stream = hipStream_t(NULL);
    hipStream_t* stream_ptr = &stream;
    m_unique_stream_idx = int(-1);
    m_device_id = 0;
    if(default_ctx_created) m_all_streams.insert({m_unique_stream_idx,&stream_ptr});
    default_ctx_created = true;
}

LaunchCtx::LaunchCtx(int device){
    hipGetDeviceCount(&m_device_count);
    if (m_device_count < device) {std::cout << "Error: requested device was not found" << std::endl; return;}
    int temp_device; hipGetDevice(&temp_device);
    hipSetDevice(device);
    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    hipStream_t* stream_ptr = &stream;
    m_unique_stream_idx = m_object_counter++;
    m_all_streams.insert({m_unique_stream_idx,&stream_ptr});
    hipSetDevice(temp_device);
}

LaunchCtx::~LaunchCtx(){
    //Never destroy the default stream everything will break if you do.
    if (!is_default_stream())
    {
        hipStreamDestroy(stream);
        m_all_streams.erase(m_unique_stream_idx);
    }
}


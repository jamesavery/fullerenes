#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <cooperative_groups/scan.h>
#include "fullerenes/gpu/cu_array.hh"
#include "hip/hip_runtime.h"
#include "fullerenes/gpu/cuda_definitions.h"
#include <ostream>
#include <iostream>

template <typename T>
CuArray<T>::CuArray() : size_(0), capacity_(0), initialized_(false){}

template <typename T>
CuArray<T>::CuArray(const size_t size) : size_(size), capacity_(size){
    hipMallocManaged(&data, size*sizeof(T));
    initialized_ = true;
    //cudaMemSet(data,0, size*sizeof(T));
}

template <typename T>
CuArray<T>::CuArray(const CuArray<T>& other) : size_(other.size_), capacity_(other.capacity_){
    hipMallocManaged(&data, size_*sizeof(T));
    initialized_ = true;
}

template <typename T>
CuArray<T>::CuArray(const size_t size, const T& value) : size_(size), capacity_(size){
    hipMallocManaged(&data, size*sizeof(T));
    for (size_t i = 0; i < size; ++i){
        data[i] = value;
    }
    initialized_ = true;
}

template <typename T>
void CuArray<T>::resize(const size_t size){
    if (!initialized_){
        hipMallocManaged(&data, size*sizeof(T));
        initialized_ = true;
    } else{
        T* new_ptr;
        hipMallocManaged(&new_ptr, size*sizeof(T));
        hipMemcpy(new_ptr, data, std::min(size_, size)*sizeof(T), hipMemcpyHostToHost);
        hipFree(data);
        data = new_ptr;
    }
    size_ = size;   
}

template <typename T>
void CuArray<T>::fill(const T& value){
    for (size_t i = 0; i < size_; ++i){
        data[i] = value;
    }
}

template <typename T>
size_t CuArray<T>::size(){
    return size_;
}

template <typename T>
__device__ __host__ T& CuArray<T>::operator[] (const size_t i){
    return data[i];
}

template <typename T>
CuArray<T>& CuArray<T>::operator=(const CuArray<T>& other){
    if (this != &other){
        hipMallocManaged(&data, other.size_*sizeof(T));
        size_ = other.size_;
        capacity_ = other.capacity_;
        hipMemcpy(data, other.data, size_*sizeof(T), hipMemcpyHostToHost );
    }
    return *this;
}

template <typename T>
std::ostream& operator<<(std::ostream& os, const CuArray<T>& input){
    os << "[" << input.data[0];
    for (size_t i = 1; i < input.size_; ++i){
        os << "," << input.data[i];
    }
    os << "]\n";
    return os;
}

template <typename T>
CuArray<T>::~CuArray(){
    if(initialized_) hipFree(data);
}



template <typename T>
void CuArray<T>::to_device(const int device){
    static int attr;
    static bool attr_set = false;
    if (!attr_set){
        hipDeviceGetAttribute(&attr,hipDeviceAttributeMemoryPoolsSupported, device);
        attr_set = true;
    }
    if (attr) {
        hipSetDevice(device);
        hipMemPrefetchAsync(data, size_*sizeof(T), device);
        hipDeviceSynchronize(); //Ensures that the data is copied before the function returns. Enabled for benchmarking purposes, remove for performance
    }
} 

template <typename T>
void CuArray<T>::to_host(const int device){
    static int attr;
    static bool attr_set = false;
    if (!attr_set){
        hipDeviceGetAttribute(&attr,hipDeviceAttributeMemoryPoolsSupported, device);
        attr_set = true;
    }
    if (attr) {
        hipSetDevice(device);
        hipMemPrefetchAsync(data, size_*sizeof(T), hipCpuDeviceId);
        hipDeviceSynchronize(); //Ensures that the data is copied before the function returns. Enabled for benchmarking purposes, remove for performance
    }
}

//Primitive way of handling the fact that templated code in this translation unit wont be generated unless explicitly instantiated somewhere.
int declare_cu_arrays(){
    CuArray<float> f1_a;                    CuArray<float> f1(1);           f1_a = f1;f1[0] = {};   std::cout << f1;  f1.to_device(0); f1.to_host(0);  CuArray<float> f1b(1,1.0f);  f1.size(); f1.resize(2); f1.fill(1.0f);
    CuArray<double> f2_a;                   CuArray<double> f2(1);          f2_a = f2;f2[0] = {};   std::cout << f2; f2.to_device(0); f2.to_host(0);    CuArray<double> f2b(1,1.0); f2.size(); f2.resize(2); f2.fill(1.0);
    CuArray<int> f7_a;                      CuArray<int> f7(1);             f7_a = f7;f7[0] = {};   std::cout << f7; f7.to_device(0); f7.to_host(0);      CuArray<int> f7b(1,1); f7.size(); f7.resize(2); f7.fill(1);
    CuArray<size_t> f8_a;                   CuArray<size_t> f8(1);          f8_a = f8;f8[0] = {};   std::cout << f8; f8.to_device(0); f8.to_host(0);  CuArray<size_t> f8b(1,1); f8.size(); f8.resize(2); f8.fill(1);
    CuArray<uint8_t> f9_a;                  CuArray<uint8_t> f9(1);         f9_a = f9;f9[0] = {};   std::cout << f9; f9.to_device(0); f9.to_host(0);   CuArray<uint8_t> f9b(1,1); f9.size(); f9.resize(2); f9.fill(1);
    CuArray<unsigned char> f9_2_a;          CuArray<unsigned char> f9_2(1); f9_2_a = f9_2;f9_2[0] = {}; std::cout << f9_2; f9_2.to_device(0); f9_2.to_host(0);  CuArray<unsigned char> f9_2b(1,1); f9_2.size(); f9_2.resize(2); f9_2.fill(1);
    CuArray<uint16_t> f10_a;                CuArray<uint16_t> f10(1);       f10_a = f10;f10[0] = {};  std::cout << f10; f10.to_device(0); f10.to_host(0); CuArray<uint16_t> f10b(1,1); f10.size(); f10.resize(2); f10.fill(1);
    CuArray<unsigned short> f10_2_a;        CuArray<unsigned short> f10_2(1); f10_2_a = f10_2;f10_2[0] = {};  std::cout << f10_2; f10_2.to_device(0); f10_2.to_host(0); CuArray<unsigned short> f10_2b(1,1); f10_2.size(); f10_2.resize(2); f10_2.fill(1);
    CuArray<uint32_t> f11_a;                CuArray<uint32_t> f11(1);       f11_a = f11;f11[0] = {};  std::cout << f11; f11.to_device(0); f11.to_host(0); CuArray<uint32_t> f11b(1,1); f11.size(); f11.resize(2); f11.fill(1);
    CuArray<uint64_t> f12_a;                CuArray<uint64_t> f12(1);       f12_a = f12;f12[0] = {};  std::cout << f12; f12.to_device(0); f12.to_host(0); CuArray<uint64_t> f12b(1,1); f12.size(); f12.resize(2); f12.fill(1);
    CuArray<char> f13_a;                    CuArray<char> f13(1);           f13_a = f13;f13[0] = {};  std::cout << f13; f13.to_device(0); f13.to_host(0); CuArray<char> f13b(1,1); f13.size(); f13.resize(2); f13.fill(1);
    CuArray<bool> f14_a;                    CuArray<bool> f14(1);           f14_a = f14;f14[0] = {};  std::cout << f14; f14.to_device(0); f14.to_host(0); CuArray<bool> f14b(1,1); f14.size(); f14.resize(2); f14.fill(1);
    return 1;
}

#include "iostream"
#include "fullerenes/gpu/launch_ctx.hh"
#include "hip/hip_runtime_api.h"

LaunchCtx& LaunchCtx::operator=(const LaunchCtx& other){
    if (this != &other){
        m_device_id = other.m_device_id;
        hipSetDevice(m_device_id);
        hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
        hipEventCreateWithFlags(&m_start, hipEventBlockingSync);
        hipEventCreateWithFlags(&m_stop, hipEventBlockingSync);
        
    }
    return *this;
}

int LaunchCtx::get_device_id() const {
    return m_device_id;
}

bool LaunchCtx::is_finished() const{
    return hipStreamQuery(stream) == hipSuccess;
}

bool LaunchCtx::is_default_stream() const{
    return m_unique_stream_idx == -1;
}

void LaunchCtx::wait() const {
    hipStreamSynchronize(stream);
}

void LaunchCtx::wait_all(){
    for (auto& it: m_all_streams) hipStreamSynchronize(**it.second);
}   

int LaunchCtx::get_device_count(){
    int count;
    hipGetDeviceCount(&count);
    return count;
}

int LaunchCtx::get_stream_count(){
    return m_all_streams.size();
}

void LaunchCtx::clear_allocations(){
    for (int i = 0; i < get_device_count(); ++i){
        hipSetDevice(i);
        hipDeviceSynchronize();
        hipDeviceReset();
    }
}

void LaunchCtx::start_timer(){
    hipEventRecord(m_start, stream);
}

std::chrono::nanoseconds LaunchCtx::stop_timer(){
    float elapsed_time = 0.0f;
    hipEventRecord(m_stop, stream);
    hipEventSynchronize(m_stop);
    hipEventElapsedTime(&elapsed_time, m_start, m_stop); //elapsed_time is in ms
    return std::chrono::nanoseconds((int) (elapsed_time*1e6));
}

LaunchCtx::LaunchCtx(){
    hipGetDeviceCount(&m_device_count);
    if (m_device_count < 1) {
        std::cout << "Error: no CUDA enabled devices found" << std::endl; 
        return;
    }
    stream = hipStream_t(NULL);
    hipStream_t* stream_ptr = &stream;
    hipEventCreateWithFlags(&m_start, hipEventBlockingSync);
    hipEventCreateWithFlags(&m_stop, hipEventBlockingSync);
    m_unique_stream_idx = int(-1);
    m_device_id = 0;
    if(!default_ctx_created) m_all_streams.insert({m_unique_stream_idx,&stream_ptr});
    default_ctx_created = true;
}

LaunchCtx::LaunchCtx(int device){
    static int m_device_count = get_device_count();
    static std::vector<bool> first_call(16, true);
    if (m_device_count < device) {std::cout << "Error: requested device was not found" << std::endl; return;}
    int temp_device; hipGetDevice(&temp_device);
    m_device_id = device;
    hipSetDevice(device);
    if(first_call[device]) hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    hipStream_t* stream_ptr = &stream;
    hipEventCreateWithFlags(&m_start, hipEventBlockingSync);
    hipEventCreateWithFlags(&m_stop, hipEventBlockingSync);
    m_unique_stream_idx = m_object_counter++;
    m_all_streams.insert({m_unique_stream_idx,&stream_ptr});
    hipSetDevice(temp_device);
}

LaunchCtx::~LaunchCtx(){
    //Never destroy the default stream everything will break if you do.
    if (!is_default_stream())
    {
        hipStreamDestroy(stream);
        m_all_streams.erase(m_unique_stream_idx);
    }
    hipEventDestroy(m_start); hipEventDestroy(m_stop);
}


#include "hip/hip_runtime.h"
#include "fullerenes/gpu/isomerspace_forcefield.hh"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include ""
#include <stdio.h>
#//include <hip/hip_runtime_api.h>  // TODO: Get rid of this (not in nvcc std include dirs)
#define getLastCudaError(x) 
#include <iostream>
#include <fstream>
#include <chrono>

namespace IsomerspaceForcefield {

typedef device_real_t real_t;
typedef device_node_t node_t;

#include "C60ih.cu"
#include "coord3d.cu"
#include "helper_functions.cu"

using namespace std::literals;
namespace cg = cooperative_groups;



__device__ struct ArcData{
    //All parameter arrays are indexed by a binary sum, 0,1,2,3,4,...
    //Pentagons = 0
    //Hexagons = 1
    //PPP = 0, {HPP, PHP, PPH} = 1, {PHH, HPH, HHP} = 2, {HHH} = 3
    const real_t optimal_corner_cos_angles[2] = {-0.3090169944, -0.5}; 
    const real_t optimal_bond_lengths[3] = {1.479, 1.458, 1.401}; 
    const real_t optimal_dih_cos_angles[8] = {0.79465455715, 0.87290360705, 0.87290360705, 0.9410338473, 0.816287936, 0.913965949, 0.913965949, 1}; 

    const real_t angle_forces[2] = {207.924,216.787}; 
    const real_t bond_forces[3] = {260.0, 353.377, 518.992}; 
    const real_t dih_forces[4] = {35.0,65.0,3.772,270.0}; 
    
    __device__ ArcData(const node_t a, const uint8_t j, const node_t* neighbours, const coord3d* X, const uint8_t* face_right, const node_t* next_on_face, const node_t* prev_on_face){   
        real_t r_rmp;
        coord3d ap, am, ab, ac, ad, mp;
        //printf("Index: %d \n", a*3 + j);

        //Compute the arcs ab, ac, ad, bp, bm, ap, am, mp, bc and cd
        ab = (X[neighbours[a*3 + j]] - X[a]);  r_rab = bond_length(ab); ab_hat = r_rab * ab;
        ac = (X[neighbours[a*3 + (j+1)%3]] - X[a]); r_rac = bond_length(ac); ac_hat = r_rac * ac;
        ad = (X[neighbours[a*3 + (j+2)%3]] - X[a]); r_rad = bond_length(ad); ad_hat = r_rad * ad;
        
        coord3d bp = (X[next_on_face[a*3 + j]] - X[neighbours[a*3 + j]]); bp_hat = unit_vector(bp);
        coord3d bm = (X[prev_on_face[a*3 + j]] - X[neighbours[a*3 + j]]); bm_hat = unit_vector(bm);

        ap = bp + ab; r_rap = bond_length(ap); ap_hat = r_rap * ap;
        am = bm + ab; r_ram = bond_length(am); am_hat = r_ram * am;
        mp = bp - bm; r_rmp = bond_length(mp); mp_hat = r_rmp * mp;

        bc_hat = unit_vector(ac - ab);
        cd_hat = unit_vector(ad - ac);

        //Compute inverses of some arcs, these are subject to be omitted if the equations are adapted appropriately with inversion of signs.
        ba_hat = -ab_hat;
        mb_hat = -bm_hat;
        pa_hat = -ap_hat;
        pb_hat = -bp_hat;
        
        uint8_t f_r = face_right[a * 3 + j] - 5;
        uint8_t f_l = face_right[a * 3 + (2 + j)%3] - 5;

        uint8_t face_sum = face_right[a * 3] - 5 + face_right[a * 3 + 1] - 5 + face_right[a * 3 + 2] - 5;
        uint8_t dihedral_face_sum = face_right[neighbours[a*3 + j] * 3]-5 + face_right[neighbours[a*3 + j] * 3 + 1]-5 +  face_right[neighbours[a*3 + j] * 3 + 2]-5;
        uint8_t dihedral_index_a = face_index(f_l,face_right[a * 3 + (1 + j)%3] - 5,f_r);
        uint8_t dihedral_index_m =  face_index(face_right[a * 3 + (1 + j)%3] - 5, f_r, f_l);
        uint8_t dihedral_index_p = face_index(f_r,f_l, face_right[a * 3 + (1 + j)%3] - 5);

        outer_dih0_a = optimal_dih_cos_angles[dihedral_index_a];
        outer_dih0_m = optimal_dih_cos_angles[dihedral_index_m];
        outer_dih0_p = optimal_dih_cos_angles[dihedral_index_p];
        //Load equillibirium distance, angles and dihedral angles from face information.
        r0 = optimal_bond_lengths[ f_l + f_r ];
        angle0 = optimal_corner_cos_angles[ f_r ];
        inner_dih0 = optimal_dih_cos_angles[ face_sum ];
        outer_angle_m0 = optimal_corner_cos_angles[ f_l ];
        outer_angle_p0 = optimal_corner_cos_angles[ f_r ];


        //Load force constants from neighbouring face information.
        f_bond = bond_forces[ f_l + f_r ];
        f_inner_angle = angle_forces[ f_l ];
        f_inner_dihedral = dih_forces[ face_sum];
        f_outer_angle_m = angle_forces[ f_r ];
        f_outer_angle_p = angle_forces[ f_l ];
        f_outer_dihedral = dih_forces[ dihedral_face_sum];
    }
    __device__ real_t harmonic_energy(const real_t p0, const real_t p) const{
        return (real_t)0.5*(p-p0)*(p-p0);
    }
    __device__ __forceinline__ coord3d  harmonic_energy_gradient(const real_t p0, const real_t p, const coord3d gradp) const{
        return (p-p0)*gradp;     
    }

    __device__ real_t bond() const {return (real_t)1.0/r_rab;}
    __device__ real_t angle() const {return dot(ab_hat,ac_hat);}
    //Returns the inner dihedral angle for the current arc. Used here only for energy calculation, 
    //otherwise embedded in dihedral computation because the planes and angles that make up the dihedral angle computation are required for derivative computation.
    __device__ __forceinline__ real_t dihedral() const 
    { 
        coord3d nabc, nbcd; real_t cos_b, cos_c, r_sin_b, r_sin_c;
        cos_b = dot(ba_hat,bc_hat); r_sin_b = rsqrt((real_t)1.0 - cos_b*cos_b); nabc = cross(ba_hat, bc_hat) * r_sin_b;
        cos_c = dot(-bc_hat,cd_hat); r_sin_c = rsqrt((real_t)1.0 - cos_c*cos_c); nbcd = cross(-bc_hat,cd_hat) * r_sin_c;
        return dot(nabc, nbcd);
    }
    
    // Chain rule terms for angle calculation
    //Computes gradient related to bending term. ~24 FLOPs
    __device__ coord3d inner_angle_gradient() const
    {
        real_t cos_angle = angle(); //Inner angle of arcs ab,ac.
        coord3d grad = cos_angle * (ab_hat * r_rab + ac_hat * r_rac) - ab_hat * r_rac - ac_hat* r_rab; //Derivative of inner angle: Eq. 21. 
        return f_inner_angle * harmonic_energy_gradient(angle0, cos_angle, grad); //Harmonic Energy Gradient: Eq. 21. multiplied by harmonic term.
    }
    //Computes gradient related to bending of outer angles. ~20 FLOPs
    __device__ coord3d outer_angle_gradient_m() const
    {
        real_t cos_angle = -dot(ab_hat, bm_hat); //Compute outer angle. ab,bm
        coord3d grad = (bm_hat + ab_hat * cos_angle) * r_rab; //Derivative of outer angles Eq. 30. Buster Thesis
        return f_outer_angle_m * harmonic_energy_gradient(outer_angle_m0,cos_angle,grad); //Harmonic Energy Gradient: Eq. 30 multiplied by harmonic term.
    }
    __device__ coord3d outer_angle_gradient_p() const
    {
        real_t cos_angle = -dot(ab_hat, bp_hat); //Compute outer angle. ab,bp
        coord3d grad = (bp_hat + ab_hat * cos_angle) * r_rab; //Derivative of outer angles Eq. 28. Buster Thesis
        return f_outer_angle_p * harmonic_energy_gradient(outer_angle_p0,cos_angle,grad); //Harmonic Energy Gradient: Eq. 28 multiplied by harmonic term.
    }
    // Chain rule terms for dihedral calculation
    //Computes gradient related to dihedral/out-of-plane term. ~75 FLOPs
    __device__ coord3d inner_dihedral_gradient() const
    {
        coord3d nabc, nbcd; real_t cos_b, cos_c, r_sin_b, r_sin_c;
        cos_b = dot(ba_hat,bc_hat); r_sin_b = rsqrtf((real_t)1.0 - cos_b*cos_b); nabc = cross(ba_hat, bc_hat) * r_sin_b;
        cos_c = dot(-bc_hat,cd_hat); r_sin_c = rsqrtf((real_t)1.0 - cos_c*cos_c); nbcd = cross(-bc_hat,cd_hat) * r_sin_c;

        real_t cos_beta = dot(nabc, nbcd); //Inner dihedral angle from planes abc,bcd.
        
        real_t cot_b = cos_b * r_sin_b * r_sin_b; //cos(b)/sin(b)^2

        //Derivative w.r.t. inner dihedral angle F and G in Eq. 26
        coord3d grad = cross(bc_hat, nbcd) * r_sin_b * r_rab - ba_hat * cos_beta * r_rab + (cot_b * cos_beta * r_rab) * (bc_hat - ba_hat * cos_b);

        return f_inner_dihedral * harmonic_energy_gradient(inner_dih0, cos_beta, grad); //Eq. 26.
    }

    //Computes gradient from dihedral angles constituted by the planes bam, amp ~162 FLOPs
    __device__ coord3d outer_a_dihedral_gradient() const
    {
        coord3d nbam_hat, namp_hat; real_t cos_a, cos_m, r_sin_a, r_sin_m;

        cos_a = dot(ab_hat,am_hat); r_sin_a = rsqrtf((real_t)1.0 - cos_a*cos_a); nbam_hat = cross(ab_hat,am_hat) * r_sin_a;
        cos_m = dot(-am_hat,mp_hat); r_sin_m = rsqrtf((real_t)1.0 - cos_m*cos_m); namp_hat = cross(-am_hat,mp_hat) * r_sin_m;
        
        real_t cos_beta = dot(nbam_hat, namp_hat); //Outer Dihedral angle bam, amp
        real_t cot_a = cos_a * r_sin_a * r_sin_a;
        real_t cot_m = cos_m * r_sin_m * r_sin_m;

        //Derivative w.r.t. outer dihedral angle, factorized version of Eq. 31.
        coord3d grad = cross(mp_hat,nbam_hat)*r_ram*r_sin_m - (cross(namp_hat,ab_hat)*r_ram + cross(am_hat,namp_hat)*r_rab)*r_sin_a +
                        cos_beta*(ab_hat*r_rab + r_ram * ((real_t)2.0*am_hat + cot_m*(mp_hat+cos_m*am_hat)) - cot_a*(r_ram*(ab_hat - am_hat*cos_a) + r_rab*(am_hat-ab_hat*cos_a)));
        
        //Eq. 31 multiplied by harmonic term.
        return f_outer_dihedral * harmonic_energy_gradient(outer_dih0_a, cos_beta, grad);
    }

    //Computes gradient from dihedral angles constituted by the planes nbmp, nmpa ~92 FLOPs
    __device__ coord3d outer_m_dihedral_gradient() const
    {
        coord3d nbmp_hat, nmpa_hat; real_t cos_m, cos_p, r_sin_m, r_sin_p;
        cos_m = dot(mb_hat,mp_hat); r_sin_m = rsqrtf((real_t)1.0 - cos_m*cos_m); nbmp_hat = cross(mb_hat,mp_hat) * r_sin_m;
        cos_p = dot(-mp_hat,pa_hat); r_sin_p = rsqrtf((real_t)1.0 - cos_p*cos_p); nmpa_hat = cross(-mp_hat,pa_hat) * r_sin_p;
        
        //Cosine to the outer dihedral angle constituted by the planes bmp and mpa
        real_t cos_beta = dot(nbmp_hat, nmpa_hat); //Outer dihedral angle bmp,mpa.
        real_t cot_p = cos_p * r_sin_p * r_sin_p;
        
        //Derivative w.r.t. outer dihedral angle, factorized version of Eq. 32.
        coord3d grad = r_rap * (cot_p*cos_beta * (-mp_hat - pa_hat*cos_p) - cross(nbmp_hat, mp_hat)*r_sin_p - pa_hat*cos_beta );

        //Eq. 32 multiplied by harmonic term.
        return f_outer_dihedral * harmonic_energy_gradient(outer_dih0_m, cos_beta, grad);
    }

    //Computes gradient from dihedral angles constituted by the planes bpa, pam ~162 FLOPs
    __device__ coord3d outer_p_dihedral_gradient() const
    {
        coord3d nbpa_hat, npam_hat; real_t cos_p, cos_a, r_sin_p, r_sin_a;
        cos_a = dot(ap_hat,am_hat); r_sin_a = rsqrtf((real_t)1.0 - cos_a*cos_a); npam_hat = cross(ap_hat,am_hat) * r_sin_a;
        cos_p = dot(pb_hat,-ap_hat); r_sin_p = rsqrtf((real_t)1.0 - cos_p*cos_p); nbpa_hat = cross(pb_hat,-ap_hat) * r_sin_p;

        real_t cos_beta = dot(nbpa_hat, npam_hat); //Outer dihedral angle bpa, pam.
        real_t cot_p = cos_p * r_sin_p * r_sin_p;
        real_t cot_a = cos_a * r_sin_a * r_sin_a;

        //Derivative w.r.t. outer dihedral angle, factorized version of Eq. 33.
        coord3d grad = cross(npam_hat,pb_hat)*r_rap*r_sin_p - (cross(am_hat,nbpa_hat)*r_rap + cross(nbpa_hat,ap_hat)*r_ram)*r_sin_a +
                        cos_beta*(am_hat*r_ram + r_rap * ((real_t)2.0*ap_hat + cot_p*(pb_hat+cos_p*ap_hat)) - cot_a*(r_rap*(am_hat - ap_hat*cos_a) + r_ram*(ap_hat-am_hat*cos_a)));
        
        //Eq. 33 multiplied by harmonic term.
        return f_outer_dihedral * harmonic_energy_gradient(outer_dih0_p, cos_beta, grad);
    }
    // Internal coordinate gradients
    __device__ coord3d bond_length_gradient() const { return - f_bond * harmonic_energy_gradient(r0,bond(),ab_hat);}
    //Sum of angular gradient components.
    __device__ coord3d angle_gradient() const { return inner_angle_gradient() + outer_angle_gradient_p() + outer_angle_gradient_m();}
    //Sum of inner and outer dihedral gradient components.
    __device__ coord3d dihedral_gradient() const { return inner_dihedral_gradient() + outer_a_dihedral_gradient() + outer_m_dihedral_gradient() + outer_p_dihedral_gradient();}
    //coord3d flatness()             const { return ;  }   
    

    //Harmonic energy contribution from bond stretching, angular bending and dihedral angle bending.
    __device__ __forceinline__ real_t energy() const {return ((real_t)0.5)*f_bond *harmonic_energy(bond(),r0)+f_inner_angle* harmonic_energy(angle(),angle0)+f_inner_dihedral* harmonic_energy(dihedral(),inner_dih0);}
    //Sum of bond, angular and dihedral gradient components.
    __device__ coord3d gradient() const{ return bond_length_gradient()+ angle_gradient() + dihedral_gradient();}


    //Force constants for all paremeters.
    real_t  
        f_outer_dihedral,
        f_inner_dihedral,
        f_inner_angle,
        f_outer_angle_p,
        f_outer_angle_m,
        f_bond;
    
    //Residual lengths of arcs ab, ac, am, ap.
    real_t
        r_rab,
        r_rac,
        r_rad,
        r_ram,
        r_rap;

    //Equillibrium parameters.
    real_t
        r0,
        angle0,
        outer_angle_m0,
        outer_angle_p0,
        inner_dih0,
        outer_dih0_a,
        outer_dih0_m,
        outer_dih0_p;

    //Base Arcs,
    coord3d
        ab,
        ac,
        ad;

    /*
    All normalized arcs required to perform energy & gradient calculations.
    Note that all these arcs are cyclical the arc ab becomes: ab->ac->ad,  the arc ac becomes: ac->ad->ab , the arc bc becomes: bc->cd->db (For iterations 0, 1, 2)
    As such the naming convention here is related to the arcs as they are used in the 0th iteration. */
    coord3d 
        ab_hat,
        ac_hat,
        ad_hat,
        bp_hat,
        bm_hat,
        am_hat,
        ap_hat,
        ba_hat,
        bc_hat,
        cd_hat,
        mp_hat,
        mb_hat,
        pa_hat,
        pb_hat;
};

__device__ coord3d gradient(const coord3d* X, const node_t node_id, const BookkeepingData &dat) {
    coord3d grad = {0.0, 0.0, 0.0};
    for (uint8_t j = 0; j < 3; j++ ){
        ArcData arc = ArcData(node_id, j, dat.neighbours, X, dat.face_right, dat.next_on_face, dat.prev_on_face);
        grad += arc.gradient();
    }
    return grad;
}

__device__ real_t energy(const coord3d* X, const node_t node_id, const BookkeepingData &dat, real_t* reduction_array, node_t N) {
    real_t node_energy = (real_t)0.0;

    for (uint8_t j = 0; j < 3; j++ ){
        ArcData arc = ArcData(node_id, j, dat.neighbours, X, dat.face_right, dat.next_on_face, dat.prev_on_face);
        node_energy += arc.energy();
    }

    cg::sync(cg::this_thread_block());
    reduction_array[threadIdx.x] = node_energy;
    reduction(reduction_array, N);
    return reduction_array[0];
}

__device__ void golden_section_search(coord3d* X, coord3d* direction, coord3d* new_direction,coord3d* X1, coord3d* X2, real_t* reduction_array, real_t a, real_t b, const node_t node_id, const node_t N, const BookkeepingData &dat){
    real_t tau = (sqrtf(5) - 1) / 2;
    cg::thread_block block = cg::this_thread_block();
    //Actual coordinates resulting from each traversal 
    //Line search x - values;
    real_t x1,  x2, dfc;
    x1 = (a + (1 - tau) * (b - a));
    x2 = (a + tau * (b - a));

    X1[node_id] = X[node_id] + x1 * direction[node_id];
    X2[node_id] = X[node_id] + x2 * direction[node_id];
    cg::sync(block);

    real_t f1 = energy(X1, node_id, dat, reduction_array, N);
    real_t f2 = energy(X2, node_id, dat, reduction_array, N);

    for (uint8_t i = 0; i < 20; i++){
        if (f1 > f2){
            a = x1;
            x1 = x2;
            f1 = f2;
            x2 = a + tau * (b - a);
            cg::sync(block);
            X2[node_id] = X[node_id] + x2 * direction[node_id];
            cg::sync(block);
            f2 = energy(X2, node_id, dat, reduction_array, N);
        }else
        {
            b = x2;
            x2 = x1;
            f2 = f1;
            x1 = a + (1 - tau) * (b - a);
            cg::sync(block);
            X1[node_id] = X[node_id] + x1 * direction[node_id];
            cg::sync(block);
            f1 = energy(X1, node_id, dat, reduction_array, N);
        }
    }
    //Line search coefficient
    real_t alfa = (a+b)/2;
    cg::sync(block);
    X1[node_id] = X[node_id] + alfa*direction[node_id];
    cg::sync(block);
    new_direction[node_id] = -gradient(X1,node_id,dat);
}

__global__ void conjugate_gradient(coord3d* d_X, coord3d* d_X_temp, coord3d* d_X1, coord3d* d_X2, coord3d* d_delta_x0, coord3d* d_delta_x1, coord3d* d_direction, node_t* d_neighbours, node_t* d_next_on_face, node_t* d_prev_on_face, uint8_t* d_face_right, size_t N){
    extern __shared__ real_t reduction_array[];

    size_t iter_count = 0;
    size_t max_iter = N*3;
    real_t beta = 0.0;
    real_t dnorm = 0;
    real_t r0_norm;
    real_t direction_norm = 0.0;
    size_t gradient_evals = 0;
    size_t energy_evals = 0;
    size_t node_id = threadIdx.x;

    size_t offset = blockIdx.x * blockDim.x;
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    cg::thread_block block = cg::this_thread_block();
    cg::grid_group grid = cg::this_grid();
    
    coord3d* X = &d_X[offset];
    coord3d* X_temp = &d_X_temp[offset];
    coord3d* X1 = &d_X1[offset];
    coord3d* X2 = &d_X2[offset];
    coord3d* delta_x0 = &d_delta_x0[offset];
    coord3d* delta_x1 = &d_delta_x1[offset];
    coord3d* direction = &d_direction[offset];
    
    const node_t* neighbours = &d_neighbours[3*offset];
    const uint8_t* face_right = &d_face_right[3*offset];
    const node_t* next_on_face = &d_next_on_face[3*offset];
    const node_t* prev_on_face = &d_prev_on_face[3*offset];

    BookkeepingData local_bookkeeping = BookkeepingData(neighbours,face_right,next_on_face,prev_on_face);   
    direction[node_id] = gradient(X, node_id ,local_bookkeeping);
    
    gradient_evals ++;
    
    
    reduction_array[threadIdx.x] = dot(direction[node_id],direction[node_id]);
    reduction(reduction_array, N);
    dnorm = sqrtf(reduction_array[0]);
    direction[node_id] = -direction[node_id]/dnorm;
    
    X_temp[node_id] = X[node_id];
    delta_x0[node_id] = direction[node_id];

    for (node_t i = 0; i < max_iter; i++)
    {   
        beta = 0.0; direction_norm = 0.0; dnorm=0.0; r0_norm = 0.0;
        cg::sync(grid);
        golden_section_search(X, direction, delta_x1, X_temp, X2,reduction_array, 0, 1, node_id, N, local_bookkeeping);

        gradient_evals++;
        energy_evals += 22;
        //Polak Ribiere method
        reduction_array[threadIdx.x] = dot(delta_x0[node_id], delta_x0[node_id]); reduction(reduction_array, N); r0_norm = reduction_array[0];
        cg::sync(block);
        reduction_array[threadIdx.x] = dot(delta_x1[node_id], (delta_x1[node_id] - delta_x0[node_id])); reduction(reduction_array, N); beta = reduction_array[0] / r0_norm;
        cg::sync(block);
        if (energy(X_temp, node_id, local_bookkeeping, reduction_array, N) > energy(X, node_id, local_bookkeeping, reduction_array, N))
        {   
            X_temp[node_id] =  X[node_id];
            delta_x1[node_id] =  delta_x0[node_id];
            beta = 0.0;
        }
        else
        {   
            X[node_id] = X_temp[node_id];
            delta_x0[node_id] = delta_x1[node_id];
        }
        direction[node_id] = delta_x1[node_id] + beta*direction[node_id];
        

        //Calculate gradient and residual gradient norms..
        cg::sync(block);
        reduction_array[threadIdx.x] = dot(direction[node_id],direction[node_id]); reduction(reduction_array, N); direction_norm = sqrtf(reduction_array[0]);
        cg::sync(block);
        reduction_array[threadIdx.x] = dot(delta_x1[node_id],delta_x1[node_id]);  reduction(reduction_array, N); dnorm = sqrtf(reduction_array[0]);
        cg::sync(block);
        //Normalize gradient.
        direction[node_id] /= direction_norm;
        iter_count++;
    }
}

size_t computeBatchSize(size_t N){
    hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties,0);

    /** Compiling with --maxrregcount=64   is necessary to easily (singular blocks / fullerene) parallelize fullerenes of size 20-1024 !**/
    int fullerenes_per_block;
    
    /** Needs 3 storage arrays for coordinates and 1 for reductions **/
    int sharedMemoryPerBlock = sizeof(coord3d)* 3 * (N + 1) + sizeof(real_t)*N;

    /** Calculates maximum number of resident fullerenes on a single Streaming Multiprocessor, multiply with multi processor count to get total batch size**/
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&fullerenes_per_block, conjugate_gradient, N, sharedMemoryPerBlock);

    return (size_t)(properties.multiProcessorCount*fullerenes_per_block);
}

void OptimizeBatch(real_t* h_X, node_t* h_cubic_neighbours, node_t* h_next_on_face, node_t* h_prev_on_face, uint8_t* h_face_right, const size_t N, const size_t batch_size){
    bool concurrent_kernels = false;
    bool single_block_fullerenes = true;
    dim3 dimBlock = dim3(N, 1, 1);
    dim3 dimGrid = dim3(batch_size, 1, 1);

    size_t* d_N;
    bool* d_single_block_fullerenes;


    coord3d* d_X;
    coord3d* d_X_temp;
    coord3d* d_X1;
    coord3d* d_X2;
    coord3d* d_delta_x0;
    coord3d* d_delta_x1;
    coord3d* d_direction;

    node_t* d_neighbours;
    uint8_t* d_face_right;
    node_t* d_next_on_face;
    node_t* d_prev_on_face;
    real_t* d_gdata;

    hipError_t error;
    error = hipMalloc(&d_X, sizeof(coord3d)*N*batch_size);
    error = hipMalloc(&d_X_temp, sizeof(coord3d)*N*batch_size);
    error = hipMalloc(&d_X1, sizeof(coord3d)*N*batch_size);
    error = hipMalloc(&d_X2, sizeof(coord3d)*N*batch_size);
    error = hipMalloc(&d_delta_x0, sizeof(coord3d)*N*batch_size);
    error = hipMalloc(&d_delta_x1, sizeof(coord3d)*N*batch_size);
    error = hipMalloc(&d_direction, sizeof(coord3d)*N*batch_size);
    
    error = hipMalloc(&d_neighbours, sizeof(node_t)*3*N*batch_size);
    error = hipMalloc(&d_next_on_face, sizeof(node_t)*3*N*batch_size);
    error = hipMalloc(&d_prev_on_face, sizeof(node_t)*3*N*batch_size);
    error = hipMalloc(&d_face_right, sizeof(uint8_t)*3*N*batch_size);
    error = hipMalloc(&d_gdata, sizeof(real_t)*dimGrid.x);
    error = hipMalloc(&d_N, sizeof(size_t)); hipMemcpy(d_N, &N, sizeof(size_t), hipMemcpyHostToDevice);
    error = hipMalloc(&d_single_block_fullerenes, sizeof(bool)); hipMemcpy(d_single_block_fullerenes, &single_block_fullerenes, sizeof(bool), hipMemcpyHostToDevice);

    error = hipMemcpy(d_X, h_X, sizeof(coord3d)*N*batch_size , hipMemcpyHostToDevice);
    error = hipMemcpy(d_neighbours, h_cubic_neighbours, sizeof(node_t)*3*N*batch_size, hipMemcpyHostToDevice);
    error = hipMemcpy(d_next_on_face, h_next_on_face, sizeof(node_t)*3*N*batch_size, hipMemcpyHostToDevice);
    error = hipMemcpy(d_prev_on_face, h_prev_on_face, sizeof(node_t)*3*N*batch_size, hipMemcpyHostToDevice);
    error = hipMemcpy(d_face_right, h_face_right, sizeof(uint8_t)*3*N*batch_size, hipMemcpyHostToDevice);

    auto start = std::chrono::system_clock::now();

    if (!concurrent_kernels)
    {
        void* kernelArgs[] = {
        (void*)&d_X,
        (void*)&d_X_temp,
        (void*)&d_X1,
        (void*)&d_X2,
        (void*)&d_delta_x0, 
        (void*)&d_delta_x1,
        (void*)&d_direction,
        (void*)&d_neighbours,
        (void*)&d_next_on_face,
        (void*)&d_prev_on_face,
        (void*)&d_face_right,
        (void*)&N,
        };
        hipLaunchCooperativeKernel((void*)conjugate_gradient, dimGrid, dimBlock, kernelArgs, sizeof(coord3d)*3*(N+1) + sizeof(real_t)*N, NULL);
    } 
    hipDeviceSynchronize();
    auto end = std::chrono::system_clock::now();
    std::cout << "Elapsed time: " << (end-start)/ 1ms << "ms\n" ;
    std::cout << "Estimated Performance " << ((real_t)(411*N*batch_size*3*N*22  + 2106*N*batch_size*3*N)/(std::chrono::duration_cast<std::chrono::microseconds>(end-start)).count()) * 1.0e6 << "FLOP/s \n";


    hipMemcpy(h_X, d_X, sizeof(coord3d)*batch_size*N, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    hipFree(d_X); hipFree(d_X2); hipFree(d_neighbours); hipFree(d_next_on_face); hipFree(d_prev_on_face);
    hipFree(d_X_temp); hipFree(d_face_right); hipFree(d_gdata); hipFree(d_delta_x0); hipFree(d_delta_x1); hipFree(d_direction);
}

};


#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <cooperative_groups/scan.h>
#include "fullerenes/gpu/kernels.hh"
namespace gpu_kernels{
namespace isomerspace_tutte{
#include "device_includes.cu"

template hipError_t tutte_layout<GPU, float, uint16_t>(IsomerBatch<GPU>& B, const size_t max_iterations, const LaunchCtx& ctx, const LaunchPolicy policy);
template hipError_t tutte_layout<GPU, double, uint16_t>(IsomerBatch<GPU>& B, const size_t max_iterations, const LaunchCtx& ctx, const LaunchPolicy policy);

//WIP: Lets try to find some pentagon-pentagon distances
template <Device U> __device__
std::array<device_node_t, 12> multiple_source_shortest_paths(const IsomerBatch<U>& B, device_node_t* distances, const size_t isomer_idx){
   /*  DEVICE_TYPEDEFS;
    
    DeviceCubicGraph FG = DeviceCubicGraph(&B.cubic_neighbours[isomer_idx*blockDim.x*3]);
    node_t outer_face[6]; memset(outer_face, 0, sizeof(node_t)*6); //Do not rely on uninitialized memory it will only be zero on first touch.
    uint8_t Nface = FG.get_face_oriented(0, FG.cubic_neighbours[0],outer_face);
    distances[threadIdx.x] = node_t(NODE_MAX);    
    BLOCK_SYNC
    if (threadIdx.x < Nface)  distances[outer_face[threadIdx.x]] = 0;
    BLOCK_SYNC
    if (threadIdx.x == 0){
        CuDeque<node_t> queue = CuDeque<node_t>(distances + blockDim.x, blockDim.x);
        for (size_t i = 0; i < Nface; i++) queue.push_back(outer_face[i]);
        while (!queue.empty())
        {   
            node_t v = queue.pop_front();
            for (size_t i = 0; i < 3; i++)
            {   
                node_t w = FG.cubic_neighbours[v*3 + i];
                if(distances[w] == NODE_MAX) {
                distances[w] = distances[v]+1;
                queue.push_back(w);
                }
            }
        }
    }
    BLOCK_SYNC
    device_node_t distance = distances[threadIdx.x];
    BLOCK_SYNC
    return distance; */
}


template <Device U, typename T, typename K> __global__
void tutte_layout_(IsomerBatch<U> B, const size_t iterations){
    TEMPLATE_TYPEDEFS(T,K);
    SMEM(T);

    clear_cache(smem, Block_Size_Pow_2);
    for (int isomer_idx = blockIdx.x; isomer_idx < B.isomer_capacity; isomer_idx+= gridDim.x){
    if (B.statuses[isomer_idx] != IsomerStatus::EMPTY){
    size_t offset = isomer_idx * blockDim.x;

    DeviceCubicGraph FG(&B.cubic_neighbours[offset*3]); 
    real_t* base_pointer        = smem + Block_Size_Pow_2;
    coord2d* xys        = reinterpret_cast<coord2d*>(base_pointer);
    coord2d* newxys     = reinterpret_cast<coord2d*>(base_pointer) + blockDim.x;


    node3 ns            = (reinterpret_cast<node3*>(B.cubic_neighbours) + offset)[threadIdx.x];

    xys[threadIdx.x]    = {real_t(0.0), real_t(0.0)};

    node_t outer_face[6];
    node_t outer_face_vertex   = 0;
    uint8_t Nface = FG.get_face_oriented(0,FG.cubic_neighbours[0], outer_face);    
    /* node_t* int_smem = reinterpret_cast<node_t*>(smem);
    uint8_t local_fsize = FG.face_size(threadIdx.x, FG.cubic_neighbours[threadIdx.x*3]);
    node_t is_pentagon = node_t(local_fsize == uint8_t(5) ? 1 : 0);
    ex_scan(int_smem, is_pentagon, blockDim.x);
    if(threadIdx.x > 0 && (int_smem[threadIdx.x] == node_t(1)) && (int_smem[threadIdx.x - 1] == node_t(0))) int_smem[0] = node_t(threadIdx.x - 1);
    BLOCK_SYNC;
    node_t lowest_index_pentagon = int_smem[0];
    uint8_t Nface = FG.get_face_oriented(lowest_index_pentagon, FG.cubic_neighbours[lowest_index_pentagon*3],outer_face);
    BLOCK_SYNC; */ //This code finds the first pentagon and then uses that as the outer face. 
    reinterpret_cast<bool*>(smem)[threadIdx.x] =  false; BLOCK_SYNC;
    if(threadIdx.x < Nface){
      outer_face_vertex = outer_face[threadIdx.x];
      reinterpret_cast<bool*>(smem)[outer_face_vertex] =  true; 
    }
    BLOCK_SYNC;
    bool fixed = reinterpret_cast<bool*>(smem)[threadIdx.x];

    if(threadIdx.x < Nface) xys[outer_face_vertex] = {SIN((real_t)threadIdx.x*(real_t)2.0*real_t(M_PI)/real_t(Nface)),COS((real_t)threadIdx.x*(real_t)2.0*real_t(M_PI)/real_t(Nface))};
    BLOCK_SYNC
    bool converged          = false;
    real_t max_change       = real_t(0.0);
    if(fixed) newxys[threadIdx.x] = xys[threadIdx.x];
    for (size_t i = 0; i < iterations && !converged; i++)
    {   
        max_change = real_t(0.0);
        BLOCK_SYNC
        coord2d neighbour_sum   = {real_t(0.0),real_t(0.0)};    
        for (uint8_t j = 0; j < 3; j++) neighbour_sum += xys[d_get(ns,j)];

        // Calculate the new position of the point
        if(!fixed) newxys[threadIdx.x] = xys[threadIdx.x]*real_t(0.15) + (neighbour_sum/real_t(3.))*real_t(0.85);
        real_t neighbour_dist = 0.0f;

        // Calculate the distance between neighbours
        for (uint8_t j = 0; j < 3; j++) neighbour_dist += norm(xys[threadIdx.x] - xys[d_get(ns,j)])/real_t(3);
        
        BLOCK_SYNC
        real_t relative_change = 0.0f;

        // Calculate the relative change
        if (neighbour_dist > (real_t)0.0f && !fixed){ 
            relative_change = norm(xys[threadIdx.x] - newxys[threadIdx.x])/neighbour_dist;
        }

        // Reduce the relative change to find the maximum change
        real_t iteration_max = reduction_max(smem, relative_change);
        if (iteration_max > max_change) max_change = iteration_max;

        converged = max_change <= 100*numeric_limits<real_t>::epsilon();

        // Update the position of the point
        xys[threadIdx.x] = newxys[threadIdx.x];
    }
    BLOCK_SYNC;
    (reinterpret_cast<std::array<real_t,2>*>(B.xys) + offset )[threadIdx.x]  =  xys[threadIdx.x];
    }
    }
}

float kernel_time = 0.0;
std::chrono::microseconds time_spent(){
    return std::chrono::microseconds((int) (kernel_time*1000.f));
}

void reset_time(){
    kernel_time = 0.0;
}

template <Device U, typename T, typename K>
hipError_t tutte_layout(IsomerBatch<U>& B, const size_t max_iterations, const LaunchCtx& ctx, const LaunchPolicy policy){
    TEMPLATE_TYPEDEFS(T,K);

    hipSetDevice(B.get_device_id());
    static std::vector<bool> first_call(16, true);
    static hipEvent_t start[16], stop[16];
    float single_kernel_time = 0.0;
    auto dev = B.get_device_id();
    if(first_call[dev]) {hipEventCreate(&start[dev]); hipEventCreate(&stop[dev]);}

    //If launch ploicy is synchronous then wait.
    if(policy == LaunchPolicy::SYNC){ ctx.wait();}
    else if(policy == LaunchPolicy::ASYNC && !first_call[dev]){
        //Records time from previous kernel call
        hipEventElapsedTime(&single_kernel_time, start[dev], stop[dev]);
        kernel_time += single_kernel_time;
    }
    size_t smem = sizeof(coord2d)*B.n_atoms*2 + sizeof(real_t)*Block_Size_Pow_2;
    static LaunchDims dims((void*)tutte_layout_<U,T,K>, B.n_atoms, smem, B.isomer_capacity);
    dims.update_dims((void*)tutte_layout_<U,T,K>, B.n_atoms, smem, B.isomer_capacity);
    void* kargs[]{(void*)&B,(void*)&max_iterations};

    hipEventRecord(start[dev], ctx.stream);
    hipError_t error = safeCudaKernelCall((void*)tutte_layout_<U,T,K>, dims.get_grid(), dims.get_block(), kargs, smem, ctx.stream);  
    hipEventRecord(stop[dev], ctx.stream);
    
    if(policy == LaunchPolicy::SYNC) {
        ctx.wait();
        hipEventElapsedTime(&single_kernel_time, start[dev], stop[dev]);
        kernel_time += single_kernel_time;
    }
    printLastCudaError("Tutte: ");
    first_call[dev] = false;
    return error;
}

}}

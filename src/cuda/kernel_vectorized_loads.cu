#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include ""
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <fstream>
#include <chrono>
#include "C43520ih.cu"
#include "coord3d.cu"
#include "helper_functions.cu"

using namespace std::literals;
namespace cg = cooperative_groups;

typedef uint16_t node_t; 

__device__ struct ArcData{
    //124 FLOPs;
    __device__ ArcData(const node_t a, const uint8_t j, const coord3d_a* __restrict__ X, const BookkeepingData& bdat){   
        this->j = j;   

        real_t r_rmp;
        coord3d_a ap, am, ab, ac, ad, mp;
        coord3d_a X_a = X[a]; coord3d_a X_b = X[bdat.neighbours[j]];
        //printf("Index: %d \n", a*3 + j);

        //Compute the arcs ab, ac, ad, bp, bm, ap, am, mp, bc and cd
        ab = (X_b - X_a);  r_rab = bond_length(ab); ab_hat = r_rab * ab;
        ac = (X[bdat.neighbours[(j+1)%3]] - X_a); r_rac = bond_length(ac); ac_hat = r_rac * ac;
        ad = (X[bdat.neighbours[(j+2)%3]] - X_a); r_rad = bond_length(ad); ad_hat = r_rad * ad;
        
        coord3d_a bp = (X[bdat.next_on_face[j]] - X_b); bp_hat = unit_vector(bp);
        coord3d_a bm = (X[bdat.prev_on_face[j]] - X_b); bm_hat = unit_vector(bm);

        ap = bp + ab; r_rap = bond_length(ap); ap_hat = r_rap * ap;
        am = bm + ab; r_ram = bond_length(am); am_hat = r_ram * am;
        mp = bp - bm; r_rmp = bond_length(mp); mp_hat = r_rmp * mp;

        bc_hat = unit_vector(ac - ab);
        cd_hat = unit_vector(ad - ac);

        //Compute inverses of some arcs, these are subject to be omitted if the equations are adapted appropriately with inversion of signs.
        ba_hat = -ab_hat;
        mb_hat = -bm_hat;
        pa_hat = -ap_hat;
        pb_hat = -bp_hat;
    }

    //3 FLOPs
    __device__ real_t harmonic_energy(const real_t p0, const real_t p) const{
        return (real_t)0.5*(p-p0)*(p-p0);
    }
    //4 FLOPs
    __device__ coord3d_a  harmonic_energy_gradient(const real_t p0, const real_t p, const coord3d_a gradp) const{
        return (p-p0)*gradp;     
    }

    //1 FLOP
    __device__ real_t bond() const {return (real_t)1.0/r_rab;}

    //5 FLOPs
    __device__ real_t angle() const {return dot(ab_hat,ac_hat);}

    //Returns the inner dihedral angle for the current arc. Used here only for energy calculation, 
    //otherwise embedded in dihedral computation because the planes and angles that make up the dihedral angle computation are required for derivative computation.
    //50 FLOPs
    __device__ real_t dihedral() const 
    { 
        coord3d_a nabc, nbcd; real_t cos_b, cos_c, r_sin_b, r_sin_c;
        cos_b = dot(ba_hat,bc_hat); r_sin_b = rsqrt((real_t)1.0 - cos_b*cos_b); nabc = cross(ba_hat, bc_hat) * r_sin_b;
        cos_c = dot(-bc_hat,cd_hat); r_sin_c = rsqrt((real_t)1.0 - cos_c*cos_c); nbcd = cross(-bc_hat,cd_hat) * r_sin_c;
        return dot(nabc, nbcd);
    }
    
    // Chain rule terms for angle calculation
    //Computes gradient related to bending term. ~24 FLOPs
    __device__ coord3d_a inner_angle_gradient(const Constants& c) const
    {   
        real_t cos_angle = angle(); //Inner angle of arcs ab,ac.
        coord3d_a grad = cos_angle * (ab_hat * r_rab + ac_hat * r_rac) - ab_hat * r_rac - ac_hat* r_rab; //Derivative of inner angle: Eq. 21. 
        return get(c.f_inner_angle,j) * harmonic_energy_gradient(get(c.angle0,j), cos_angle, grad); //Harmonic Energy Gradient: Eq. 21. multiplied by harmonic term.
    }
    //Computes gradient related to bending of outer angles. ~20 FLOPs
    __device__ coord3d_a outer_angle_gradient_m(const Constants& c) const
    {
        real_t cos_angle = -dot(ab_hat, bm_hat); //Compute outer angle. ab,bm
        coord3d_a grad = (bm_hat + ab_hat * cos_angle) * r_rab; //Derivative of outer angles Eq. 30. Buster Thesis
        return get(c.f_outer_angle_m,j) * harmonic_energy_gradient(get(c.outer_angle_m0,j),cos_angle,grad); //Harmonic Energy Gradient: Eq. 30 multiplied by harmonic term.
    }
    __device__ coord3d_a outer_angle_gradient_p(const Constants& c) const
    {
        real_t cos_angle = -dot(ab_hat, bp_hat); //Compute outer angle. ab,bp
        coord3d_a grad = (bp_hat + ab_hat * cos_angle) * r_rab; //Derivative of outer angles Eq. 28. Buster Thesis
        return get(c.f_outer_angle_p,j) * harmonic_energy_gradient(get(c.outer_angle_p0,j),cos_angle,grad); //Harmonic Energy Gradient: Eq. 28 multiplied by harmonic term.
    }
    // Chain rule terms for dihedral calculation
    //Computes gradient related to dihedral/out-of-plane term. ~75 FLOPs
    __device__ coord3d_a inner_dihedral_gradient(const Constants& c) const
    {
        coord3d_a nabc, nbcd; real_t cos_b, cos_c, r_sin_b, r_sin_c;
        cos_b = dot(ba_hat,bc_hat); r_sin_b = rsqrtf((real_t)1.0 - cos_b*cos_b); nabc = cross(ba_hat, bc_hat) * r_sin_b;
        cos_c = dot(-bc_hat,cd_hat); r_sin_c = rsqrtf((real_t)1.0 - cos_c*cos_c); nbcd = cross(-bc_hat,cd_hat) * r_sin_c;

        real_t cos_beta = dot(nabc, nbcd); //Inner dihedral angle from planes abc,bcd.
        real_t cot_b = cos_b * r_sin_b * r_sin_b; //cos(b)/sin(b)^2

        //Derivative w.r.t. inner dihedral angle F and G in Eq. 26
        coord3d_a grad = cross(bc_hat, nbcd) * r_sin_b * r_rab - ba_hat * cos_beta * r_rab + (cot_b * cos_beta * r_rab) * (bc_hat - ba_hat * cos_b);

        return get(c.f_inner_dihedral,j) * harmonic_energy_gradient(get(c.inner_dih0,j), cos_beta, grad); //Eq. 26.
    }

    //Computes gradient from dihedral angles constituted by the planes bam, amp ~162 FLOPs
    __device__ coord3d_a outer_a_dihedral_gradient(const Constants& c) const
    {
        coord3d_a nbam_hat, namp_hat; real_t cos_a, cos_m, r_sin_a, r_sin_m;

        cos_a = dot(ab_hat,am_hat); r_sin_a = rsqrtf((real_t)1.0 - cos_a*cos_a); nbam_hat = cross(ab_hat,am_hat) * r_sin_a;
        cos_m = dot(-am_hat,mp_hat); r_sin_m = rsqrtf((real_t)1.0 - cos_m*cos_m); namp_hat = cross(-am_hat,mp_hat) * r_sin_m;
        
        real_t cos_beta = dot(nbam_hat, namp_hat); //Outer Dihedral angle bam, amp
        real_t cot_a = cos_a * r_sin_a * r_sin_a;
        real_t cot_m = cos_m * r_sin_m * r_sin_m;

        //Derivative w.r.t. outer dihedral angle, factorized version of Eq. 31.
        coord3d_a grad = cross(mp_hat,nbam_hat)*r_ram*r_sin_m - (cross(namp_hat,ab_hat)*r_ram + cross(am_hat,namp_hat)*r_rab)*r_sin_a +
                        cos_beta*(ab_hat*r_rab + r_ram * ((real_t)2.0*am_hat + cot_m*(mp_hat+cos_m*am_hat)) - cot_a*(r_ram*(ab_hat - am_hat*cos_a) + r_rab*(am_hat-ab_hat*cos_a)));
        
        //Eq. 31 multiplied by harmonic term.
        return get(c.f_outer_dihedral,j) * harmonic_energy_gradient(get(c.outer_dih0,j), cos_beta, grad);
    }

    //Computes gradient from dihedral angles constituted by the planes nbmp, nmpa ~92 FLOPs
    __device__ coord3d_a outer_m_dihedral_gradient(const Constants& c) const
    {
        coord3d_a nbmp_hat, nmpa_hat; real_t cos_m, cos_p, r_sin_m, r_sin_p;
        cos_m = dot(mb_hat,mp_hat); r_sin_m = rsqrtf((real_t)1.0 - cos_m*cos_m); nbmp_hat = cross(mb_hat,mp_hat) * r_sin_m;
        cos_p = dot(-mp_hat,pa_hat); r_sin_p = rsqrtf((real_t)1.0 - cos_p*cos_p); nmpa_hat = cross(-mp_hat,pa_hat) * r_sin_p;
        
        //Cosine to the outer dihedral angle constituted by the planes bmp and mpa
        real_t cos_beta = dot(nbmp_hat, nmpa_hat); //Outer dihedral angle bmp,mpa.
        real_t cot_p = cos_p * r_sin_p * r_sin_p;
        
        //Derivative w.r.t. outer dihedral angle, factorized version of Eq. 32.
        coord3d_a grad = r_rap * (cot_p*cos_beta * (-mp_hat - pa_hat*cos_p) - cross(nbmp_hat, mp_hat)*r_sin_p - pa_hat*cos_beta );

        //Eq. 32 multiplied by harmonic term.
        return get(c.f_outer_dihedral,j) * harmonic_energy_gradient(get(c.outer_dih0,j), cos_beta, grad);
    }

    //Computes gradient from dihedral angles constituted by the planes bpa, pam ~162 FLOPs
    __device__ coord3d_a outer_p_dihedral_gradient(const Constants& c) const
    {
        coord3d_a nbpa_hat, npam_hat; real_t cos_p, cos_a, r_sin_p, r_sin_a;
        cos_a = dot(ap_hat,am_hat); r_sin_a = rsqrtf((real_t)1.0 - cos_a*cos_a); npam_hat = cross(ap_hat,am_hat) * r_sin_a;
        cos_p = dot(pb_hat,-ap_hat); r_sin_p = rsqrtf((real_t)1.0 - cos_p*cos_p); nbpa_hat = cross(pb_hat,-ap_hat) * r_sin_p;

        real_t cos_beta = dot(nbpa_hat, npam_hat); //Outer dihedral angle bpa, pam.
        real_t cot_p = cos_p * r_sin_p * r_sin_p;
        real_t cot_a = cos_a * r_sin_a * r_sin_a;

        //Derivative w.r.t. outer dihedral angle, factorized version of Eq. 33.
        coord3d_a grad = cross(npam_hat,pb_hat)*r_rap*r_sin_p - (cross(am_hat,nbpa_hat)*r_rap + cross(nbpa_hat,ap_hat)*r_ram)*r_sin_a +
                        cos_beta*(am_hat*r_ram + r_rap * ((real_t)2.0*ap_hat + cot_p*(pb_hat+cos_p*ap_hat)) - cot_a*(r_rap*(am_hat - ap_hat*cos_a) + r_ram*(ap_hat-am_hat*cos_a)));
        
        //Eq. 33 multiplied by harmonic term.
        return get(c.f_outer_dihedral,j) * harmonic_energy_gradient(get(c.outer_dih0,j), cos_beta, grad);
    }
    // Internal coordinate gradients
    __device__ coord3d_a bond_length_gradient(const Constants& c) const { return - get(c.f_bond,j) * harmonic_energy_gradient(get(c.r0,j),bond(),ab_hat);}
    //Sum of angular gradient components.
    __device__ coord3d_a angle_gradient(const Constants& c) const { return inner_angle_gradient(c) + outer_angle_gradient_p(c) + outer_angle_gradient_m(c);}
    //Sum of inner and outer dihedral gradient components.
    __device__ coord3d_a dihedral_gradient(const Constants& c) const { return inner_dihedral_gradient(c) + outer_a_dihedral_gradient(c) + outer_m_dihedral_gradient(c) + outer_p_dihedral_gradient(c);}
    //coord3d_a flatness()             const { return ;  }   
    
    //Harmonic energy contribution from bond stretching, angular bending and dihedral angle bending.
    //71 FLOPs
    __device__ real_t energy(const Constants& c) const {return (real_t)0.5 *get(c.f_bond,j) *harmonic_energy(bond(),get(c.r0,j))+ get(c.f_inner_angle,j)* harmonic_energy(angle(),get(c.angle0,j)) + get(c.f_inner_dihedral,j)* harmonic_energy(dihedral(),get(c.inner_dih0,j));}
    //Sum of bond, angular and dihedral gradient components.
    __device__ coord3d_a gradient(const Constants& c) const{return bond_length_gradient(c) + angle_gradient(c) + dihedral_gradient(c);}
    
    uint8_t j;

    //Residual lengths of arcs ab, ac, am, ap.
    real_t
        r_rab,
        r_rac,
        r_rad,
        r_ram,
        r_rap;

    //Base Arcs,
    coord3d_a
        ab,
        ac,
        ad;

    /*
    All normalized arcs required to perform energy & gradient calculations.
    Note that all these arcs are cyclical the arc ab becomes: ab->ac->ad,  the arc ac becomes: ac->ad->ab , the arc bc becomes: bc->cd->db (For iterations 0, 1, 2)
    As such the naming convention here is related to the arcs as they are used in the 0th iteration. */
    coord3d_a 
        ab_hat,
        ac_hat,
        ad_hat,
        bp_hat,
        bm_hat,
        am_hat,
        ap_hat,
        ba_hat,
        bc_hat,
        cd_hat,
        mp_hat,
        mb_hat,
        pa_hat,
        pb_hat;
};

__device__ coord3d_a gradient(const coord3d_a* __restrict__ X, const node_t node_id, const BookkeepingData &dat, const Constants &constants) {
    coord3d_a grad = {0.0, 0.0, 0.0};

    for (uint8_t j = 0; j < 3; j++ ){
        ArcData arc = ArcData::ArcData(node_id, j, X, dat);
        grad += arc.gradient(constants);
    }
    return grad;
}

__device__ real_t energy(const coord3d_a* __restrict__ X, const node_t node_id, const BookkeepingData &dat, const Constants &constants, real_t* __restrict__ reduction_array, real_t* __restrict__ gdata, const node_t N, bool single_block_fullerenes) {
    real_t arc_energy = (real_t)0.0;

    //(71 + 124) * 3 * N  = 585*N FLOPs
    for (uint8_t j = 0; j < 3; j++ ){
        ArcData arc = ArcData::ArcData(node_id, j, X, dat);
        arc_energy += arc.energy(constants);
    }
    cg::sync(cg::this_thread_block());
    reduction_array[threadIdx.x] = arc_energy;
    // (/N // 32) * log2(32) = N//32  * 5 FLOPs 
    reduction(reduction_array,gdata, N, single_block_fullerenes); 
     return reduction_array[0];
}

__device__ void golden_section_search(coord3d_a* __restrict__ X, coord3d_a& direction, coord3d_a& new_direction,coord3d_a* __restrict__ X1, coord3d_a* __restrict__ X2, real_t* __restrict__ reduction_array, real_t* __restrict__ gdata, const node_t node_id, const node_t N, const BookkeepingData& dat, const Constants& constants, cg::thread_group sync_group, bool single_block_fullerenes){
    real_t tau = (sqrtf(5) - 1) / 2;
    //Actual coordinates resulting from each traversal 
    //Line search x - values;
    real_t a = 0.0; real_t b = 1.0;
    real_t x1,  x2, dfc;


    x1 = (a + (1 - tau) * (b - a));
    x2 = (a + tau * (b - a));

    X1[node_id] = X[node_id] + x1 * direction;
    X2[node_id] = X[node_id] + x2 * direction;


    cg::sync(sync_group);

    real_t f1 = energy(X1, node_id, dat, constants, reduction_array, gdata, N, single_block_fullerenes);
    real_t f2 = energy(X2, node_id, dat, constants, reduction_array, gdata, N, single_block_fullerenes);

    for (uint8_t i = 0; i < 20; i++){
        if (f1 > f2){
            a = x1;
            x1 = x2;
            f1 = f2;
            x2 = a + tau * (b - a);
            cg::sync(sync_group);
            X2[node_id] = X[node_id] + x2 * direction;
            cg::sync(sync_group);
            f2 = energy(X2, node_id, dat, constants, reduction_array, gdata, N, single_block_fullerenes);
        }else
        {
            b = x2;
            x2 = x1;
            f2 = f1;
            x1 = a + (1 - tau) * (b - a);
            cg::sync(sync_group);
            X1[node_id] = X[node_id] + x1 * direction;
            cg::sync(sync_group);
            f1 = energy(X1, node_id, dat, constants, reduction_array, gdata, N, single_block_fullerenes);
        }
    }
    //Line search coefficient
    real_t alfa = (a+b)/2;
    cg::sync(sync_group);
    X1[node_id] = X[node_id] + alfa*direction;
    cg::sync(sync_group);
    new_direction = -gradient(X1,node_id,dat, constants);
}

__global__ void conjugate_gradient(coord3d* d_X_in, coord3d_a* d_X, coord3d_a* d_X_temp, coord3d_a* d_X2, const node_t* d_neighbours, const node_t* d_next_on_face, const node_t* d_prev_on_face, const uint8_t* d_face_right, real_t* gdata, const size_t N, const bool single_block_fullerenes){
    extern __shared__ real_t smem[];
    

    cg::grid_group grid = cg::this_grid();
    

    coord3d_a* sX;
    coord3d_a* sX_temp;
    coord3d_a* sX2;

    coord3d_a delta_x0, delta_x1, direction;
    node_t node_id;
    size_t offset;

    size_t iter_count = 0;
    size_t max_iter = N*2.6;
    size_t gradient_evals = 0;
    size_t energy_evals = 0;
    
    real_t beta, dnorm, r0_norm, direction_norm;
    beta = dnorm = r0_norm = direction_norm = 0.0;

    //If fullerenes are localized to individual blocks then use block threadIdx else use grid ID and use 1 grid per fullerene with concurrent launches.
    //If fullerenes are localized to individual blocks then use block size to determine pointer in array, else assume concurrent kernel launches with pointers to individual fullerenes.
    if (single_block_fullerenes){
        node_id = threadIdx.x;
        offset = blockIdx.x * blockDim.x;
    } else
    {
        node_id = blockDim.x * blockIdx.x + threadIdx.x;
        offset = 0;
    }
    
    coord3d* X_in = &d_X_in[offset];
    coord3d_a* X = &d_X[offset];
    coord3d_a* X_temp = &d_X_temp[offset];
    coord3d_a* X2 = &d_X2[offset];
    
    align16(X_in,X,N);

    if (single_block_fullerenes)
    {
        sX =&reinterpret_cast<coord3d_a*>(smem)[(int)ceil(N/4) ];
        sX_temp =&reinterpret_cast<coord3d_a*>(smem)[(int)ceil(N/4) + N];
        sX2 =&reinterpret_cast<coord3d_a*>(smem)[(int)ceil(N/4) +2*N];  
        sX[node_id] = X[node_id];
        sX_temp[node_id] = sX[node_id];

        X = &sX[0]; X_temp = &sX_temp[0]; X2 = &sX2[0];
    } else {
        X_temp[node_id] = X[node_id];
    }

    
    //Pre-compute force constants and store in registers.
    BookkeepingData bookit = BookkeepingData::BookkeepingData(&d_neighbours[3*offset],&d_face_right[3*offset],&d_next_on_face[3*offset],&d_prev_on_face[3*offset]);
    Constants constants = compute_constants(bookit, node_id);

    //Load constant bookkeeping data into registers.
    const node_t neighbours[3] = {d_neighbours[3*(offset+node_id)],d_neighbours[3*(offset+node_id) + 1],d_neighbours[3*(offset+node_id) + 2]};
    const uint8_t face_right[3] = {d_face_right[3*(offset+node_id)],d_face_right[3*(offset+node_id) + 1],d_face_right[3*(offset+node_id) + 2]};;
    const node_t next_on_face[3] = {d_next_on_face[3*(offset+node_id)],d_next_on_face[3*(offset+node_id) + 1],d_next_on_face[3*(offset+node_id) + 2]};
    const node_t prev_on_face[3] = {d_prev_on_face[3*(offset+node_id)],d_prev_on_face[3*(offset+node_id) + 1],d_prev_on_face[3*(offset+node_id) + 2]};
    BookkeepingData bookkeeping = BookkeepingData::BookkeepingData(&neighbours[0],&face_right[0],&next_on_face[0],&prev_on_face[0]);   

    cg::sync(grid);
    direction = gradient(X, node_id ,bookkeeping, constants);
    gradient_evals ++;
    
    smem[threadIdx.x] = dot(direction,direction);

    reduction(smem,gdata,N,single_block_fullerenes);
    dnorm = sqrtf(smem[0]);
    direction = -direction/dnorm;
    

    delta_x0 = direction;
    
    for (size_t i = 0; i < max_iter; i++)
    {   
        beta = 0.0; direction_norm = 0.0; dnorm=0.0; r0_norm = 0.0;
        cg::sync(grid);
        if (single_block_fullerenes){golden_section_search(X, direction, delta_x1, X_temp, X2, smem, gdata, node_id, N, bookkeeping, constants, cg::this_thread_block(), single_block_fullerenes);} 
        else { golden_section_search(X, direction, delta_x1, X_temp, X2, smem, gdata, node_id, N, bookkeeping, constants, cg::this_grid(), single_block_fullerenes);}
        
        
        cg::sync(grid);

        gradient_evals++;
        energy_evals += 42;
        //Polak Ribiere method
        
        smem[threadIdx.x] = dot(delta_x0, delta_x0); reduction(smem,gdata,N,single_block_fullerenes); r0_norm = smem[0];
        cg::sync(grid);
        smem[threadIdx.x] = dot(delta_x1, (delta_x1 - delta_x0)); reduction(smem,gdata,N,single_block_fullerenes); beta = smem[0] / r0_norm;
        cg::sync(grid);
        real_t E1 = energy(X_temp, node_id, bookkeeping, constants, smem, gdata, N, single_block_fullerenes);
        cg::sync(grid);
        real_t E2 = energy(X, node_id, bookkeeping, constants, smem, gdata, N, single_block_fullerenes);
        cg::sync(grid);
        if (E1> E2)
        {   
            cg::sync(grid);
            X_temp[node_id] =  X[node_id];
            delta_x1 =  delta_x0;
            beta = 0.0;
        }
        else
        {   
            cg::sync(grid);
            X[node_id] = X_temp[node_id];
            delta_x0 = delta_x1;
        }
        direction = delta_x1 + beta*direction;
        //Calculate gradient and residual gradient norms..
        cg::sync(grid);
        smem[threadIdx.x] = dot(direction,direction); 
        cg::sync(grid);
        reduction(smem,gdata,N,single_block_fullerenes); 
        cg::sync(grid);
        direction_norm = sqrtf(smem[0]);
        cg::sync(grid);
        smem[threadIdx.x] = dot(delta_x1,delta_x1); 
        cg::sync(grid);
        reduction(smem,gdata,N,single_block_fullerenes);
        cg::sync(grid);
        dnorm = sqrtf(smem[0]);
        cg::sync(grid);
        //Normalize gradient.
        direction /= direction_norm;
        iter_count++;
        
        if (node_id == 0)
        {
            //printf("%e \n", E1);
        }
        
    }
    
    cg::sync(grid);
    real_t test = energy(X_temp, node_id, bookkeeping, constants, smem, gdata, N, single_block_fullerenes);
    cg::sync(grid);
    
    if ((node_id == 0))
    {
        //printf("Energy at end %e \n", test);
        /* code */
    }
}


int main(){

    const size_t N = 43520;
    int maxActiveBlocks;
    size_t sharedMemoryPerBlock = sizeof(coord3d_a)* 3 * (N + 1) + sizeof(real_t)*N;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxActiveBlocks, conjugate_gradient, N, sharedMemoryPerBlock);
    hipDeviceProp_t GPU_properties;
    hipGetDeviceProperties(&GPU_properties,0);
    bool use_L1_cache = (GPU_properties.sharedMemPerBlock > (sharedMemoryPerBlock ) && (maxActiveBlocks > 0) );
    std::cout << use_L1_cache << "\n";
    bool single_block_fullerenes = maxActiveBlocks > 0;
    size_t num_molecules = maxActiveBlocks*GPU_properties.multiProcessorCount;
    dim3 dimBlock, dimGrid;

    const real_t* h_X;
    const node_t* h_neighbours;
    const node_t* h_next_on_face;
    const node_t* h_prev_on_face;
    const uint8_t* h_face_right;

    if (single_block_fullerenes)
    {
        dimBlock = dim3::dim3(N, 1, 1);
        dimGrid = dim3::dim3(num_molecules, 1, 1);
        h_X = reinterpret_cast<real_t*>(synthetic_array<real_t>(N, num_molecules, &X[0]));
        h_neighbours = reinterpret_cast<node_t*>(synthetic_array<node_t>(N, num_molecules, &cubic_neighbours[0]));
        h_next_on_face = reinterpret_cast<node_t*>(synthetic_array<node_t>(N, num_molecules, &next_on_face[0]));
        h_prev_on_face = reinterpret_cast<node_t*>(synthetic_array<node_t>(N, num_molecules, &prev_on_face[0]));
        h_face_right = reinterpret_cast<uint8_t*>(synthetic_array<uint8_t>(N, num_molecules, &face_right[0]));
    } else
    {
        size_t blocksize = optimize_block_size(N,GPU_properties,conjugate_gradient);
        dimBlock = dim3::dim3(blocksize, 1, 1);
        dimGrid = dim3::dim3(ceil(N/blocksize), 1, 1);
        num_molecules = floor((GPU_properties.maxThreadsPerBlock*GPU_properties.multiProcessorCount)/(dimBlock.x*dimGrid.x) );
        std::cout << num_molecules << "\n";
        GPU_properties.asyncEngineCount;
        h_X = &X[0]; 
        h_neighbours = &cubic_neighbours[0];
        h_next_on_face = &next_on_face[0];
        h_prev_on_face = &prev_on_face[0];
        h_face_right = &face_right[0];
        
    }
    if (!use_L1_cache) {sharedMemoryPerBlock = sizeof(real_t)*dimBlock.x*2;}
    std::cout << dimBlock.x << "\n";
    std::cout << dimGrid.x << "\n";


    


    size_t* d_N;
    bool* d_single_block_fullerenes;


    coord3d* d_X_in;
    coord3d_a* d_X;
    coord3d_a* d_X_temp;
    coord3d_a* d_X2;
    coord3d_a* d_delta_x0;
    coord3d_a* d_delta_x1;
    coord3d_a* d_direction;


    node_t* d_neighbours;
    uint8_t* d_face_right;
    node_t* d_next_on_face;
    node_t* d_prev_on_face;
    real_t* d_gdata;

    hipError_t error;
    error = hipMalloc(&d_X_in, sizeof(coord3d)*N*num_molecules);
    error = hipMalloc(&d_X, sizeof(coord3d_a)*N*num_molecules);
    error = hipMalloc(&d_X_temp, sizeof(coord3d_a)*N*num_molecules);
    error = hipMalloc(&d_X2, sizeof(coord3d_a)*N*num_molecules);
    
    error = hipMalloc(&d_neighbours, sizeof(node_t)*3*N*num_molecules);
    error = hipMalloc(&d_next_on_face, sizeof(node_t)*3*N*num_molecules);
    error = hipMalloc(&d_prev_on_face, sizeof(node_t)*3*N*num_molecules);
    error = hipMalloc(&d_face_right, sizeof(uint8_t)*3*N*num_molecules);
    error = hipMalloc(&d_gdata, sizeof(real_t)*dimGrid.x);
    error = hipMalloc(&d_N, sizeof(size_t)); hipMemcpy(d_N, &N, sizeof(size_t), hipMemcpyHostToDevice);
    error = hipMalloc(&d_single_block_fullerenes, sizeof(bool)); hipMemcpy(d_single_block_fullerenes, &single_block_fullerenes, sizeof(bool), hipMemcpyHostToDevice);

    getLastCudaError("One or more Mallocs Failed! \n");

    error = hipMemcpy(d_X_in, h_X, sizeof(coord3d)*N*num_molecules , hipMemcpyHostToDevice);
    error = hipMemcpy(d_neighbours, h_neighbours, sizeof(node_t)*3*N*num_molecules, hipMemcpyHostToDevice);
    error = hipMemcpy(d_next_on_face, h_next_on_face, sizeof(node_t)*3*N*num_molecules, hipMemcpyHostToDevice);
    error = hipMemcpy(d_prev_on_face, h_prev_on_face, sizeof(node_t)*3*N*num_molecules, hipMemcpyHostToDevice);
    error = hipMemcpy(d_face_right, h_face_right, sizeof(uint8_t)*3*N*num_molecules, hipMemcpyHostToDevice);

    getLastCudaError("Memcpy Failed! \n");
    
    BookkeepingData bpointers = BookkeepingData::BookkeepingData(d_neighbours, d_face_right, d_next_on_face, d_prev_on_face);

    void *kernelArgs[] = {
        (void*)&d_X_in,
        (void*)&d_X,
        (void*)&d_X_temp,
        (void*)&d_X2,
        (void*)&d_neighbours,
        (void*)&d_next_on_face,
        (void*)&d_prev_on_face,
        (void*)&d_face_right,
        (void*)&d_gdata,
        (void*)&N,
        (void*)&single_block_fullerenes
    };

    
    
    


    auto start = std::chrono::system_clock::now();
    checkCudaErrors(hipLaunchCooperativeKernel((void*)conjugate_gradient, dimGrid, dimBlock, kernelArgs, sharedMemoryPerBlock, NULL));
    hipDeviceSynchronize();


    printf("Max Number of Blocks / multiprocesser: %d \n", maxActiveBlocks);
    printf("Number of MultiProcessers : %d \n", GPU_properties.multiProcessorCount);
    auto end = std::chrono::system_clock::now();
    std::cout << "Elapsed time: " << (end-start)/ 1ms << "ms\n" ;
    getLastCudaError("Failed to launch kernel: ");

}



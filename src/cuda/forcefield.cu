#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <cooperative_groups/scan.h>
#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include "fullerenes/gpu/cuda_definitions.h"
#include "fullerenes/gpu/cu_array.hh"
#include "fullerenes/gpu/isomer_batch.hh"
#include "fullerenes/gpu/kernels.hh"

namespace gpu_kernels{
namespace isomerspace_forcefield{
#include "device_includes.cu"

// This struct was made to reduce signature cluttering of device functions, it is simply a container for default arguments which are shared between functions
template <ForcefieldType T>
struct ForceField{
    DEVICE_TYPEDEFS
    
    const NodeNeighbours node_graph;         //Contains face-information and neighbour-information. Both of which are constant in the lifespan of this struct. 
    const Constants constants;          //Contains force-constants and equillibrium-parameters. Constant in the lifespan of this struct.

    size_t node_id = threadIdx.x;
    real_t* sdata;                      //Pointer to start of L1 cache array, used exclusively for reduction.

    __device__ ForceField(  const NodeNeighbours &G,
                            const Constants &c, 
                            real_t* sdata): node_graph(G), constants(c), sdata(sdata) {}



struct FaceData{
    coord3d Xa;
    symMat3 A;
    coord3d n_f; //normalized normal vector to face-plane
    real_t lambda_f; //Smallest eigenvalue defining the flatness of the face
    coord3d lambdas;
    coord3d centroid;
    device_node3 face_neighbours;
    //84 + 107 FLOPS
    INLINE FaceData(const coord3d* X, const NodeNeighbours& G){
        face_neighbours = G.face_neighbours;
        Xa = X[threadIdx.x];
        //There are only blockDim.x/2 + 2 faces. (Nf  =  N/2 + 1)
        if(threadIdx.x < blockDim.x/2 + 2){
            coord3d Xf[6] = {X[G.face_nodes[0]], X[G.face_nodes[1]] , X[G.face_nodes[2]] , X[G.face_nodes[3]] , X[G.face_nodes[4]] };
            //If pentagon set to 0 otherwise get the 6th node coordinates.
            if(G.face_size == 6){Xf[5] = X[G.face_nodes[5]];} else {Xf[5] = {(real_t)0., (real_t)0., (real_t)0.};}
            centroid = (Xf[0] + Xf[1] + Xf[2] + Xf[3] + Xf[4] + Xf[5]) / (device_real_t)G.face_size;
            //Centralise coordinate system to centroid of the face
            Xf[0] -= centroid; Xf[1] -= centroid; Xf[2] -= centroid; Xf[3] -= centroid; Xf[4] -= centroid;  
            if(G.face_size == 6){Xf[5] -= centroid;}
            auto a = Xf[0][0] * Xf[0][0] + Xf[1][0] * Xf[1][0] + Xf[2][0] * Xf[2][0] + Xf[3][0] * Xf[3][0] + Xf[4][0] * Xf[4][0] + Xf[5][0] * Xf[5][0],
                 b = Xf[0][0] * Xf[0][1] + Xf[1][0] * Xf[1][1] + Xf[2][0] * Xf[2][1] + Xf[3][0] * Xf[3][1] + Xf[4][0] * Xf[4][1] + Xf[5][0] * Xf[5][1],
                 c = Xf[0][0] * Xf[0][2] + Xf[1][0] * Xf[1][2] + Xf[2][0] * Xf[2][2] + Xf[3][0] * Xf[3][2] + Xf[4][0] * Xf[4][2] + Xf[5][0] * Xf[5][2],
                 d = Xf[0][1] * Xf[0][1] + Xf[1][1] * Xf[1][1] + Xf[2][1] * Xf[2][1] + Xf[3][1] * Xf[3][1] + Xf[4][1] * Xf[4][1] + Xf[5][1] * Xf[5][1],
                 e = Xf[0][1] * Xf[0][2] + Xf[1][1] * Xf[1][2] + Xf[2][1] * Xf[2][2] + Xf[3][1] * Xf[3][2] + Xf[4][1] * Xf[4][2] + Xf[5][1] * Xf[5][2],
                 f = Xf[0][2] * Xf[0][2] + Xf[1][2] * Xf[1][2] + Xf[2][2] * Xf[2][2] + Xf[3][2] * Xf[3][2] + Xf[4][2] * Xf[4][2] + Xf[5][2] * Xf[5][2];
            //Xf * Xf^T In closed form.
            A = symMat3(a,b,c,d,e,f);

            //A is positive-semi-definite so all eigenvalues are non-negative
            lambdas = A.eigenvalues();
            lambda_f = d_min(d_min(lambdas[0], lambdas[1]), lambdas[2]);  
        }
    }
    //3 FLOPs
        /**
     * Computes the harmonic energy contribution of one term.
     *
     * @param[in] p0 Equillibrium parameter
     * @param[in] p Current parameter
     * @return Hooke's law harmonic energy contribution of the term
     */
    INLINE real_t harmonic_energy(const real_t p0, const real_t p) const{
        return (real_t)0.5*(p-p0)*(p-p0);
    }


    /** @brief Compute the flatness of the threadIdx^th face in the isomer
     *  @return The flatness of the threadIdx^th face in the isomer
     */
    INLINE real_t flatness() const {return threadIdx.x < blockDim.x/2 + 2 ? lambda_f : (real_t)0.;}

    //4 FLOPs
    /**
     * Computes the harmonic energy gradient contribution of one term.
     *
     * @param[in] p0 Equillibrium parameter
     * @param[in] p Current parameter
     * @param[in] gradp Gradient of the parameter w.r.t. the particle position
     * @return Hooke's law harmonic energy gradient contribution of the term
     */
    INLINE coord3d  harmonic_energy_gradient(const real_t p0, const real_t p, const coord3d gradp) const{
        return (p-p0)*gradp;     
    }


    /**
 * @brief Compute the flatness energy contribution of the threadIdx^th face in the isomer.
 *
 * @param c The forcefield constants for the threadIdx^th node.
 * @return The flatness energy.
 */
    INLINE real_t flatness_energy(const Constants& c) const {
        return c.f_flat * harmonic_energy(flatness(),(real_t)0.);
    }

    /**
     * @brief Compute the gradient of the flatness w.r.t to the threadIdx^th atom in the isomer.
     * @param c The forcefield constants for the threadIdx^th node.
     * @param cache A pointer to a cache of minimum size Nf * 2 * sizeof(coord3d) bytes.
     * @return The flatness energy gradient.
     */
    INLINE coord3d flatness_gradient(const Constants& c, coord3d* cache) const {
        coord3d* centroids = reinterpret_cast<coord3d*>(cache);
        coord3d* norms = reinterpret_cast<coord3d*>(cache + blockDim.x/2 + 2);
        if(threadIdx.x < blockDim.x/2 + 2){
            centroids[threadIdx.x] = centroid;
            norms[threadIdx.x] = A.eigenvector(lambda_f);
        }
        BLOCK_SYNC

        coord3d grad = {(real_t)0., (real_t)0., (real_t)0.};
        for(unsigned char j = 0; j < 3; j++) grad += dot(Xa - centroids[d_get(face_neighbours,j)], norms[d_get(face_neighbours,j)]) * norms[d_get(face_neighbours,j)];
        return c.f_flat * (real_t)2. * grad;
    }
};

//Container for all energy and gradient evaluations with respect to an arc, eg. AB, AC or AD.
struct ArcData{
    //124 FLOPs;
    uint8_t j;
    /**
     * @brief Construct a new ArcData object
     * @param j The index of the arc, eg. 0 for ab, 1 for ac and 2 for ad.
     * @param X The coordinates of all nodes in the isomer.
     * @param G The neighbour information for the threadIdx^th node.
     * @return A new ArcData object.
    */
    INLINE ArcData(const uint8_t j, const coord3d* __restrict__ X, const NodeNeighbours& G){  
        __builtin_assume(j < 3); 
        this->j = j;   
        node_t a = threadIdx.x;
        real_t r_rmp;
        coord3d ap, am, ab, ac, ad, mp, db, bc;
        coord3d X_a = X[a], X_b = X[d_get(G.cubic_neighbours,j)];
        //Compute the arcs ab, ac, ad, bp, bm, ap, am, mp, bc and cd
        ab = (X_b - X_a);  r_rab = bond_length(ab); ab_hat = r_rab * ab;
        ac = (X[d_get(G.cubic_neighbours,(j+1)%3)] - X_a); r_rac = bond_length(ac); ac_hat = r_rac * ac; rab = non_resciprocal_bond_length(ab);
        ad = (X[d_get(G.cubic_neighbours,(j+2)%3)] - X_a); r_rad = bond_length(ad); ad_hat = r_rad * ad;
        db = (X_b - X[d_get(G.cubic_neighbours,(j+2)%3)]); r_rdb = bond_length(db); db_hat = r_rdb * db;
        coord3d bp = (X[d_get(G.next_on_face,j)] - X_b); 
        coord3d bm = (X[d_get(G.prev_on_face,j)] - X_b); 
        
        r_rbp = bond_length(bp); bp_hat = bp * r_rbp;
        r_rbm = bond_length(bm); bm_hat = bm * r_rbm;

        ap = bp + ab; r_rap = bond_length(ap); ap_hat = r_rap * ap;
        am = bm + ab; r_ram = bond_length(am); am_hat = r_ram * am;
        mp = bp - bm; r_rmp = bond_length(mp); mp_hat = r_rmp * mp;
        bc = ac - ab; r_rbc = bond_length(bc); bc_hat = r_rbc * bc;
        cd_hat = unit_vector(ad - ac);

        //Compute inverses of some arcs, these are subject to be omitted if the equations are adapted appropriately with inversion of signs.
        ba_hat = -ab_hat;
        mb_hat = -bm_hat;
        pa_hat = -ap_hat;
        pb_hat = -bp_hat;
    }

    //3 FLOPs
    /**
     * @brief Compute the harmonic energy contribution from one parameter.
     * @param p0 The equillibrium value of the parameter.
     * @param p The current value of the parameter.
    */
    INLINE real_t harmonic_energy(const real_t p0, const real_t p) const{
        return (real_t)0.5*(p-p0)*(p-p0);
    }
    //4 FLOPs
    /**
     * @brief Compute the harmonic energy gradient contribution from one parameter.
     * @param p0 The equillibrium value of the parameter.
     * @param p The current value of the parameter.
     * @param gradp The gradient of the parameter with respect to the node position.
    */
    INLINE coord3d  harmonic_energy_gradient(const real_t p0, const real_t p, const coord3d gradp) const{
        return (p-p0)*gradp;     
    }

    INLINE mat3 harmonic_energy_hessian(const real_t p0, const real_t p, const coord3d grad_a, const coord3d grad_b, const mat3& hessp) const{
        return hessp*(p-p0) + tensor_product(grad_a,grad_b);
    }

    //1 FLOP
    /**
     * @brief Compute the bond length of the main arc ab or ac or ad. for j = 0, 1 or 2 respectively.
     * @return The bond length.
    */
    INLINE real_t bond() const {return rab;}
    //5 FLOPs
    /**
     * @brief Compute the cosine of the angle between the main arc and the next arc, (ab,ac), (ac,ad), (ad,ab). For j = 0, 1 or 2 respectively.
     * @return The cosine of the angle.
    */
    INLINE real_t angle() const {return dot(ab_hat,ac_hat);}

    INLINE real_t normalized_angle_err() const {return acos((float)(float)dot(ab_hat,ac_hat));}

    //Returns outer angle m, used only diagnostically.
    INLINE real_t outer_angle_m() const {return -dot(ab_hat, bm_hat);} //Compute outer angle. ab,bm

    //Returns outer angle p, used only diagnostically.
    INLINE real_t outer_angle_p() const{return -dot(ab_hat, bp_hat);} //Compute outer angle. ab,bp

    //Returns the inner dihedral angle for the current arc. Used here only for energy calculation, 
    //otherwise embedded in dihedral computation because the planes and angles that make up the dihedral angle computation are required for derivative computation.
    //50 FLOPs
    /**
     * @brief Compute the dihedral angle between the planes (abc,bcd), (acd,bcd) and (abd,bcd). For j = 0, 1 or 2 respectively. 
     * @return The dihedral angle.
    */
    INLINE real_t dihedral() const 
    { 
        coord3d nabc, nbcd; real_t cos_b, cos_c, r_sin_b, r_sin_c;
        cos_b = dot(ba_hat,bc_hat);  r_sin_b = (real_t)1.0/SQRT((real_t)1.0 - cos_b*cos_b); nabc = cross(ba_hat, bc_hat) * r_sin_b;
        cos_c = dot(-bc_hat,cd_hat); r_sin_c = (real_t)1.0/SQRT((real_t)1.0 - cos_c*cos_c); nbcd = cross(-bc_hat,cd_hat) * r_sin_c;
        return dot(nabc, nbcd);
    }
    //Returns the Outer-dihedral-a wrt. current arc, only accessed diagnostically (internal coordinate).
    /**
     * @brief Compute the dihedral angle between the planes $(b-a-b_m, a-b_m-b_p)$, $(c-a-c_m, a-c_m-c_p)$ and $(d-a-d_m, a-d_m-d_p)$. For j = 0, 1 or 2 respectively.
     * @return The dihedral angle.
    */
    INLINE real_t outer_dihedral_a() const
    {
        coord3d nbam_hat, namp_hat; real_t cos_a, cos_m, r_sin_a, r_sin_m;
        cos_a = dot(ab_hat,am_hat); r_sin_a = (real_t)1.0/SQRT((real_t)1.0 - cos_a*cos_a); nbam_hat = cross(ab_hat,am_hat) * r_sin_a;
        cos_m = dot(-am_hat,mp_hat); r_sin_m = (real_t)1.0/SQRT((real_t)1.0 - cos_m*cos_m); namp_hat = cross(-am_hat,mp_hat) * r_sin_m;
        real_t cos_beta = dot(nbam_hat, namp_hat); //Outer Dihedral angle bam, amp
        return cos_beta;
    }
    //Returns the Outer-dihedral-m wrt. current arc, only accessed diagnostically (internal coordinate).
    /**
     * @brief Compute the dihedral angle between the planes $(b-b_m-b_p, b_m-b_p-a)$, $(c-c_m-c_p, c_m-c_p-a)$ and $(d-d_m-d_p, d_m-d_p-a)$. For j = 0, 1 or 2 respectively.
     * @return The dihedral angle.
    */
    INLINE real_t outer_dihedral_m() const
    {
        coord3d nbmp_hat, nmpa_hat; real_t cos_m, cos_p, r_sin_m, r_sin_p;
        cos_m = dot(mb_hat,mp_hat);  r_sin_m = (real_t)1.0/SQRT((real_t)1.0 - cos_m*cos_m); nbmp_hat = cross(mb_hat,mp_hat) * r_sin_m;
        cos_p = dot(-mp_hat,pa_hat); r_sin_p = (real_t)1.0/SQRT((real_t)1.0 - cos_p*cos_p); nmpa_hat = cross(-mp_hat,pa_hat) * r_sin_p;
        //Cosine to the outer dihedral angle constituted by the planes bmp and mpa
        real_t cos_beta = dot(nbmp_hat, nmpa_hat); //Outer dihedral angle bmp,mpa.
        return cos_beta;    
    }
    //Returns the Outer-dihedral-p wrt. current arc, only accessed diagnostically (internal coordinate).
    /**
     * @brief Compute the dihedral angle between the planes $(b-b_p-a, b_p-a-b_m)$, $(c-c_p-a, c_p-a-c_m)$ and $(d-d_p-a, d_p-a-d_m)$. For j = 0, 1 or 2 respectively.
     * @return The dihedral angle.
    */
    INLINE real_t outer_dihedral_p() const
    {
        coord3d nbpa_hat, npam_hat; real_t cos_p, cos_a, r_sin_p, r_sin_a;
        cos_a = dot(ap_hat,am_hat);  r_sin_a = (real_t)1.0/SQRT((real_t)1.0 - cos_a*cos_a); npam_hat = cross(ap_hat,am_hat)  * r_sin_a;
        cos_p = dot(pb_hat,-ap_hat); r_sin_p = (real_t)1.0/SQRT((real_t)1.0 - cos_p*cos_p); nbpa_hat = cross(pb_hat,-ap_hat) * r_sin_p;
        real_t cos_beta = dot(nbpa_hat, npam_hat); //Outer dihedral angle bpa, pam.
        //Eq. 33 multiplied by harmonic term.
        return cos_beta;
    }
    
    // Chain rule terms for angle calculation
    //Computes gradient related to bending term. ~24 FLOPs
    /**
     * @brief Compute the gradient of the bending term.
     * @param c The constants for the threadIdx^th node.
     * @return The gradient of the bending term.
    */
    INLINE coord3d inner_angle_gradient(const Constants& c) const
    {   
        real_t cos_angle = angle(); //Inner angle of arcs ab,ac.
        coord3d grad = cos_angle * (ab_hat * r_rab + ac_hat * r_rac) - ab_hat * r_rac - ac_hat* r_rab; //Derivative of inner angle: Eq. 21. 
        return d_get(c.f_inner_angle,j) * harmonic_energy_gradient(d_get(c.angle0,j), cos_angle, grad); //Harmonic Energy Gradient: Eq. 21. multiplied by harmonic term.
    }

    INLINE mat3 inner_angle_hessian_a(const Constants& c) const{
        real_t cos_angle = angle(); //Inner angle of arcs ab,ac.
        coord3d grad_a = (ab_hat * angle() - ac_hat) * r_rab + (ac_hat * angle() - ab_hat) * r_rac; 
        
        //TensorProduct[abh, 1/abn * (abh*cost - ach) + 1/acn * (ach*cost-abh)] + cost/abn * (TensorProduct[abh,abh] - IdentityMatrix[3]) - 1/acn * (TensorProduct[ach,ach] - IdentityMatrix[3])
        auto G = tensor_product(ab_hat, r_rab * (ab_hat * angle() - ac_hat) + r_rac * (ac_hat * angle() - ab_hat))
                + angle()*r_rab * (tensor_product(ab_hat, ab_hat) - identity3()) 
                - r_rac * (tensor_product(ac_hat, ac_hat) - identity3());

        //F := TensorProduct[ach, 1/abn * (abh*cost - ach) + 1/acn * (ach*cost-abh)] + cost/acn * (TensorProduct[ach,ach] - IdentityMatrix[3]) - 1/abn * (TensorProduct[abh,abh] - IdentityMatrix[3])
        auto F = tensor_product(ac_hat, r_rab * (ab_hat * angle() - ac_hat) + r_rac * (ac_hat * angle() - ab_hat)) + angle()*r_rac * (tensor_product(ac_hat, ac_hat) - identity3()) - r_rab * (tensor_product(ab_hat, ab_hat) - identity3());

        auto P1 = tensor_product(ab_hat * angle() - ac_hat, ab_hat * r_rab*r_rab);
        auto P2 = r_rab*G;
        auto P3 = tensor_product(ac_hat * angle() - ab_hat, ac_hat * r_rac*r_rac);
        auto P4 = r_rac*F;
        return d_get(c.f_inner_angle,j) * harmonic_energy_hessian(d_get(c.angle0,j), cos_angle, grad_a, grad_a, P1+P2+P3+P4); //Harmonic Energy Hessian
    }

    INLINE mat3 inner_angle_hessian_b(const Constants& c) const{
        coord3d grad_a = (ab_hat * angle() - ac_hat) * r_rab + (ac_hat * angle() - ab_hat) * r_rac; 
        coord3d grad_b = (ac_hat - ab_hat * angle()) * r_rab;
        auto G = tensor_product(ab_hat, r_rab * (ac_hat - ab_hat * angle())) + angle()*r_rab * (identity3() - tensor_product(ab_hat, ab_hat));
        auto F = tensor_product(ac_hat, r_rab * (ac_hat - ab_hat * angle())) - r_rab * (identity3() - tensor_product(ab_hat, ab_hat));
        auto P1 = tensor_product(ab_hat * angle() - ac_hat, -ab_hat * r_rab*r_rab);
        
        auto P2 = r_rab*G;
        auto P4 = r_rac*F;
        return d_get(c.f_inner_angle,j) * harmonic_energy_hessian(d_get(c.angle0,j), angle(), grad_a, grad_b, P1+P2+P4); //Harmonic Energy Hessian
    }

    INLINE mat3 inner_angle_hessian_c(const Constants& c) const{
        auto grad_a = (ab_hat * angle() - ac_hat) * r_rab + (ac_hat * angle() - ab_hat) * r_rac;
        auto grad_c = (ab_hat - angle() * ac_hat) * r_rac;
        auto G = tensor_product(ab_hat, r_rac * (ab_hat - ac_hat * angle())) - r_rac * (identity3() - tensor_product(ac_hat, ac_hat));
        auto F = tensor_product(ac_hat, r_rac * (ab_hat - ac_hat * angle())) + angle()*r_rac * (identity3() - tensor_product(ac_hat, ac_hat));
        auto P2 = r_rab*G;
        auto P3 = tensor_product(ac_hat * angle() - ab_hat, -ac_hat * r_rac*r_rac);
        auto P4 = r_rac*F;
        return d_get(c.f_inner_angle,j) * harmonic_energy_hessian(d_get(c.angle0,j), angle(), grad_a, grad_c, P2+P3+P4); //Harmonic Energy Hessian
    }

    INLINE mat3 outer_angle_hessian_m_a(const Constants& c) const{
        auto cost = dot(ba_hat, bm_hat); //Compute outer angle. ab,bm
        auto gradba = r_rab * (identity3() - tensor_product(ba_hat, ba_hat));
        auto grad_a = (bm_hat - ba_hat * cost) * r_rab;
        auto P1 = tensor_product(bm_hat - ba_hat * cost, -ba_hat * r_rab*r_rab);
        auto P2 = -r_rab * (tensor_product(ba_hat, grad_a) + cost * gradba);
        return d_get(c.f_outer_angle_m,j) * harmonic_energy_hessian(d_get(c.outer_angle_m0,j), cost, grad_a, grad_a, P1+P2); //Harmonic Energy Hessian
    }

    INLINE mat3 outer_angle_hessian_m_b(const Constants& c) const{
        auto cost = dot(ba_hat, bm_hat); //Compute outer angle. ba,bm
        auto gradba = r_rab * (tensor_product(ba_hat, ba_hat) - identity3());
        auto gradbm = r_rbm * (tensor_product(bm_hat, bm_hat) - identity3());
        auto grad_b = r_rab * (ba_hat * cost - bm_hat) + r_rbm * (bm_hat * cost - ba_hat);
        auto grad_a = (bm_hat - ba_hat * cost) * r_rab;
        auto P1 = tensor_product(bm_hat - ba_hat * cost, ba_hat * r_rab*r_rab);
        auto P3 = r_rab * (gradbm - (tensor_product(ba_hat, grad_b) + cost * gradba));
        return d_get(c.f_outer_angle_m,j) * harmonic_energy_hessian(d_get(c.outer_angle_m0,j), cost, grad_a, grad_b, P1+P3); //Harmonic Energy Hessian
    }

    INLINE mat3 outer_angle_hessian_m_m(const Constants& c) const{
        auto cost = dot(ba_hat, bm_hat); //Compute outer angle. ba,bm
        auto gradbm = r_rbm * (identity3() - tensor_product(bm_hat, bm_hat));
        auto grad_a = (bm_hat - ba_hat * cost) * r_rab;
        auto grad_m = r_rbm * (ba_hat - bm_hat * cost);
        auto P1 = r_rab * (gradbm - tensor_product(ba_hat, grad_m));
        return d_get(c.f_outer_angle_m,j) * harmonic_energy_hessian(d_get(c.outer_angle_m0,j), cost, grad_a, grad_m, P1); //Harmonic Energy Hessian
    }

    INLINE mat3 outer_angle_hessian_p_a(const Constants& c) const{
        auto cost = dot(ba_hat, bp_hat); //Compute outer angle. ba,bp
        auto gradba = r_rab * (identity3() - tensor_product(ba_hat, ba_hat));
        auto grad_a = r_rab * (bp_hat - ba_hat * cost);
        auto P1 = tensor_product(bp_hat - ba_hat * cost, -ba_hat * r_rab*r_rab);    
        auto P2 = -r_rab * (tensor_product(ba_hat, grad_a) + cost * gradba);
        return d_get(c.f_outer_angle_p,j) * harmonic_energy_hessian(d_get(c.outer_angle_p0,j), cost, grad_a, grad_a, P1+P2); //Harmonic Energy Hessian
    }

    INLINE mat3 outer_angle_hessian_p_b(const Constants& c) const{
        auto cost = dot(ba_hat, bp_hat); //Compute outer angle. ba,bp
        auto gradba = r_rab * (tensor_product(ba_hat, ba_hat) - identity3());
        auto gradbp = r_rbp * (tensor_product(bp_hat, bp_hat) - identity3());
        auto grad_b = r_rab * (ba_hat * cost - bp_hat) + r_rbp * (bp_hat * cost - ba_hat);
        auto grad_a = r_rab * (bp_hat - ba_hat * cost);
        auto P1 = tensor_product(bp_hat - ba_hat * cost, ba_hat * r_rab*r_rab);
        auto P3 = r_rab * (gradbp - (tensor_product(ba_hat, grad_b) + cost * gradba));
        return d_get(c.f_outer_angle_p,j) * harmonic_energy_hessian(d_get(c.outer_angle_p0,j), cost, grad_a, grad_b, P1 + P3); //Harmonic Energy Hessian
    }

    INLINE mat3 outer_angle_hessian_p_p(const Constants& c) const{
        auto cost = dot(ba_hat, bp_hat); //Compute outer angle. ba,bp
        auto gradbp = r_rbp * (identity3() - tensor_product(bp_hat, bp_hat));
        auto grad_a = r_rab * (bp_hat - ba_hat * cost);
        auto grad_p = r_rbp * (ba_hat - bp_hat * cost);
        auto P1 = r_rab * (gradbp - tensor_product(ba_hat, grad_p));
        return d_get(c.f_outer_angle_p,j) * harmonic_energy_hessian(d_get(c.outer_angle_p0,j), cost, grad_a, grad_p, P1); //Harmonic Energy Hessian
    }   

    INLINE auto dihedral_hessian_terms(const Constants& c) const{
        auto cb_hat = -bc_hat;
        auto cost1 = dot(ab_hat, cb_hat);
        auto cost2 = dot(cb_hat, db_hat);
        auto sint1 = SQRT(1 - cost1*cost1);
        auto sint2 = SQRT(1 - cost2*cost2);
        auto cot1 = cost1/sint1;
        auto csc1 = device_real_t(1.)/sint1;
        auto csc2 = device_real_t(1.)/sint2;
        auto nabc = cross(ab_hat, cb_hat) * csc1;
        auto nbcd = cross(db_hat, cb_hat) * csc2;
        auto cosb = dot(nabc, nbcd);
        auto Coeff = cosb * csc1 * r_rab;
        auto F1 = ab_hat * sint1;
        auto F2 = cross(cb_hat, nbcd) / cosb;
        auto F3 = cot1 * (ab_hat * cost1 - cb_hat);
        auto F = F1 - F2 + F3;
        auto GradACosb = cosb * r_rab * csc1 * (ab_hat * sint1 - cross(cb_hat, nbcd) / cosb + cot1 * (ab_hat * cost1 - cb_hat)); 
        return std::tuple{cb_hat, cost1, cost2, sint1, sint2, cot1, csc1, csc2, nabc, nbcd, cosb, Coeff, F, GradACosb};
    }

    // $\nabla_a(\nabla_a(\cos(\theta)))$
    INLINE mat3 dihedral_hessian_a(const Constants& c) const{
        auto [cb_hat, cost1, cost2, sint1, sint2, cot1, csc1, csc2, nabc, nbcd, cosb, Coeff, F, GradACosb] = dihedral_hessian_terms(c);
        auto GradARab = ab_hat * r_rab * r_rab;
        auto GradAabh = (tensor_product(ab_hat, ab_hat) - identity3()) * r_rab;
        auto GradASint1 = -(ab_hat * cost1 - cb_hat) * cost1 * r_rab * csc1;
        auto GradAcsc1 = -GradASint1 * csc1 * csc1;
        auto GradAcost1 = (ab_hat * cost1 - cb_hat) * r_rab;
        auto GradAcot1 = (sint1 * GradAcost1 - cost1 * GradASint1) * csc1 * csc1;
        auto GradACoeff = GradACosb * r_rab * csc1 + cosb * (GradARab * csc1 + GradAcsc1 * r_rab);
        auto GradAF1 = GradAabh * sint1 + tensor_product(ab_hat, GradASint1);
        auto GradAF2 = tensor_product(cross(cb_hat, nbcd), -GradACosb / (cosb * cosb));
        auto GradAF3 = tensor_product(ab_hat * cost1 - cb_hat, GradAcot1) + cot1 * (tensor_product(ab_hat, GradAcost1) + cost1 * GradAabh);
        auto GradAF = GradAF1 - GradAF2 + GradAF3;
        auto GradAGradCosb = tensor_product(F, GradACoeff) + Coeff * GradAF;
        return GradAGradCosb;

    }

    // $\nabla_b(\nabla_a(\cos(\beta)))$
    INLINE mat3 dihedral_hessian_b(const Constants& c) const{
        auto [cb_hat, cost1, cost2, sint1, sint2, cot1, csc1, csc2, nabc, nbcd, cosb, Coeff, F, GradACosb] = dihedral_hessian_terms(c);
        auto grad_b_sint1 = -((cb_hat - ab_hat*cost1)*r_rab + (ab_hat - cb_hat*cost1)*r_rbc)*cost1 * csc1;
        auto grad_b_sint2 = -((cb_hat - db_hat*cost2)*r_rdb + (db_hat - cb_hat*cost2)*r_rbc)*cost2 * csc2;
        auto grad_b_ab_cross_cb_dot_nbcd = (r_rbc * (cross(nbcd, ab_hat) - dot(nbcd, cross(ab_hat, cb_hat))*cb_hat) - r_rab * (cross(nbcd, cb_hat) - dot(nbcd, cross(cb_hat, ab_hat))*ab_hat)); 
        auto grad_b_db_cross_cb_dot_nabc = (r_rbc * (cross(nabc, db_hat) - dot(nabc, cross(db_hat, cb_hat))*cb_hat) - r_rdb * (cross(nabc, cb_hat) - dot(nabc, cross(cb_hat, db_hat))*db_hat));
        auto P1 = (grad_b_ab_cross_cb_dot_nbcd*sint1 - (dot(nbcd, cross(ab_hat, cb_hat)))*grad_b_sint1)*csc1*csc1;
        auto P2 = (grad_b_db_cross_cb_dot_nabc*sint2 - (dot(nabc, cross(db_hat, cb_hat)))*grad_b_sint2)*csc2*csc2;
        auto grad_b = P1 + P2;
        auto GradBRab = -ab_hat*r_rab*r_rab;
        auto GradBabh = (identity3() - tensor_product(ab_hat, ab_hat))*r_rab;
        auto GradBcbh = (identity3() - tensor_product(cb_hat, cb_hat))*r_rbc;
        auto GradBdbh = (identity3() - tensor_product(db_hat, db_hat))*r_rdb;
        auto GradBnbcd = ((cross(db_hat, GradBcbh) - cross(cb_hat, GradBdbh))* sint2 - tensor_product(cross(db_hat,cb_hat), grad_b_sint2))*csc2*csc2;
        auto GradBcsc1 = -grad_b_sint1 * csc1*csc1;
        auto GradBcost1 = (cb_hat - ab_hat*cost1)*r_rab + (ab_hat - cb_hat*cost1)*r_rbc;
        auto GradBcot1 = (sint1 * GradBcost1 - cost1 * grad_b_sint1)*csc1*csc1;
        auto GradBCoeff = grad_b*r_rab*csc1 + cosb*(GradBRab * csc1 + GradBcsc1*r_rab);
        auto GradBF1 = GradBabh*sint1 + tensor_product(ab_hat, grad_b_sint1);
        auto GradBF2 = tensor_product(cross(cb_hat,nbcd), -grad_b/(cosb*cosb)) + (cross(cb_hat, GradBnbcd) - cross(nbcd, GradBcbh))/cosb;
        auto GradBF3 = tensor_product(ab_hat*cost1-cb_hat, GradBcot1) + cot1*(tensor_product(ab_hat,GradBcost1) + cost1*GradBabh - GradBcbh);
        auto GradBF = GradBF1 - GradBF2 + GradBF3;
        auto GradBGradCosb = tensor_product(F, GradBCoeff) + Coeff*GradBF;
        return GradBGradCosb;
    }

    // $\nabla_c(\nabla_a(\cos(\theta)))$
    INLINE mat3 dihedral_hessian_c(const Constants& c) const{
        auto [cb_hat, cost1, cost2, sint1, sint2, cot1, csc1, csc2, nabc, nbcd, cosb, Coeff, F, GradACosb] = dihedral_hessian_terms(c);
        auto grad_c_sint1 = -(cb_hat*cost1 - ab_hat)*cost1*csc1*r_rbc;
        auto grad_c_sint2 = -(cb_hat*cost2 - db_hat)*cost2*csc2*r_rbc;
        auto grad_c_ab_cross_cb_dot_nabc = r_rbc * (dot(nabc, cross(db_hat, cb_hat))*cb_hat - cross(nabc, db_hat));
        auto grad_c_db_cross_cb_dot_nbcd = r_rbc * (dot(nbcd, cross(ab_hat, cb_hat))*cb_hat - cross(nbcd, ab_hat));
        auto P1 = (grad_c_ab_cross_cb_dot_nabc*sint2 - (dot(nabc, cross(db_hat, cb_hat)))*grad_c_sint2)*csc2*csc2;
        auto P2 = (grad_c_db_cross_cb_dot_nbcd*sint1 - (dot(nbcd, cross(ab_hat, cb_hat)))*grad_c_sint1)*csc1*csc1;
        auto grad_c = P1 + P2;

        auto GradCcbh   = (tensor_product(cb_hat, cb_hat) - identity3())*r_rbc;
        auto GradCcsc1  = -grad_c_sint1 * csc1*csc1;
        auto GradCcost1 = (cb_hat*cost1 - ab_hat)*r_rbc;
        auto GradCcot1  = (sint1 * GradCcost1 - cost1 * grad_c_sint1)*csc1*csc1;
        auto GradCnbcd  = (cross(db_hat, GradCcbh)*sint2 - tensor_product(cross(db_hat, cb_hat), grad_c_sint2))*csc2*csc2;
        auto GradCCoeff = grad_c * r_rab * csc1 + cosb*(GradCcsc1*r_rab);
        auto GradCF1    = tensor_product(ab_hat, grad_c_sint1);
        auto GradCF2    = tensor_product(cross(cb_hat, nbcd), -grad_c/(cosb*cosb)) + (cross(cb_hat, GradCnbcd) - cross(nbcd, GradCcbh))/cosb;
        auto GradCF3    = tensor_product(ab_hat*cost1-cb_hat, GradCcot1) + cot1*(tensor_product(ab_hat,GradCcost1) - GradCcbh);
        auto GradCF     = GradCF1 - GradCF2 + GradCF3;
        auto GradCGradCosb = tensor_product(F, GradCCoeff) + Coeff*GradCF;
        return GradCGradCosb;
    }

    // $\nabla_d(\nabla_a(\cos(\theta)))$
    INLINE mat3 dihedral_hessian_d(const Constants& c) const{
        auto [cb_hat, cost1, cost2, sint1, sint2, cot1, csc1, csc2, nabc, nbcd, cosb, Coeff, F, GradACosb] = dihedral_hessian_terms(c);
        auto GradDSint2 = -(db_hat*cost2 - cb_hat)*cost2*csc2*r_rdb;
        auto GradDDbCrossCbDotNabc = -r_rdb * (dot(nabc, cross(cb_hat, db_hat))*db_hat - cross(nabc, cb_hat));
        auto grad_d = (GradDDbCrossCbDotNabc*sint2 - (dot(nabc, cross(db_hat, cb_hat)))*GradDSint2)*csc2*csc2;

        auto GradDdbh = (tensor_product(db_hat, db_hat) - identity3())*r_rdb;
        auto GradDnbcd = (-cross(cb_hat, GradDdbh)*sint2 - tensor_product(cross(db_hat, cb_hat), GradDSint2))*csc2*csc2;
        auto GradDCoeff = grad_d * r_rab * csc1;
        auto GradDF2 = tensor_product(cross(cb_hat, nbcd), -grad_d/(cosb*cosb)) + cross(cb_hat, GradDnbcd)/cosb;
        auto GradDF = - GradDF2;
        auto GradDGradCosb = tensor_product(F, GradDCoeff) + Coeff*GradDF;
        return GradDGradCosb;
    }

    INLINE auto outer_dihedral_hessian_a_terms(const Constants& c) const{
        auto ma_hat = -am_hat;
        auto cosa = dot(ab_hat, am_hat);
        auto cosm = dot(ma_hat, mp_hat);
        auto sina = sqrt(1 - cosa*cosa);
        auto sinm = sqrt(1 - cosm*cosm);
        auto cota = cosa/sina;
        auto cotm = cosm/sinm;
        auto csca = real_t(1.)/sina;
        auto cscm = real_t(1.)/sinm;
        auto nbam = cross(ab_hat, am_hat)*csca;
        auto namp = cross(ma_hat, mp_hat)*cscm;
        auto cosb = dot(nbam, namp);
        auto F1 = ab_hat*cosb;
        auto F2 = cross(am_hat, namp)*csca;
        auto F3 = am_hat*cosb;
        auto F4 = cross(namp, ab_hat)*csca;
        auto G1 = ab_hat*cosa * r_rab;
        auto G2 = am_hat * r_rab;
        auto G3 = am_hat*cosa * r_ram;
        auto G4 = ab_hat * r_ram;
        auto H1 = cross(mp_hat, nbam);
        auto H2 = ma_hat*cosb*sinm;
        auto H3 = cotm*cosb*(mp_hat - ma_hat*cosm);
        auto C1 = cota*cosb*csca;
        auto C2 = r_ram * cscm;
        auto GradAcosb = (F1 - F2)*r_rab + (F3 - F4)*r_ram + C1*(G1 - G2 + G3 - G4) + C2*(H1 - H2 + H3);
        return std::tuple(cosa, cosm, sina, sinm, cota, cotm, csca, cscm, nbam, namp, cosb, F1, F2, F3, F4, G1, G2, G3, G4, H1, H2, H3, C1, C2, GradAcosb);
    }

    INLINE mat3 outer_dihedral_hessian_a_a(const Constants& c) const{
        auto [cosa, cosm, sina, sinm, cota, cotm, csca, cscm, nbam, namp, cosb, F1, F2, F3, F4, G1, G2, G3, G4, H1, H2, H3, C1, C2, GradAcosb] = outer_dihedral_hessian_a_terms(c);
        auto ma_hat = -am_hat;

        auto GradAcosa = (am_hat*cosa - ab_hat)*r_ram + (ab_hat*cosa - am_hat)*r_rab;
        auto GradAsina = -cosa*csca * GradAcosa;
        auto GradAcota = (sina * GradAcosa - cosa * GradAsina)*csca*csca;
        auto GradAcsca = -GradAsina*csca*csca;
        auto GradAcosm = (mp_hat - ma_hat*cosm)*r_ram;
        auto GradAsinm = -cosm*cscm * GradAcosm;
        auto GradAcotm = (sinm * GradAcosm - cosm * GradAsinm)*cscm*cscm;
        auto GradAcscm = -GradAsinm*cscm*cscm;
        
        auto GradAab = (tensor_product(ab_hat, ab_hat) - identity3())*r_rab;
        auto GradArabn = ab_hat*r_rab*r_rab;
        auto GradAam = (tensor_product(am_hat, am_hat) - identity3())*r_ram;
        auto GradAramn = am_hat*r_ram*r_ram;
        auto GradAma = (identity3() - tensor_product(ma_hat,ma_hat))*r_ram;
        auto GradAnbam = ((cross(ab_hat, GradAam) - cross(am_hat, GradAab))*sina  - tensor_product(cross(ab_hat, am_hat), GradAsina))*csca*csca;
        auto GradAnamp = (( - cross(mp_hat, GradAma))*sinm - tensor_product(cross(ma_hat, mp_hat), GradAsinm))*cscm*cscm;
        auto GradAF1 = tensor_product(ab_hat, GradAcosb) + cosb*GradAab;
        auto GradAF2 = ((cross(am_hat, GradAnamp) - cross(namp, GradAam))*sina - tensor_product(cross(am_hat, namp), GradAsina))*csca*csca;
        auto GradAF3 = tensor_product(am_hat, GradAcosb) + cosb*GradAam;
        auto GradAF4 = ((cross(namp, GradAab) - cross(ab_hat, GradAnamp))*sina - tensor_product(cross(namp, ab_hat), GradAsina))*csca*csca;

        auto GradAG1 = tensor_product(ab_hat, (GradAcosa*r_rab + GradArabn*cosa)) + cosa*r_rab * GradAab;
        auto GradAG2 = tensor_product(am_hat, GradArabn) + GradAam*r_rab;
        auto GradAG3 = tensor_product(am_hat, (GradAcosa*r_ram + GradAramn*cosa)) + cosa*r_ram * GradAam;
        auto GradAG4 = tensor_product(ab_hat, GradAramn) + GradAab*r_ram;

        auto GradAH1 = cross(mp_hat,GradAnbam);
        auto GradAH2 = tensor_product(ma_hat, (cosb*GradAsinm + GradAcosb*sinm)) + cosb*sinm*GradAma;
        auto GradAH3 = tensor_product(mp_hat - ma_hat*cosm, (GradAcotm*cosb + GradAcosb*cotm)) + cotm*cosb*(- (GradAma*cosm + tensor_product(ma_hat,GradAcosm)));

        auto GradAC1 = GradAcota * cosb*csca + cota* (GradAcosb*csca + cosb*GradAcsca);
        auto GradAC2 = GradAramn*cscm + GradAcscm*r_ram;

        auto GradAF = r_rab * (GradAF1 - GradAF2)  + tensor_product(F1 - F2,GradArabn) + r_ram * (GradAF3 - GradAF4) + tensor_product(F3 - F4,GradAramn);
        auto GradAG = C1 * (GradAG1 - GradAG2 + GradAG3 - GradAG4) + tensor_product(G1 - G2 + G3 - G4, GradAC1);
        auto GradAH = C2 * (GradAH1 - GradAH2 + GradAH3) + tensor_product(H1 - H2 + H3, GradAC2);

        auto GradGradAcosb = GradAF + GradAG + GradAH;
        return GradGradAcosb;
    }

    INLINE mat3 outer_dihedral_hessian_a_b(const Constants& c) const{
        auto [cosa, cosm, sina, sinm, cota, cotm, csca, cscm, nbam, namp, cosb, F1, F2, F3, F4, G1, G2, G3, G4, H1, H2, H3, C1, C2, GradAcosb] = outer_dihedral_hessian_a_terms(c);
        /*         
        GradBab := (IdentityMatrix[3] - TensorProduct[abh, abh])/abn
        GradBrabn := -abh/abn^2
        GradBam := {0,0,0}
        GradBramn := {0,0,0}
        GradBma := {0,0,0}
        GradBmp := {0,0,0}
        GradBsinm := {0,0,0}
        GradBcotm := {0,0,0}
        GradBcota := (sina * GradBcosa - cosa * GradBsina)/sina^2
        GradBcsca := -GradBsina/(sina^2)
        GradBcscm := {0,0,0}
        GradBnbam := (( - CPL[amh, GradBab])*sina  - TensorProduct[Cross[abh, amh], GradBsina])/sina^2
        GradBnamp := 0

        GradBF1 := TensorProduct[abh, GradBcosb] + cosb*GradBab
        GradBF2 := (- TensorProduct[Cross[amh, namp], GradBsina])/sina^2
        GradBF3 := TensorProduct[amh, GradBcosb] 
        GradBF4 := ((CPL[namp, GradBab] )*sina - TensorProduct[Cross[namp, abh], GradBsina])/sina^2

        GradBG1 := TensorProduct[abh, (GradBcosa/abn + GradBrabn*cosa)] + cosa/abn * GradBab
        GradBG2 := TensorProduct[amh, GradBrabn]
        GradBG3 := TensorProduct[amh, (GradBcosa/amn)]
        GradBG4 := GradBab/amn

        GradBH1 := CPL[mph, GradBnbam] 
        GradBH2 := TensorProduct[mah, GradBcosb*sinm] 
        GradBH3 := TensorProduct[mph - mah*cosm, (GradBcosb*cotm)] + cotm*cosb*( - (TensorProduct[mah,GradBcosm]))

        GradBC1 := GradBcota * cosb/sina + cota* (GradBcosb/sina + cosb*GradBcsca)
        GradBC2 := 0 */

        auto GradBab := (IdentityMatrix[3] - TensorProduct[ab_hat, ab_hat])/r_rab;
        

    }

    //Computes gradient related to bending of outer angles. ~20 FLOPs
    /**
     * @brief Compute the gradient of the outer angle-m term.
     * @param c The constants for the threadIdx^th node.
     * @return The gradient of the outer angle-m term.
    */
    INLINE coord3d outer_angle_gradient_m(const Constants& c) const
    {
        real_t cos_angle = -dot(ab_hat, bm_hat); //Compute outer angle. ab,bm
        coord3d grad = (bm_hat + ab_hat * cos_angle) * r_rab; //Derivative of outer angles Eq. 30. Buster Thesis
        return d_get(c.f_outer_angle_m,j) * harmonic_energy_gradient(d_get(c.outer_angle_m0,j),cos_angle,grad); //Harmonic Energy Gradient: Eq. 30 multiplied by harmonic term.
    }

    /**
     * @brief Compute the gradient of the outer angle-p term.
     * @param c The constants for the threadIdx^th node.
     * @return The gradient of the outer angle-p term.
    */
    INLINE coord3d outer_angle_gradient_p(const Constants& c) const
    {   
        real_t cos_angle = -dot(ab_hat, bp_hat); //Compute outer angle. ab,bp
        coord3d grad = (bp_hat + ab_hat * cos_angle) * r_rab; //Derivative of outer angles Eq. 28. Buster Thesis
        return d_get(c.f_outer_angle_p,j) * harmonic_energy_gradient(d_get(c.outer_angle_p0,j),cos_angle,grad); //Harmonic Energy Gradient: Eq. 28 multiplied by harmonic term.
    }
    // Chain rule terms for dihedral calculation
    //Computes gradient related to dihedral/out-of-plane term. ~75 FLOPs
    /**
     * @brief Compute the gradient of the inner dihedral term.
     * @param c The constants for the threadIdx^th node.
     * @return The gradient of the inner dihedral term.
    */
    INLINE coord3d inner_dihedral_gradient(const Constants& c) const
    {
        coord3d nabc, nbcd; real_t cos_b, cos_c, r_sin_b, r_sin_c;
        cos_b = dot(ba_hat,bc_hat); r_sin_b = (real_t)1.0/SQRT((real_t)1.0 - cos_b*cos_b); nabc = cross(ba_hat, bc_hat) * r_sin_b;
        cos_c = dot(-bc_hat,cd_hat); r_sin_c = (real_t)1.0/SQRT((real_t)1.0 - cos_c*cos_c); nbcd = cross(-bc_hat,cd_hat) * r_sin_c;

        real_t cos_beta = dot(nabc, nbcd); //Inner dihedral angle from planes abc,bcd.
        real_t cot_b = cos_b * r_sin_b * r_sin_b; //cos(b)/sin(b)^2

        //Derivative w.r.t. inner dihedral angle F and G in Eq. 26
        coord3d grad = cross(bc_hat, nbcd) * r_sin_b * r_rab - ba_hat * cos_beta * r_rab + (cot_b * cos_beta * r_rab) * (bc_hat - ba_hat * cos_b);
        return d_get(c.f_inner_dihedral,j) * harmonic_energy_gradient(d_get(c.inner_dih0,j), cos_beta, grad); //Eq. 26.
    }

    //Computes gradient from dihedral angles constituted by the planes bam, amp ~162 FLOPs
    /**
     * @brief Compute the gradient of the outer dihedral-a term.
     * @param c The constants for the threadIdx^th node.
     * @return The gradient of the outer dihedral-a term.
    */
    INLINE coord3d outer_dihedral_gradient_a(const Constants& c) const
    {
        coord3d nbam_hat, namp_hat; real_t cos_a, cos_m, r_sin_a, r_sin_m;

        cos_a = dot(ab_hat,am_hat); r_sin_a = (real_t)1.0/SQRT((real_t)1.0 - cos_a*cos_a); nbam_hat = cross(ab_hat,am_hat) * r_sin_a;
        cos_m = dot(-am_hat,mp_hat); r_sin_m = (real_t)1.0/SQRT((real_t)1.0 - cos_m*cos_m); namp_hat = cross(-am_hat,mp_hat) * r_sin_m;
        
        real_t cos_beta = dot(nbam_hat, namp_hat); //Outer Dihedral angle bam, amp
        real_t cot_a = cos_a * r_sin_a * r_sin_a;
        real_t cot_m = cos_m * r_sin_m * r_sin_m;

        //Derivative w.r.t. outer dihedral angle, factorized version of Eq. 31.
        coord3d grad = cross(mp_hat,nbam_hat)*r_ram*r_sin_m - (cross(namp_hat,ab_hat)*r_ram + cross(am_hat,namp_hat)*r_rab)*r_sin_a +
                        cos_beta*(ab_hat*r_rab + r_ram * ((real_t)2.0*am_hat + cot_m*(mp_hat+cos_m*am_hat)) - cot_a*(r_ram*(ab_hat - am_hat*cos_a) + r_rab*(am_hat-ab_hat*cos_a)));
        
        //Eq. 31 multiplied by harmonic term.
        return d_get(c.f_outer_dihedral,j) * harmonic_energy_gradient(d_get(c.outer_dih0_a,j), cos_beta, grad);
    }

    //Computes gradient from dihedral angles constituted by the planes nbmp, nmpa ~92 FLOPs
    /**
     * @brief Compute the gradient of the outer dihedral-m term.
     * @param c The constants for the threadIdx^th node.
     * @return The gradient of the outer dihedral-m term.
    */
    INLINE coord3d outer_dihedral_gradient_m(const Constants& c) const
    {
        coord3d nbmp_hat, nmpa_hat; real_t cos_m, cos_p, r_sin_m, r_sin_p;
        cos_m = dot(mb_hat,mp_hat);  r_sin_m = (real_t)1.0/SQRT((real_t)1.0 - cos_m*cos_m); nbmp_hat = cross(mb_hat,mp_hat) * r_sin_m;
        cos_p = dot(-mp_hat,pa_hat); r_sin_p = (real_t)1.0/SQRT((real_t)1.0 - cos_p*cos_p); nmpa_hat = cross(-mp_hat,pa_hat) * r_sin_p;
        
        //Cosine to the outer dihedral angle constituted by the planes bmp and mpa
        real_t cos_beta = dot(nbmp_hat, nmpa_hat); //Outer dihedral angle bmp,mpa.
        real_t cot_p = cos_p * r_sin_p * r_sin_p;
        
        //Derivative w.r.t. outer dihedral angle, factorized version of Eq. 32.
        coord3d grad = r_rap * (cot_p*cos_beta * (-mp_hat - pa_hat*cos_p) - cross(nbmp_hat, mp_hat)*r_sin_p - pa_hat*cos_beta );

        //Eq. 32 multiplied by harmonic term.
        return d_get(c.f_outer_dihedral,j) * harmonic_energy_gradient(d_get(c.outer_dih0_m,j), cos_beta, grad);
    }

    //Computes gradient from dihedral angles constituted by the planes bpa, pam ~162 FLOPs
    /**
     * @brief Compute the gradient of the outer dihedral-p term.
     * @param c The constants for the threadIdx^th node.
     * @return The gradient of the outer dihedral-p term.
    */
    INLINE coord3d outer_dihedral_gradient_p(const Constants& c) const
    {
        coord3d nbpa_hat, npam_hat; real_t cos_p, cos_a, r_sin_p, r_sin_a;
        cos_a = dot(ap_hat,am_hat);  r_sin_a = (real_t)1.0/SQRT((real_t)1.0 - cos_a*cos_a); npam_hat = cross(ap_hat,am_hat)  * r_sin_a;
        cos_p = dot(pb_hat,-ap_hat); r_sin_p = (real_t)1.0/SQRT((real_t)1.0 - cos_p*cos_p); nbpa_hat = cross(pb_hat,-ap_hat) * r_sin_p;

        real_t cos_beta = dot(nbpa_hat, npam_hat); //Outer dihedral angle bpa, pam.
        real_t cot_p = cos_p * r_sin_p * r_sin_p;
        real_t cot_a = cos_a * r_sin_a * r_sin_a;

        //Derivative w.r.t. outer dihedral angle, factorized version of Eq. 33.
        coord3d grad = cross(npam_hat,pb_hat)*r_rap*r_sin_p - (cross(am_hat,nbpa_hat)*r_rap + cross(nbpa_hat,ap_hat)*r_ram)*r_sin_a +
                        cos_beta*(am_hat*r_ram + r_rap * ((real_t)2.0*ap_hat + cot_p*(pb_hat+cos_p*ap_hat)) - cot_a*(r_rap*(am_hat - ap_hat*cos_a) + r_ram*(ap_hat-am_hat*cos_a)));
        
        //Eq. 33 multiplied by harmonic term.
        return d_get(c.f_outer_dihedral,j) * harmonic_energy_gradient(d_get(c.outer_dih0_p,j), cos_beta, grad);
    }

    // Internal coordinate gradients
    /**
     * @brief Compute the gradient of the bond length term.
     * @param c The constants for the threadIdx^th node.
     * @return The gradient of the bond length term.
    */
    INLINE coord3d bond_length_gradient(const Constants& c) const {
        return d_get(c.f_bond,j) * harmonic_energy_gradient(bond(),d_get(c.r0,j),ab_hat); 
    }
    //Sum of angular gradient components.
    /**
     * @brief Compute the sum of the gradients of the bending terms.
     * @param c The constants for the threadIdx^th node.
     * @return The sum of the gradients of the bending terms.
    */
    INLINE coord3d angle_gradient(const Constants& c) const { return inner_angle_gradient(c) + outer_angle_gradient_p(c) + outer_angle_gradient_m(c);}
    //Sum of inner and outer dihedral gradient components.
    /**
     * @brief Compute the sum of the gradients of the dihedral terms.
     * @param c The constants for the threadIdx^th node.
     * @return The sum of the gradients of the dihedral terms.
    */
    INLINE coord3d dihedral_gradient(const Constants& c) const { 
        switch (T)
        {
        case PEDERSEN:
            return inner_dihedral_gradient(c) + outer_dihedral_gradient_a(c) + outer_dihedral_gradient_m(c) + outer_dihedral_gradient_p(c);
        case WIRZ:
            return inner_dihedral_gradient(c);
        default:
            return inner_dihedral_gradient(c) + outer_dihedral_gradient_a(c) + outer_dihedral_gradient_m(c) + outer_dihedral_gradient_p(c);
        }
        return inner_dihedral_gradient(c) + outer_dihedral_gradient_a(c) + outer_dihedral_gradient_m(c) + outer_dihedral_gradient_p(c);
    }
    /**
     * @brief Compute the energy contribution of the bond length term.
     * @param c The constants for the threadIdx^th node.
     * @return The energy contribution of the bond length term.
    */
    INLINE real_t bond_energy(const Constants& c) const {
        return (real_t)0.5 *d_get(c.f_bond,j) *harmonic_energy(bond(),d_get(c.r0,j));
    }
    /**
     * @brief Compute the total energy contribution of the bending terms.
     * @param c The constants for the threadIdx^th node.
     * @return The energy contribution of the bending terms.
    */
    INLINE real_t bend_energy(const Constants& c) const {
        return d_get(c.f_inner_angle,j)* harmonic_energy(angle(),d_get(c.angle0,j));
    }

    /**
     * @brief Compute the total energy contribution of the dihedral terms.
     * @param c The constants for the threadIdx^th node.
     * @return The energy contribution of the dihedral terms.
    */
    INLINE real_t dihedral_energy(const Constants& c) const {
        return d_get(c.f_inner_dihedral,j)* harmonic_energy(dihedral(),d_get(c.inner_dih0,j));
    }
    //Harmonic energy contribution from bond stretching, angular bending and dihedral angle bending.
    //71 FLOPs
    /**
     * @brief Compute the total energy contribution of the bond length, bending and dihedral terms.
     * @param c The constants for the threadIdx^th node.
     * @return The energy contribution of the bond length, bending and dihedral terms.
    */
    INLINE real_t energy(const Constants& c) const {
        switch (T)
        {
        case FLAT_BOND:
            return bond_energy(c);
            break;
        default:
            return bond_energy(c) + bend_energy(c) + dihedral_energy(c);
            break;
        }
    }
    //Sum of bond, angular and dihedral gradient components.
    /**
     * @brief Compute the total gradient of the bond length, bending and dihedral terms w.r.t. the coordinates of the threadIdx^th node.
     * @param c The constants for the threadIdx^th node.
     * @return The gradient of the bond length, bending and dihedral terms w.r.t. the coordinates of the threadIdx^th node.
    */
    INLINE coord3d gradient(const Constants& c) const{
        switch (T)
        {
        case FLAT_BOND:
            return bond_length_gradient(c);
        default:
            return bond_length_gradient(c) + angle_gradient(c) + dihedral_gradient(c);
            break;
        }
        return bond_length_gradient(c) + angle_gradient(c) + dihedral_gradient(c);
    }

    //Returns Gradient of the Gradient w.r.t. the coordinates of the threadIdx^th node.
    INLINE mat3 hessian_a(const Constants& c) const {
        return inner_angle_hessian_b(c);
    }


    //Reciprocal lengths of arcs ab, ac, am, ap.
    real_t
        rab,
        r_rab,
        r_rac,
        r_rad,
        r_ram,
        r_rbm,
        r_rbp,
        r_rbc,
        r_rdb,
        r_rap;

    //Base Arcs,
    coord3d
        ab,
        ac,
        ad;

    //All normalized arcs required to perform energy & gradient calculations.
    //Note that all these arcs are cyclical the arc ab becomes: ab->ac->ad,  the arc ac becomes: ac->ad->ab , the arc bc becomes: bc->cd->db (For iterations 0, 1, 2)
    //As such the naming convention here is related to the arcs as they are used in the 0th iteration.
    coord3d 
        ab_hat,
        ac_hat,
        ad_hat,
        bp_hat,
        bm_hat,
        am_hat,
        ap_hat,
        ba_hat,
        bc_hat,
        cd_hat,
        db_hat,
        mp_hat,
        mb_hat,
        pa_hat,
        pb_hat;

    coord3d
        face_center, //Center of the face to the left of the arc a->b, a->b, a->c
        face_offset; //Difference between the node coordinates X and the face-center coordinates face_center
    
    coord3d A[3];
    
};

INLINE hessian_t hessian(const coord3d* X) const {
    BLOCK_SYNC
    hessian_t hess(node_graph);
    for (uint8_t j = 0; j < 1; j++ ){
        ArcData arc = ArcData(j, X, node_graph);
        hess.A[0] += arc.outer_dihedral_hessian_a_a(constants);
    }
    if(threadIdx.x + blockIdx.x == 0){
        printf("Outer Neighbours (p, m): %d %d \n", d_get(node_graph.next_on_face,0), d_get(node_graph.prev_on_face,1));
        printf("Node Coordinates: a, b, c, d, m, p \n");
        auto XA = X[threadIdx.x];
        auto XB = X[d_get(node_graph.cubic_neighbours,0)];
        auto XC = X[d_get(node_graph.cubic_neighbours,1)];
        auto XD = X[d_get(node_graph.cubic_neighbours,2)];
        auto XM = X[d_get(node_graph.prev_on_face,0)];
        auto XP = X[d_get(node_graph.next_on_face,0)];
        printf("%f %f %f \n", XA[0], XA[1], XA[2]);
        printf("%f %f %f \n", XB[0], XB[1], XB[2]);
        printf("%f %f %f \n", XC[0], XC[1], XC[2]);
        printf("%f %f %f \n", XD[0], XD[1], XD[2]);
        printf("%f %f %f \n", XM[0], XM[1], XM[2]);
        printf("%f %f %f \n", XP[0], XP[1], XP[2]);
        printf("Hessian:\n");
        for (uint8_t i = 0; i < 3; i++){
            for (uint8_t j = 0; j < 3; j++){
                printf("%f ", hess.A[0].A[i*3 + j]);
            }
            printf("\n");
        }
    }
    return hess;
}


/**
 * @brief Compute the total gradient of the bond, flatness, bending and dihedral terms w.r.t. the coordinates of the threadIdx^th node.
 * @param c The constants for the threadIdx^th node.
 * @return The gradient of the bond, flatness, bending and dihedral terms w.r.t. the coordinates of the threadIdx^th node.
*/
INLINE coord3d gradient(coord3d* X) const {
    BLOCK_SYNC
    coord3d grad = {0.0, 0.0, 0.0};
    for (uint8_t j = 0; j < 3; j++ ){
        ArcData arc = ArcData(j, X, node_graph);
        grad += arc.gradient(constants);
    }
    switch (T)
    {
    case FLATNESS_ENABLED: {
        FaceData face(X, node_graph);
        auto face_grad = face.flatness_gradient(constants, reinterpret_cast<coord3d*>(sdata + Block_Size_Pow_2) + blockDim.x*2);
        return grad + face_grad;
        break;
        }
    case FLAT_BOND:{
        FaceData face(X, node_graph);
        auto face_grad = face.flatness_gradient(constants, reinterpret_cast<coord3d*>(sdata + Block_Size_Pow_2) + blockDim.x*2);
        return grad + face_grad;
        break;
        }
    default:
        return grad;
        break;
    }
}

/**
 * @brief Compute the total energy of the bond, flatness, bending and dihedral terms from all nodes in the isomer.
 * @param c The constants for the threadIdx^th node.
 * @return Total energy.
*/
INLINE real_t energy(coord3d* X) const {
    BLOCK_SYNC
    real_t arc_energy = (real_t)0.0;

    //(71 + 124) * 3 * N  = 585*N FLOPs
    for (uint8_t j = 0; j < 3; j++ ){
        ArcData arc = ArcData(j, X, node_graph);
        arc_energy += arc.energy(constants);
    }
    switch (T)
    {
    case FLATNESS_ENABLED: {
        FaceData face(X, node_graph);
        return reduction(sdata, arc_energy + face.flatness_energy(constants));
        break;
        }
    case FLAT_BOND: {
        FaceData face(X, node_graph);
        return reduction(sdata, arc_energy + face.flatness_energy(constants));
        break;
        }
    default:
        return reduction(sdata, arc_energy);
        break;
    }
}

INLINE real_t gradnorm(coord3d* X, coord3d& d)const {
    return reduction(sdata, dot(-gradient(X),d));
}

//Bracketing method designed to find upper bound for linesearch method that matches 
//reference python implementation by Buster.
INLINE real_t FindLineSearchBound(coord3d* X, coord3d& r0, coord3d* X1) const{
    real_t bound        = 1e-5;
    bool negative_grad  = true;
    size_t iter         = 0;
    while (negative_grad && iter < 1000)
    {   
        bound *= (real_t)1.5;
        X1[node_id] = X[node_id] + bound * r0;
        real_t gradsum = reduction(sdata, dot(gradient(X1),r0));
        negative_grad = (gradsum < (real_t)0);
    }
    return bound;
}


// finds the minimum point of a function f(x) using the bisection method
// X is the current point, X1 is the next point, X2 is the previous point
// r0 is the direction of the line search
// returns the minimum point
INLINE real_t Bisection(coord3d* X, coord3d& r0, coord3d* X1, coord3d* X2){
    real_t dfc = 1; size_t count = 0;
    real_t c; real_t a = 0.0; real_t b = FindLineSearchBound(X,r0,X1);
    coord3d d;
    while (ABS(dfc) > (real_t)1e-10 && count < 1000){
        count++;
        c =  (a+b)/(real_t)2;
        X1[node_id] = X[node_id] + c*r0;
        d  =  gradient(X1);
        dfc = reduction(sdata,dot(d,r0)); 

        if (dfc < (real_t)0.0){
            a = c;
        }
        else{
            b = c;
        }
    }
    return c;
}

//Brents Method for line-search using fixed number of iterations.
/**
 * @brief Brent's method for line-search.
 * @param X The coordinates of the nodes.
 * @param r0 The direction of the line-search.
 * @param X1 memory for storing temporary coordinates at a and s.
 * @param X2 memory for storing temporary coordinates at b.
 * @return The step-size s.
*/
INLINE real_t BrentsMethod(coord3d* X, coord3d& r0, coord3d* X1, coord3d* X2)const{
    real_t a,b,s,d;
    a = (real_t)0.0; //b = (real_t)1.0; 

    //To match python reference implementation by Buster.
    b = FindLineSearchBound(X,r0,X1);

    X1[node_id] = X[node_id] + a * r0;
    X2[node_id] = X[node_id] + b * r0;

    real_t f_a = gradnorm(X1,r0);
    real_t f_b = gradnorm(X2,r0);

    if (f_a*f_b > (real_t)0)
    {
        return b;
    }
    if (ABS(f_a) < ABS(f_b)){swap_reals(a,b); swap_reals(f_a,f_b);}

    real_t c = a; real_t f_c = f_a;
    bool flag = true;

    for (uint8_t i = 0; i < 30; i++)
    {   
        // Inverse quadratic interpolation
        if ( (f_a != f_c) && (f_b != f_c) )
        {
            s = a*f_a*f_c / ((f_a - f_b)*(f_a - f_c)) + b * f_a * f_c / ((f_b-f_a)*(f_b-f_c)) + c*f_a*f_b/((f_c-f_a)*(f_c-f_b));
        }else // Secant Method
        {
            s = b - f_b*(b-a)/(f_b-f_a);
        }
        
        bool condition_1 = !(s > (((real_t)3.0*a + b)/(real_t)4.0) && s < b);
        bool condition_2 = flag && (ABS(s-b) >= ABS(b-c)/(real_t)2.0);
        bool condition_3 = !flag && (ABS(s-b) >= ABS(c-d)/(real_t)2.0);
        bool condition_4 = flag && (ABS(b-c) < (real_t)5e-8);
        bool condition_5 = !flag && (ABS(c-d) < (real_t)5e-8);

        if (condition_1 || condition_2 || condition_3 || condition_4 || condition_5)
        {
            s = (a+b) / (real_t)2.0; // Bisection Method
            flag = true;
        }else
        {
            flag = false;
        }
        X1[node_id] = X[node_id] + s * r0;
        real_t f_s = gradnorm(X1,r0);
        d = c;
        c = b; f_c = f_b;
        if (f_a*f_s < (real_t)0)
        {
            b = s; f_b = f_s;
        }else
        {
            a = s; f_a = f_s;
        }
        if (ABS(f_a) < ABS(f_b))
        {
            swap_reals(a,b); swap_reals(f_a,f_b);
        }
    }
    return b;
}

//Golden Section Search, using fixed iterations.
/**
 * @brief Golden Section Search for line-search.
 * @param X The coordinates of the nodes.
 * @param r0 The direction of the line-search.
 * @param X1 memory for storing temporary coordinates at x1.
 * @param X2 memory for storing temporary coordinates at x2.
 * @return The step-size alpha
*/
INLINE real_t GSS(coord3d* X, const coord3d& r0, coord3d* X1, coord3d* X2) const{
    const real_t tau = (real_t)0.6180339887;
    //Line search x - values;
    real_t a = 0.0; real_t b = (real_t)1.0;
    
    real_t x1,  x2;
    x1 = (a + ((real_t)1. - tau) * (b - a));
    x2 = (a + tau * (b - a));
    //Actual coordinates resulting from each traversal 
    X1[node_id] = X[node_id] + x1 * r0;
    X2[node_id] = X[node_id] + x2 * r0;

    real_t f1 = energy(X1);
    real_t f2 = energy(X2);

    for (uint8_t i = 0; i < 20; i++){
        if (f1 > f2){
            a = x1;
            x1 = x2;
            f1 = f2;
            x2 = a + tau * (b - a);
            X2[node_id] = X[node_id] + x2 * r0;
            f2 = energy(X2);
        }else
        {
            b = x2;
            x2 = x1;
            f2 = f1;
            x1 = a + ((real_t)1.0 - tau) * (b - a);
            X1[node_id] = X[node_id] + x1 * r0;
            f1 = energy(X1);
        }
    }
    if (f1 > energy(X)) {return (real_t)0.0;}
    //Line search coefficient
    real_t alpha = (a+b)/(real_t)2.0;
    return alpha;
}

/**
 * @brief Conjugate Gradient Method for energy minimization.
 * @param X The coordinates of the nodes.
 * @param X1 memory for storing temporary coordinates.
 * @param X2 memory for storing temporary coordinates.
 * @param MaxIter The maximum number of iterations.
*/
INLINE  void CG(coord3d* X, coord3d* X1, coord3d* X2, const size_t MaxIter){
    real_t alpha, beta, g0_norm2, s_norm;
    coord3d g0,g1,s;
    g0 = gradient(X);
    s = -g0;
    //Normalize To match reference python implementation by Buster.
    #if USE_MAX_NORM==1
        s_norm = reduction_max(sdata, max(max(s.x,s.y),s.z));
    #else
        s_norm = SQRT(reduction(sdata, dot(s,s)));
    #endif
    s /= s_norm;

    for (size_t i = 0; i < MaxIter; i++){
        alpha = LINESEARCH_METHOD(X,s,X1,X2);
        if (alpha > (real_t)0.0){X1[node_id] = X[node_id] + alpha * s;}
        g1 = gradient(X1);
        //Polak Ribiere method
        g0_norm2 = reduction(sdata, dot(g0, g0));
        beta = d_max(reduction(sdata, dot(g1, (g1 - g0))) / g0_norm2,(real_t)0.0);

        if (alpha > (real_t)0.0){X[node_id] = X1[node_id];}else{ g1 = g0; beta = (real_t) 0.0;}
        s = -g1 + beta*s;
        g0 = g1;
        //Normalize Search Direction using MaxNorm or 2Norm
        #if USE_MAX_NORM==1
            s_norm = reduction_max(sdata, max(max(s.x,s.y),s.z));
        #else
            s_norm = SQRT(reduction(sdata, dot(s,s)));
        #endif
        s /= s_norm;
    }
}
};

/**
 * @brief Checks if the isomer_idx^th isomer has converged, isomer_idx is different for each block.
 * @param B The isomer batch.
 * @param isomer_idx The index of the isomer to check.
 * @param max_iterations The maximum number of iterations, if the isomer has not converged after this many iterations, it is marked as failed.
 * @tparam T The forcefield type.
 * @return void
*/
template <ForcefieldType T>
__device__ void check_batch(IsomerBatch &B, const size_t isomer_idx, const size_t max_iterations){
    DEVICE_TYPEDEFS
    extern __shared__ real_t smem[];
    clear_cache(smem,Block_Size_Pow_2);

    if (isomer_idx < B.isomer_capacity){ //Avoid illegal memory access
    if (B.statuses[isomer_idx] == IsomerStatus::NOT_CONVERGED){
    size_t offset = isomer_idx * blockDim.x;
    Constants constants     = Constants(B, isomer_idx);
    NodeNeighbours node_graph    = NodeNeighbours(B, isomer_idx, smem);
    ForceField FF           = ForceField<T>(node_graph, constants, smem);
    coord3d* X              = reinterpret_cast<coord3d*>(smem + B.n_atoms);
    assign(X[threadIdx.x],reinterpret_cast<std::array<float,3>*>(B.X+offset*3)[threadIdx.x]);
    
    coord3d rel_bond_err, rel_angle_err, rel_dihedral_err;
    BLOCK_SYNC
    for (uint8_t j = 0; j < 3; j++){
        auto arc            = ForceField<T>::ArcData(j, X, node_graph);
        #if USE_CONSTANT_INDICES
        d_set(rel_bond_err,      j, ABS(ABS(arc.bond()       - constants.r0(j))        /constants.r0(j)));
        d_set(rel_angle_err,     j, ABS(ABS(arc.angle()      - constants.angle0(j))    /constants.angle0(j)));
        d_set(rel_dihedral_err,  j, ABS(ABS(arc.dihedral()   - constants.inner_dih0(j))/constants.inner_dih0(j)));
        #else
        d_set(rel_bond_err,      j, ABS(ABS(arc.bond()       - d_get(constants.r0,j))        /d_get(constants.r0,j)));
        d_set(rel_angle_err,     j, ABS(ABS(arc.angle()      - d_get(constants.angle0,j))    /d_get(constants.angle0,j)));
        d_set(rel_dihedral_err,  j, ABS(ABS(arc.dihedral()   - d_get(constants.inner_dih0,j))/d_get(constants.inner_dih0,j)));
        #endif
    }

    real_t bond_max         = reduction_max(smem, max(rel_bond_err));
    //real_t angle_max        = reduction_max(smem, max(rel_angle_err));
    //real_t dihedral_max     = reduction_max(smem, max(rel_dihedral_err));
    //real_t bond_rms         = SQRT(reduction(smem,dot(rel_bond_err,rel_bond_err))/(device_real_t)blockDim.x);
    //real_t angle_rms        = SQRT(reduction(smem,dot(rel_angle_err,rel_angle_err))/(device_real_t)blockDim.x);
    //real_t dihedral_rms     = SQRT(reduction(smem,dot(rel_dihedral_err,rel_dihedral_err))/(device_real_t)blockDim.x);
    //real_t bond_mean        = reduction(smem,sum(rel_bond_err))/(device_real_t)blockDim.x;
    //real_t angle_mean       = reduction(smem,sum(rel_angle_err))/(device_real_t)blockDim.x;
    //real_t dihedral_mean    = reduction(smem,sum(rel_dihedral_err))/(device_real_t)blockDim.x;
    real_t grad_norm        = SQRT(reduction(smem,dot(FF.gradient(X), FF.gradient(X))))/(real_t)blockDim.x;
    //real_t grad_rms         = SQRT(reduction(smem,dot(FF.gradient(X), FF.gradient(X)))/(device_real_t)blockDim.x);
    //real_t grad_max         = reduction_max(smem, SQRT(dot(FF.gradient(X), FF.gradient(X)))     );
    //real_t energy           = FF.energy(X); 
    
    bool converged = ((grad_norm < (real_t)1e-2)  && (bond_max < (real_t)0.1) && !ISNAN(grad_norm)) ;
    //if(threadIdx.x + isomer_idx == 0){printf("%d", (int)num_converged); printf("/ %d Fullerenes Converged in Batch \n", (int)gridDim.x);}
    if(threadIdx.x == 0 && B.statuses[isomer_idx] != IsomerStatus::EMPTY){
        if (converged)
        {
            B.statuses[isomer_idx] = IsomerStatus::CONVERGED;
        } else if (B.iterations[isomer_idx] >= max_iterations || ISNAN(grad_norm)  ) {
            B.statuses[isomer_idx] = IsomerStatus::FAILED;
        }
       
    }}}
}

//Compute Hessians for all isomers in the batch
template <ForcefieldType T> __global__ void compute_hessians_(IsomerBatch B){
    DEVICE_TYPEDEFS
    extern __shared__ real_t smem[];
    clear_cache(smem,Block_Size_Pow_2);
    auto limit = ((B.isomer_capacity + gridDim.x - 1) / gridDim.x ) * gridDim.x;  //Fast ceiling integer division.
    for (int isomer_idx = blockIdx.x; isomer_idx < limit; isomer_idx += gridDim.x){
    BLOCK_SYNC
    if (isomer_idx < B.isomer_capacity){ //Avoid illegal memory access
    if (B.statuses[isomer_idx] == IsomerStatus::CONVERGED){
    size_t offset = isomer_idx * blockDim.x;
    Constants constants     = Constants(B, isomer_idx);
    NodeNeighbours node_graph    = NodeNeighbours(B, isomer_idx, smem);
    ForceField<T> FF           = ForceField<T>(node_graph, constants, smem);
    coord3d* X              = reinterpret_cast<coord3d*>(smem + B.n_atoms);
    assign(X[threadIdx.x],reinterpret_cast<std::array<float,3>*>(B.X+offset*3)[threadIdx.x]);
    FF.hessian(X);

    }}}
}

/**
 * @brief Forcefield Optimizes a batch of isomers.
 * @param B IsomerBatch
 * @param iterations Number of iterations to run
 * @param max_iterations Maximum number of iterations, to compare against, isomers are marked as failed if this is exceeded.
 * @return void
*/
template <ForcefieldType T>
__global__ void optimise_(IsomerBatch B, const size_t iterations, const size_t max_iterations){
    DEVICE_TYPEDEFS
    extern __shared__ real_t smem[];
    clear_cache(smem,Block_Size_Pow_2);
    auto limit = ((B.isomer_capacity + gridDim.x - 1) / gridDim.x ) * gridDim.x;  //Fast ceiling integer division.
    for (int isomer_idx = blockIdx.x; isomer_idx < limit; isomer_idx += gridDim.x){
    BLOCK_SYNC
    if (isomer_idx < B.isomer_capacity){ //Avoid illegal memory access
    if (B.statuses[isomer_idx] == IsomerStatus::NOT_CONVERGED)
    {
        real_t* base_pointer        = smem + Block_Size_Pow_2;
        size_t offset               = isomer_idx * B.n_atoms;
        size_t node_id              = threadIdx.x;
        size_t N                    = B.n_atoms;


        //Pre-compute force constants and store in registers.
        Constants constants = Constants(B, isomer_idx);
        NodeNeighbours nodeG     = NodeNeighbours(B, isomer_idx, smem);

        //Set VRAM pointer to start of each fullerene, as opposed to at the start of the isomerbatch.


        //Assign a section of L1 cache to each set of cartesian coordinates X, X1 and X2.
        coord3d* sX =reinterpret_cast<coord3d*>(base_pointer);
        coord3d* X1 =reinterpret_cast<coord3d*>(base_pointer+3*N);
        coord3d* X2 =reinterpret_cast<coord3d*>(base_pointer+6*N);  

        assign(sX[node_id],reinterpret_cast<std::array<float,3>*>(B.X+3*offset)[node_id]); //Copy cartesian coordinates from L1 Cache to DRAM.
        coord3d* X           = sX;       //Switch coordinate pointer from DRAM to L1 Cache.

        //Create forcefield struct and use optimization algorithm to optimise the fullerene 
        ForceField FF = ForceField<T>(nodeG, constants, smem);
        FF.CG(X,X1,X2,iterations-1);
        BLOCK_SYNC
        auto E0 = FF.energy(X);
        FF.CG(X,X1,X2,1);
        auto E1 = FF.energy(X);
        if (ABS(E1 - E0)/(real_t)blockDim.x < (real_t)1e-5){
            B.statuses[isomer_idx] = IsomerStatus::CONVERGED;
        }
        //Copy data back from L1 cache to DRAM 
        assign(reinterpret_cast<std::array<float,3>*>(B.X)[offset + threadIdx.x], X[threadIdx.x]);

        if (threadIdx.x == 0) {B.iterations[isomer_idx] += iterations;}
    }}
    //Check the convergence of isomers and assign status accordingly.
    BLOCK_SYNC
    check_batch<T>(B, isomer_idx, max_iterations);
    }
}


#if USE_CONSTANT_INDICES
    #define GET_STAT(fun_1, fun_2, param_fun, equillibrium_param, err_fun) \
            template <ForcefieldType T>\
            __global__ void fun_1(const IsomerBatch B, CuArray<float> bond_rms){\
                DEVICE_TYPEDEFS\
                extern __shared__ real_t smem[];\
                clear_cache(smem,Block_Size_Pow_2);\
                for (int isomer_idx = blockIdx.x; isomer_idx < B.isomer_capacity; isomer_idx+= gridDim.x){\
                if(B.statuses[isomer_idx] != IsomerStatus::EMPTY){\
                    coord3d rel_err;\
                    size_t offset = isomer_idx * blockDim.x;  \
                    Constants constants     = Constants(B, isomer_idx);\
                    NodeNeighbours node_graph    = NodeNeighbours(B, isomer_idx, smem);\
                    ForceField FF           = ForceField<T>(node_graph, constants, smem);\
                    coord3d* X              = reinterpret_cast<coord3d*>(B.X+offset*3);\
                    for (uint8_t j = 0; j < 3; j++){\
                        auto arc            = ForceField<T>::ArcData(j, X, node_graph);\
                        d_set(rel_err,      j, ABS(ABS(param_fun       - equillibrium_param(j))        /equillibrium_param(j)));\
                    }\
                    bond_rms.data[isomer_idx]         = err_fun;\
                }}\
            }\
            template <ForcefieldType T>\
            hipError_t fun_2(const IsomerBatch& B, CuArray<float>& bond_rms){\
                hipDeviceSynchronize();\
                hipSetDevice(B.get_device_id());\
                size_t smem = sizeof(device_coord3d)* (3*B.n_atoms + 4) + sizeof(device_real_t)*Block_Size_Pow_2;\
                static LaunchDims dims((void*)fun_1<T>, B.n_atoms, smem, B.isomer_capacity);\
                dims.update_dims((void*)fun_1<T>, B.n_atoms, smem, B.isomer_capacity);\
                void* kargs[]{(void*)&B, (void*)&bond_rms};\
                auto error = safeCudaKernelCall((void*)fun_1<T>, dims.get_grid(), dims.get_block(), kargs, smem);\
                hipDeviceSynchronize();\
                return error;\
            }
#else
    #define GET_STAT(fun_1, fun_2, param_fun, equillibrium_param, err_fun) \
        template <ForcefieldType T>\
        __global__ void fun_1(const IsomerBatch B, CuArray<float> bond_rms){\
            DEVICE_TYPEDEFS\
            extern __shared__ real_t smem[];\
            clear_cache(smem,Block_Size_Pow_2);\
            for (int isomer_idx = blockIdx.x; isomer_idx < B.isomer_capacity; isomer_idx+= gridDim.x){\
            if(B.statuses[isomer_idx] != IsomerStatus::EMPTY){\
                coord3d rel_err;\
                size_t offset = isomer_idx * blockDim.x;  \
                Constants constants     = Constants(B, isomer_idx);\
                NodeNeighbours node_graph    = NodeNeighbours(B, isomer_idx, smem);\
                ForceField FF           = ForceField<T>(node_graph, constants, smem);\
                coord3d* X              = reinterpret_cast<coord3d*>(B.X+offset*3);\
                for (uint8_t j = 0; j < 3; j++){\
                    auto arc            = ForceField<T>::ArcData(j, X, node_graph);\
                    d_set(rel_err,      j, ABS(ABS(param_fun       - d_get(equillibrium_param,j))        /d_get(equillibrium_param,j)));\
                }\
                bond_rms.data[isomer_idx]         = err_fun;\
            }}\
        }\
        template <ForcefieldType T>\
        hipError_t fun_2(const IsomerBatch& B, CuArray<float>& bond_rms){\
            hipDeviceSynchronize();\
            hipSetDevice(B.get_device_id());\
            size_t smem = sizeof(device_coord3d)* (3*B.n_atoms + 4) + sizeof(device_real_t)*Block_Size_Pow_2;\
            static LaunchDims dims((void*)fun_1<T>, B.n_atoms, smem, B.isomer_capacity);\
            dims.update_dims((void*)fun_1<T>, B.n_atoms, smem, B.isomer_capacity);\
            void* kargs[]{(void*)&B, (void*)&bond_rms};\
            auto error = safeCudaKernelCall((void*)fun_1<T>, dims.get_grid(), dims.get_block(), kargs, smem);\
            hipDeviceSynchronize();\
            return error;\
        }
#endif



#define GET_MEAN(fun_1, fun_2, param_fun, err_fun) \
    template <ForcefieldType T>\
    __global__ void fun_1(const IsomerBatch B, CuArray<float> bond_rms){\
        DEVICE_TYPEDEFS\
        extern __shared__ real_t smem[];\
        clear_cache(smem,Block_Size_Pow_2);\
        for (int isomer_idx = blockIdx.x; isomer_idx < B.isomer_capacity; isomer_idx+= gridDim.x){\
        if(B.statuses[isomer_idx] != IsomerStatus::EMPTY){\
            coord3d rel_err;\
            size_t offset = isomer_idx * blockDim.x;  \
            Constants constants     = Constants(B, isomer_idx);\
            NodeNeighbours node_graph    = NodeNeighbours(B, isomer_idx, smem);\
            ForceField FF           = ForceField<T>(node_graph, constants, smem);\
            coord3d* X              = reinterpret_cast<coord3d*>(B.X+offset*3);\
            for (uint8_t j = 0; j < 3; j++){\
                auto arc            = ForceField<T>::ArcData(j, X, node_graph);\
                d_set(rel_err,      j, ABS(param_fun));\
            }\
            bond_rms.data[isomer_idx]         = err_fun;\
        }}\
    }\
    template <ForcefieldType T>\
    hipError_t fun_2(const IsomerBatch& B, CuArray<float>& bond_rms){\
        hipDeviceSynchronize();\
        hipSetDevice(B.get_device_id());\
        size_t smem = sizeof(device_coord3d)* (3*B.n_atoms + 4) + sizeof(device_real_t)*Block_Size_Pow_2;\
        static LaunchDims dims((void*)fun_1<T>, B.n_atoms, smem, B.isomer_capacity);\
        dims.update_dims((void*)fun_1<T>, B.n_atoms, smem, B.isomer_capacity);\
        void* kargs[]{(void*)&B, (void*)&bond_rms};\
        auto error = safeCudaKernelCall((void*)fun_1<T>, dims.get_grid(), dims.get_block(), kargs, smem);\
        hipDeviceSynchronize();\
        return error;\
    }

#define GET_RRMSE(fun_1, fun_2, param_fun, err_fun) \
    template <ForcefieldType T>\
    __global__ void fun_1(const IsomerBatch B, CuArray<float> bond_rms){\
        DEVICE_TYPEDEFS\
        extern __shared__ real_t smem[];\
        clear_cache(smem,Block_Size_Pow_2);\
        for (int isomer_idx = blockIdx.x; isomer_idx < B.isomer_capacity; isomer_idx+= gridDim.x){\
        if(B.statuses[isomer_idx] != IsomerStatus::EMPTY){\
            coord3d top;\
            coord3d bot;\
            size_t offset = isomer_idx * blockDim.x;  \
            Constants constants     = Constants(B, isomer_idx);\
            NodeNeighbours node_graph    = NodeNeighbours(B, isomer_idx, smem);\
            ForceField FF           = ForceField<T>(node_graph, constants, smem);\
            coord3d* X              = reinterpret_cast<coord3d*>(B.X+offset*3);\
            for (uint8_t j = 0; j < 3; j++){\
                auto arc            = ForceField<T>::ArcData(j, X, node_graph);\
                d_set(top,      j, param_fun -  err_fun);\
                d_set(bot,      j, err_fun);\
            }\
            bond_rms.data[isomer_idx]         = SQRT((reduction(smem, dot(top,top))/reduction(smem,dot(bot,bot)))/device_real_t(blockDim.x*3));\
        }}\
    }\
    template <ForcefieldType T>\
    hipError_t fun_2(const IsomerBatch& B, CuArray<float>& bond_rms){\
        hipDeviceSynchronize();\
        hipSetDevice(B.get_device_id());\
        size_t smem = sizeof(device_coord3d)* (3*B.n_atoms + 4) + sizeof(device_real_t)*Block_Size_Pow_2;\
        static LaunchDims dims((void*)fun_1<T>, B.n_atoms, smem, B.isomer_capacity);\
        dims.update_dims((void*)fun_1<T>, B.n_atoms, smem, B.isomer_capacity);\
        void* kargs[]{(void*)&B, (void*)&bond_rms};\
        auto error = safeCudaKernelCall((void*)fun_1<T>, dims.get_grid(), dims.get_block(), kargs, smem);\
        hipDeviceSynchronize();\
        return error;\
    }

#define GET_RMSE(fun_1, fun_2, param_fun, err_fun) \
    template <ForcefieldType T>\
    __global__ void fun_1(const IsomerBatch B, CuArray<float> bond_rms){\
        DEVICE_TYPEDEFS\
        extern __shared__ real_t smem[];\
        clear_cache(smem,Block_Size_Pow_2);\
        for (int isomer_idx = blockIdx.x; isomer_idx < B.isomer_capacity; isomer_idx+= gridDim.x){\
        if(B.statuses[isomer_idx] != IsomerStatus::EMPTY){\
            coord3d top;\
            size_t offset = isomer_idx * blockDim.x;  \
            Constants constants     = Constants(B, isomer_idx);\
            NodeNeighbours node_graph    = NodeNeighbours(B, isomer_idx, smem);\
            ForceField FF           = ForceField<T>(node_graph, constants, smem);\
            coord3d* X              = reinterpret_cast<coord3d*>(B.X+offset*3);\
            for (uint8_t j = 0; j < 3; j++){\
                auto arc            = ForceField<T>::ArcData(j, X, node_graph);\
                d_set(top,      j, param_fun - err_fun);\
            }\
            bond_rms.data[isomer_idx]         = SQRT(reduction(smem, dot(top,top))/device_real_t(blockDim.x*3));\
        }}\
    }\
    template <ForcefieldType T>\
    hipError_t fun_2(const IsomerBatch& B, CuArray<float>& bond_rms){\
        hipDeviceSynchronize();\
        hipSetDevice(B.get_device_id());\
        size_t smem = sizeof(device_coord3d)* (3*B.n_atoms + 4) + sizeof(device_real_t)*Block_Size_Pow_2;\
        static LaunchDims dims((void*)fun_1<T>, B.n_atoms, smem, B.isomer_capacity);\
        dims.update_dims((void*)fun_1<T>, B.n_atoms, smem, B.isomer_capacity);\
        void* kargs[]{(void*)&B, (void*)&bond_rms};\
        auto error = safeCudaKernelCall((void*)fun_1<T>, dims.get_grid(), dims.get_block(), kargs, smem);\
        hipDeviceSynchronize();\
        return error;\
    }

#define GET_INTERNAL(fun_1, fun_2, param_fun) \
    __global__ void fun_1(const IsomerBatch B, CuArray<float> bond_rms){\
        DEVICE_TYPEDEFS\
        extern __shared__ real_t smem[];\
        clear_cache(smem,Block_Size_Pow_2);\
        for (int isomer_idx = blockIdx.x; isomer_idx < B.isomer_capacity; isomer_idx+= gridDim.x){\
        if(B.statuses[isomer_idx] != IsomerStatus::EMPTY){\
            coord3d rel_err;\
            size_t offset = isomer_idx * blockDim.x;  \
            Constants constants     = Constants(B, isomer_idx);\
            NodeNeighbours node_graph    = NodeNeighbours(B, isomer_idx, smem);\
            ForceField FF           = ForceField<FORCEFIELD_VERSION>(node_graph, constants, smem);\
            coord3d* X              = reinterpret_cast<coord3d*>(B.X+offset*3);\
            for (uint8_t j = 0; j < 3; j++){\
                auto arc            = ForceField<FORCEFIELD_VERSION>::ArcData(j, X, node_graph);\
                d_set(rel_err,      j, param_fun);\
            }\
            reinterpret_cast<coord3d*>(bond_rms.data)[isomer_idx*B.n_atoms + threadIdx.x]         =  {rel_err[0], rel_err[1], rel_err[2]};\
        }}\
    }\
    hipError_t fun_2(const IsomerBatch& B, CuArray<float>& bond_rms){\
        hipDeviceSynchronize();\
        hipSetDevice(B.get_device_id());\
        size_t smem = sizeof(device_coord3d)* (3*B.n_atoms + 4) + sizeof(device_real_t)*Block_Size_Pow_2;\
        static LaunchDims dims((void*)fun_1, B.n_atoms, smem, B.isomer_capacity);\
        dims.update_dims((void*)fun_1, B.n_atoms, smem, B.isomer_capacity);\
        void* kargs[]{(void*)&B, (void*)&bond_rms};\
        auto error = safeCudaKernelCall((void*)fun_1, dims.get_grid(), dims.get_block(), kargs, smem);\
        hipDeviceSynchronize();\
        return error;\
    }

GET_INTERNAL(get_bonds_,get_bonds, arc.bond())
GET_INTERNAL(get_angles_,get_angles, arc.angle())
GET_INTERNAL(get_dihedrals_,get_dihedrals, arc.dihedral())

GET_RRMSE(get_bond_rrmse_,get_bond_rrmse, arc.bond(), d_get(constants.r0,j))
GET_RRMSE(get_angle_rrmse_,get_angle_rrmse, acos((float)arc.angle()), acos((float)d_get(constants.angle0,j)))
GET_RRMSE(get_dihedral_rrmse_,get_dihedral_rrmse, acos((float)arc.dihedral()), acos((float)d_get(constants.inner_dih0,j)))

GET_RMSE(get_bond_rmse_,get_bond_rmse, arc.bond(), d_get(constants.r0,j))
GET_RMSE(get_angle_rmse_,get_angle_rmse, acos((float)arc.angle()), acos((float)d_get(constants.angle0,j)))
GET_RMSE(get_dihedral_rmse_,get_dihedral_rmse, acos((float)arc.dihedral()), acos((float)d_get(constants.inner_dih0,j)))

GET_STAT(get_bond_max_,get_bond_max, arc.bond(), constants.r0, reduction_max(smem, max(rel_err)))
GET_STAT(get_angle_rms_,get_angle_rms, arc.angle(), constants.angle0, SQRT(reduction(smem,dot(rel_err,rel_err))/(device_real_t)blockDim.x);)
GET_STAT(get_angle_max_,get_angle_max, arc.angle(), constants.angle0, reduction_max(smem, max(rel_err)))
GET_STAT(get_dihedral_max_,get_dihedral_max, arc.dihedral(), constants.inner_dih0, reduction_max(smem, max(rel_err)))
GET_STAT(get_energies_,get_energies, arc.dihedral(), constants.inner_dih0, FF.energy(X))
GET_STAT(get_gradient_norm_,get_gradient_norm, arc.dihedral(), constants.inner_dih0, SQRT(reduction(smem,dot(FF.gradient(X), FF.gradient(X))))/(device_real_t)blockDim.x)
GET_STAT(get_gradient_rms_,get_gradient_rms, arc.dihedral(), constants.inner_dih0, SQRT(reduction(smem,dot(FF.gradient(X), FF.gradient(X)))/(device_real_t)blockDim.x))
GET_STAT(get_gradient_max_,get_gradient_max, arc.dihedral(), constants.inner_dih0, reduction_max(smem, SQRT(dot(FF.gradient(X), FF.gradient(X)))     ))

GET_MEAN(get_bond_mae_, get_bond_mae, ABS(arc.bond() - d_get(constants.r0,j)), reduction(smem,sum(rel_err)/(device_real_t)3.0f)/(device_real_t)blockDim.x)
GET_MEAN(get_angle_mae_, get_angle_mae, ABS(acos((float)arc.angle()) - acos((float)d_get(constants.angle0,j))), reduction(smem,sum(rel_err)/(device_real_t)3.0f)/(device_real_t)blockDim.x)
GET_MEAN(get_dihedral_mae_, get_dihedral_mae, ABS(acos((float)arc.dihedral()) - acos((float)d_get(constants.inner_dih0,j))), reduction(smem,sum(rel_err)/(device_real_t)3.0f)/(device_real_t)blockDim.x)

GET_MEAN(get_energy_, get_energy, arc.bond(), FF.energy(X)/(device_real_t)blockDim.x)
GET_MEAN(get_bond_mean_,get_bond_mean, arc.bond(), reduction(smem,sum(rel_err)/(device_real_t)3.0f)/(device_real_t)blockDim.x)
GET_MEAN(get_angle_mean_,get_angle_mean, arc.angle(), reduction(smem,sum(rel_err)/(device_real_t)3.0f)/(device_real_t)blockDim.x)
GET_MEAN(get_dihedral_mean_,get_dihedral_mean, arc.dihedral(), reduction(smem,sum(rel_err)/(device_real_t)3.0f)/(device_real_t)blockDim.x)
GET_MEAN(get_gradient_mean_,get_gradient_mean, d_get(FF.gradient(X),j), reduction(smem,sum(rel_err)/(device_real_t)3.0f)/(device_real_t)blockDim.x)
GET_MEAN(get_flat_mean_,get_flat_mean, d_get(FF.gradient(X),j), reduction(smem,ForceField<FORCEFIELD_VERSION>::FaceData(X, node_graph).flatness())/device_real_t(blockDim.x/2 + 2 ))
GET_MEAN(get_flat_rmse_,get_flat_rmse, d_get(FF.gradient(X),j), SQRT(reduction(smem,ForceField<FORCEFIELD_VERSION>::FaceData(X, node_graph).flatness() * ForceField<FORCEFIELD_VERSION>::FaceData(X, node_graph).flatness())/device_real_t(blockDim.x/2 + 2 )) )
GET_MEAN(get_flat_max_,get_flat_max, d_get(FF.gradient(X),j), reduction_max(smem,ForceField<FORCEFIELD_VERSION>::FaceData(X, node_graph).flatness())   )

int optimal_batch_size(const int N, const int device_id) {
    hipSetDevice(device_id);
    static size_t smem = sizeof(device_coord3d)*3*N + sizeof(device_real_t)*Block_Size_Pow_2;
    static LaunchDims dims((void*)optimise_<FORCEFIELD_VERSION>, N, smem);
    dims.update_dims((void*)optimise_<FORCEFIELD_VERSION>, N, smem);
    return dims.get_grid().x;
}

float kernel_time = 0.0;
std::chrono::microseconds time_spent(){
    return std::chrono::microseconds((int) (kernel_time*1000.f));
}

void reset_time(){
    kernel_time = 0.0;
}

template <ForcefieldType T>
hipError_t optimise(IsomerBatch& B, const size_t iterations, const size_t max_iterations, const LaunchCtx& ctx, const LaunchPolicy policy){
    hipSetDevice(B.get_device_id());
    static std::vector<bool> first_call(16, true);
    static hipEvent_t start[16], stop[16];
    float single_kernel_time = 0.0;
    auto dev = B.get_device_id();
    if(first_call[dev]) {hipEventCreate(&start[dev]); hipEventCreate(&stop[dev]);}
        
    //If launch ploicy is synchronous then wait.
    if(policy == LaunchPolicy::SYNC) {ctx.wait();}
    else if(policy == LaunchPolicy::ASYNC && !first_call[dev]){
        //Records time from previous kernel call
        hipEventElapsedTime(&single_kernel_time, start[dev], stop[dev]);
        kernel_time += single_kernel_time;
    }

    size_t smem = sizeof(device_coord3d)* (3*B.n_atoms + 4) + sizeof(device_real_t)*Block_Size_Pow_2;
    static LaunchDims dims((void*)optimise_<T>, B.n_atoms, smem, B.isomer_capacity);
    dims.update_dims((void*)optimise_<T>, B.n_atoms, smem, B.isomer_capacity);
    void* kargs[]{(void*)&B, (void*)&iterations, (void*)&max_iterations};

    hipEventRecord(start[dev], ctx.stream);
    auto error = safeCudaKernelCall((void*)optimise_<T>, dims.get_grid(), dims.get_block(), kargs, smem, ctx.stream);
    hipEventRecord(stop[dev], ctx.stream);
    
    if(policy == LaunchPolicy::SYNC) {
        ctx.wait();
        hipEventElapsedTime(&single_kernel_time, start[dev], stop[dev]);
        kernel_time += single_kernel_time;
    }
    printLastCudaError("Forcefield: ");
    first_call[dev] = false;
    return error;
}

template <ForcefieldType T>
hipError_t compute_hessians(IsomerBatch& B, const LaunchCtx& ctx, const LaunchPolicy policy){
    hipSetDevice(B.get_device_id());
    static std::vector<bool> first_call(16, true);
    static hipEvent_t start[16], stop[16];
    float single_kernel_time = 0.0;
    auto dev = B.get_device_id();
    if(first_call[dev]) {hipEventCreate(&start[dev]); hipEventCreate(&stop[dev]);}
        
    //If launch ploicy is synchronous then wait.
    if(policy == LaunchPolicy::SYNC) {ctx.wait();}
    else if(policy == LaunchPolicy::ASYNC && !first_call[dev]){
        //Records time from previous kernel call
        hipEventElapsedTime(&single_kernel_time, start[dev], stop[dev]);
        kernel_time += single_kernel_time;
    }

    size_t smem = sizeof(device_coord3d)* (3*B.n_atoms + 4) + sizeof(device_real_t)*Block_Size_Pow_2;
    static LaunchDims dims((void*)compute_hessians_<T>, B.n_atoms, smem, B.isomer_capacity);
    dims.update_dims((void*)compute_hessians_<T>, B.n_atoms, smem, B.isomer_capacity);
    void* kargs[]{(void*)&B};

    hipEventRecord(start[dev], ctx.stream);
    auto error = safeCudaKernelCall((void*)compute_hessians_<T>, dims.get_grid(), dims.get_block(), kargs, smem, ctx.stream);
    hipEventRecord(stop[dev], ctx.stream);
    
    if(policy == LaunchPolicy::SYNC) {
        ctx.wait();
        hipEventElapsedTime(&single_kernel_time, start[dev], stop[dev]);
        kernel_time += single_kernel_time;
    }
    printLastCudaError("Forcefield: ");
    first_call[dev] = false;
    return error;
}

int declare_generics(){
    IsomerBatch B(20,1,DEVICE_BUFFER);
    CuArray<float> arr(1);
    optimise<PEDERSEN>(B,100,100);
    compute_hessians<PEDERSEN>(B);
    /* get_angle_max<PEDERSEN>(B,arr);
    get_bond_max<PEDERSEN>(B,arr);
    get_dihedral_max<PEDERSEN>(B,arr);
    get_angle_mae<PEDERSEN>(B,arr);
    get_bond_mae<PEDERSEN>(B,arr);
    get_dihedral_mae<PEDERSEN>(B,arr);
    get_angle_rrmse<PEDERSEN>(B,arr);
    get_bond_rrmse<PEDERSEN>(B,arr);
    get_dihedral_rrmse<PEDERSEN>(B,arr);
    get_angle_rmse<PEDERSEN>(B,arr);
    get_bond_rmse<PEDERSEN>(B,arr);
    get_dihedral_rmse<PEDERSEN>(B,arr);
    get_angle_mean<PEDERSEN>(B,arr);
    get_bond_mean<PEDERSEN>(B,arr);
    get_flat_mean<PEDERSEN>(B,arr);
    get_flat_max<PEDERSEN>(B,arr);
    get_flat_rmse<PEDERSEN>(B,arr);
    get_dihedral_mean<PEDERSEN>(B,arr);
    get_gradient_max<PEDERSEN>(B,arr);
    get_gradient_rms<PEDERSEN>(B,arr);
    get_gradient_mean<PEDERSEN>(B,arr);
    get_gradient_norm<PEDERSEN>(B,arr);
    get_energies<PEDERSEN>(B,arr);

    optimise<FLATNESS_ENABLED>(B,100,100);
    compute_hessians<FLATNESS_ENABLED>(B);
    get_angle_max<FLATNESS_ENABLED>(B,arr);
    get_bond_max<FLATNESS_ENABLED>(B,arr);
    get_dihedral_max<FLATNESS_ENABLED>(B,arr);
    get_angle_mae<FLATNESS_ENABLED>(B,arr);
    get_bond_mae<FLATNESS_ENABLED>(B,arr);
    get_dihedral_mae<FLATNESS_ENABLED>(B,arr);
    get_angle_rrmse<FLATNESS_ENABLED>(B,arr);
    get_bond_rrmse<FLATNESS_ENABLED>(B,arr);
    get_dihedral_rrmse<FLATNESS_ENABLED>(B,arr);
    get_angle_rmse<FLATNESS_ENABLED>(B,arr);
    get_bond_rmse<FLATNESS_ENABLED>(B,arr);
    get_dihedral_rmse<FLATNESS_ENABLED>(B,arr);
    get_angle_mean<FLATNESS_ENABLED>(B,arr);
    get_bond_mean<FLATNESS_ENABLED>(B,arr);
    get_dihedral_mean<FLATNESS_ENABLED>(B,arr);
    get_flat_mean<FLATNESS_ENABLED>(B,arr);
    get_flat_max<FLATNESS_ENABLED>(B,arr);
    get_flat_rmse<FLATNESS_ENABLED>(B,arr);
    get_gradient_max<FLATNESS_ENABLED>(B,arr);
    get_gradient_rms<FLATNESS_ENABLED>(B,arr);
    get_gradient_mean<FLATNESS_ENABLED>(B,arr);
    get_gradient_norm<FLATNESS_ENABLED>(B,arr);
    get_energies<FLATNESS_ENABLED>(B,arr);

    optimise<WIRZ>(B,100,100);
    get_angle_max<WIRZ>(B,arr);
    get_bond_max<WIRZ>(B,arr);
    get_dihedral_max<WIRZ>(B,arr);
    get_angle_mae<WIRZ>(B,arr);
    get_bond_mae<WIRZ>(B,arr);
    get_dihedral_mae<WIRZ>(B,arr);
    get_angle_rrmse<WIRZ>(B,arr);
    get_bond_rrmse<WIRZ>(B,arr);
    get_dihedral_rrmse<WIRZ>(B,arr);
    get_angle_rmse<WIRZ>(B,arr);
    get_bond_rmse<WIRZ>(B,arr);
    get_dihedral_rmse<WIRZ>(B,arr);
    get_angle_mean<WIRZ>(B,arr);
    get_bond_mean<WIRZ>(B,arr);
    get_dihedral_mean<WIRZ>(B,arr);
    get_flat_mean<WIRZ>(B,arr);
    get_flat_max<WIRZ>(B,arr);
    get_flat_rmse<WIRZ>(B,arr);
    get_gradient_max<WIRZ>(B,arr);
    get_gradient_rms<WIRZ>(B,arr);
    get_gradient_mean<WIRZ>(B,arr);
    get_gradient_norm<WIRZ>(B,arr);
    get_energies<WIRZ>(B,arr);

    optimise<FLAT_BOND>(B,100,100);
    compute_hessians<FLAT_BOND>(B);
    get_angle_max<FLAT_BOND>(B,arr);
    get_bond_max<FLAT_BOND>(B,arr);
    get_dihedral_max<FLAT_BOND>(B,arr);
    get_angle_mae<FLAT_BOND>(B,arr);
    get_bond_mae<FLAT_BOND>(B,arr);
    get_dihedral_mae<FLAT_BOND>(B,arr);
    get_angle_rrmse<FLAT_BOND>(B,arr);
    get_bond_rrmse<FLAT_BOND>(B,arr);
    get_dihedral_rrmse<FLAT_BOND>(B,arr);
    get_angle_rmse<FLAT_BOND>(B,arr);
    get_bond_rmse<FLAT_BOND>(B,arr);
    get_dihedral_rmse<FLAT_BOND>(B,arr);
    get_angle_mean<FLAT_BOND>(B,arr);
    get_bond_mean<FLAT_BOND>(B,arr);
    get_dihedral_mean<FLAT_BOND>(B,arr);
    get_flat_mean<FLAT_BOND>(B,arr);
    get_flat_max<FLAT_BOND>(B,arr);
    get_flat_rmse<FLAT_BOND>(B,arr);
    get_gradient_max<FLAT_BOND>(B,arr);
    get_gradient_rms<FLAT_BOND>(B,arr);
    get_gradient_mean<FLAT_BOND>(B,arr);
    get_gradient_norm<FLAT_BOND>(B,arr);
    get_energies<FLAT_BOND>(B,arr); 
 */
    return 1;
}


}}
